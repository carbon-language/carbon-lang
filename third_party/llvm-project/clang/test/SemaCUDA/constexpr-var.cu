#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -x hip %s \
// RUN:   -fsyntax-only -verify
// RUN: %clang_cc1 -triple x86_64 -x hip %s \
// RUN:   -fsyntax-only -verify=host

// host-no-diagnostics

#include "Inputs/hip/hip_runtime.h"

// Test constexpr var initialized with address of a const var.
// Both are promoted to device side.

namespace Test1 {
const int a = 1;

struct B {
    static constexpr const int *p = &a;
    __device__ static constexpr const int *const p2 = &a;
};

// Const variable 'a' is treated as __constant__ on device side,
// therefore its address can be used as initializer for another
// device variable.

__device__ void f() {
  int y = a;
  constexpr const int *x = B::p;
  constexpr const int *z = B::p2;
}
}

// Test constexpr var initialized with address of a non-cost var.
// Neither is promoted to device side.

namespace Test2 {
int a = 1;
// expected-note@-1{{host variable declared here}}

struct B {
    static constexpr int *const p = &a;
    // expected-note@-1{{const variable cannot be emitted on device side due to dynamic initialization}}
};

__device__ void f() {
  int y = a;
  // expected-error@-1{{reference to __host__ variable 'a' in __device__ function}}
  const int *const *x = &B::p;
  // expected-error@-1{{reference to __host__ variable 'p' in __device__ function}}
  // ToDo: use of non-promotable constexpr variable in device compilation should be treated as
  // ODR-use and diagnosed.
  const int *const z = B::p;
}
}

// Test constexpr device var initialized with address of a non-const host var, __shared var,
// __managed__ var, __device__ var, __constant__ var, texture var, surface var.

namespace Test3 {
struct textureReference {
  int desc;
};

enum ReadMode {
  ElementType = 0,
  NormalizedFloat = 1
};

template <typename T, int dim = 1, enum ReadMode mode = ElementType>
struct __attribute__((device_builtin_texture_type)) texture : public textureReference {
};

struct surfaceReference {
  int desc;
};

template <typename T, int dim = 1>
struct __attribute__((device_builtin_surface_type)) surface : public surfaceReference {
};

// Partial specialization over `void`.
template<int dim>
struct __attribute__((device_builtin_surface_type)) surface<void, dim> : public surfaceReference {
};

texture<float, 2, ElementType> tex;
surface<void, 2> surf;

int a = 1;
__shared__ int b;
__managed__ int c = 1;
__device__ int d = 1;
__constant__ int e = 1;
struct B {
    __device__ static constexpr int *const p1 = &a;
    // expected-error@-1{{dynamic initialization is not supported for __device__, __constant__, __shared__, and __managed__ variables}}
    __device__ static constexpr int *const p2 = &b;
    // expected-error@-1{{dynamic initialization is not supported for __device__, __constant__, __shared__, and __managed__ variables}}
    __device__ static constexpr int *const p3 = &c;
    // expected-error@-1{{dynamic initialization is not supported for __device__, __constant__, __shared__, and __managed__ variables}}
    __device__ static constexpr int *const p4 = &d;
    __device__ static constexpr int *const p5 = &e;
    __device__ static constexpr texture<float, 2, ElementType> *const p6 = &tex;
    __device__ static constexpr surface<void, 2> *const p7 = &surf;
};
}
