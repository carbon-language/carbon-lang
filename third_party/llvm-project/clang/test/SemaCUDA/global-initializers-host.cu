// RUN: %clang_cc1 %s --std=c++11 -triple x86_64-linux-unknown -fsyntax-only -o - -verify

#include "Inputs/hip/hip_runtime.h"

// Check that we get an error if we try to call a __device__ function from a
// module initializer.

struct S {
  __device__ S() {}
  // expected-note@-1 {{'S' declared here}}
};

S s;
// expected-error@-1 {{reference to __device__ function 'S' in global initializer}}

struct T {
  __host__ __device__ T() {}
};
T t;  // No error, this is OK.

struct U {
  __host__ U() {}
  __device__ U(int) {}
  // expected-note@-1 {{'U' declared here}}
};
U u(42);
// expected-error@-1 {{reference to __device__ function 'U' in global initializer}}

__device__ int device_fn() { return 42; }
// expected-note@-1 {{'device_fn' declared here}}
int n = device_fn();
// expected-error@-1 {{reference to __device__ function 'device_fn' in global initializer}}
