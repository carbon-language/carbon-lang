// RUN: %clang_cc1 -std=c++11 -fsyntax-only -verify -isystem %S/Inputs %s
// RUN: %clang_cc1 -std=c++11 -fsyntax-only -verify -isystem %S/Inputs %s \
// RUN:            -fcuda-is-device
// RUN: %clang_cc1 -std=c++11 -fsyntax-only -verify -isystem %S/Inputs \
// RUN:            -fopenmp %s
// RUN: %clang_cc1 -std=c++11 -fsyntax-only -verify -isystem %S/Inputs \
// RUN:            -fopenmp %s -fcuda-is-device

#include "Inputs/hip/hip_runtime.h"

// Declares one function and pulls it into namespace ns:
//
//   __device__ int OverloadMe();
//   namespace ns { using ::OverloadMe; }
//
// Clang cares that this is done in a system header.
#include <overload.h>

// Opaque type used to determine which overload we're invoking.
struct HostReturnTy {};

// These shouldn't become host+device because they already have attributes.
__host__ constexpr int HostOnly() { return 0; }
// expected-note@-1 0+ {{not viable}}
__device__ constexpr int DeviceOnly() { return 0; }
// expected-note@-1 0+ {{not viable}}

constexpr int HostDevice() { return 0; }

// This should be a host-only function, because there's a previous __device__
// overload in <overload.h>.
constexpr HostReturnTy OverloadMe() { return HostReturnTy(); }

namespace ns {
// The "using" statement in overload.h should prevent OverloadMe from being
// implicitly host+device.
constexpr HostReturnTy OverloadMe() { return HostReturnTy(); }
}  // namespace ns

// This is an error, because NonSysHdrOverload was not defined in a system
// header.
__device__ int NonSysHdrOverload() { return 0; }
// expected-note@-1 {{conflicting __device__ function declared here}}
constexpr int NonSysHdrOverload() { return 0; }
// expected-error@-1 {{constexpr function 'NonSysHdrOverload' without __host__ or __device__ attributes}}

// Variadic device functions are not allowed, so this is just treated as
// host-only.
constexpr void Variadic(const char*, ...);
// expected-note@-1 {{call to __host__ function from __device__ function}}

__host__ void HostFn() {
  HostOnly();
  DeviceOnly(); // expected-error {{no matching function}}
  HostReturnTy x = OverloadMe();
  HostReturnTy y = ns::OverloadMe();
  Variadic("abc", 42);
}

__device__ void DeviceFn() {
  HostOnly(); // expected-error {{no matching function}}
  DeviceOnly();
  int x = OverloadMe();
  int y = ns::OverloadMe();
  Variadic("abc", 42); // expected-error {{no matching function}}
}

__host__ __device__ void HostDeviceFn() {
#ifdef __CUDA_ARCH__
  int y = OverloadMe();
#else
  constexpr HostReturnTy y = OverloadMe();
#endif
}
