// RUN: %clang_cc1 -fsyntax-only -verify %s -DHOST -triple x86_64-unknown-linux-gnu -Wuninitialized
// RUN: %clang_cc1 -fsyntax-only -verify %s -DHOST -DHOST_USED -triple x86_64-unknown-linux-gnu -Wuninitialized
// RUN: %clang_cc1 -fsyntax-only -fcuda-is-device -verify %s -DDEVICE_NOT_USED -triple nvptx-unknown-cuda -Wuninitialized
// RUN: %clang_cc1 -fsyntax-only -fcuda-is-device -verify %s -DDEVICE -triple nvptx-unknown-cuda -Wuninitialized
// RUN: %clang_cc1 -fsyntax-only -fcuda-is-device -verify %s -DDEVICE -DDEVICE_USED -triple nvptx-unknown-cuda -Wuninitialized

// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

#if (defined(HOST) && !defined(HOST_USED)) || defined(DEVICE_NOT_USED)
// expected-no-diagnostics
#endif

#include "Inputs/hip/hip_runtime.h"

static __device__ __host__ void t1(int r) {
  __asm__("PR3908 %[lf] %[xx] %[li] %[r]"
          : [ r ] "+r"(r)
          : [ lf ] "mx"(0), [ li ] "mr"(0), [ xx ] "x"((double)(0)));
}

static __device__ __host__ unsigned t2(signed char input) {
  unsigned output;
  __asm__("xyz"
          : "=a"(output)
          : "0"(input));
  return output;
}

static __device__ __host__ double t3(double x) {
  long double result;
  __asm __volatile("frndint"
                   : "=t"(result)
                   : "0"(x));
  return result;
}

static __device__ __host__ unsigned char t4(unsigned char a, unsigned char b) {
  unsigned int la = a;
  unsigned int lb = b;
  unsigned int bigres;
  unsigned char res;
  __asm__("0:\n1:\n"
          : [ bigres ] "=la"(bigres)
          : [ la ] "0"(la), [ lb ] "c"(lb)
          : "edx", "cc");
  res = bigres;
  return res;
}

static __device__ __host__ void t5(void) {
  __asm__ __volatile__(
      "finit"
      :
      :
      : "st", "st(1)", "st(2)", "st(3)",
        "st(4)", "st(5)", "st(6)", "st(7)",
        "fpsr", "fpcr");
}

typedef long long __m256i __attribute__((__vector_size__(32)));
static __device__ __host__ void t6(__m256i *p) {
  __asm__ volatile("vmovaps  %0, %%ymm0" ::"m"(*(__m256i *)p)
                   : "ymm0");
}

static __device__ __host__ void t7(__m256i *p) {
  __asm__ volatile("vmovaps  %0, %%ymm0" ::"m"(*(__m256i *)p)
                   : "r0");
}

#ifdef DEVICE
__device__ int m() {
  t1(0);
  t2(0);
  t3(0);
  t4(0, 0);
  t5();
  t6(0);
#ifdef DEVICE_USED
  t7(0);
#endif // DEVICE_USED
  return 0;
}
#endif // DEVICE

#ifdef HOST
__host__ int main() {
  t1(0);
  t2(0);
  t3(0);
  t4(0, 0);
  t5();
  t6(0);
#ifdef HOST_USED
  t7(0);
#endif // HOST_USED
  return 0;
}
#endif // HOST

#if defined(HOST_USED)
// expected-error@69 {{unknown register name 'r0' in asm}}
// expected-note@96 {{called by 'main'}}
#elif defined(DEVICE)
// expected-error@19 {{invalid input constraint 'mx' in asm}}
// expected-error@25 {{invalid output constraint '=a' in asm}}
// expected-error@33 {{invalid output constraint '=t' in asm}}
// expected-error@44 {{invalid output constraint '=la' in asm}}
// expected-error@56 {{unknown register name 'st' in asm}}
// expected-error@64 {{unknown register name 'ymm0' in asm}}
// expected-note@74 {{called by 'm'}}
// expected-note@75 {{called by 'm'}}
// expected-note@76 {{called by 'm'}}
// expected-note@77 {{called by 'm'}}
// expected-note@78 {{called by 'm'}}
// expected-note@79 {{called by 'm'}}
#endif
