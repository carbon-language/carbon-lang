#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -std=c++11 -fcuda-is-device -fsyntax-only -verify=dev %s
// RUN: %clang_cc1 -std=c++11 -fsyntax-only -verify=host %s

// host-no-diagnostics

#include "Inputs/hip/hip_runtime.h"

int func();

struct A {
  int x;
  static int host_var;
};

int A::host_var; // dev-note {{host variable declared here}}

namespace X {
  int host_var; // dev-note {{host variable declared here}}
}

// struct with non-empty ctor.
struct B1 {
  int x;
  B1() { x = 1; }
};

// struct with non-empty dtor.
struct B2 {
  int x;
  B2() {}
  ~B2() { x = 0; }
};

static int static_host_var; // dev-note {{host variable declared here}}

__device__ int global_dev_var;
__constant__ int global_constant_var;
__shared__ int global_shared_var;

int global_host_var; // dev-note 8{{host variable declared here}}
const int global_const_var = 1;
constexpr int global_constexpr_var = 1;

int global_host_array[2] = {1, 2}; // dev-note {{host variable declared here}}
const int global_const_array[2] = {1, 2};
constexpr int global_constexpr_array[2] = {1, 2};

A global_host_struct_var{1}; // dev-note 2{{host variable declared here}}
const A global_const_struct_var{1};
constexpr A global_constexpr_struct_var{1};

// Check const host var initialized with non-empty ctor is not allowed in
// device function.
const B1 b1; // dev-note {{const variable cannot be emitted on device side due to dynamic initialization}}

// Check const host var having non-empty dtor is not allowed in device function.
const B2 b2; // dev-note {{const variable cannot be emitted on device side due to dynamic initialization}}

// Check const host var initialized by non-constant initializer is not allowed
// in device function.
const int b3 = func(); // dev-note {{const variable cannot be emitted on device side due to dynamic initialization}}

template<typename F>
__global__ void kernel(F f) { f(); } // dev-note2 {{called by 'kernel<(lambda}}

__device__ void dev_fun(int *out) {
  // Check access device variables are allowed.
  int &ref_dev_var = global_dev_var;
  int &ref_constant_var = global_constant_var;
  int &ref_shared_var = global_shared_var;
  *out = ref_dev_var;
  *out = ref_constant_var;
  *out = ref_shared_var;
  *out = global_dev_var;
  *out = global_constant_var;
  *out = global_shared_var;

  // Check access of non-const host variables are not allowed.
  *out = global_host_var; // dev-error {{reference to __host__ variable 'global_host_var' in __device__ function}}
  *out = global_const_var;
  *out = global_constexpr_var;
  *out = b1.x; // dev-error {{reference to __host__ variable 'b1' in __device__ function}}
  *out = b2.x; // dev-error {{reference to __host__ variable 'b2' in __device__ function}}
  *out = b3; // dev-error {{reference to __host__ variable 'b3' in __device__ function}}
  global_host_var = 1; // dev-error {{reference to __host__ variable 'global_host_var' in __device__ function}}

  // Check reference of non-constexpr host variables are not allowed.
  int &ref_host_var = global_host_var; // dev-error {{reference to __host__ variable 'global_host_var' in __device__ function}}
  const int &ref_const_var = global_const_var;
  const int &ref_constexpr_var = global_constexpr_var;
  *out = ref_host_var;
  *out = ref_constexpr_var;
  *out = ref_const_var;

  // Check access member of non-constexpr struct type host variable is not allowed.
  *out = global_host_struct_var.x; // dev-error {{reference to __host__ variable 'global_host_struct_var' in __device__ function}}
  *out = global_const_struct_var.x;
  *out = global_constexpr_struct_var.x;
  global_host_struct_var.x = 1; // dev-error {{reference to __host__ variable 'global_host_struct_var' in __device__ function}}

  // Check address taking of non-constexpr host variables is not allowed.
  int *p = &global_host_var; // dev-error {{reference to __host__ variable 'global_host_var' in __device__ function}}
  const int *cp = &global_const_var;
  const int *cp2 = &global_constexpr_var;

  // Check access elements of non-constexpr host array is not allowed.
  *out = global_host_array[1]; // dev-error {{reference to __host__ variable 'global_host_array' in __device__ function}}
  *out = global_const_array[1];
  *out = global_constexpr_array[1];

  // Check ODR-use of host variables in namespace is not allowed.
  *out = X::host_var; // dev-error {{reference to __host__ variable 'host_var' in __device__ function}}

  // Check ODR-use of static host varables in class or file scope is not allowed.
  *out = A::host_var; // dev-error {{reference to __host__ variable 'host_var' in __device__ function}}
  *out = static_host_var; // dev-error {{reference to __host__ variable 'static_host_var' in __device__ function}}

  // Check function-scope static variable is allowed.
  static int static_var;
  *out = static_var;

  // Check non-ODR use of host varirables are allowed.
  *out = sizeof(global_host_var);
  *out = sizeof(global_host_struct_var.x);
  decltype(global_host_var) var1;
  decltype(global_host_struct_var.x) var2;
}

__global__ void global_fun(int *out) {
  int &ref_host_var = global_host_var; // dev-error {{reference to __host__ variable 'global_host_var' in __global__ function}}
  int &ref_dev_var = global_dev_var;
  int &ref_constant_var = global_constant_var;
  int &ref_shared_var = global_shared_var;
  const int &ref_constexpr_var = global_constexpr_var;
  const int &ref_const_var = global_const_var;

  *out = global_host_var; // dev-error {{reference to __host__ variable 'global_host_var' in __global__ function}}
  *out = global_dev_var;
  *out = global_constant_var;
  *out = global_shared_var;
  *out = global_constexpr_var;
  *out = global_const_var;

  *out = ref_host_var;
  *out = ref_dev_var;
  *out = ref_constant_var;
  *out = ref_shared_var;
  *out = ref_constexpr_var;
  *out = ref_const_var;
}

__host__ __device__ void host_dev_fun(int *out) {
  int &ref_host_var = global_host_var; // dev-error {{reference to __host__ variable 'global_host_var' in __host__ __device__ function}}
  int &ref_dev_var = global_dev_var;
  int &ref_constant_var = global_constant_var;
  int &ref_shared_var = global_shared_var;
  const int &ref_constexpr_var = global_constexpr_var;
  const int &ref_const_var = global_const_var;

  *out = global_host_var; // dev-error {{reference to __host__ variable 'global_host_var' in __host__ __device__ function}}
  *out = global_dev_var;
  *out = global_constant_var;
  *out = global_shared_var;
  *out = global_constexpr_var;
  *out = global_const_var;

  *out = ref_host_var;
  *out = ref_dev_var;
  *out = ref_constant_var;
  *out = ref_shared_var;
  *out = ref_constexpr_var;
  *out = ref_const_var;
}

inline __host__ __device__ void inline_host_dev_fun(int *out) {
  int &ref_host_var = global_host_var;
  int &ref_dev_var = global_dev_var;
  int &ref_constant_var = global_constant_var;
  int &ref_shared_var = global_shared_var;
  const int &ref_constexpr_var = global_constexpr_var;
  const int &ref_const_var = global_const_var;

  *out = global_host_var;
  *out = global_dev_var;
  *out = global_constant_var;
  *out = global_shared_var;
  *out = global_constexpr_var;
  *out = global_const_var;

  *out = ref_host_var;
  *out = ref_dev_var;
  *out = ref_constant_var;
  *out = ref_shared_var;
  *out = ref_constexpr_var;
  *out = ref_const_var;
}

void dev_lambda_capture_by_ref(int *out) {
  int &ref_host_var = global_host_var;
  kernel<<<1,1>>>([&]() {
  int &ref_dev_var = global_dev_var;
  int &ref_constant_var = global_constant_var;
  int &ref_shared_var = global_shared_var;
  const int &ref_constexpr_var = global_constexpr_var;
  const int &ref_const_var = global_const_var;

  *out = global_host_var; // dev-error {{reference to __host__ variable 'global_host_var' in __host__ __device__ function}}
                          // dev-error@-1 {{capture host variable 'out' by reference in device or host device lambda function}}
  *out = global_dev_var;
  *out = global_constant_var;
  *out = global_shared_var;
  *out = global_constexpr_var;
  *out = global_const_var;

  *out = ref_host_var; // dev-error {{capture host variable 'ref_host_var' by reference in device or host device lambda function}}
  *out = ref_dev_var;
  *out = ref_constant_var;
  *out = ref_shared_var;
  *out = ref_constexpr_var;
  *out = ref_const_var;
  });
}

void dev_lambda_capture_by_copy(int *out) {
  int &ref_host_var = global_host_var;
  kernel<<<1,1>>>([=]() {
  int &ref_dev_var = global_dev_var;
  int &ref_constant_var = global_constant_var;
  int &ref_shared_var = global_shared_var;
  const int &ref_constexpr_var = global_constexpr_var;
  const int &ref_const_var = global_const_var;

  *out = global_host_var; // dev-error {{reference to __host__ variable 'global_host_var' in __host__ __device__ function}}
  *out = global_dev_var;
  *out = global_constant_var;
  *out = global_shared_var;
  *out = global_constexpr_var;
  *out = global_const_var;

  *out = ref_host_var;
  *out = ref_dev_var;
  *out = ref_constant_var;
  *out = ref_shared_var;
  *out = ref_constexpr_var;
  *out = ref_const_var;
  });
}

// Texture references are special. As far as C++ is concerned they are host
// variables that are referenced from device code. However, they are handled
// very differently by the compiler under the hood and such references are
// allowed. Compiler should produce no warning here, but it should diagnose the
// same case without the device_builtin_texture_type attribute.
template <class, int = 1, int = 1>
struct __attribute__((device_builtin_texture_type)) texture {
  static texture<int> ref;
  __device__ void c() {
    auto &x = ref;
  }
};

template <class, int = 1, int = 1>
struct  not_a_texture {
  static not_a_texture<int> ref;
  __device__ void c() {
    auto &x = ref; // dev-error {{reference to __host__ variable 'ref' in __device__ function}}
  }
};

template<>
not_a_texture<int> not_a_texture<int>::ref; // dev-note {{host variable declared here}}

__device__ void test_not_a_texture() {
  not_a_texture<int> inst;
  inst.c(); // dev-note {{in instantiation of member function 'not_a_texture<int, 1, 1>::c' requested here}}
}

// Test static variable in host function used by device function.
void test_static_var_host() {
  for (int i = 0; i < 10; i++) {
    static int x; // dev-note {{host variable declared here}}
    struct A {
      __device__ int f() {
        return x; // dev-error{{reference to __host__ variable 'x' in __device__ function}}
      }
    };
  }
}

// Test static variable in device function used by device function.
__device__ void test_static_var_device() {
  for (int i = 0; i < 10; i++) {
    static int x;
    int y = x;
    struct A {
      __device__ int f() {
        return x;
      }
    };
  }
}
