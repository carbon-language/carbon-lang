#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fcuda-is-device -fsyntax-only -verify %s

#include "Inputs/hip/hip_runtime.h"

// We should emit an error for hd_fn's use of a VLA.  This would have been
// legal if hd_fn were never codegen'ed on the device, so we should also print
// out a callstack showing how we determine that hd_fn is known-emitted.
//
// Compare to no-call-stack-for-deferred-err.cu.

inline __host__ __device__ void hd_fn(int n);
inline __device__ void device_fn2() { hd_fn(42); } // expected-note {{called by 'device_fn2'}}

__global__ void kernel() { device_fn2(); } // expected-note {{called by 'kernel'}}

inline __host__ __device__ void hd_fn(int n) {
  int vla[n]; // expected-error {{variable-length array}}
}
