// RUN: %clang_cc1 -fsyntax-only -verify %s

#include "Inputs/hip/hip_runtime.h"

//------------------------------------------------------------------------------
// Test 1: collision between two bases

struct A1_with_host_ctor {
  A1_with_host_ctor() {}
};

struct B1_with_device_ctor {
  __device__ B1_with_device_ctor() {}
};

struct C1_with_collision : A1_with_host_ctor, B1_with_device_ctor {
};

// expected-note@-3 {{candidate constructor (the implicit default constructor}} not viable
// expected-note@-4 {{implicit default constructor inferred target collision: call to both __host__ and __device__ members}}
// expected-note@-5 {{candidate constructor (the implicit copy constructor}} not viable
// expected-note@-6 {{candidate constructor (the implicit move constructor) not viable}}

void hostfoo1() {
  C1_with_collision c; // expected-error {{no matching constructor}}
}

//------------------------------------------------------------------------------
// Test 2: collision between two fields

struct C2_with_collision {
  A1_with_host_ctor aa;
  B1_with_device_ctor bb;
};

// expected-note@-5 {{candidate constructor (the implicit default constructor}} not viable
// expected-note@-6 {{implicit default constructor inferred target collision: call to both __host__ and __device__ members}}
// expected-note@-7 {{candidate constructor (the implicit copy constructor}} not viable
// expected-note@-8 {{candidate constructor (the implicit move constructor) not viable}}

void hostfoo2() {
  C2_with_collision c; // expected-error {{no matching constructor}}

}

//------------------------------------------------------------------------------
// Test 3: collision between a field and a base

struct C3_with_collision : A1_with_host_ctor {
  B1_with_device_ctor bb;
};

// expected-note@-4 {{candidate constructor (the implicit default constructor}} not viable
// expected-note@-5 {{implicit default constructor inferred target collision: call to both __host__ and __device__ members}}
// expected-note@-6 {{candidate constructor (the implicit copy constructor}} not viable
// expected-note@-7 {{candidate constructor (the implicit move constructor) not viable}}

void hostfoo3() {
  C3_with_collision c; // expected-error {{no matching constructor}}
}
