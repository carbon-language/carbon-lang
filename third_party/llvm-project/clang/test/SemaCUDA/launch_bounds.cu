// RUN: %clang_cc1 -std=c++11 -fsyntax-only -verify %s

#include "Inputs/hip/hip_runtime.h"

__launch_bounds__(128, 7) void Test2Args(void);
__launch_bounds__(128) void Test1Arg(void);

__launch_bounds__(0xffffffff) void TestMaxArg(void);
__launch_bounds__(0x100000000) void TestTooBigArg(void); // expected-error {{integer constant expression evaluates to value 4294967296 that cannot be represented in a 32-bit unsigned integer type}}
__launch_bounds__(0x10000000000000000) void TestWayTooBigArg(void); // expected-error {{integer literal is too large to be represented in any integer type}}

__launch_bounds__(-128, 7) void TestNegArg1(void); // expected-warning {{'launch_bounds' attribute parameter 0 is negative and will be ignored}}
__launch_bounds__(128, -7) void TestNegArg2(void); // expected-warning {{'launch_bounds' attribute parameter 1 is negative and will be ignored}}

__launch_bounds__(1, 2, 3) void Test3Args(void); // expected-error {{'launch_bounds' attribute takes no more than 2 arguments}}
__launch_bounds__() void TestNoArgs(void); // expected-error {{'launch_bounds' attribute takes at least 1 argument}}

int TestNoFunction __launch_bounds__(128, 7); // expected-warning {{'launch_bounds' attribute only applies to Objective-C methods, functions, and function pointers}}

__launch_bounds__(true) void TestBool(void);
__launch_bounds__(128.0) void TestFP(void); // expected-error {{'launch_bounds' attribute requires parameter 0 to be an integer constant}}
__launch_bounds__((void*)0) void TestNullptr(void); // expected-error {{'launch_bounds' attribute requires parameter 0 to be an integer constant}}

int nonconstint = 256;
__launch_bounds__(nonconstint) void TestNonConstInt(void); // expected-error {{'launch_bounds' attribute requires parameter 0 to be an integer constant}}

const int constint = 512;
__launch_bounds__(constint) void TestConstInt(void);
__launch_bounds__(constint * 2 + 3) void TestConstIntExpr(void);

template <int a, int b> __launch_bounds__(a, b) void TestTemplate2Args(void) {}
template void TestTemplate2Args<128,7>(void);

template <int a> __launch_bounds__(a) void TestTemplate1Arg(void) {}
template void TestTemplate1Arg<128>(void);

template <class a>
__launch_bounds__(a) void TestTemplate1ArgClass(void) {} // expected-error {{'a' does not refer to a value}}
// expected-note@-2 {{declared here}}

template <int a, int b, int c>
__launch_bounds__(a + b, c + constint) void TestTemplateExpr(void) {}
template void TestTemplateExpr<128+constint, 3, 7>(void);

template <int... Args>
__launch_bounds__(Args) void TestTemplateVariadicArgs(void) {} // expected-error {{expression contains unexpanded parameter pack 'Args'}}

template <int... Args>
__launch_bounds__(1, Args) void TestTemplateVariadicArgs2(void) {} // expected-error {{expression contains unexpanded parameter pack 'Args'}}
