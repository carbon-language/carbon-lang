#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -fcxx-exceptions -fcuda-is-device -fsyntax-only -std=c++17 -verify %s

#include "Inputs/hip/hip_runtime.h"

// Error, instantiated on device.
inline __host__ __device__ void hasInvalid() {
  throw NULL;
  // expected-error@-1 2{{cannot use 'throw' in __host__ __device__ function}}
}

inline __host__ __device__ void hasInvalid2() {
  throw NULL;
  // expected-error@-1 2{{cannot use 'throw' in __host__ __device__ function}}
}

inline __host__ __device__ void hasInvalidDiscarded() {
  // This is only used in the discarded statements below, so this should not diagnose.
  throw NULL;
}

static __device__ void use0() {
  hasInvalid(); // expected-note {{called by 'use0'}}
  hasInvalid(); // expected-note {{called by 'use0'}}

  if constexpr (true) {
    hasInvalid2(); // expected-note {{called by 'use0'}}
  } else {
    hasInvalidDiscarded();
  }

  if constexpr (false) {
    hasInvalidDiscarded();
  } else {
    hasInvalid2(); // expected-note {{called by 'use0'}}
  }

  if constexpr (false) {
    hasInvalidDiscarded();
  }
}

// To avoid excessive diagnostic messages, deferred diagnostics are only
// emitted the first time a function is called.
static __device__ void use1() {
  use0(); // expected-note 4{{called by 'use1'}}
  use0();
}

static __device__ void use2() {
  use1(); // expected-note 4{{called by 'use2'}}
  use1();
}

static __device__ void use3() {
  use2(); // expected-note 4{{called by 'use3'}}
  use2();
}

__global__ void use4() {
  use3(); // expected-note 4{{called by 'use4'}}
  use3();
}
