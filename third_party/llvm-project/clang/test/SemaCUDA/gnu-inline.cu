// RUN: %clang_cc1 -fsyntax-only -verify %s

#include "Inputs/hip/hip_runtime.h"

// expected-no-diagnostics

// Check that we can handle gnu_inline functions when compiling in CUDA mode.

void foo();
extern inline __attribute__((gnu_inline)) void bar() { foo(); }
