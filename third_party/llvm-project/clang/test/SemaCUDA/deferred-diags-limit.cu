#include "hip/hip_runtime.h"
// RUN: not %clang_cc1 -fcxx-exceptions -fcuda-is-device -fsyntax-only \
// RUN:   -ferror-limit 2 2>&1 %s | FileCheck %s

#include "Inputs/hip/hip_runtime.h"

// CHECK: cannot use 'throw' in __host__ __device__ function
// CHECK: cannot use 'throw' in __host__ __device__ function
// CHECK-NOT: cannot use 'throw' in __host__ __device__ function
// CHECK: too many errors emitted, stopping now

inline __host__ __device__ void hasInvalid() {
  throw NULL;
}

__global__ void use0() {
  hasInvalid();
  hasInvalid();
  hasInvalid();
  hasInvalid();
}
