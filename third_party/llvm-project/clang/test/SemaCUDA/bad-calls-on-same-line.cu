// RUN: %clang_cc1 -fsyntax-only -verify %s

// The hd function template is instantiated three times.
//
// Two of those instantiations call a device function, which is an error when
// compiling for host.  Clang should report both errors.

#include "Inputs/hip/hip_runtime.h"

template <typename T>
struct Selector {};

template <>
struct Selector<int> {
  __host__ void f() {}
};

template <>
struct Selector<float> {
  __device__ void f() {} // expected-note {{declared here}}
};

template <>
struct Selector<double> {
  __device__ void f() {} // expected-note {{declared here}}
};

template <typename T>
inline __host__ __device__ void hd() {
  Selector<T>().f();
  // expected-error@-1 2 {{reference to __device__ function}}
}

void host_fn() {
  hd<int>();
  hd<double>();
  // expected-note@-1 {{called by 'host_fn'}}
  hd<float>();
  // expected-note@-1 {{called by 'host_fn'}}
}
