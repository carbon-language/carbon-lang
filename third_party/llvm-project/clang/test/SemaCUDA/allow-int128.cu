
#include <hip/hip_runtime.h>
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa \
// RUN:   -aux-triple x86_64-unknown-linux-gnu \
// RUN:   -fcuda-is-device -verify -fsyntax-only %s
// RUN: %clang_cc1 -triple nvptx \
// RUN:   -aux-triple x86_64-unknown-linux-gnu \
// RUN:   -fcuda-is-device -verify -fsyntax-only %s

// expected-no-diagnostics
#define __device__ __attribute__((device))

__int128 h_glb;
__device__ __int128 d_unused;
__device__ __int128 d_glb;
__device__ __int128 bar() {
  return d_glb;
}
