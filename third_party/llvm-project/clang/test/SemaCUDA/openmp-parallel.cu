// RUN: %clang_cc1 -fopenmp -fsyntax-only -verify %s
// RUN: %clang_cc1 -fopenmp -fexceptions -fsyntax-only -verify %s

#include "Inputs/hip/hip_runtime.h"

__device__ void foo(int) {} // expected-note {{candidate function not viable: call to __device__ function from __host__ function}}
// expected-note@-1 {{'foo' declared here}}

int main() {
  #pragma omp parallel
  for (int i = 0; i < 100; i++) {
    foo(1); // expected-error {{no matching function for call to 'foo'}}
    new int;
  }

  auto Lambda = []() {
    #pragma omp parallel
    for (int i = 0; i < 100; i++) {
      foo(1); // expected-error {{reference to __device__ function 'foo' in __host__ __device__ function}}
      new int;
    }
  };
  Lambda(); // expected-note {{called by 'main'}}
}
