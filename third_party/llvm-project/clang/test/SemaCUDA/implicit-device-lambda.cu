#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -std=c++11 -fcuda-is-device -verify=dev,expected -fsyntax-only \
// RUN:   -verify-ignore-unexpected=warning -verify-ignore-unexpected=note %s
// RUN: %clang_cc1 -std=c++11 -verify -fsyntax-only \
// RUN:   -verify-ignore-unexpected=warning -verify-ignore-unexpected=note %s

#include "Inputs/hip/hip_runtime.h"

__device__ void device_fn() {
  auto f1 = [&] {};
  f1(); // implicitly __device__

  auto f2 = [&] __device__ {};
  f2();

  auto f3 = [&] __host__ {};
  f3();  // expected-error {{no matching function}}

  auto f4 = [&] __host__ __device__ {};
  f4();

  // Now do it all again with '()'s in the lambda declarations: This is a
  // different parse path.
  auto g1 = [&]() {};
  g1(); // implicitly __device__

  auto g2 = [&]() __device__ {};
  g2();

  auto g3 = [&]() __host__ {};
  g3();  // expected-error {{no matching function}}

  auto g4 = [&]() __host__ __device__ {};
  g4();

  // Once more, with the '()'s in a different place.
  auto h1 = [&]() {};
  h1(); // implicitly __device__

  auto h2 = [&] __device__ () {};
  h2();

  auto h3 = [&] __host__ () {};
  h3();  // expected-error {{no matching function}}

  auto h4 = [&] __host__ __device__ () {};
  h4();
}

// Behaves identically to device_fn.
__global__ void kernel_fn() {
  auto f1 = [&] {};
  f1(); // implicitly __device__

  auto f2 = [&] __device__ {};
  f2();

  auto f3 = [&] __host__ {};
  f3();  // expected-error {{no matching function}}

  auto f4 = [&] __host__ __device__ {};
  f4();

  // No need to re-test all the parser contortions we test in the device
  // function.
}

__host__ void host_fn() {
  auto f1 = [&] {};
  f1(); // implicitly __host__ (i.e., no magic)

  auto f2 = [&] __device__ {};
  f2();  // expected-error {{no matching function}}

  auto f3 = [&] __host__ {};
  f3();

  auto f4 = [&] __host__ __device__ {};
  f4();
}

__host__ __device__ void hd_fn() {
  auto f1 = [&] {};
  f1(); // implicitly __host__ __device__

  auto f2 = [&] __device__ {};
  f2();
#ifndef __CUDA_ARCH__
  // expected-error@-2 {{reference to __device__ function}}
#endif

  auto f3 = [&] __host__ {};
  f3();
#ifdef __CUDA_ARCH__
  // expected-error@-2 {{reference to __host__ function}}
#endif

  auto f4 = [&] __host__ __device__ {};
  f4();
}

// The special treatment above only applies to lambdas.
__device__ void foo() {
  struct X {
    void foo() {}
  };
  X x;
  x.foo(); // dev-error {{reference to __host__ function 'foo' in __device__ function}}
}
