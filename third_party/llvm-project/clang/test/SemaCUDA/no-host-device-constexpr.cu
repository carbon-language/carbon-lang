// RUN: %clang_cc1 -std=c++11 -fsyntax-only -fno-cuda-host-device-constexpr -verify %s
// RUN: %clang_cc1 -std=c++11 -fsyntax-only -fno-cuda-host-device-constexpr -fcuda-is-device -verify %s

#include "Inputs/hip/hip_runtime.h"

// Check that, with -fno-cuda-host-device-constexpr, constexpr functions are
// host-only, and __device__ constexpr functions are still device-only.

constexpr int f() { return 0; } // expected-note {{not viable}}
__device__ constexpr int g() { return 0; } // expected-note {{not viable}}

void __device__ foo() {
  f(); // expected-error {{no matching function}}
  g();
}

void __host__ foo() {
  f();
  g(); // expected-error {{no matching function}}
}
