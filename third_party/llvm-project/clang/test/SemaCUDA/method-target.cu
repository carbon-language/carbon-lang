// RUN: %clang_cc1 -fsyntax-only -verify=host,expected %s
// RUN: %clang_cc1 -fcuda-is-device -fsyntax-only -verify=dev,expected %s

#include "Inputs/hip/hip_runtime.h"

//------------------------------------------------------------------------------
// Test 1: host method called from device function

struct S1 {
  void method() {} // dev-note {{'method' declared here}}
};

__device__ void foo1(S1& s) {
  s.method(); // dev-error {{reference to __host__ function 'method' in __device__ function}}
}

//------------------------------------------------------------------------------
// Test 2: host method called from device function, for overloaded method

struct S2 {
  void method(int) {} // expected-note {{candidate function not viable: call to __host__ function from __device__ function}}
  void method(float) {} // expected-note {{candidate function not viable: call to __host__ function from __device__ function}}
};

__device__ void foo2(S2& s, int i, float f) {
  s.method(f); // expected-error {{no matching member function}}
}

//------------------------------------------------------------------------------
// Test 3: device method called from host function

struct S3 {
  __device__ void method() {} // host-note {{'method' declared here}}
};

void foo3(S3& s) {
  s.method(); // host-error {{reference to __device__ function 'method' in __host__ function}}
}

//------------------------------------------------------------------------------
// Test 4: device method called from host&device function

struct S4 {
  __device__ void method() {}  // host-note {{'method' declared here}}
};

__host__ __device__ void foo4(S4& s) {
  s.method(); // host-error {{reference to __device__ function 'method' in __host__ __device__ function}}
}

//------------------------------------------------------------------------------
// Test 5: overloaded operators

struct S5 {
  S5() {}
  S5& operator=(const S5&) {return *this;} // expected-note {{candidate function not viable}}
};

__device__ void foo5(S5& s, S5& t) {
  s = t; // expected-error {{no viable overloaded '='}}
}

//------------------------------------------------------------------------------
// Test 6: call method through pointer

struct S6 {
  void method() {} // dev-note {{'method' declared here}};
};

__device__ void foo6(S6* s) {
  s->method(); // dev-error {{reference to __host__ function 'method' in __device__ function}}
}
