// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target
// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fcuda-is-device -fsyntax-only \
// RUN:   -verify -DEXPECT_VA_ARG_ERR -DEXPECT_VARARG_ERR %s
// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fcuda-is-device -fsyntax-only \
// RUN:   -fcuda-allow-variadic-functions -verify -DEXPECT_VA_ARG_ERR %s
// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -fsyntax-only -verify \
// RUN:   -DEXPECT_VARARG_ERR %s

#include <stdarg.h>
#include "Inputs/hip/hip_runtime.h"

__device__ void foo() {
  va_list list;
  va_arg(list, int);
#ifdef EXPECT_VA_ARG_ERR
  // expected-error@-2 {{CUDA device code does not support va_arg}}
#endif
}

void bar() {
  va_list list;
  va_arg(list, int);  // OK: host-only
}

__device__ void baz() {
#if !defined(__CUDA_ARCH__)
  va_list list;
  va_arg(list, int);  // OK: only seen when compiling for host
#endif
}

__device__ void vararg(const char* x, ...) {}
#ifdef EXPECT_VARARG_ERR
// expected-error@-2 {{CUDA device code does not support variadic functions}}
#endif

template <typename T>
__device__ void vararg(T t, ...) {}
#ifdef EXPECT_VARARG_ERR
// expected-error@-2 {{CUDA device code does not support variadic functions}}
#endif

extern "C" __device__ int printf(const char* fmt, ...);  // OK, special case.

// Definition of printf not allowed.
extern "C" __device__ int printf(const char* fmt, ...) { return 0; }
#ifdef EXPECT_VARARG_ERR
// expected-error@-2 {{CUDA device code does not support variadic functions}}
#endif

namespace ns {
__device__ int printf(const char* fmt, ...);
#ifdef EXPECT_VARARG_ERR
// expected-error@-2 {{CUDA device code does not support variadic functions}}
#endif
}
