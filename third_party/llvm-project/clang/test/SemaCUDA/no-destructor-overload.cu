// RUN: %clang_cc1 -fsyntax-only -verify %s
// RUN: %clang_cc1 -fcuda-is-device -fsyntax-only -verify %s

#include "Inputs/hip/hip_runtime.h"

// We don't allow destructors to be overloaded.  Making this work would be a
// giant change to clang, and the use cases seem quite limited.

struct A {
  ~A() {} // expected-note {{previous definition is here}}
  __device__ ~A() {} // expected-error {{destructor cannot be redeclared}}
};

struct B {
  __host__ ~B() {} // expected-note {{previous definition is here}}
  __host__ __device__ ~B() {} // expected-error {{destructor cannot be redeclared}}
};

struct C {
  __host__ __device__ ~C() {} // expected-note {{previous definition is here}}
  __host__ ~C() {} // expected-error {{destructor cannot be redeclared}}
};

struct D {
  __device__ ~D() {} // expected-note {{previous definition is here}}
  __host__ __device__ ~D() {} // expected-error {{destructor cannot be redeclared}}
};

struct E {
  __host__ __device__ ~E() {} // expected-note {{previous definition is here}}
  __device__ ~E() {} // expected-error {{destructor cannot be redeclared}}
};

