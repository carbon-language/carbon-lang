// expected-no-diagnostics

// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -fsyntax-only -verify %s
// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fsyntax-only -fcuda-is-device -verify %s

#include "Inputs/hip/hip_runtime.h"

struct S {
  __host__ static void operator delete(void*, size_t) {}
  __device__ static void operator delete(void*, size_t) {}
};

__host__ __device__ void test(S* s) {
  // This shouldn't be ambiguous -- we call the host overload in host mode and
  // the device overload in device mode.
  delete s;
}

// Code should work with no explicit declarations/definitions of
// allocator functions.
__host__ __device__ void test_default_global_delete_hd(int *ptr) {
  // Again, there should be no ambiguity between which operator delete we call.
  ::delete ptr;
}

__device__ void test_default_global_delete(int *ptr) {
  // Again, there should be no ambiguity between which operator delete we call.
  ::delete ptr;
}
__host__ void test_default_global_delete(int *ptr) {
  // Again, there should be no ambiguity between which operator delete we call.
  ::delete ptr;
}

// It should work with only some of allocators (re-)declared.
__device__ void operator delete(void *ptr);

__host__ __device__ void test_partial_global_delete_hd(int *ptr) {
  // Again, there should be no ambiguity between which operator delete we call.
  ::delete ptr;
}

__device__ void test_partial_global_delete(int *ptr) {
  // Again, there should be no ambiguity between which operator delete we call.
  ::delete ptr;
}
__host__ void test_partial_global_delete(int *ptr) {
  // Again, there should be no ambiguity between which operator delete we call.
  ::delete ptr;
}


// We should be able to define both host and device variants.
__host__ void operator delete(void *ptr) {}
__device__ void operator delete(void *ptr) {}

__host__ __device__ void test_overloaded_global_delete_hd(int *ptr) {
  // Again, there should be no ambiguity between which operator delete we call.
  ::delete ptr;
}

__device__ void test_overloaded_global_delete(int *ptr) {
  // Again, there should be no ambiguity between which operator delete we call.
  ::delete ptr;
}
__host__ void test_overloaded_global_delete(int *ptr) {
  // Again, there should be no ambiguity between which operator delete we call.
  ::delete ptr;
}
