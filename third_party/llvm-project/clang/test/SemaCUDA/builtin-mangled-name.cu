#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -triple x86_64-unknown-gnu-linux -aux-triple amdgcn-amd-amdhsa \
// RUN:   -verify -fsyntax-only -x hip %s

#include "Inputs/hip/hip_runtime.h"

__global__ void kern1();
int y;

void fun1() {
  int x;
  const char *p;
  p = __builtin_get_device_side_mangled_name();
  // expected-error@-1 {{invalid argument: symbol must be a device-side function or global variable}}
  p = __builtin_get_device_side_mangled_name(kern1, kern1);
  // expected-error@-1 {{invalid argument: symbol must be a device-side function or global variable}}
  p = __builtin_get_device_side_mangled_name(1);
  // expected-error@-1 {{invalid argument: symbol must be a device-side function or global variable}}
  p = __builtin_get_device_side_mangled_name(x);
  // expected-error@-1 {{invalid argument: symbol must be a device-side function or global variable}}
  p = __builtin_get_device_side_mangled_name(fun1);
  // expected-error@-1 {{invalid argument: symbol must be a device-side function or global variable}}
  p = __builtin_get_device_side_mangled_name(y);
  // expected-error@-1 {{invalid argument: symbol must be a device-side function or global variable}}
}
