#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -std=c++11 -fsyntax-only -verify=host \
// RUN:   -verify-ignore-unexpected=note %s
// RUN: %clang_cc1 -std=c++11 -fcuda-is-device -fsyntax-only -verify=dev \
// RUN:   -verify-ignore-unexpected=note %s

// Check that we can reference (get a function pointer to) a __global__
// function from the host side, but not the device side.  (We don't yet support
// device-side kernel launches.)

// host-no-diagnostics

#include "Inputs/hip/hip_runtime.h"

struct Dummy {};

__global__ void kernel() {}

typedef void (*fn_ptr_t)();

__host__ __device__ fn_ptr_t get_ptr_hd() {
  return kernel;
  // dev-error@-1 {{reference to __global__ function}}
}
__host__ fn_ptr_t get_ptr_h() {
  return kernel;
}
__device__ fn_ptr_t get_ptr_d() {
  return kernel;  // dev-error {{reference to __global__ function}}
}
