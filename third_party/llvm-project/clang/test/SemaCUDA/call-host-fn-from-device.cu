#include "hip/hip_runtime.h"
// RUN: %clang_cc1 %s --std=c++11 -triple nvptx-unknown-unknown -fcuda-is-device \
// RUN:   -emit-llvm -o /dev/null -verify -verify-ignore-unexpected=note

// Note: This test won't work with -fsyntax-only, because some of these errors
// are emitted during codegen.

#include "Inputs/hip/hip_runtime.h"

extern "C" void host_fn() {}
// expected-note@-1 7 {{'host_fn' declared here}}

struct Dummy {};

struct S {
  S() {}
  // expected-note@-1 2 {{'S' declared here}}
  ~S() { host_fn(); }
  // expected-note@-1 {{'~S' declared here}}
  int x;
};

struct T {
  __host__ __device__ void hd() { host_fn(); }
  // expected-error@-1 {{reference to __host__ function 'host_fn' in __host__ __device__ function}}

  // No error; this is (implicitly) inline and is never called, so isn't
  // codegen'ed.
  __host__ __device__ void hd2() { host_fn(); }

  __host__ __device__ void hd3();

  void h() {}
  // expected-note@-1 {{'h' declared here}}

  void operator+();
  // expected-note@-1 {{'operator+' declared here}}

  void operator-(const T&) {}
  // expected-note@-1 {{'operator-' declared here}}

  operator Dummy() { return Dummy(); }
  // expected-note@-1 {{'operator Dummy' declared here}}

  __host__ void operator delete(void *) { host_fn(); };
  __device__ void operator delete(void*, __SIZE_TYPE__);
};

struct U {
  __device__ void operator delete(void*, __SIZE_TYPE__) = delete;
  __host__ __device__ void operator delete(void*);
};

__host__ __device__ void T::hd3() {
  host_fn();
  // expected-error@-1 {{reference to __host__ function 'host_fn' in __host__ __device__ function}}
}

template <typename T> __host__ __device__ void hd2() { host_fn(); }
// expected-error@-1 {{reference to __host__ function 'host_fn' in __host__ __device__ function}}
__global__ void kernel() { hd2<int>(); }

__host__ __device__ void hd() { host_fn(); }
// expected-error@-1 {{reference to __host__ function 'host_fn' in __host__ __device__ function}}

template <typename T> __host__ __device__ void hd3() { host_fn(); }
// expected-error@-1 {{reference to __host__ function 'host_fn' in __host__ __device__ function}}
__device__ void device_fn() { hd3<int>(); }

// No error because this is never instantiated.
template <typename T> __host__ __device__ void hd4() { host_fn(); }

__host__ __device__ void local_var() {
  S s;
  // expected-error@-1 {{reference to __host__ function 'S' in __host__ __device__ function}}
}

__host__ __device__ void placement_new(char *ptr) {
  ::new(ptr) S();
  // expected-error@-1 {{reference to __host__ function 'S' in __host__ __device__ function}}
}

__host__ __device__ void explicit_destructor(S *s) {
  s->~S();
  // expected-error@-1 {{reference to __host__ function '~S' in __host__ __device__ function}}
}

__host__ __device__ void class_specific_delete(T *t, U *u) {
  delete t; // ok, call sized device delete even though host has preferable non-sized version
  delete u; // ok, call non-sized HD delete rather than sized D delete
}

__host__ __device__ void hd_member_fn() {
  T t;
  // Necessary to trigger an error on T::hd.  It's (implicitly) inline, so
  // isn't codegen'ed until we call it.
  t.hd();
}

__host__ __device__ void h_member_fn() {
  T t;
  t.h();
  // expected-error@-1 {{reference to __host__ function 'h' in __host__ __device__ function}}
}

__host__ __device__ void fn_ptr() {
  auto* ptr = &host_fn;
  // expected-error@-1 {{reference to __host__ function 'host_fn' in __host__ __device__ function}}
}

template <typename T>
__host__ __device__ void fn_ptr_template() {
  auto* ptr = &host_fn;  // Not an error because the template isn't instantiated.
}

__host__ __device__ void unaryOp() {
  T t;
  (void) +t; // expected-error {{reference to __host__ function 'operator+' in __host__ __device__ function}}
}

__host__ __device__ void binaryOp() {
  T t;
  (void) (t - t); // expected-error {{reference to __host__ function 'operator-' in __host__ __device__ function}}
}

__host__ __device__ void implicitConversion() {
  T t;
  Dummy d = t; // expected-error {{reference to __host__ function 'operator Dummy' in __host__ __device__ function}}
}

template <typename T>
struct TmplStruct {
  template <typename U> __host__ __device__ void fn() {}
};

template <>
template <>
__host__ __device__ void TmplStruct<int>::fn<int>() { host_fn(); }
// expected-error@-1 {{reference to __host__ function 'host_fn' in __host__ __device__ function}}

__device__ void double_specialization() { TmplStruct<int>().fn<int>(); }
