// RUN: %clang_cc1 %s --std=c++11 -triple nvptx-unknown-unknown -fcuda-is-device \
// RUN:   -emit-llvm -o /dev/null -verify=device
// RUN: %clang_cc1 %s --std=c++11 -triple nvptx-unknown-unknown \
// RUN:   -emit-llvm -o /dev/null -verify=host
// RUN: %clang_cc1 %s --std=c++17 -triple nvptx-unknown-unknown -fcuda-is-device \
// RUN:   -emit-llvm -o /dev/null -verify=device
// RUN: %clang_cc1 %s --std=c++17 -triple nvptx-unknown-unknown \
// RUN:   -emit-llvm -o /dev/null -verify=host

#include "Inputs/hip/hip_runtime.h"
extern __host__ void host_fn();
extern __device__ void dev_fn();
extern __host__ __device__ void hd_fn();

struct H1D1 {
  __host__ void operator delete(void *) { host_fn(); };
  __device__ void operator delete(void *) { dev_fn(); };
};

struct h1D1 {
  __host__ void operator delete(void *) = delete;
  // host-note@-1 {{'operator delete' has been explicitly marked deleted here}}
  __device__ void operator delete(void *) { dev_fn(); };
};

struct H1d1 {
  __host__ void operator delete(void *) { host_fn(); };
  __device__ void operator delete(void *) = delete;
  // device-note@-1 {{'operator delete' has been explicitly marked deleted here}}
};

struct H1D2 {
  __host__ void operator delete(void *) { host_fn(); };
  __device__ void operator delete(void *, __SIZE_TYPE__) { dev_fn(); };
};

struct H2D1 {
  __host__ void operator delete(void *, __SIZE_TYPE__) { host_fn(); };
  __device__ void operator delete(void *) { dev_fn(); };
};

struct H2D2 {
  __host__ void operator delete(void *, __SIZE_TYPE__) { host_fn(); };
  __device__ void operator delete(void *, __SIZE_TYPE__) { dev_fn(); };
};

struct H1D1D2 {
  __host__ void operator delete(void *) { host_fn(); };
  __device__ void operator delete(void *) { dev_fn(); };
  __device__ void operator delete(void *, __SIZE_TYPE__) { dev_fn(); };
};

struct H1H2D1 {
  __host__ void operator delete(void *) { host_fn(); };
  __host__ void operator delete(void *, __SIZE_TYPE__) { host_fn(); };
  __device__ void operator delete(void *) { dev_fn(); };
};

struct H1H2D2 {
  __host__ void operator delete(void *) { host_fn(); };
  __host__ void operator delete(void *, __SIZE_TYPE__) { host_fn(); };
  __device__ void operator delete(void *, __SIZE_TYPE__) { dev_fn(); };
};

struct H1H2D1D2 {
  __host__ void operator delete(void *) { host_fn(); };
  __host__ void operator delete(void *, __SIZE_TYPE__) { host_fn(); };
  __device__ void operator delete(void *) { dev_fn(); };
  __device__ void operator delete(void *, __SIZE_TYPE__) { dev_fn(); };
};


template <typename T>
__host__ __device__ void test_hd(void *p) {
  T *t = (T *)p;
  delete t;
  // host-error@-1 {{attempt to use a deleted function}}
  // device-error@-2 {{attempt to use a deleted function}}
}

__host__ __device__ void tests_hd(void *t) {
  test_hd<H1D1>(t);
  test_hd<h1D1>(t);
  // host-note@-1 {{in instantiation of function template specialization 'test_hd<h1D1>' requested here}}
  test_hd<H1d1>(t);
  // device-note@-1 {{in instantiation of function template specialization 'test_hd<H1d1>' requested here}}
  test_hd<H1D2>(t);
  test_hd<H2D1>(t);
  test_hd<H2D2>(t);
  test_hd<H1D1D2>(t);
  test_hd<H1H2D1>(t);
  test_hd<H1H2D1>(t);
  test_hd<H1H2D2>(t);
  test_hd<H1H2D1D2>(t);
}

// This should produce no errors.  Defaulted destructor should be treated as HD,
// which allows referencing host-only `operator delete` with a deferred
// diagnostics that would fire if we ever attempt to codegen it on device..
struct H {
  virtual ~H() = default;
  static void operator delete(void *) {}
};
H h;
