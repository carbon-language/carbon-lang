// RUN: %clang_cc1 -std=gnu++11 -fsyntax-only -verify %s -Wno-defaulted-function-deleted

#include "Inputs/hip/hip_runtime.h"

//------------------------------------------------------------------------------
// Test 1: infer default ctor to be host.

struct A1_with_host_ctor {
  A1_with_host_ctor() {}
};

// The implicit default constructor is inferred to be host because it only needs
// to invoke a single host constructor (A1_with_host_ctor's). So we'll encounter
// an error when calling it from a __device__ function, but not from a __host__
// function.
struct B1_with_implicit_default_ctor : A1_with_host_ctor {
};

// expected-note@-3 {{call to __host__ function from __device__}}
// expected-note@-4 {{candidate constructor (the implicit copy constructor) not viable}}
// expected-note@-5 {{candidate constructor (the implicit move constructor) not viable}}

void hostfoo() {
  B1_with_implicit_default_ctor b;
}

__device__ void devicefoo() {
  B1_with_implicit_default_ctor b; // expected-error {{no matching constructor}}
}

//------------------------------------------------------------------------------
// Test 2: infer default ctor to be device.

struct A2_with_device_ctor {
  __device__ A2_with_device_ctor() {}
};

struct B2_with_implicit_default_ctor : A2_with_device_ctor {
};

// expected-note@-3 {{call to __device__ function from __host__}}
// expected-note@-4 {{candidate constructor (the implicit copy constructor) not viable}}
// expected-note@-5 {{candidate constructor (the implicit move constructor) not viable}}

void hostfoo2() {
  B2_with_implicit_default_ctor b;  // expected-error {{no matching constructor}}
}

__device__ void devicefoo2() {
  B2_with_implicit_default_ctor b;
}

//------------------------------------------------------------------------------
// Test 3: infer copy ctor

struct A3_with_device_ctors {
  __host__ A3_with_device_ctors() {}
  __device__ A3_with_device_ctors(const A3_with_device_ctors&) {}
};

struct B3_with_implicit_ctors : A3_with_device_ctors {
};
// expected-note@-2 2{{call to __device__ function from __host__ function}}
// expected-note@-3 {{default constructor}}


void hostfoo3() {
  B3_with_implicit_ctors b;  // this is OK because the inferred default ctor
                             // here is __host__
  B3_with_implicit_ctors b2 = b; // expected-error {{no matching constructor}}

}

//------------------------------------------------------------------------------
// Test 4: infer default ctor from a field, not a base

struct A4_with_host_ctor {
  A4_with_host_ctor() {}
};

struct B4_with_implicit_default_ctor {
  A4_with_host_ctor field;
};

// expected-note@-4 {{call to __host__ function from __device__}}
// expected-note@-5 {{candidate constructor (the implicit copy constructor) not viable}}
// expected-note@-6 {{candidate constructor (the implicit move constructor) not viable}}

void hostfoo4() {
  B4_with_implicit_default_ctor b;
}

__device__ void devicefoo4() {
  B4_with_implicit_default_ctor b; // expected-error {{no matching constructor}}
}

//------------------------------------------------------------------------------
// Test 5: copy ctor with non-const param

struct A5_copy_ctor_constness {
  __host__ A5_copy_ctor_constness() {}
  __host__ A5_copy_ctor_constness(A5_copy_ctor_constness&) {}
};

struct B5_copy_ctor_constness : A5_copy_ctor_constness {
};

// expected-note@-3 {{candidate constructor (the implicit copy constructor) not viable: call to __host__ function from __device__ function}}
// expected-note@-4 {{candidate constructor (the implicit default constructor) not viable}}

void hostfoo5(B5_copy_ctor_constness& b_arg) {
  B5_copy_ctor_constness b = b_arg;
}

__device__ void devicefoo5(B5_copy_ctor_constness& b_arg) {
  B5_copy_ctor_constness b = b_arg; // expected-error {{no matching constructor}}
}

//------------------------------------------------------------------------------
// Test 6: explicitly defaulted ctor: since they are spelled out, they have
// a host/device designation explicitly so no inference needs to be done.

struct A6_with_device_ctor {
  __device__ A6_with_device_ctor() {}
};

struct B6_with_defaulted_ctor : A6_with_device_ctor {
  __host__ B6_with_defaulted_ctor() = default;
};

// expected-note@-3 {{candidate constructor not viable: call to __host__ function from __device__ function}}
// expected-note@-5 {{candidate constructor (the implicit copy constructor) not viable}}
// expected-note@-6 {{candidate constructor (the implicit move constructor) not viable}}

__device__ void devicefoo6() {
  B6_with_defaulted_ctor b; // expected-error {{no matching constructor}}
}

//------------------------------------------------------------------------------
// Test 7: copy assignment operator

struct A7_with_copy_assign {
  A7_with_copy_assign() {}
  __device__ A7_with_copy_assign& operator=(const A7_with_copy_assign&) {}
};

struct B7_with_copy_assign : A7_with_copy_assign {
};

// expected-note@-3 {{candidate function (the implicit copy assignment operator) not viable: call to __device__ function from __host__ function}}
// expected-note@-4 {{candidate function (the implicit move assignment operator) not viable: call to __device__ function from __host__ function}}

void hostfoo7() {
  B7_with_copy_assign b1, b2;
  b1 = b2; // expected-error {{no viable overloaded '='}}
}

//------------------------------------------------------------------------------
// Test 8: move assignment operator

// definitions for std::move
namespace std {
inline namespace foo {
template <class T> struct remove_reference { typedef T type; };
template <class T> struct remove_reference<T&> { typedef T type; };
template <class T> struct remove_reference<T&&> { typedef T type; };

template <class T> typename remove_reference<T>::type&& move(T&& t);
}
}

struct A8_with_move_assign {
  A8_with_move_assign() {}
  __device__ A8_with_move_assign& operator=(A8_with_move_assign&&) {}
  __device__ A8_with_move_assign& operator=(const A8_with_move_assign&) {}
};

struct B8_with_move_assign : A8_with_move_assign {
};

// expected-note@-3 {{candidate function (the implicit copy assignment operator) not viable: call to __device__ function from __host__ function}}
// expected-note@-4 {{candidate function (the implicit move assignment operator) not viable: call to __device__ function from __host__ function}}

void hostfoo8() {
  B8_with_move_assign b1, b2;
  b1 = std::move(b2); // expected-error {{no viable overloaded '='}}
}
