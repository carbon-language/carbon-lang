#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -fsyntax-only -verify %s

#include "Inputs/hip/hip_runtime.h"

__global__ void g1(int x) {}

template <typename T> void t1(T arg) {
  g1<<<arg, arg>>>(1);
}

void h1(int x) {}
int h2(int x) { return 1; }

int main(void) {
  g1<<<1, 1>>>(42);
  g1(42); // expected-error {{call to global function 'g1' not configured}}
  g1<<<1>>>(42); // expected-error {{too few execution configuration arguments to kernel function call}}
  g1<<<1, 1, 0, 0, 0>>>(42); // expected-error {{too many execution configuration arguments to kernel function call}}

  t1(1);

  h1<<<1, 1>>>(42); // expected-error {{kernel call to non-global function 'h1'}}

  int (*fp)(int) = h2;
  fp<<<1, 1>>>(42); // expected-error {{must have void return type}}

  g1<<<undeclared, 1>>>(42); // expected-error {{use of undeclared identifier 'undeclared'}}
}

// Make sure we can call static member kernels.
template <typename > struct a0 {
  template <typename T> static __global__ void Call(T);
};
struct a1 {
  template <typename T> static __global__ void Call(T);
};
template <typename T> struct a2 {
  static __global__ void Call(T);
};
struct a3 {
  static __global__ void Call(int);
  static __global__ void Call(void*);
};

struct b {
  template <typename c> void d0(c arg) {
    a0<c>::Call<<<0, 0>>>(arg);
    a1::Call<<<0,0>>>(arg);
    a2<c>::Call<<<0,0>>>(arg);
    a3::Call<<<0, 0>>>(arg);
  }
  void d1(void* arg) {
    a0<void*>::Call<<<0, 0>>>(arg);
    a1::Call<<<0,0>>>(arg);
    a2<void*>::Call<<<0,0>>>(arg);
    a3::Call<<<0, 0>>>(arg);
  }
  void e() { d0(1); }
};
