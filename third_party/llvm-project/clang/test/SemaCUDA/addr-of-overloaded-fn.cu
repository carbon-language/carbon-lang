// expected-no-diagnostics

// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -fsyntax-only -verify %s
// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fsyntax-only -fcuda-is-device -verify %s

#include "Inputs/hip/hip_runtime.h"

__host__ void overload() {}
__device__ void overload() {}

__host__ __device__ void test_hd() {
  // This should not be ambiguous -- we choose the host or the device overload
  // depending on whether or not we're compiling for host or device.
  void (*x)() = overload;
}

// These also shouldn't be ambiguous, but they're an easier test than the HD
// function above.
__host__ void test_host() {
  void (*x)() = overload;
}
__device__ void test_device() {
  void (*x)() = overload;
}
