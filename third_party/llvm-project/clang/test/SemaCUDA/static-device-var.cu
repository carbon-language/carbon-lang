#include "hip/hip_runtime.h"
// REQUIRES: x86-registered-target
// REQUIRES: amdgpu-registered-target

// RUN: %clang_cc1 -triple nvptx -fcuda-is-device -std=c++11 \
// RUN:    -emit-llvm -o - %s -fsyntax-only -verify=dev,com

// RUN: %clang_cc1 -triple x86_64-gnu-linux -std=c++11 \
// RUN:    -emit-llvm -o - %s -fsyntax-only -verify=host,com

// Checks allowed usage of file-scope and function-scope static variables.

#include "Inputs/hip/hip_runtime.h"

// Checks static variables are allowed in device functions.

__device__ void f1() {
  const static int b = 123;
  static int a;
}

// Checks static variables are allowd in global functions.

__global__ void k1() {
  const static int b = 123;
  static int a;
}

// Checks static device and constant variables are allowed in device and
// host functions, and static host variables are not allowed in device
// functions.

static __device__ int x;
static __constant__ int y;
static int z; // dev-note {{host variable declared here}}

__global__ void kernel(int *a) {
  a[0] = x;
  a[1] = y;
  a[2] = z;
  // dev-error@-1 {{reference to __host__ variable 'z' in __global__ function}}
}

// Check dynamic initialization of static device variable is not allowed.

namespace TestStaticVarInLambda {
class A {
public:
  A(char *);
};
class B {
public:
  __device__ B(char *);
};
void fun() {
  (void) [](char *c) {
    static A var1(c);
    static __device__ B var2(c);
    // com-error@-1 {{dynamic initialization is not supported for __device__, __constant__, __shared__, and __managed__ variables}}
    (void) var1;
    (void) var2;
  };
}
}

int* getDeviceSymbol(int *x);

void foo() {
  getDeviceSymbol(&x);
  getDeviceSymbol(&y);
}
