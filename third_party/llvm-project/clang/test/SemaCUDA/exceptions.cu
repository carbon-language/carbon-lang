#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -fcxx-exceptions -fcuda-is-device -fsyntax-only -verify %s
// RUN: %clang_cc1 -fcxx-exceptions -fsyntax-only -verify %s

#include "Inputs/hip/hip_runtime.h"

void host() {
  throw NULL;
  try {} catch(void*) {}
}
__device__ void device() {
  throw NULL;
  // expected-error@-1 {{cannot use 'throw' in __device__ function}}
  try {} catch(void*) {}
  // expected-error@-1 {{cannot use 'try' in __device__ function}}
}
__global__ void kernel() {
  throw NULL;
  // expected-error@-1 {{cannot use 'throw' in __global__ function}}
  try {} catch(void*) {}
  // expected-error@-1 {{cannot use 'try' in __global__ function}}
}

// Check that it's an error to use 'try' and 'throw' from a __host__ __device__
// function if and only if it's codegen'ed for device.

__host__ __device__ void hd1() {
  throw NULL;
  try {} catch(void*) {}
#ifdef __CUDA_ARCH__
  // expected-error@-3 {{cannot use 'throw' in __host__ __device__ function}}
  // expected-error@-3 {{cannot use 'try' in __host__ __device__ function}}
#endif
}

// No error, never instantiated on device.
inline __host__ __device__ void hd2() {
  throw NULL;
  try {} catch(void*) {}
}
void call_hd2() { hd2(); }

// Error, instantiated on device.
inline __host__ __device__ void hd3() {
  throw NULL;
  try {} catch(void*) {}
#ifdef __CUDA_ARCH__
  // expected-error@-3 {{cannot use 'throw' in __host__ __device__ function}}
  // expected-error@-3 {{cannot use 'try' in __host__ __device__ function}}
#endif
}

__device__ void call_hd3() { hd3(); }
#ifdef __CUDA_ARCH__
// expected-note@-2 {{called by 'call_hd3'}}
#endif
