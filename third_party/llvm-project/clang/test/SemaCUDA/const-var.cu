#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -x hip %s \
// RUN:   -fsyntax-only -verify
// RUN: %clang_cc1 -triple x86_64 -x hip %s \
// RUN:   -fsyntax-only -verify=host

// host-no-diagnostics

#include "Inputs/hip/hip_runtime.h"

// Test const var initialized with address of a const var.
// Both are promoted to device side.

namespace Test1 {
const int a = 1;

struct B {
    static const int *const p;
    __device__ static const int *const p2;
};
const int *const B::p = &a;
// Const variable 'a' is treated as __constant__ on device side,
// therefore its address can be used as initializer for another
// device variable.
__device__ const int *const B::p2 = &a;

__device__ void f() {
  int y = a;
  const int *x = B::p;
  const int *z = B::p2;
}
}

// Test const var initialized with address of a non-cost var.
// Neither is promoted to device side.

namespace Test2 {
int a = 1;
// expected-note@-1{{host variable declared here}}

struct B {
    static int *const p;
};
int *const B::p = &a;
// expected-note@-1{{const variable cannot be emitted on device side due to dynamic initialization}}

__device__ void f() {
  int y = a;
  // expected-error@-1{{reference to __host__ variable 'a' in __device__ function}}
  const int *x = B::p;
  // expected-error@-1{{reference to __host__ variable 'p' in __device__ function}}
}
}

// Test device var initialized with address of a non-const host var, __shared var,
// __managed__ var, __device__ var, __constant__ var, texture var, surface var.

namespace Test3 {
struct textureReference {
  int desc;
};

enum ReadMode {
  ElementType = 0,
  NormalizedFloat = 1
};

template <typename T, int dim = 1, enum ReadMode mode = ElementType>
struct __attribute__((device_builtin_texture_type)) texture : public textureReference {
};

struct surfaceReference {
  int desc;
};

template <typename T, int dim = 1>
struct __attribute__((device_builtin_surface_type)) surface : public surfaceReference {
};

// Partial specialization over `void`.
template<int dim>
struct __attribute__((device_builtin_surface_type)) surface<void, dim> : public surfaceReference {
};

texture<float, 2, ElementType> tex;
surface<void, 2> surf;

int a = 1;
__shared__ int b;
__managed__ int c = 1;
__device__ int d = 1;
__constant__ int e = 1;
struct B {
    __device__ static int *const p1;
    __device__ static int *const p2;
    __device__ static int *const p3;
    __device__ static int *const p4;
    __device__ static int *const p5;
    __device__ static texture<float, 2, ElementType> *const p6;
    __device__ static surface<void, 2> *const p7;
};
__device__ int *const B::p1 = &a;
// expected-error@-1{{dynamic initialization is not supported for __device__, __constant__, __shared__, and __managed__ variables}}
__device__ int *const B::p2 = &b;
// expected-error@-1{{dynamic initialization is not supported for __device__, __constant__, __shared__, and __managed__ variables}}
__device__ int *const B::p3 = &c;
// expected-error@-1{{dynamic initialization is not supported for __device__, __constant__, __shared__, and __managed__ variables}}
__device__ int *const B::p4 = &d;
__device__ int *const B::p5 = &e;
__device__ texture<float, 2, ElementType> *const B::p6 = &tex;
__device__ surface<void, 2> *const B::p7 = &surf;
}
