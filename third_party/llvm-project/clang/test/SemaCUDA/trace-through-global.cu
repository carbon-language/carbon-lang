#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -fsyntax-only -verify %s

// Check that it's OK for kernels to call HD functions that call device-only
// functions.

#include "Inputs/hip/hip_runtime.h"

__device__ void device_fn(int) {}
// expected-note@-1 2 {{declared here}}

inline __host__ __device__ int hd1() {
  device_fn(0);  // expected-error {{reference to __device__ function}}
  return 0;
}

inline __host__ __device__ int hd2() {
  // No error here because hd2 is only referenced from a kernel.
  device_fn(0);
  return 0;
}

inline __host__ __device__ void hd3(int) {
  device_fn(0);  // expected-error {{reference to __device__ function 'device_fn'}}
}
inline __host__ __device__ void hd3(double) {}

inline __host__ __device__ void hd4(int) {}
inline __host__ __device__ void hd4(double) {
  device_fn(0);  // No error; this function is never called.
}

__global__ void kernel(int) { hd2(); }

template <typename T>
void launch_kernel() {
  kernel<<<0, 0>>>(T());

  // Notice that these two diagnostics are different: Because the call to hd1
  // is not dependent on T, the call to hd1 comes from 'launch_kernel', while
  // the call to hd3, being dependent, comes from 'launch_kernel<int>'.
  hd1(); // expected-note {{called by 'launch_kernel<int>'}}
  hd3(T()); // expected-note {{called by 'launch_kernel<int>'}}
}

void host_fn() {
  launch_kernel<int>();
  // expected-note@-1 2 {{called by 'host_fn'}}
}
