
#include <hip/hip_runtime.h>
// Verify that we do check for constraints in device-side inline
// assembly. Passing an illegal input/output constraint and look 
// for corresponding error
// RUN: %clang_cc1 -triple nvptx-unknown-cuda -fsyntax-only -fcuda-is-device -verify %s

__attribute__((device)) void df() {
  short h;
  int a;
  // asm with PTX constraints. Some of them are PTX-specific.
  __asm__("output constraints"
          : "=h"(h), // .u16 reg, OK
            "=a"(a)  // expected-error {{invalid output constraint '=a' in asm}}
          :          // None
          );
  __asm__("input constraints"
          :           // None
          : "f"(0.0), // .f32 reg, OK
            "d"(0.0), // .f64 reg, OK
            "h"(0),   // .u16 reg, OK
            "r"(0),   // .u32 reg, OK
            "l"(0),   // .u64 reg, OK
            "a"(0)    // expected-error {{invalid input constraint 'a' in asm}}
          );
}
