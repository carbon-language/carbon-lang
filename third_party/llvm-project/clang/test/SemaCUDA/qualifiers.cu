#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -fsyntax-only -verify %s
// RUN: %clang_cc1 -triple nvptx-unknown-cuda -fsyntax-only -verify -fcuda-is-device %s
//
// We run clang_cc1 with 'not' because source file contains
// intentional errors. CC1 failure is expected and must be ignored
// here. We're interested in what ends up in AST and that's what
// FileCheck verifies.
// RUN: not %clang_cc1 -triple x86_64-unknown-linux-gnu -fsyntax-only -ast-dump %s \
// RUN:   | FileCheck %s --check-prefix=CHECK-ALL --check-prefix=CHECK-HOST
// RUN: not %clang_cc1 -triple nvptx-unknown-cuda -fsyntax-only -ast-dump -fcuda-is-device %s \
// RUN:   | FileCheck %s --check-prefix=CHECK-ALL --check-prefix=CHECK-DEVICE

#include "Inputs/hip/hip_runtime.h"

// Host (x86) supports TLS and device-side compilation should ignore
// host variables. No errors in either case.
int __thread host_tls_var;
// CHECK-ALL: host_tls_var 'int' tls

#if defined(__CUDA_ARCH__)
// NVPTX does not support TLS
__device__ int __thread device_tls_var; // expected-error {{thread-local storage is not supported for the current target}}
// CHECK-DEVICE: device_tls_var 'int' tls
__shared__ int __thread shared_tls_var; // expected-error {{thread-local storage is not supported for the current target}}
// CHECK-DEVICE: shared_tls_var 'int' tls
#else
// Device-side vars should not produce any errors during host-side
// compilation.
__device__ int __thread device_tls_var;
// CHECK-HOST: device_tls_var 'int' tls
__shared__ int __thread shared_tls_var;
// CHECK-HOST: shared_tls_var 'int' tls
#endif

__global__ void g1(int x) {}
__global__ int g2(int x) { // expected-error {{must have void return type}}
  return 1;
}
