#include "hip/hip_runtime.h"
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// RUN: %clang_cc1 -std=c++14 -triple x86_64-unknown-linux-gnu -fsyntax-only \
// RUN:   -verify=host,hostdefer,devdefer,expected %s
// RUN: %clang_cc1 -std=c++14 -triple nvptx64-nvidia-cuda -fsyntax-only \
// RUN:   -fcuda-is-device -verify=dev,devnodeferonly,hostdefer,devdefer,expected %s
// RUN: %clang_cc1 -fgpu-exclude-wrong-side-overloads -fgpu-defer-diag -DDEFER=1 \
// RUN:    -std=c++14 -triple x86_64-unknown-linux-gnu -fsyntax-only \
// RUN:    -verify=host,hostdefer,expected %s
// RUN: %clang_cc1 -fgpu-exclude-wrong-side-overloads -fgpu-defer-diag -DDEFER=1 \
// RUN:    -std=c++14 -triple nvptx64-nvidia-cuda -fsyntax-only -fcuda-is-device \
// RUN:    -verify=dev,devdeferonly,devdefer,expected %s

#include "Inputs/hip/hip_runtime.h"

// Opaque return types used to check that we pick the right overloads.
struct HostReturnTy {};
struct HostReturnTy2 {};
struct DeviceReturnTy {};
struct DeviceReturnTy2 {};
struct HostDeviceReturnTy {};
struct TemplateReturnTy {};

typedef HostReturnTy (*HostFnPtr)();
typedef DeviceReturnTy (*DeviceFnPtr)();
typedef HostDeviceReturnTy (*HostDeviceFnPtr)();
typedef void (*GlobalFnPtr)();  // __global__ functions must return void.

// CurrentReturnTy is {HostReturnTy,DeviceReturnTy} during {host,device}
// compilation.
#ifdef __CUDA_ARCH__
typedef DeviceReturnTy CurrentReturnTy;
#else
typedef HostReturnTy CurrentReturnTy;
#endif

// CurrentFnPtr is a function pointer to a {host,device} function during
// {host,device} compilation.
typedef CurrentReturnTy (*CurrentFnPtr)();

// Host and unattributed functions can't be overloaded.
__host__ void hh() {} // expected-note {{previous definition is here}}
void hh() {} // expected-error {{redefinition of 'hh'}}

// H/D overloading is OK.
__host__ HostReturnTy dh() { return HostReturnTy(); }
__device__ DeviceReturnTy dh() { return DeviceReturnTy(); }

// H/HD and D/HD are not allowed.
__host__ __device__ int hdh() { return 0; } // expected-note {{previous declaration is here}}
__host__ int hdh() { return 0; }
// expected-error@-1 {{__host__ function 'hdh' cannot overload __host__ __device__ function 'hdh'}}

__host__ int hhd() { return 0; }            // expected-note {{previous declaration is here}}
__host__ __device__ int hhd() { return 0; }
// expected-error@-1 {{__host__ __device__ function 'hhd' cannot overload __host__ function 'hhd'}}

__host__ __device__ int hdd() { return 0; } // expected-note {{previous declaration is here}}
__device__ int hdd() { return 0; }
// expected-error@-1 {{__device__ function 'hdd' cannot overload __host__ __device__ function 'hdd'}}

__device__ int dhd() { return 0; }          // expected-note {{previous declaration is here}}
__host__ __device__ int dhd() { return 0; }
// expected-error@-1 {{__host__ __device__ function 'dhd' cannot overload __device__ function 'dhd'}}

// Same tests for extern "C" functions.
extern "C" __host__ int chh() { return 0; } // expected-note {{previous definition is here}}
extern "C" int chh() { return 0; }          // expected-error {{redefinition of 'chh'}}

// H/D overloading is OK.
extern "C" __device__ DeviceReturnTy cdh() { return DeviceReturnTy(); }
extern "C" __host__ HostReturnTy cdh() { return HostReturnTy(); }

// H/HD and D/HD overloading is not allowed.
extern "C" __host__ __device__ int chhd1() { return 0; } // expected-note {{previous declaration is here}}
extern "C" __host__ int chhd1() { return 0; }
// expected-error@-1 {{__host__ function 'chhd1' cannot overload __host__ __device__ function 'chhd1'}}

extern "C" __host__ int chhd2() { return 0; } // expected-note {{previous declaration is here}}
extern "C" __host__ __device__ int chhd2() { return 0; }
// expected-error@-1 {{__host__ __device__ function 'chhd2' cannot overload __host__ function 'chhd2'}}

// Helper functions to verify calling restrictions.
__device__ DeviceReturnTy d() { return DeviceReturnTy(); }
// host-note@-1 1+ {{'d' declared here}}
// hostdefer-note@-2 1+ {{candidate function not viable: call to __device__ function from __host__ function}}
// expected-note@-3 0+ {{candidate function not viable: call to __device__ function from __host__ __device__ function}}

__host__ HostReturnTy h() { return HostReturnTy(); }
// dev-note@-1 1+ {{'h' declared here}}
// devdefer-note@-2 1+ {{candidate function not viable: call to __host__ function from __device__ function}}
// expected-note@-3 0+ {{candidate function not viable: call to __host__ function from __host__ __device__ function}}
// devdefer-note@-4 1+ {{candidate function not viable: call to __host__ function from __global__ function}}

__global__ void g() {}
// dev-note@-1 1+ {{'g' declared here}}
// devdefer-note@-2 1+ {{candidate function not viable: call to __global__ function from __device__ function}}
// expected-note@-3 0+ {{candidate function not viable: call to __global__ function from __host__ __device__ function}}
// devdefer-note@-4 1+ {{candidate function not viable: call to __global__ function from __global__ function}}

extern "C" __device__ DeviceReturnTy cd() { return DeviceReturnTy(); }
// host-note@-1 1+ {{'cd' declared here}}
// hostdefer-note@-2 1+ {{candidate function not viable: call to __device__ function from __host__ function}}
// expected-note@-3 0+ {{candidate function not viable: call to __device__ function from __host__ __device__ function}}

extern "C" __host__ HostReturnTy ch() { return HostReturnTy(); }
// dev-note@-1 1+ {{'ch' declared here}}
// devdefer-note@-2 1+ {{candidate function not viable: call to __host__ function from __device__ function}}
// expected-note@-3 0+ {{candidate function not viable: call to __host__ function from __host__ __device__ function}}
// devdefer-note@-4 1+ {{candidate function not viable: call to __host__ function from __global__ function}}

__host__ void hostf() {
  DeviceFnPtr fp_d = d;         // host-error {{reference to __device__ function 'd' in __host__ function}}
  DeviceReturnTy ret_d = d();   // hostdefer-error {{no matching function for call to 'd'}}
  DeviceFnPtr fp_cd = cd;       // host-error {{reference to __device__ function 'cd' in __host__ function}}
  DeviceReturnTy ret_cd = cd(); // hostdefer-error {{no matching function for call to 'cd'}}

  HostFnPtr fp_h = h;
  HostReturnTy ret_h = h();
  HostFnPtr fp_ch = ch;
  HostReturnTy ret_ch = ch();

  HostFnPtr fp_dh = dh;
  HostReturnTy ret_dh = dh();
  HostFnPtr fp_cdh = cdh;
  HostReturnTy ret_cdh = cdh();

  GlobalFnPtr fp_g = g;
  g(); // expected-error {{call to global function 'g' not configured}}
  g<<<0, 0>>>();
}

__device__ void devicef() {
  DeviceFnPtr fp_d = d;
  DeviceReturnTy ret_d = d();
  DeviceFnPtr fp_cd = cd;
  DeviceReturnTy ret_cd = cd();

  HostFnPtr fp_h = h;         // dev-error {{reference to __host__ function 'h' in __device__ function}}
  HostReturnTy ret_h = h();   // devdefer-error {{no matching function for call to 'h'}}
  HostFnPtr fp_ch = ch;       // dev-error {{reference to __host__ function 'ch' in __device__ function}}
  HostReturnTy ret_ch = ch(); // devdefer-error {{no matching function for call to 'ch'}}

  DeviceFnPtr fp_dh = dh;
  DeviceReturnTy ret_dh = dh();
  DeviceFnPtr fp_cdh = cdh;
  DeviceReturnTy ret_cdh = cdh();

  GlobalFnPtr fp_g = g; // dev-error {{reference to __global__ function 'g' in __device__ function}}
  g(); // devdefer-error {{no matching function for call to 'g'}}
  g<<<0,0>>>(); // dev-error {{reference to __global__ function 'g' in __device__ function}}
}

__global__ void globalf() {
  DeviceFnPtr fp_d = d;
  DeviceReturnTy ret_d = d();
  DeviceFnPtr fp_cd = cd;
  DeviceReturnTy ret_cd = cd();

  HostFnPtr fp_h = h;         // dev-error {{reference to __host__ function 'h' in __global__ function}}
  HostReturnTy ret_h = h();   // devdefer-error {{no matching function for call to 'h'}}
  HostFnPtr fp_ch = ch;       // dev-error {{reference to __host__ function 'ch' in __global__ function}}
  HostReturnTy ret_ch = ch(); // devdefer-error {{no matching function for call to 'ch'}}

  DeviceFnPtr fp_dh = dh;
  DeviceReturnTy ret_dh = dh();
  DeviceFnPtr fp_cdh = cdh;
  DeviceReturnTy ret_cdh = cdh();

  GlobalFnPtr fp_g = g; // dev-error {{reference to __global__ function 'g' in __global__ function}}
  g(); // devdefer-error {{no matching function for call to 'g'}}
  g<<<0,0>>>(); // dev-error {{reference to __global__ function 'g' in __global__ function}}
}

__host__ __device__ void hostdevicef() {
  DeviceFnPtr fp_d = d;
  DeviceReturnTy ret_d = d();
  DeviceFnPtr fp_cd = cd;
  DeviceReturnTy ret_cd = cd();
#if !defined(__CUDA_ARCH__)
  // expected-error@-5 {{reference to __device__ function 'd' in __host__ __device__ function}}
  // expected-error@-5 {{reference to __device__ function 'd' in __host__ __device__ function}}
  // expected-error@-5 {{reference to __device__ function 'cd' in __host__ __device__ function}}
  // expected-error@-5 {{reference to __device__ function 'cd' in __host__ __device__ function}}
#endif

  HostFnPtr fp_h = h;
  HostReturnTy ret_h = h();
  HostFnPtr fp_ch = ch;
  HostReturnTy ret_ch = ch();
#if defined(__CUDA_ARCH__)
  // expected-error@-5 {{reference to __host__ function 'h' in __host__ __device__ function}}
  // expected-error@-5 {{reference to __host__ function 'h' in __host__ __device__ function}}
  // devdefer-error@-5 {{reference to __host__ function 'ch' in __host__ __device__ function}}
  // expected-error@-5 {{reference to __host__ function 'ch' in __host__ __device__ function}}
#endif

  CurrentFnPtr fp_dh = dh;
  CurrentReturnTy ret_dh = dh();
  CurrentFnPtr fp_cdh = cdh;
  CurrentReturnTy ret_cdh = cdh();

  GlobalFnPtr fp_g = g;
#if defined(__CUDA_ARCH__)
  // expected-error@-2 {{reference to __global__ function 'g' in __host__ __device__ function}}
#endif

  g();
#if defined (__CUDA_ARCH__)
  // expected-error@-2 {{reference to __global__ function 'g' in __host__ __device__ function}}
#else
  // expected-error@-4 {{call to global function 'g' not configured}}
#endif

  g<<<0,0>>>();
#if defined(__CUDA_ARCH__)
  // expected-error@-2 {{reference to __global__ function 'g' in __host__ __device__ function}}
#endif
}

// Test for address of overloaded function resolution in the global context.
HostFnPtr fp_h = h;
HostFnPtr fp_ch = ch;
CurrentFnPtr fp_dh = dh;
CurrentFnPtr fp_cdh = cdh;
GlobalFnPtr fp_g = g;


// Test overloading of destructors
// Can't mix H and unattributed destructors
struct d_h {
  ~d_h() {} // expected-note {{previous definition is here}}
  __host__ ~d_h() {} // expected-error {{destructor cannot be redeclared}}
};

// HD is OK
struct d_hd {
  __host__ __device__ ~d_hd() {}
};

// Test overloading of member functions
struct m_h {
  void operator delete(void *ptr); // expected-note {{previous declaration is here}}
  __host__ void operator delete(void *ptr); // expected-error {{class member cannot be redeclared}}
};

// D/H overloading is OK
struct m_dh {
  __device__ void operator delete(void *ptr);
  __host__ void operator delete(void *ptr);
};

// HD by itself is OK
struct m_hd {
  __device__ __host__ void operator delete(void *ptr);
};

struct m_hhd {
  __host__ void operator delete(void *ptr) {} // expected-note {{previous declaration is here}}
  __host__ __device__ void operator delete(void *ptr) {}
  // expected-error@-1 {{__host__ __device__ function 'operator delete' cannot overload __host__ function 'operator delete'}}
};

struct m_hdh {
  __host__ __device__ void operator delete(void *ptr) {} // expected-note {{previous declaration is here}}
  __host__ void operator delete(void *ptr) {}
  // expected-error@-1 {{__host__ function 'operator delete' cannot overload __host__ __device__ function 'operator delete'}}
};

struct m_dhd {
  __device__ void operator delete(void *ptr) {} // expected-note {{previous declaration is here}}
  __host__ __device__ void operator delete(void *ptr) {}
  // expected-error@-1 {{__host__ __device__ function 'operator delete' cannot overload __device__ function 'operator delete'}}
};

struct m_hdd {
  __host__ __device__ void operator delete(void *ptr) {} // expected-note {{previous declaration is here}}
  __device__ void operator delete(void *ptr) {}
  // expected-error@-1 {{__device__ function 'operator delete' cannot overload __host__ __device__ function 'operator delete'}}
};

// __global__ functions can't be overloaded based on attribute
// difference.
struct G {
  friend void friend_of_g(G &arg); // expected-note {{previous declaration is here}}
private:
  int x; // expected-note {{declared private here}}
};
__global__ void friend_of_g(G &arg) { int x = arg.x; }
// expected-error@-1 {{__global__ function 'friend_of_g' cannot overload __host__ function 'friend_of_g'}}
// expected-error@-2 {{'x' is a private member of 'G'}}
void friend_of_g(G &arg) { int x = arg.x; }

// HD functions are sometimes allowed to call H or D functions -- this
// is an artifact of the source-to-source splitting performed by nvcc
// that we need to mimic. During device mode compilation in nvcc, host
// functions aren't present at all, so don't participate in
// overloading. But in clang, H and D functions are present in both
// compilation modes. Clang normally uses the target attribute as a
// tiebreaker between overloads with otherwise identical priority, but
// in order to match nvcc's behavior, we sometimes need to wholly
// discard overloads that would not be present during compilation
// under nvcc.

template <typename T> TemplateReturnTy template_vs_function(T arg) {
  return TemplateReturnTy();
}
__device__ DeviceReturnTy template_vs_function(float arg) {
  return DeviceReturnTy();
}

// Here we expect to call the templated function during host compilation, even
// if -fcuda-disable-target-call-checks is passed, and even though C++ overload
// rules prefer the non-templated function.
__host__ __device__ void test_host_device_calls_template(void) {
#ifdef __CUDA_ARCH__
  typedef DeviceReturnTy ExpectedReturnTy;
#else
  typedef TemplateReturnTy ExpectedReturnTy;
#endif

  ExpectedReturnTy ret1 = template_vs_function(1.0f);
  ExpectedReturnTy ret2 = template_vs_function(2.0);
}

// Calls from __host__ and __device__ functions should always call the
// overloaded function that matches their mode.
__host__ void test_host_calls_template_fn() {
  TemplateReturnTy ret1 = template_vs_function(1.0f);
  TemplateReturnTy ret2 = template_vs_function(2.0);
}

__device__ void test_device_calls_template_fn() {
  DeviceReturnTy ret1 = template_vs_function(1.0f);
  DeviceReturnTy ret2 = template_vs_function(2.0);
}

// If we have a mix of HD and H-only or D-only candidates in the overload set,
// normal C++ overload resolution rules apply first.
template <typename T> TemplateReturnTy template_vs_hd_function(T arg)
// devnodeferonly-note@-1{{'template_vs_hd_function<int>' declared here}}
{
  return TemplateReturnTy();
}
__host__ __device__ HostDeviceReturnTy template_vs_hd_function(float arg) {
  return HostDeviceReturnTy();
}

__host__ __device__ void test_host_device_calls_hd_template() {
#if __CUDA_ARCH__ && DEFER
  typedef HostDeviceReturnTy ExpectedReturnTy;
#else
  typedef TemplateReturnTy ExpectedReturnTy;
#endif
  HostDeviceReturnTy ret1 = template_vs_hd_function(1.0f);
  ExpectedReturnTy ret2 = template_vs_hd_function(1);
  // devnodeferonly-error@-1{{reference to __host__ function 'template_vs_hd_function<int>' in __host__ __device__ function}}
}

__host__ void test_host_calls_hd_template() {
  HostDeviceReturnTy ret1 = template_vs_hd_function(1.0f);
  TemplateReturnTy ret2 = template_vs_hd_function(1);
}

__device__ void test_device_calls_hd_template() {
  HostDeviceReturnTy ret1 = template_vs_hd_function(1.0f);
  // Host-only function template is not callable with strict call checks,
  // so for device side HD function will be the only choice.
  HostDeviceReturnTy ret2 = template_vs_hd_function(1);
}

// Check that overloads still work the same way on both host and
// device side when the overload set contains only functions from one
// side of compilation.
__device__ DeviceReturnTy device_only_function(int arg) { return DeviceReturnTy(); }
__device__ DeviceReturnTy2 device_only_function(float arg) { return DeviceReturnTy2(); }
#ifndef __CUDA_ARCH__
  // expected-note@-3 2{{'device_only_function' declared here}}
  // expected-note@-3 2{{'device_only_function' declared here}}
#endif
__host__ HostReturnTy host_only_function(int arg) { return HostReturnTy(); }
__host__ HostReturnTy2 host_only_function(float arg) { return HostReturnTy2(); }
#ifdef __CUDA_ARCH__
  // expected-note@-3 2{{'host_only_function' declared here}}
  // expected-note@-3 2{{'host_only_function' declared here}}
#endif

__host__ __device__ void test_host_device_single_side_overloading() {
  DeviceReturnTy ret1 = device_only_function(1);
  DeviceReturnTy2 ret2 = device_only_function(1.0f);
#ifndef __CUDA_ARCH__
  // expected-error@-3 {{reference to __device__ function 'device_only_function' in __host__ __device__ function}}
  // expected-error@-3 {{reference to __device__ function 'device_only_function' in __host__ __device__ function}}
#endif
  HostReturnTy ret3 = host_only_function(1);
  HostReturnTy2 ret4 = host_only_function(1.0f);
#ifdef __CUDA_ARCH__
  // expected-error@-3 {{reference to __host__ function 'host_only_function' in __host__ __device__ function}}
  // expected-error@-3 {{reference to __host__ function 'host_only_function' in __host__ __device__ function}}
#endif
}

// wrong-sided overloading should not cause diagnostic unless it is emitted.
// This inline function is not emitted.
inline __host__ __device__ void test_host_device_wrong_side_overloading_inline_no_diag() {
  DeviceReturnTy ret1 = device_only_function(1);
  DeviceReturnTy2 ret2 = device_only_function(1.0f);
  HostReturnTy ret3 = host_only_function(1);
  HostReturnTy2 ret4 = host_only_function(1.0f);
}

// wrong-sided overloading should cause diagnostic if it is emitted.
// This inline function is emitted since it is called by an emitted function.
inline __host__ __device__ void test_host_device_wrong_side_overloading_inline_diag() {
  DeviceReturnTy ret1 = device_only_function(1);
  DeviceReturnTy2 ret2 = device_only_function(1.0f);
#ifndef __CUDA_ARCH__
  // expected-error@-3 {{reference to __device__ function 'device_only_function' in __host__ __device__ function}}
  // expected-error@-3 {{reference to __device__ function 'device_only_function' in __host__ __device__ function}}
#endif
  HostReturnTy ret3 = host_only_function(1);
  HostReturnTy2 ret4 = host_only_function(1.0f);
#ifdef __CUDA_ARCH__
  // expected-error@-3 {{reference to __host__ function 'host_only_function' in __host__ __device__ function}}
  // expected-error@-3 {{reference to __host__ function 'host_only_function' in __host__ __device__ function}}
#endif
}

__host__ __device__ void test_host_device_wrong_side_overloading_inline_diag_caller() {
  test_host_device_wrong_side_overloading_inline_diag();
  // expected-note@-1 {{called by 'test_host_device_wrong_side_overloading_inline_diag_caller'}}
}

// Verify that we allow overloading function templates.
template <typename T> __host__ T template_overload(const T &a) { return a; };
template <typename T> __device__ T template_overload(const T &a) { return a; };

__host__ void test_host_template_overload() {
  template_overload(1); // OK. Attribute-based overloading picks __host__ variant.
}
__device__ void test_device_template_overload() {
  template_overload(1); // OK. Attribute-based overloading picks __device__ variant.
}

// Two classes with `operator-` defined. One of them is device only.
struct C1;
struct C2;
__device__
int operator-(const C1 &x, const C1 &y);
int operator-(const C2 &x, const C2 &y);

template <typename T>
__host__ __device__ int constexpr_overload(const T &x, const T &y) {
  return x - y;
}

// Verify that function overloading doesn't prune candidate wrongly.
int test_constexpr_overload(C2 &x, C2 &y) {
  return constexpr_overload(x, y);
}

// Verify no ambiguity for new operator.
void *a = new int;
__device__ void *b = new int;
// expected-error@-1{{dynamic initialization is not supported for __device__, __constant__, __shared__, and __managed__ variables.}}

// Verify no ambiguity for new operator.
template<typename _Tp> _Tp&& f();
template<typename _Tp, typename = decltype(new _Tp(f<_Tp>()))>
void __test();

void foo() {
  __test<int>();
}

// Test resolving implicit host device candidate vs wrong-sided candidate.
// In device compilation, implicit host device caller choose implicit host
// device candidate and wrong-sided candidate with equal preference.
// Resolution result should not change with/without pragma.
namespace ImplicitHostDeviceVsWrongSided {
HostReturnTy callee(double x);
#pragma clang force_cuda_host_device begin
HostDeviceReturnTy callee(int x);
inline HostReturnTy implicit_hd_caller() {
  return callee(1.0);
}
#pragma clang force_cuda_host_device end
}

// Test resolving implicit host device candidate vs same-sided candidate.
// In host compilation, implicit host device caller choose implicit host
// device candidate and same-sided candidate with equal preference.
// Resolution result should not change with/without pragma.
namespace ImplicitHostDeviceVsSameSide {
HostReturnTy callee(int x);
#pragma clang force_cuda_host_device begin
HostDeviceReturnTy callee(double x);
inline HostDeviceReturnTy implicit_hd_caller() {
  return callee(1.0);
}
#pragma clang force_cuda_host_device end
}

// Test resolving explicit host device candidate vs. wrong-sided candidate.
// When -fgpu-defer-diag is off, wrong-sided candidate is not excluded, therefore
// the first callee is chosen.
// When -fgpu-defer-diag is on, wrong-sided candidate is excluded, therefore
// the second callee is chosen.
namespace ExplicitHostDeviceVsWrongSided {
HostReturnTy callee(double x);
__host__ __device__ HostDeviceReturnTy callee(int x);
#if __CUDA_ARCH__ && DEFER
typedef HostDeviceReturnTy ExpectedRetTy;
#else
typedef HostReturnTy ExpectedRetTy;
#endif
inline __host__ __device__ ExpectedRetTy explicit_hd_caller() {
  return callee(1.0);
}
}

// In the implicit host device function 'caller', the second 'callee' should be
// chosen since it has better match, even though it is an implicit host device
// function whereas the first 'callee' is a host function. A diagnostic will be
// emitted if the first 'callee' is chosen since deduced return type cannot be
// used before it is defined.
namespace ImplicitHostDeviceByConstExpr {
template <class a> a b;
auto callee(...);
template <class d> constexpr auto callee(d) -> decltype(0);
struct e {
  template <class ad, class... f> static auto g(ad, f...) {
    return h<e, decltype(b<f>)...>;
  }
  struct i {
    template <class, class... f> static constexpr auto caller(f... k) {
      return callee(k...);
    }
  };
  template <class, class... f> static auto h() {
    return i::caller<int, f...>;
  }
};
class l {
  l() {
    e::g([] {}, this);
  }
};
}

// Implicit HD candidate competes with device candidate.
// a and b have implicit HD copy ctor. In copy ctor of b, ctor of a is resolved.
// copy ctor of a should win over a(short), otherwise there will be ambiguity
// due to conversion operator.
namespace TestImplicitHDWithD {
  struct a {
    __device__ a(short);
    __device__ operator unsigned() const;
    __device__ operator int() const;
  };
  struct b {
    a d;
  };
  void f(b g) { b e = g; }
}

// Implicit HD candidate competes with host candidate.
// a and b have implicit HD copy ctor. In copy ctor of b, ctor of a is resolved.
// copy ctor of a should win over a(short), otherwise there will be ambiguity
// due to conversion operator.
namespace TestImplicitHDWithH {
  struct a {
    a(short);
    __device__ operator unsigned() const;
    __device__ operator int() const;
  };
  struct b {
    a d;
  };
  void f(b g) { b e = g; }
}

// Implicit HD candidate competes with HD candidate.
// a and b have implicit HD copy ctor. In copy ctor of b, ctor of a is resolved.
// copy ctor of a should win over a(short), otherwise there will be ambiguity
// due to conversion operator.
namespace TestImplicitHDWithHD {
  struct a {
    __host__ __device__ a(short);
    __device__ operator unsigned() const;
    __device__ operator int() const;
  };
  struct b {
    a d;
  };
  void f(b g) { b e = g; }
}

// HD candidate competes with H candidate.
// HD has type mismatch whereas H has type match.
// In device compilation, H wins when -fgpu-defer-diag is off and HD wins
// when -fgpu-defer-diags is on. In both cases the diagnostic should be
// deferred.
namespace TestDeferNoMatchingFuncNotEmitted {
  template <typename> struct a {};
  namespace b {
    struct c : a<int> {};
    template <typename d> void ag(d);
  } // namespace b
  template <typename ae>
  __host__ __device__ void ag(a<ae>) {
    ae e;
    ag(e);
  }
  void f() { (void)ag<b::c>; }
}

namespace TestDeferNoMatchingFuncEmitted {
  template <typename> struct a {};
  namespace b {
    struct c : a<int> {};
    template <typename d> void ag(d);
    // devnodeferonly-note@-1{{'ag<TestDeferNoMatchingFuncEmitted::b::c>' declared here}}
  } // namespace b
  template <typename ae>
  __host__ __device__ void ag(a<ae>) {
    ae e;
    ag(e);
    // devnodeferonly-error@-1{{reference to __host__ function 'ag<TestDeferNoMatchingFuncEmitted::b::c>' in __host__ __device__ function}}
    // devdeferonly-error@-2{{no matching function for call to 'ag'}}
    // devdeferonly-note@-3{{called by 'ag<TestDeferNoMatchingFuncEmitted::b::c>'}}
  }
  __host__ __device__ void f() { (void)ag<b::c>; }
  // devnodeferonly-note@-1{{called by 'f'}}
  // devdeferonly-note@-2{{called by 'f'}}
}

// Two HD candidates compete with H candidate.
// HDs have type mismatch whereas H has type match.
// In device compilation, H wins when -fgpu-defer-diag is off and two HD win
// when -fgpu-defer-diags is on. In both cases the diagnostic should be
// deferred.
namespace TestDeferAmbiguityNotEmitted {
  template <typename> struct a {};
  namespace b {
    struct c : a<int> {};
    template <typename d> void ag(d, int);
  } // namespace b
  template <typename ae>
  __host__ __device__ void ag(a<ae>, float) {
    ae e;
    ag(e, 1);
  }
  template <typename ae>
  __host__ __device__ void ag(a<ae>, double) {
  }
  void f() {
    b::c x;
    ag(x, 1);
  }
}

namespace TestDeferAmbiguityEmitted {
  template <typename> struct a {};
  namespace b {
    struct c : a<int> {};
    template <typename d> void ag(d, int);
    // devnodeferonly-note@-1{{'ag<TestDeferAmbiguityEmitted::b::c>' declared here}}
  } // namespace b
  template <typename ae>
  __host__ __device__ void ag(a<ae>, float) {
    // devdeferonly-note@-1{{candidate function [with ae = int]}}
    ae e;
    ag(e, 1);
  }
  template <typename ae>
  __host__ __device__ void ag(a<ae>, double) {
    // devdeferonly-note@-1{{candidate function [with ae = int]}}
  }
  __host__ __device__ void f() {
    b::c x;
    ag(x, 1);
    // devnodeferonly-error@-1{{reference to __host__ function 'ag<TestDeferAmbiguityEmitted::b::c>' in __host__ __device__ function}}
    // devdeferonly-error@-2{{call to 'ag' is ambiguous}}
  }
}

// Implicit HD functions compute with H function and D function.
// In host compilation, foo(0.0, 2) should resolve to X::foo<double, int>.
// In device compilation, foo(0.0, 2) should resolve to foo(double, int).
// In either case there should be no ambiguity.
namespace TestImplicitHDWithHAndD {
  namespace X {
    inline double foo(double, double) { return 0;}
    inline constexpr float foo(float, float) { return 1;}
    inline constexpr long double foo(long double, long double) { return 2;}
    template<typename _Tp, typename _Up> inline constexpr double foo(_Tp, _Up) { return 3;}
  };
  using X::foo;
  inline __device__ double foo(double, double) { return 4;}
  inline __device__ float foo(float, int) { return 5;}
  inline __device__ float foo(int, int) { return 6;}
  inline __device__ double foo(double, int) { return 7;}
  inline __device__ float foo(float, float) { return 9;}
  template<typename _Tp, typename _Up> inline __device__ double foo(_Tp, _Up) { return 10;}

  int g() {
    return [](){
    return foo(0.0, 2);
    }();
  }
}
