// RUN: %clang_cc1 -std=c++11 -triple x86_64-unknown-linux-gnu -fsyntax-only -verify %s
// RUN: %clang_cc1 -std=c++11 -triple nvptx64-nvidia-cuda -fsyntax-only -fcuda-is-device -verify %s

#include "Inputs/hip/hip_runtime.h"

struct HType {}; // expected-note-re 6 {{candidate constructor {{.*}} not viable: no known conversion from 'DType'}}
struct DType {}; // expected-note-re 6 {{candidate constructor {{.*}} not viable: no known conversion from 'HType'}}
struct HDType {};

template <typename T> __host__ HType overload_h_d(T a) { return HType(); }
// expected-note@-1 2 {{candidate template ignored: could not match 'HType' against 'DType'}}
// expected-note@-2 2 {{candidate template ignored: target attributes do not match}}
template <typename T> __device__ DType overload_h_d(T a) { return DType(); }
// expected-note@-1 2 {{candidate template ignored: could not match 'DType' against 'HType'}}
// expected-note@-2 2 {{candidate template ignored: target attributes do not match}}

// Check explicit instantiation.
template  __device__ __host__ DType overload_h_d(int a); // There's no HD template...
// expected-error@-1 {{explicit instantiation of 'overload_h_d' does not refer to a function template, variable template, member function, member class, or static data member}}
template  __device__ __host__ HType overload_h_d(int a); // There's no HD template...
// expected-error@-1 {{explicit instantiation of 'overload_h_d' does not refer to a function template, variable template, member function, member class, or static data member}}
template  __device__ DType overload_h_d(int a); // OK. instantiates D
template  __host__ HType overload_h_d(int a); // OK. instantiates H

// Check explicit specialization.
template  <> __device__ __host__ DType overload_h_d(long a); // There's no HD template...
// expected-error@-1 {{no function template matches function template specialization 'overload_h_d'}}
template  <> __device__ __host__ HType overload_h_d(long a); // There's no HD template...
// expected-error@-1 {{no function template matches function template specialization 'overload_h_d'}}
template  <> __device__ DType overload_h_d(long a); // OK. instantiates D
template  <> __host__ HType overload_h_d(long a); // OK. instantiates H


// Can't overload HD template with H or D template, though
// non-template functions are OK.
template <typename T> __host__ __device__ HDType overload_hd(T a) { return HDType(); }
// expected-note@-1 {{previous declaration is here}}
// expected-note@-2 2 {{candidate template ignored: could not match 'HDType' against 'HType'}}
template <typename T> __device__ HDType overload_hd(T a);
// expected-error@-1 {{__device__ function 'overload_hd' cannot overload __host__ __device__ function 'overload_hd'}}
__device__ HDType overload_hd(int a); // OK.

// Verify that target attributes are taken into account when we
// explicitly specialize or instantiate function templates.
template <> __host__ HType overload_hd(int a);
// expected-error@-1 {{no function template matches function template specialization 'overload_hd'}}
template __host__ HType overload_hd(long a);
// expected-error@-1 {{explicit instantiation of 'overload_hd' does not refer to a function template, variable template, member function, member class, or static data member}}
__host__ HType overload_hd(int a); // OK

template <typename T> __host__ T overload_h(T a); // expected-note {{previous declaration is here}}
template <typename T> __host__ __device__ T overload_h(T a);
// expected-error@-1 {{__host__ __device__ function 'overload_h' cannot overload __host__ function 'overload_h'}}
template <typename T> __device__ T overload_h(T a); // OK. D can overload H.

template <typename T> __host__ HType overload_h_d2(T a) { return HType(); }
template <typename T> __host__ __device__ HDType overload_h_d2(T a) { return HDType(); }
template <typename T1, typename T2 = int> __device__ DType overload_h_d2(T1 a) { T1 x; T2 y; return DType(); }

// constexpr functions are implicitly HD, but explicit
// instantiation/specialization must use target attributes as written.
template <typename T> constexpr T overload_ce_implicit_hd(T a) { return a+1; }
// expected-note@-1 3 {{candidate template ignored: target attributes do not match}}

// These will not match the template.
template __host__ __device__ int overload_ce_implicit_hd(int a);
// expected-error@-1 {{explicit instantiation of 'overload_ce_implicit_hd' does not refer to a function template, variable template, member function, member class, or static data member}}
template <> __host__ __device__ long overload_ce_implicit_hd(long a);
// expected-error@-1 {{no function template matches function template specialization 'overload_ce_implicit_hd'}}
template <> __host__ __device__ constexpr long overload_ce_implicit_hd(long a);
// expected-error@-1 {{no function template matches function template specialization 'overload_ce_implicit_hd'}}

// These should work, because template matching ignores the implicit
// HD attributes the compiler gives to constexpr functions/templates,
// so 'overload_ce_implicit_hd' template will match __host__ functions
// only.
template __host__ int overload_ce_implicit_hd(int a);
template <> __host__ long overload_ce_implicit_hd(long a);

template float overload_ce_implicit_hd(float a);
template <> float* overload_ce_implicit_hd(float *a);
template <> constexpr double overload_ce_implicit_hd(double a) { return a + 3.0; };

__host__ void hf() {
  overload_hd(13);
  overload_ce_implicit_hd('h');        // Implicitly instantiated
  overload_ce_implicit_hd(1.0f);       // Explicitly instantiated
  overload_ce_implicit_hd(2.0);        // Explicitly specialized

  HType h = overload_h_d(10);
  HType h2i = overload_h_d2<int>(11);
  HType h2ii = overload_h_d2<int>(12);

  // These should be implicitly instantiated from __host__ template returning HType.
  DType d = overload_h_d(20);          // expected-error {{no viable conversion from 'HType' to 'DType'}}
  DType d2i = overload_h_d2<int>(21);  // expected-error {{no viable conversion from 'HType' to 'DType'}}
  DType d2ii = overload_h_d2<int>(22); // expected-error {{no viable conversion from 'HType' to 'DType'}}
}
__device__ void df() {
  overload_hd(23);
  overload_ce_implicit_hd('d');        // Implicitly instantiated
  overload_ce_implicit_hd(1.0f);       // Explicitly instantiated
  overload_ce_implicit_hd(2.0);        // Explicitly specialized

  // These should be implicitly instantiated from __device__ template returning DType.
  HType h = overload_h_d(10);          // expected-error {{no viable conversion from 'DType' to 'HType'}}
  HType h2i = overload_h_d2<int>(11);  // expected-error {{no viable conversion from 'DType' to 'HType'}}
  HType h2ii = overload_h_d2<int>(12); // expected-error {{no viable conversion from 'DType' to 'HType'}}

  DType d = overload_h_d(20);
  DType d2i = overload_h_d2<int>(21);
  DType d2ii = overload_h_d2<int>(22);
}
