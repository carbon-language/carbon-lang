// RUN: %clang_cc1 -std=c++11 -fsyntax-only -verify %s -Wno-defaulted-function-deleted

#include "Inputs/hip/hip_runtime.h"

//------------------------------------------------------------------------------
// Test 1: infer inherited default ctor to be host.

struct A1_with_host_ctor {
  A1_with_host_ctor() {}
};
// expected-note@-3 {{candidate constructor (the implicit copy constructor) not viable}}
// expected-note@-4 {{candidate constructor (the implicit move constructor) not viable}}

// The inherited default constructor is inferred to be host, so we'll encounter
// an error when calling it from a __device__ function, but not from a __host__
// function.
struct B1_with_implicit_default_ctor : A1_with_host_ctor {
  using A1_with_host_ctor::A1_with_host_ctor;
};

// expected-note@-4 {{call to __host__ function from __device__}}
// expected-note@-5 {{candidate constructor (the implicit copy constructor) not viable}}
// expected-note@-6 {{candidate constructor (the implicit move constructor) not viable}}
// expected-note@-6 2{{constructor from base class 'A1_with_host_ctor' inherited here}}

void hostfoo() {
  B1_with_implicit_default_ctor b;
}

__device__ void devicefoo() {
  B1_with_implicit_default_ctor b; // expected-error {{no matching constructor}}
}

//------------------------------------------------------------------------------
// Test 2: infer inherited default ctor to be device.

struct A2_with_device_ctor {
  __device__ A2_with_device_ctor() {}
};
// expected-note@-3 {{candidate constructor (the implicit copy constructor) not viable}}
// expected-note@-4 {{candidate constructor (the implicit move constructor) not viable}}

struct B2_with_implicit_default_ctor : A2_with_device_ctor {
  using A2_with_device_ctor::A2_with_device_ctor;
};

// expected-note@-4 {{call to __device__ function from __host__}}
// expected-note@-5 {{candidate constructor (the implicit copy constructor) not viable}}
// expected-note@-6 {{candidate constructor (the implicit move constructor) not viable}}
// expected-note@-6 2{{constructor from base class 'A2_with_device_ctor' inherited here}}

void hostfoo2() {
  B2_with_implicit_default_ctor b;  // expected-error {{no matching constructor}}
}

__device__ void devicefoo2() {
  B2_with_implicit_default_ctor b;
}

//------------------------------------------------------------------------------
// Test 3: infer inherited copy ctor

struct A3_with_device_ctors {
  __host__ A3_with_device_ctors() {}
  __device__ A3_with_device_ctors(const A3_with_device_ctors&) {}
};

struct B3_with_implicit_ctors : A3_with_device_ctors {
  using A3_with_device_ctors::A3_with_device_ctors;
};
// expected-note@-3 2{{call to __device__ function from __host__ function}}
// expected-note@-4 {{default constructor}}


void hostfoo3() {
  B3_with_implicit_ctors b;  // this is OK because the inferred inherited default ctor
                             // here is __host__
  B3_with_implicit_ctors b2 = b; // expected-error {{no matching constructor}}

}

//------------------------------------------------------------------------------
// Test 4: infer inherited default ctor from a field, not a base

struct A4_with_host_ctor {
  A4_with_host_ctor() {}
};

struct B4_with_inherited_host_ctor : A4_with_host_ctor{
  using A4_with_host_ctor::A4_with_host_ctor;
};

struct C4_with_implicit_default_ctor {
  B4_with_inherited_host_ctor field;
};

// expected-note@-4 {{call to __host__ function from __device__}}
// expected-note@-5 {{candidate constructor (the implicit copy constructor) not viable}}
// expected-note@-6 {{candidate constructor (the implicit move constructor) not viable}}

void hostfoo4() {
  C4_with_implicit_default_ctor b;
}

__device__ void devicefoo4() {
  C4_with_implicit_default_ctor b; // expected-error {{no matching constructor}}
}

//------------------------------------------------------------------------------
// Test 5: inherited copy ctor with non-const param

struct A5_copy_ctor_constness {
  __host__ A5_copy_ctor_constness() {}
  __host__ A5_copy_ctor_constness(A5_copy_ctor_constness&) {}
};

struct B5_copy_ctor_constness : A5_copy_ctor_constness {
  using A5_copy_ctor_constness::A5_copy_ctor_constness;
};

// expected-note@-4 {{candidate constructor (the implicit copy constructor) not viable: call to __host__ function from __device__ function}}
// expected-note@-5 {{candidate constructor (the implicit default constructor) not viable}}

void hostfoo5(B5_copy_ctor_constness& b_arg) {
  B5_copy_ctor_constness b = b_arg;
}

__device__ void devicefoo5(B5_copy_ctor_constness& b_arg) {
  B5_copy_ctor_constness b = b_arg; // expected-error {{no matching constructor}}
}

//------------------------------------------------------------------------------
// Test 6: explicitly defaulted ctor

struct A6_with_device_ctor {
  __device__ A6_with_device_ctor() {}
};

struct B6_with_defaulted_ctor : A6_with_device_ctor {
  using A6_with_device_ctor::A6_with_device_ctor;
  __host__ B6_with_defaulted_ctor() = default;
};

// expected-note@-3 {{explicitly defaulted function was implicitly deleted here}}
// expected-note@-6 {{default constructor of 'B6_with_defaulted_ctor' is implicitly deleted because base class 'A6_with_device_ctor' has no default constructor}}

void hostfoo6() {
  B6_with_defaulted_ctor b; // expected-error {{call to implicitly-deleted default constructor}}
}

__device__ void devicefoo6() {
  B6_with_defaulted_ctor b;
}

//------------------------------------------------------------------------------
// Test 7: inherited copy assignment operator

struct A7_with_copy_assign {
  A7_with_copy_assign() {}
  __device__ A7_with_copy_assign& operator=(const A7_with_copy_assign&) {}
};

struct B7_with_copy_assign : A7_with_copy_assign {
  using A7_with_copy_assign::A7_with_copy_assign;
};

// expected-note@-4 {{candidate function (the implicit copy assignment operator) not viable: call to __device__ function from __host__ function}}
// expected-note@-5 {{candidate function (the implicit move assignment operator) not viable: call to __device__ function from __host__ function}}

void hostfoo7() {
  B7_with_copy_assign b1, b2;
  b1 = b2; // expected-error {{no viable overloaded '='}}
}

//------------------------------------------------------------------------------
// Test 8: inherited move assignment operator

// definitions for std::move
namespace std {
inline namespace foo {
template <class T> struct remove_reference { typedef T type; };
template <class T> struct remove_reference<T&> { typedef T type; };
template <class T> struct remove_reference<T&&> { typedef T type; };

template <class T> typename remove_reference<T>::type&& move(T&& t);
}
}

struct A8_with_move_assign {
  A8_with_move_assign() {}
  __device__ A8_with_move_assign& operator=(A8_with_move_assign&&) {}
  __device__ A8_with_move_assign& operator=(const A8_with_move_assign&) {}
};

struct B8_with_move_assign : A8_with_move_assign {
  using A8_with_move_assign::A8_with_move_assign;
};

// expected-note@-4 {{candidate function (the implicit copy assignment operator) not viable: call to __device__ function from __host__ function}}
// expected-note@-5 {{candidate function (the implicit move assignment operator) not viable: call to __device__ function from __host__ function}}

void hostfoo8() {
  B8_with_move_assign b1, b2;
  b1 = std::move(b2); // expected-error {{no viable overloaded '='}}
}
