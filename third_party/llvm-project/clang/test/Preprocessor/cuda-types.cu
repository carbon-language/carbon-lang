
#include <hip/hip_runtime.h>
// Check that types, widths, __GCC_ATOMIC* macros, etc. match on the host and
// device sides of CUDA compilations.  Note that we filter out long double, as
// this is intentionally different on host and device.
//
// FIXME: We really should make __GCC_HAVE_SYNC_COMPARE_AND_SWAP identical on
// host and device, but architecturally this is difficult at the moment.

// RUN: mkdir -p %t

// RUN: %clang --cuda-host-only -nocudainc -target i386-unknown-linux-gnu -x cuda -E -dM -o - /dev/null \
// RUN:   | grep -E 'define __[^ ]*(TYPE|MAX|SIZEOF|WIDTH)|define __GCC_ATOMIC' \
// RUN:   | grep -Ev '__LDBL|_LONG_DOUBLE' > %t/i386-host-defines-filtered
// RUN: %clang --cuda-device-only -nocudainc -nocudalib -target i386-unknown-linux-gnu -x cuda -E -dM -o - /dev/null \
// RUN:   | grep -E 'define __[^ ]*(TYPE|MAX|SIZEOF|WIDTH)|define __GCC_ATOMIC' \
// RUN:   | grep -Ev '__LDBL|_LONG_DOUBLE' > %t/i386-device-defines-filtered
// RUN: diff %t/i386-host-defines-filtered %t/i386-device-defines-filtered

// RUN: %clang --cuda-host-only -nocudainc -target x86_64-unknown-linux-gnu -x cuda -E -dM -o - /dev/null \
// RUN:   | grep -E 'define __[^ ]*(TYPE|MAX|SIZEOF|WIDTH)|define __GCC_ATOMIC' \
// RUN:   | grep -Ev '__LDBL|_LONG_DOUBLE' > %t/x86_64-host-defines-filtered
// RUN: %clang --cuda-device-only -nocudainc -nocudalib -target x86_64-unknown-linux-gnu -x cuda -E -dM -o - /dev/null \
// RUN:   | grep -E 'define __[^ ]*(TYPE|MAX|SIZEOF|WIDTH)|define __GCC_ATOMIC' \
// RUN:   | grep -Ev '__LDBL|_LONG_DOUBLE' > %t/x86_64-device-defines-filtered
// RUN: diff %t/x86_64-host-defines-filtered %t/x86_64-device-defines-filtered

// RUN: %clang --cuda-host-only -nocudainc -target powerpc64-unknown-linux-gnu -x cuda -E -dM -o - /dev/null \
// RUN:   | grep -E 'define __[^ ]*(TYPE|MAX|SIZEOF|WIDTH)|define __GCC_ATOMIC' \
// RUN:   | grep -Ev '__LDBL|_LONG_DOUBLE' > %t/powerpc64-host-defines-filtered
// RUN: %clang --cuda-device-only -nocudainc -nocudalib -target powerpc64-unknown-linux-gnu -x cuda -E -dM -o - /dev/null \
// RUN:   | grep -E 'define __[^ ]*(TYPE|MAX|SIZEOF|WIDTH)|define __GCC_ATOMIC' \
// RUN:   | grep -Ev '__LDBL|_LONG_DOUBLE' > %t/powerpc64-device-defines-filtered
// RUN: diff %t/powerpc64-host-defines-filtered %t/powerpc64-device-defines-filtered

// RUN: %clang --cuda-host-only -nocudainc -target i386-windows-msvc -x cuda -E -dM -o - /dev/null \
// RUN:   | grep -E 'define __[^ ]*(TYPE|MAX|SIZEOF|WIDTH)|define __GCC_ATOMIC' \
// RUN:   | grep -Ev '__LDBL|_LONG_DOUBLE' > %t/i386-msvc-host-defines-filtered
// RUN: %clang --cuda-device-only -nocudainc -nocudalib -target i386-windows-msvc -x cuda -E -dM -o - /dev/null \
// RUN:   | grep -E 'define __[^ ]*(TYPE|MAX|SIZEOF|WIDTH)|define __GCC_ATOMIC' \
// RUN:   | grep -Ev '__LDBL|_LONG_DOUBLE' > %t/i386-msvc-device-defines-filtered
// RUN: diff %t/i386-msvc-host-defines-filtered %t/i386-msvc-device-defines-filtered

// RUN: %clang --cuda-host-only -nocudainc -target x86_64-windows-msvc -x cuda -E -dM -o - /dev/null \
// RUN:   | grep -E 'define __[^ ]*(TYPE|MAX|SIZEOF|WIDTH)|define __GCC_ATOMIC' \
// RUN:   | grep -Ev '__LDBL|_LONG_DOUBLE' > %t/x86_64-msvc-host-defines-filtered
// RUN: %clang --cuda-device-only -nocudainc -nocudalib -target x86_64-windows-msvc -x cuda -E -dM -o - /dev/null \
// RUN:   | grep -E 'define __[^ ]*(TYPE|MAX|SIZEOF|WIDTH)|define __GCC_ATOMIC' \
// RUN:   | grep -Ev '__LDBL|_LONG_DOUBLE' > %t/x86_64-msvc-device-defines-filtered
// RUN: diff %t/x86_64-msvc-host-defines-filtered %t/x86_64-msvc-device-defines-filtered
