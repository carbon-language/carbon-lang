
#include <hip/hip_runtime.h>
// RUN: %clang_cc1 -dM -E -x hip %s | FileCheck -check-prefix=CXX14 %s
// RUN: %clang_cc1 -dM -E %s | FileCheck -check-prefix=CXX14 %s
// RUN: %clang_cc1 -dM -E -std=c++98 -x hip %s | FileCheck -check-prefix=CXX98 %s
// RUN: %clang_cc1 -dM -E -std=c++98 %s | FileCheck -check-prefix=CXX98 %s

// CXX98: #define __cplusplus 199711L
// CXX14: #define __cplusplus 201402L
