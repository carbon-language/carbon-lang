
#include <hip/hip_runtime.h>
// RUN: %clang --cuda-host-only -nocudainc -target i386-unknown-linux-gnu -x cuda -E -dM -o - /dev/null | FileCheck --check-prefix HOST %s
// RUN: %clang --cuda-device-only -nocudainc -nocudalib -target i386-unknown-linux-gnu -x cuda -E -dM -o - /dev/null | FileCheck --check-prefix DEVICE-NOFAST %s
// RUN: %clang -fcuda-approx-transcendentals --cuda-device-only -nocudainc -nocudalib -target i386-unknown-linux-gnu -x cuda -E -dM -o - /dev/null | FileCheck --check-prefix DEVICE-FAST %s
// RUN: %clang -ffast-math --cuda-device-only -nocudainc -nocudalib -target i386-unknown-linux-gnu -x cuda -E -dM -o - /dev/null | FileCheck --check-prefix DEVICE-FAST %s

// HOST-NOT: __CLANG_CUDA_APPROX_TRANSCENDENTALS__
// DEVICE-NOFAST-NOT: __CLANG_CUDA_APPROX_TRANSCENDENTALS__
// DEVICE-FAST: __CLANG_CUDA_APPROX_TRANSCENDENTALS__
