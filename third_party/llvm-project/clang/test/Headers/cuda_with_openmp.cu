
#include <hip/hip_runtime.h>
// Test using -x cuda -fopenmp does not clash integrated headers.
// Reported in https://bugs.llvm.org/show_bug.cgi?id=48014
///==========================================================================///

// REQUIRES: nvptx-registered-target

// RUN: %clang -x cuda -fopenmp -c %s -o - --cuda-path=%S/../Driver/Inputs/CUDA/usr/local/cuda -nocudalib -isystem %S/Inputs/include -isystem %S/../../lib/Headers -fsyntax-only

