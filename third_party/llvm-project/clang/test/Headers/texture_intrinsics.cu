// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target
//
// RUN: %clang -std=c++11 -fsyntax-only -target x86_64-linux -nocudainc -nocudalib --cuda-gpu-arch=sm_86 --cuda-device-only -S %s
// RUN: %clang -std=c++11 -fsyntax-only -target x86_64-linux -nocudainc -nocudalib --cuda-gpu-arch=sm_86 --cuda-host-only -S %s

// Define bare minimum required for parsing the header file.
#include "Inputs/include/hip/hip_runtime.h"

// The header file is expected to compile w/o errors.  This ensures that texture
// ID hash has no collisions for known texture operations, otherwise the
// compilation would fail with an attempt to redefine a type.
#include <__clang_cuda_texture_intrinsics.h>
