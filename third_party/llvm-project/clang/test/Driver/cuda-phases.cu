
#include <hip/hip_runtime.h>
// Tests the phases generated for a CUDA offloading target for different
// combinations of:
// - Number of gpu architectures;
// - Host/device-only compilation;
// - User-requested final phase - binary or assembly.

// REQUIRES: powerpc-registered-target
// REQUIRES: nvptx-registered-target
//
// Test single gpu architecture with complete compilation.
//
// Test CUDA NVPTX phases.
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=sm_30 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=BIN %s
//
// BIN-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (host-[[T]])
// BIN-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (host-[[T]])
// BIN-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (host-[[T]])
// BIN-DAG: [[P3:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T]], (device-[[T]], [[ARCH:sm_30]])
// BIN-DAG: [[P4:[0-9]+]]: preprocessor, {[[P3]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH]])
// BIN-DAG: [[P5:[0-9]+]]: compiler, {[[P4]]}, ir, (device-[[T]], [[ARCH]])
// BIN-DAG: [[P6:[0-9]+]]: backend, {[[P5]]}, assembler, (device-[[T]], [[ARCH]])
// BIN-DAG: [[P7:[0-9]+]]: assembler, {[[P6]]}, object, (device-[[T]], [[ARCH]])
// BIN-DAG: [[P8:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE:nvptx64-nvidia-cuda]]:[[ARCH]])" {[[P7]]}, object
// BIN-DAG: [[P9:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE]]:[[ARCH]])" {[[P6]]}, assembler
// BIN-DAG: [[P10:[0-9]+]]: linker, {[[P8]], [[P9]]}, cuda-fatbin, (device-[[T]])
// BIN-DAG: [[P11:[0-9]+]]: offload, "host-[[T]] (powerpc64le-ibm-linux-gnu)" {[[P2]]}, "device-[[T]] ([[TRIPLE]])" {[[P10]]}, ir
// BIN-DAG: [[P12:[0-9]+]]: backend, {[[P11]]}, assembler, (host-[[T]])
// BIN-DAG: [[P13:[0-9]+]]: assembler, {[[P12]]}, object, (host-[[T]])
// BIN-DAG: [[P14:[0-9]+]]: linker, {[[P13]]}, image, (host-[[T]])

//
// Test single gpu architecture up to the assemble phase.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=sm_30 %s -S 2>&1 \
// RUN: | FileCheck -check-prefixes=ASM %s
// ASM-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (device-[[T]], [[ARCH:sm_30]])
// ASM-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH]])
// ASM-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (device-[[T]], [[ARCH]])
// ASM-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (device-[[T]], [[ARCH]])
// ASM-DAG: [[P4:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE:nvptx64-nvidia-cuda]]:[[ARCH]])" {[[P3]]}, assembler
// ASM-DAG: [[P5:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T]], (host-[[T]])
// ASM-DAG: [[P6:[0-9]+]]: preprocessor, {[[P5]]}, [[T]]-cpp-output, (host-[[T]])
// ASM-DAG: [[P7:[0-9]+]]: compiler, {[[P6]]}, ir, (host-[[T]])
// ASM-DAG: [[P8:[0-9]+]]: backend, {[[P7]]}, assembler, (host-[[T]])

//
// Test two gpu architectures with complete compilation.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=BIN2 %s
// BIN2-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (host-[[T]])
// BIN2-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (host-[[T]])
// BIN2-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (host-[[T]])
// BIN2-DAG: [[P3:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T]], (device-[[T]], [[ARCH1:sm_30]])
// BIN2-DAG: [[P4:[0-9]+]]: preprocessor, {[[P3]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH1]])
// BIN2-DAG: [[P5:[0-9]+]]: compiler, {[[P4]]}, ir, (device-[[T]], [[ARCH1]])
// BIN2-DAG: [[P6:[0-9]+]]: backend, {[[P5]]}, assembler, (device-[[T]], [[ARCH1]])
// BIN2-DAG: [[P7:[0-9]+]]: assembler, {[[P6]]}, object, (device-[[T]], [[ARCH1]])
// BIN2-DAG: [[P8:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE:nvptx64-nvidia-cuda]]:[[ARCH1]])" {[[P7]]}, object
// BIN2-DAG: [[P9:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE]]:[[ARCH1]])" {[[P6]]}, assembler
// BIN2-DAG: [[P10:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T]], (device-[[T]], [[ARCH2:sm_35]])
// BIN2-DAG: [[P11:[0-9]+]]: preprocessor, {[[P10]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH2]])
// BIN2-DAG: [[P12:[0-9]+]]: compiler, {[[P11]]}, ir, (device-[[T]], [[ARCH2]])
// BIN2-DAG: [[P13:[0-9]+]]: backend, {[[P12]]}, assembler, (device-[[T]], [[ARCH2]])
// BIN2-DAG: [[P14:[0-9]+]]: assembler, {[[P13]]}, object, (device-[[T]], [[ARCH2]])
// BIN2-DAG: [[P15:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE]]:[[ARCH2]])" {[[P14]]}, object
// BIN2-DAG: [[P16:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE]]:[[ARCH2]])" {[[P13]]}, assembler
// BIN2-DAG: [[P17:[0-9]+]]: linker, {[[P8]], [[P9]], [[P15]], [[P16]]}, cuda-fatbin, (device-[[T]])
// BIN2-DAG: [[P18:[0-9]+]]: offload, "host-[[T]] (powerpc64le-ibm-linux-gnu)" {[[P2]]}, "device-[[T]] ([[TRIPLE]])" {[[P17]]}, ir
// BIN2-DAG: [[P19:[0-9]+]]: backend, {[[P18]]}, assembler, (host-[[T]])
// BIN2-DAG: [[P20:[0-9]+]]: assembler, {[[P19]]}, object, (host-[[T]])
// BIN2-DAG: [[P21:[0-9]+]]: linker, {[[P20]]}, image, (host-[[T]])

//
// Test two gpu architecturess up to the assemble phase.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s -S 2>&1 \
// RUN: | FileCheck -check-prefixes=ASM2 %s
// ASM2-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (device-[[T]], [[ARCH1:sm_30]])
// ASM2-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH1]])
// ASM2-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (device-[[T]], [[ARCH1]])
// ASM2-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (device-[[T]], [[ARCH1]])
// ASM2-DAG: [[P4:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE:nvptx64-nvidia-cuda]]:[[ARCH1]])" {[[P3]]}, assembler
// ASM2-DAG: [[P5:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T]], (device-[[T]], [[ARCH2:sm_35]])
// ASM2-DAG: [[P6:[0-9]+]]: preprocessor, {[[P5]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH2]])
// ASM2-DAG: [[P7:[0-9]+]]: compiler, {[[P6]]}, ir, (device-[[T]], [[ARCH2]])
// ASM2-DAG: [[P8:[0-9]+]]: backend, {[[P7]]}, assembler, (device-[[T]], [[ARCH2]])
// ASM2-DAG: [[P9:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE]]:[[ARCH2]])" {[[P8]]}, assembler
// ASM2-DAG: [[P10:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T]], (host-[[T]])
// ASM2-DAG: [[P11:[0-9]+]]: preprocessor, {[[P10]]}, [[T]]-cpp-output, (host-[[T]])
// ASM2-DAG: [[P12:[0-9]+]]: compiler, {[[P11]]}, ir, (host-[[T]])
// ASM2-DAG: [[P13:[0-9]+]]: backend, {[[P12]]}, assembler, (host-[[T]])

//
// Test single gpu architecture with complete compilation in host-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=sm_30 %s --cuda-host-only 2>&1 \
// RUN: | FileCheck -check-prefixes=HBIN %s
// HBIN-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (host-[[T]])
// HBIN-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (host-[[T]])
// HBIN-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (host-[[T]])
// HBIN-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (host-[[T]])
// HBIN-DAG: [[P4:[0-9]+]]: assembler, {[[P3]]}, object, (host-[[T]])
// HBIN-DAG: [[P5:[0-9]+]]: linker, {[[P4]]}, image, (host-[[T]])
// HBIN-NOT: device
//
// Test single gpu architecture up to the assemble phase in host-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=sm_30 %s --cuda-host-only -S 2>&1 \
// RUN: | FileCheck -check-prefixes=HASM %s
// HASM-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (host-[[T]])
// HASM-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (host-[[T]])
// HASM-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (host-[[T]])
// HASM-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (host-[[T]])
// HASM-NOT: device

//
// Test two gpu architectures with complete compilation in host-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s --cuda-host-only 2>&1 \
// RUN: | FileCheck -check-prefixes=HBIN2 %s
// HBIN2-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (host-[[T]])
// HBIN2-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (host-[[T]])
// HBIN2-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (host-[[T]])
// HBIN2-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (host-[[T]])
// HBIN2-DAG: [[P4:[0-9]+]]: assembler, {[[P3]]}, object, (host-[[T]])
// HBIN2-DAG: [[P5:[0-9]+]]: linker, {[[P4]]}, image, (host-[[T]])
// HBIN2-NOT: device

//
// Test two gpu architectures up to the assemble phase in host-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s --cuda-host-only -S \
// RUN: 2>&1 | FileCheck -check-prefixes=HASM2 %s
// HASM2-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (host-[[T]])
// HASM2-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (host-[[T]])
// HASM2-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (host-[[T]])
// HASM2-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (host-[[T]])
// HASM2-NOT: device

//
// Test single gpu architecture with complete compilation in device-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=sm_30 %s --cuda-device-only 2>&1 \
// RUN: | FileCheck -check-prefixes=DBIN %s
// DBIN-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (device-[[T]], [[ARCH:sm_30]])
// DBIN-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH]])
// DBIN-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (device-[[T]], [[ARCH]])
// DBIN-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (device-[[T]], [[ARCH]])
// DBIN-DAG: [[P4:[0-9]+]]: assembler, {[[P3]]}, object, (device-[[T]], [[ARCH]])
// DBIN-DAG: [[P5:[0-9]+]]: offload, "device-[[T]] (nvptx64-nvidia-cuda:[[ARCH]])" {[[P4]]}, object
// DBIN-NOT: host
//
// Test single gpu architecture up to the assemble phase in device-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=sm_30 %s --cuda-device-only -S 2>&1 \
// RUN: | FileCheck -check-prefixes=DASM %s
// DASM-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (device-[[T]], [[ARCH:sm_30]])
// DASM-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH]])
// DASM-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (device-[[T]], [[ARCH]])
// DASM-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (device-[[T]], [[ARCH]])
// DASM-DAG: [[P4:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE:nvptx64-nvidia-cuda]]:[[ARCH]])" {[[P3]]}, assembler
// DASM-NOT: host

//
// Test two gpu architectures with complete compilation in device-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s --cuda-device-only 2>&1 \
// RUN: | FileCheck -check-prefixes=DBIN2 %s
// DBIN2-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (device-[[T]], [[ARCH:sm_30]])
// DBIN2-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH]])
// DBIN2-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (device-[[T]], [[ARCH]])
// DBIN2-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (device-[[T]], [[ARCH]])
// DBIN2-DAG: [[P4:[0-9]+]]: assembler, {[[P3]]}, object, (device-[[T]], [[ARCH]])
// DBIN2-DAG: [[P5:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE:nvptx64-nvidia-cuda]]:[[ARCH]])" {[[P4]]}, object
// DBIN2-DAG: [[P6:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T]], (device-[[T]], [[ARCH2:sm_35]])
// DBIN2-DAG: [[P7:[0-9]+]]: preprocessor, {[[P6]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH2]])
// DBIN2-DAG: [[P8:[0-9]+]]: compiler, {[[P7]]}, ir, (device-[[T]], [[ARCH2]])
// DBIN2-DAG: [[P9:[0-9]+]]: backend, {[[P8]]}, assembler, (device-[[T]], [[ARCH2]])
// DBIN2-DAG: [[P10:[0-9]+]]: assembler, {[[P9]]}, object, (device-[[T]], [[ARCH2]])
// DBIN2-DAG: [[P11:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE]]:[[ARCH2]])" {[[P10]]}, object
// DBIN2-NOT: host
//
// Test two gpu architectures up to the assemble phase in device-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s --cuda-device-only -S \
// RUN: 2>&1 | FileCheck -check-prefixes=DASM2 %s
// DASM2-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (device-[[T]], [[ARCH:sm_30]])
// DASM2-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH]])
// DASM2-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (device-[[T]], [[ARCH]])
// DASM2-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (device-[[T]], [[ARCH]])
// DASM2-DAG: [[P4:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE:nvptx64-nvidia-cuda]]:[[ARCH]])" {[[P3]]}, assembler
// DASM2-DAG: [[P5:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T]], (device-[[T]], [[ARCH2:sm_35]])
// DASM2-DAG: [[P6:[0-9]+]]: preprocessor, {[[P5]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH2]])
// DASM2-DAG: [[P7:[0-9]+]]: compiler, {[[P6]]}, ir, (device-[[T]], [[ARCH2]])
// DASM2-DAG: [[P8:[0-9]+]]: backend, {[[P7]]}, assembler, (device-[[T]], [[ARCH2]])
// DASM2-DAG: [[P9:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE]]:[[ARCH2]])" {[[P8]]}, assembler
// DASM2-NOT: host

//
// Test the phases generated when using the new offloading driver.
//
// RUN: %clang -### -target powerpc64le-ibm-linux-gnu -ccc-print-phases --offload-new-driver \
// RUN: --offload-arch=sm_52 --offload-arch=sm_70 %s 2>&1 | FileCheck --check-prefix=NEW_DRIVER %s
// NEW_DRIVER: 0: input, "[[INPUT:.+]]", cuda
// NEW_DRIVER: 1: preprocessor, {0}, cuda-cpp-output
// NEW_DRIVER: 2: compiler, {1}, ir
// NEW_DRIVER: 3: input, "[[INPUT]]", cuda, (device-cuda, sm_52)
// NEW_DRIVER: 4: preprocessor, {3}, cuda-cpp-output, (device-cuda, sm_52)
// NEW_DRIVER: 5: compiler, {4}, ir, (device-cuda, sm_52)
// NEW_DRIVER: 6: backend, {5}, assembler, (device-cuda, sm_52)
// NEW_DRIVER: 7: assembler, {6}, object, (device-cuda, sm_52)
// NEW_DRIVER: 8: offload, "device-cuda (nvptx64-nvidia-cuda:sm_52)" {7}, object
// NEW_DRIVER: 9: offload, "device-cuda (nvptx64-nvidia-cuda:sm_52)" {6}, assembler
// NEW_DRIVER: 10: linker, {8, 9}, cuda-fatbin, (device-cuda, sm_52)
// NEW_DRIVER: 11: offload, "device-cuda (nvptx64-nvidia-cuda:sm_52)" {10}, cuda-fatbin
// NEW_DRIVER: 12: input, "[[INPUT]]", cuda, (device-cuda, sm_70)
// NEW_DRIVER: 13: preprocessor, {12}, cuda-cpp-output, (device-cuda, sm_70)
// NEW_DRIVER: 14: compiler, {13}, ir, (device-cuda, sm_70)
// NEW_DRIVER: 15: backend, {14}, assembler, (device-cuda, sm_70)
// NEW_DRIVER: 16: assembler, {15}, object, (device-cuda, sm_70)
// NEW_DRIVER: 17: offload, "device-cuda (nvptx64-nvidia-cuda:sm_70)" {16}, object
// NEW_DRIVER: 18: offload, "device-cuda (nvptx64-nvidia-cuda:sm_70)" {15}, assembler
// NEW_DRIVER: 19: linker, {17, 18}, cuda-fatbin, (device-cuda, sm_70)
// NEW_DRIVER: 20: offload, "device-cuda (nvptx64-nvidia-cuda:sm_70)" {19}, cuda-fatbin
// NEW_DRIVER: 21: clang-offload-packager, {11, 20}, image
// NEW_DRIVER: 22: offload, " (powerpc64le-ibm-linux-gnu)" {2}, " (powerpc64le-ibm-linux-gnu)" {21}, ir
// NEW_DRIVER: 23: backend, {22}, assembler, (host-cuda)
// NEW_DRIVER: 24: assembler, {23}, object, (host-cuda)
// NEW_DRIVER: 25: clang-linker-wrapper, {24}, image, (host-cuda)
