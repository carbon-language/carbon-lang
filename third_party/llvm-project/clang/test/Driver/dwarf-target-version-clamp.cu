
#include <hip/hip_runtime.h>
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// Verify that DWARF version is properly clamped for nvptx, but not for the host.
// RUN: %clang -### -target x86_64-linux-gnu -c %s -gdwarf-5 -gembed-source 2>&1 \
// RUN: | FileCheck %s --check-prefix=DWARF-CLAMP
// RUN: %clang -### -target x86_64-linux-gnu -c %s -ggdb -gembed-source -gdwarf-5 2>&1 \
// RUN: | FileCheck %s --check-prefix=DWARF-CLAMP

// DWARF-CLAMP: "-triple" "nvptx64-nvidia-cuda"
// DWARF-CLAMP-SAME: -dwarf-version=2
// DWARF-CLAMP: "-triple" "x86_64
// DWARF-CLAMP-SAME: -dwarf-version=5
