
#include <hip/hip_runtime.h>
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// -flto=thin causes a switch to llvm-bc object files.
// RUN: %clangxx -ccc-print-phases -nocudainc -nocudalib -c %s -flto=thin 2> %t
// RUN: FileCheck -check-prefix=CHECK-COMPILE-ACTIONS < %t %s
//
// CHECK-COMPILE-ACTIONS: 2: compiler, {1}, ir, (host-cuda)
// CHECK-COMPILE-ACTIONS-NOT: lto-bc
// CHECK-COMPILE-ACTIONS: 12: backend, {11}, lto-bc, (host-cuda)

// RUN: %clangxx -ccc-print-phases -nocudainc -nocudalib %s -flto=thin 2> %t
// RUN: FileCheck -check-prefix=CHECK-COMPILELINK-ACTIONS < %t %s
//
// CHECK-COMPILELINK-ACTIONS: 0: input, "{{.*}}thinlto.cu", cuda, (host-cuda)
// CHECK-COMPILELINK-ACTIONS: 1: preprocessor, {0}, cuda-cpp-output
// CHECK-COMPILELINK-ACTIONS: 2: compiler, {1}, ir, (host-cuda)
// CHECK-COMPILELINK-ACTIONS: 3: input, "{{.*}}thinlto.cu", cuda, (device-cuda, sm_{{.*}})
// CHECK-COMPILELINK-ACTIONS: 4: preprocessor, {3}, cuda-cpp-output, (device-cuda, sm_{{.*}})
// CHECK-COMPILELINK-ACTIONS: 5: compiler, {4}, ir, (device-cuda, sm_{{.*}})
// CHECK-COMPILELINK-ACTIONS: 6: backend, {5}, assembler, (device-cuda, sm_{{.*}})
// CHECK-COMPILELINK-ACTIONS: 7: assembler, {6}, object, (device-cuda, sm_{{.*}})
// CHECK-COMPILELINK-ACTIONS: 8: offload, "device-cuda (nvptx{{.*}}-nvidia-cuda:sm_{{.*}})" {7}, object
// CHECK-COMPILELINK-ACTIONS: 9: offload, "device-cuda (nvptx{{.*}}-nvidia-cuda:sm_{{.*}})" {6}, assembler
// CHECK-COMPILELINK-ACTIONS: 10: linker, {8, 9}, cuda-fatbin, (device-cuda)
// CHECK-COMPILELINK-ACTIONS: 11: offload, "host-cuda {{.*}}" {2}, "device-cuda{{.*}}" {10}, ir
// CHECK-COMPILELINK-ACTIONS: 12: backend, {11}, lto-bc, (host-cuda)
// CHECK-COMPILELINK-ACTIONS: 13: linker, {12}, image, (host-cuda)
