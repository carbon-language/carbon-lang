
#include <hip/hip_runtime.h>
// Tests that ptxas and fatbinary are invoked correctly during CUDA
// compilation.
//
// REQUIRES: clang-driver
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// Regular compiles with -O{0,1,2,3,4,fast}.  -O4 and -Ofast map to ptxas O3.
// RUN: %clang -### -target x86_64-linux-gnu -O0 -c %s 2>&1 \
// RUN:   --offload-arch=sm_35 --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM35,OPT0 %s
// RUN: %clang -### -target x86_64-linux-gnu -O1 -c %s 2>&1 \
// RUN:   --offload-arch=sm_35 --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM35,OPT1 %s
// RUN: %clang -### -target x86_64-linux-gnu -O2 -c %s 2>&1 \
// RUN:   --offload-arch=sm_35 --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM35,OPT2 %s
// RUN: %clang -### -target x86_64-linux-gnu -O3 -c %s 2>&1 \
// RUN:   --offload-arch=sm_35 --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM35,OPT3 %s
// RUN: %clang -### -target x86_64-linux-gnu -O4 -c %s 2>&1 \
// RUN:   --offload-arch=sm_35 --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM35,OPT3 %s
// RUN: %clang -### -target x86_64-linux-gnu -Ofast -c %s 2>&1 \
// RUN:   --offload-arch=sm_35 --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM35,OPT3 %s
// Generating relocatable device code
// RUN: %clang -### -target x86_64-linux-gnu -fgpu-rdc -c %s 2>&1 \
// RUN:   --offload-arch=sm_35 --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM35,RDC %s

// With debugging enabled, ptxas should be run with with no ptxas optimizations.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-noopt-device-debug -O2 -g -c %s 2>&1 \
// RUN:   --offload-arch=sm_35 --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM35,DBG %s

// --no-cuda-noopt-device-debug overrides --cuda-noopt-device-debug.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-noopt-device-debug \
// RUN:   --offload-arch=sm_35 --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN:   --no-cuda-noopt-device-debug -O2 -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM35,OPT2 %s

// Regular compile without -O.  This should result in us passing -O0 to ptxas.
// RUN: %clang -### -target x86_64-linux-gnu -c %s 2>&1 \
// RUN:   --offload-arch=sm_35 --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM35,OPT0 %s

// Regular compiles with -Os and -Oz.  For lack of a better option, we map
// these to ptxas -O3.
// RUN: %clang -### -target x86_64-linux-gnu -Os -c %s 2>&1 \
// RUN:   --offload-arch=sm_35 --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM35,OPT2 %s
// RUN: %clang -### -target x86_64-linux-gnu -Oz -c %s 2>&1 \
// RUN:   --offload-arch=sm_35 --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM35,OPT2 %s

// Regular compile targeting sm_35.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-gpu-arch=sm_35 -c %s 2>&1 \
// RUN:   --offload-arch=sm_35 --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM35 %s
// Separate compilation targeting sm_35.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-gpu-arch=sm_35 -fgpu-rdc -c %s 2>&1 \
// RUN:   --offload-arch=sm_35 --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM35,RDC %s

// 32-bit compile.
// RUN: %clang -### -target i386-linux-gnu -c %s 2>&1 \
// RUN:   --offload-arch=sm_35 --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH32,SM35 %s
// 32-bit compile when generating relocatable device code.
// RUN: %clang -### -target i386-linux-gnu -fgpu-rdc -c %s 2>&1 \
// RUN:   --offload-arch=sm_35 --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH32,SM35,RDC %s

// Compile with -fintegrated-as.  This should still cause us to invoke ptxas.
// RUN: %clang -### -target x86_64-linux-gnu -fintegrated-as -c %s 2>&1 \
// RUN:   --offload-arch=sm_35 --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM35,OPT0 %s
// Check that we still pass -c when generating relocatable device code.
// RUN: %clang -### -target x86_64-linux-gnu -fintegrated-as -fgpu-rdc -c %s 2>&1 \
// RUN:   --offload-arch=sm_35 --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM35,RDC %s

// Check -Xcuda-ptxas and -Xcuda-fatbinary
// RUN: %clang -### -target x86_64-linux-gnu -c -Xcuda-ptxas -foo1 \
// RUN:   --offload-arch=sm_35 --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN:   -Xcuda-fatbinary -bar1 -Xcuda-ptxas -foo2 -Xcuda-fatbinary -bar2 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CHECK,SM35,PTXAS-EXTRA,FATBINARY-EXTRA %s

// MacOS spot-checks
// RUN: %clang -### -target x86_64-apple-macosx -O0 -c %s 2>&1 \
// RUN:   --offload-arch=sm_35 --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM35,OPT0 %s
// RUN: %clang -### -target x86_64-apple-macosx --cuda-gpu-arch=sm_35 -c %s 2>&1 \
// RUN:   --offload-arch=sm_35 --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM35 %s
// RUN: %clang -### -target i386-apple-macosx -c %s 2>&1 \
// RUN:   --offload-arch=sm_35 --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH32,SM35 %s

// Check relocatable device code generation on MacOS.
// RUN: %clang -### -target x86_64-apple-macosx -O0 -fgpu-rdc -c %s 2>&1 \
// RUN:   --offload-arch=sm_35 --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM35,RDC %s
// RUN: %clang -### -target x86_64-apple-macosx --cuda-gpu-arch=sm_35 -fgpu-rdc -c %s 2>&1 \
// RUN:   --offload-arch=sm_35 --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH64,SM35,RDC %s
// RUN: %clang -### -target i386-apple-macosx -fgpu-rdc -c %s 2>&1 \
// RUN:   --offload-arch=sm_35 --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN: | FileCheck -check-prefixes=CHECK,ARCH32,SM35,RDC %s

// Check that CLANG forwards the -v flag to PTXAS.
// RUN: %clang -### -save-temps -no-canonical-prefixes -v %s 2>&1 \
// RUN:   --offload-arch=sm_35 --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN: | FileCheck -check-prefix=CHK-PTXAS-VERBOSE %s

// Match clang job that produces PTX assembly.
// CHECK: "-cc1"
// ARCH64-SAME: "-triple" "nvptx64-nvidia-cuda"
// ARCH32-SAME: "-triple" "nvptx-nvidia-cuda"
// SM35-SAME: "-target-cpu" "sm_35"
// RDC-SAME: "-fgpu-rdc"
// CHECK-NOT: "-fgpu-rdc"
// SM35-SAME: "-o" "[[PTXFILE:[^"]*]]"

// Match the call to ptxas (which assembles PTX to SASS).
// CHECK: ptxas
// ARCH64-SAME: "-m64"
// ARCH32-SAME: "-m32"
// OPT0-SAME: "-O0"
// OPT0-NOT: "-g"
// OPT1-SAME: "-O1"
// OPT1-NOT: "-g"
// OPT2-SAME: "-O2"
// OPT2-NOT: "-g"
// OPT3-SAME: "-O3"
// OPT3-NOT: "-g"
// DBG-SAME: "-g" "--dont-merge-basicblocks" "--return-at-end"
// SM35-SAME: "--gpu-name" "sm_35"
// SM35-SAME: "--output-file" "[[CUBINFILE:[^"]*]]"
// CHECK-SAME: "[[PTXFILE]]"
// PTXAS-EXTRA-SAME: "-foo1"
// PTXAS-EXTRA-SAME: "-foo2"
// RDC-SAME: "-c"
// CHECK-NOT: "-c"

// Match the call to fatbinary (which combines all our PTX and SASS into one
// blob).
// CHECK: fatbinary
// CHECK-SAME-DAG: "--cuda"
// ARCH64-SAME-DAG: "-64"
// ARCH32-SAME-DAG: "-32"
// CHECK-DAG: "--create" "[[FATBINARY:[^"]*]]"
// SM35-SAME-DAG: "--image=profile=compute_35,file=[[PTXFILE]]"
// SM35-SAME-DAG: "--image=profile=sm_35,file=[[CUBINFILE]]"
// FATBINARY-EXTRA-SAME: "-bar1"
// FATBINARY-EXTRA-SAME: "-bar2"

// Match the clang job for host compilation.
// CHECK: "-cc1"
// ARCH64-SAME: "-triple" "x86_64-
// ARCH32-SAME: "-triple" "i386-
// CHECK-SAME: "-fcuda-include-gpubinary" "[[FATBINARY]]"
// RDC-SAME: "-fgpu-rdc"
// CHECK-NOT: "-fgpu-rdc"

// CHK-PTXAS-VERBOSE: ptxas{{.*}}" "-v"
