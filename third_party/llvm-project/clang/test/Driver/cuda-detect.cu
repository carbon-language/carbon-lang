
#include <hip/hip_runtime.h>
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target
//
// Check that we properly detect CUDA installation.
// RUN: %clang -v --target=i386-unknown-linux \
// RUN:   --sysroot=%S/no-cuda-there --cuda-path-ignore-env 2>&1 | FileCheck %s -check-prefix NOCUDA
// RUN: %clang -v --target=i386-apple-macosx \
// RUN:   --sysroot=%S/no-cuda-there --cuda-path-ignore-env 2>&1 | FileCheck %s -check-prefix NOCUDA
// RUN: %clang -v --target=x86_64-unknown-linux \
// RUN:   --sysroot=%S/no-cuda-there --cuda-path-ignore-env 2>&1 | FileCheck %s -check-prefix NOCUDA
// RUN: %clang -v --target=x86_64-apple-macosx \
// RUN:   --sysroot=%S/no-cuda-there --cuda-path-ignore-env 2>&1 | FileCheck %s -check-prefix NOCUDA


// RUN: %clang -v --target=i386-unknown-linux \
// RUN:   --sysroot=%S/Inputs/CUDA --cuda-path-ignore-env 2>&1 | FileCheck %s
// RUN: %clang -v --target=i386-apple-macosx \
// RUN:   --sysroot=%S/Inputs/CUDA --cuda-path-ignore-env 2>&1 | FileCheck %s

// RUN: %clang -v --target=i386-unknown-linux \
// RUN:   --cuda-path=%S/Inputs/CUDA/usr/local/cuda 2>&1 | FileCheck %s
// RUN: %clang -v --target=i386-apple-macosx \
// RUN:   --cuda-path=%S/Inputs/CUDA/usr/local/cuda 2>&1 | FileCheck %s

// Check that we don't find a CUDA installation without libdevice ...
// RUN: %clang -v --target=i386-unknown-linux \
// RUN:   --sysroot=%S/Inputs/CUDA-nolibdevice --cuda-path-ignore-env 2>&1 | FileCheck %s -check-prefix NOCUDA
// RUN: %clang -v --target=i386-apple-macosx \
// RUN:   --sysroot=%S/Inputs/CUDA-nolibdevice --cuda-path-ignore-env 2>&1 | FileCheck %s -check-prefix NOCUDA
// RUN: %clang -v --target=x86_64-unknown-linux \
// RUN:   --sysroot=%S/Inputs/CUDA-nolibdevice --cuda-path-ignore-env 2>&1 | FileCheck %s -check-prefix NOCUDA
// RUN: %clang -v --target=x84_64-apple-macosx \
// RUN:   --sysroot=%S/Inputs/CUDA-nolibdevice --cuda-path-ignore-env 2>&1 | FileCheck %s -check-prefix NOCUDA

// ... unless the user doesn't need libdevice
// RUN: %clang -v --target=i386-unknown-linux -nocudalib \
// RUN:   --sysroot=%S/Inputs/CUDA-nolibdevice --cuda-path-ignore-env 2>&1 | FileCheck %s -check-prefix NO-LIBDEVICE
// RUN: %clang -v --target=i386-apple-macosx -nocudalib \
// RUN:   --sysroot=%S/Inputs/CUDA-nolibdevice --cuda-path-ignore-env 2>&1 | FileCheck %s -check-prefix NO-LIBDEVICE
// RUN: %clang -v --target=x86_64-unknown-linux -nocudalib \
// RUN:   --sysroot=%S/Inputs/CUDA-nolibdevice --cuda-path-ignore-env 2>&1 | FileCheck %s -check-prefix NO-LIBDEVICE
// RUN: %clang -v --target=x86_64-apple-macosx -nocudalib \
// RUN:   --sysroot=%S/Inputs/CUDA-nolibdevice --cuda-path-ignore-env 2>&1 | FileCheck %s -check-prefix NO-LIBDEVICE


// Make sure we map libdevice bitcode files to proper GPUs. These
// tests use Inputs/CUDA_80 which has full set of libdevice files.
// However, libdevice mapping only matches CUDA-7.x at the moment.
// sm_2x, sm_32 -> compute_20
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_21 \
// RUN:   --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON \
// RUN:     -check-prefixes PTX42,LIBDEVICE,LIBDEVICE20
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_32 \
// RUN:   --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON \
// RUN:     -check-prefixes PTX42,LIBDEVICE,LIBDEVICE20
// sm_30, sm_6x map to compute_30.
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_30 \
// RUN:   --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON \
// RUN:     -check-prefixes PTX42,LIBDEVICE,LIBDEVICE30
// sm_5x is a special case. Maps to compute_30 for cuda-7.x only.
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_50 \
// RUN:   --cuda-path=%S/Inputs/CUDA/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON \
// RUN:     -check-prefixes PTX42,LIBDEVICE,LIBDEVICE30
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_60 \
// RUN:   --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON \
// RUN:     -check-prefixes PTX42,LIBDEVICE,LIBDEVICE30
// sm_35 and sm_37 -> compute_35
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_35 \
// RUN:   --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON -check-prefix CUDAINC \
// RUN:     -check-prefixes PTX42,LIBDEVICE,LIBDEVICE35
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_37 \
// RUN:   --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON -check-prefix CUDAINC \
// RUN:     -check-prefixes PTX42,LIBDEVICE,LIBDEVICE35
// sm_5x -> compute_50 for CUDA-8.0 and newer.
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_50 \
// RUN:   --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON \
// RUN:     -check-prefixes PTX42,LIBDEVICE,LIBDEVICE50

// CUDA-9+ uses the same libdevice for all GPU variants:
// RUN: %clang -### -v --target=x86_64-unknown-linux --cuda-gpu-arch=sm_30 \
// RUN:   --cuda-path=%S/Inputs/CUDA_90/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON64 \
// RUN:     -check-prefixes PTX60,LIBDEVICE,LIBDEVICE10
// RUN: %clang -### -v --target=x86_64-unknown-linux --cuda-gpu-arch=sm_50 \
// RUN:   --cuda-path=%S/Inputs/CUDA_90/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON64 \
// RUN:     -check-prefixes PTX60,LIBDEVICE,LIBDEVICE10
// RUN: %clang -### -v --target=x86_64-unknown-linux --cuda-gpu-arch=sm_60 \
// RUN:   --cuda-path=%S/Inputs/CUDA_90/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON64 \
// RUN:     -check-prefixes PTX60,LIBDEVICE,LIBDEVICE10


// Verify that -nocudainc prevents adding include path to CUDA headers.
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_35 \
// RUN:   -nocudainc --cuda-path=%S/Inputs/CUDA/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON -check-prefix NOCUDAINC \
// RUN:     -check-prefixes PTX42,LIBDEVICE,LIBDEVICE35
// RUN: %clang -### -v --target=i386-apple-macosx --cuda-gpu-arch=sm_35 \
// RUN:   -nocudainc --cuda-path=%S/Inputs/CUDA/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON -check-prefix NOCUDAINC \
// RUN:     -check-prefixes PTX42,LIBDEVICE,LIBDEVICE35

// We should not add any CUDA include paths if there's no valid CUDA installation
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_35 \
// RUN:   --cuda-path=%S/no-cuda-there %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON -check-prefix NOCUDAINC
// RUN: %clang -### -v --target=i386-apple-macosx --cuda-gpu-arch=sm_35 \
// RUN:   --cuda-path=%S/no-cuda-there %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON -check-prefix NOCUDAINC

// Verify that we get an error if there's no libdevice library to link with.
// NOTE: Inputs/CUDA deliberately does *not* have libdevice.compute_20  for this purpose.
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_20 \
// RUN:   --cuda-path=%S/Inputs/CUDA/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON -check-prefix MISSINGLIBDEVICE
// RUN: %clang -### -v --target=i386-apple-macosx --cuda-gpu-arch=sm_20 \
// RUN:   --cuda-path=%S/Inputs/CUDA/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON -check-prefix MISSINGLIBDEVICE

// Verify that  -nocudalib prevents linking libdevice bitcode in.
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_35 \
// RUN:   -nocudalib --cuda-path=%S/Inputs/CUDA/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON
// RUN: %clang -### -v --target=i386-apple-macosx --cuda-gpu-arch=sm_35 \
// RUN:   -nocudalib --cuda-path=%S/Inputs/CUDA/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON

// Verify that we don't add include paths, link with libdevice or
// -include __clang_cuda_runtime_wrapper.h without valid CUDA installation.
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_35 \
// RUN:   --cuda-path=%S/no-cuda-there %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON \
// RUN:     -check-prefix NOCUDAINC
// RUN: %clang -### -v --target=i386-apple-macosx --cuda-gpu-arch=sm_35 \
// RUN:   --cuda-path=%S/no-cuda-there %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON \
// RUN:     -check-prefix NOCUDAINC

// Verify that C++ include paths are passed for both host and device frontends.
// RUN: %clang -### --target=x86_64-linux-gnu %s \
// RUN: --stdlib=libstdc++ --sysroot=%S/Inputs/ubuntu_14.04_multiarch_tree2 \
// RUN: --gcc-toolchain="" 2>&1 \
// RUN: | FileCheck %s --check-prefix CHECK-CXXINCLUDE

// Verify that CUDA SDK version is propagated to the CC1 compilations.
// RUN: %clang -### -v --target=x86_64-linux-gnu --cuda-gpu-arch=sm_50 \
// RUN:   --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix CUDA80

// Verify that if no version file is found, we report the default of 7.0.
// RUN: %clang -### -v --target=x86_64-linux-gnu --cuda-gpu-arch=sm_50 \
// RUN:   --cuda-path=%S/Inputs/CUDA/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix CUDA70

// CHECK: Found CUDA installation: {{.*}}/Inputs/CUDA/usr/local/cuda
// NO-LIBDEVICE: Found CUDA installation: {{.*}}/Inputs/CUDA-nolibdevice/usr/local/cuda
// NOCUDA-NOT: Found CUDA installation:

// MISSINGLIBDEVICE: error: cannot find libdevice for sm_20;

// COMMON: "-triple" "nvptx-nvidia-cuda"
// COMMON64: "-triple" "nvptx64-nvidia-cuda"
// COMMON-SAME: "-fcuda-is-device"
// LIBDEVICE-SAME: "-mlink-builtin-bitcode"
// LIBDEVICE10-SAME: libdevice.10.bc
// LIBDEVICE20-SAME: libdevice.compute_20.10.bc
// LIBDEVICE30-SAME: libdevice.compute_30.10.bc
// LIBDEVICE35-SAME: libdevice.compute_35.10.bc
// LIBDEVICE50-SAME: libdevice.compute_50.10.bc
// PTX42-SAME: "-target-feature" "+ptx42"
// PTX60-SAME: "-target-feature" "+ptx60"
// CUDAINC-SAME: "-include" "__clang_cuda_runtime_wrapper.h"
// NOCUDAINC-NOT: "-include" "__clang_cuda_runtime_wrapper.h"
// CUDAINC-SAME: "-internal-isystem" "{{.*}}/Inputs/CUDA{{[_0-9]+}}/usr/local/cuda/include"
// NOCUDAINC-NOT: "-internal-isystem" "{{.*}}/cuda/include"
// -internal-externc-isystem flags must come *after* the cuda include flags,
// because we must search the cuda include directory first.
// CUDAINC-SAME: "-internal-externc-isystem"
// COMMON-SAME: "-x" "cuda"
// CHECK-CXXINCLUDE: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// CHECK-CXXINCLUDE-SAME: {{.*}}"-internal-isystem" "{{.+}}/include/c++/4.8"
// CHECK-CXXINCLUDE: "-cc1" "-triple" "x86_64-unknown-linux-gnu"
// CHECK-CXXINCLUDE-SAME: {{.*}}"-internal-isystem" "{{.+}}/include/c++/4.8"
// CHECK-CXXINCLUDE: ld{{.*}}"

// CUDA80: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// CUDA80-SAME: -target-sdk-version=8.0
// CUDA80: "-cc1" "-triple" "x86_64-unknown-linux-gnu"
// CUDA80-SAME: -target-sdk-version=8.0
// CUDA80: ld{{.*}}"

// CUDA70: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// CUDA70-SAME: -target-sdk-version=7.0
// CUDA70: "-cc1" "-triple" "x86_64-unknown-linux-gnu"
// CUDA70-SAME: -target-sdk-version=7.0
// CUDA70: ld{{.*}}"
