
#include <hip/hip_runtime.h>
// Tests CUDA compilation with -S and -emit-llvm.

// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// RUN: %clang -### -S -target x86_64-linux-gnu --cuda-gpu-arch=sm_20 %s 2>&1 \
// RUN:   | FileCheck -check-prefix HOST -check-prefix SM20 %s
// RUN: %clang -### -S -target x86_64-linux-gnu --cuda-host-only -o foo.s %s 2>&1 \
// RUN:   | FileCheck -check-prefix HOST %s
// RUN: %clang -### -S -target x86_64-linux-gnu --cuda-gpu-arch=sm_20 \
// RUN:   --cuda-device-only -o foo.s %s 2>&1 \
// RUN:   | FileCheck -check-prefix SM20 %s
// RUN: %clang -### -S -target x86_64-linux-gnu --cuda-gpu-arch=sm_20 \
// RUN:   --cuda-gpu-arch=sm_30 --cuda-device-only %s 2>&1 \
// RUN:   | FileCheck -check-prefix SM20 -check-prefix SM30 %s

// HOST-DAG: "-cc1" "-triple" "x86_64-unknown-linux-gnu"
// SM20-DAG: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// SM20-same: "-target-cpu" "sm_20"
// SM30-DAG: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// SM30-same: "-target-cpu" "sm_30"

// RUN: %clang -### -S -target x86_64-linux-gnu -o foo.s %s 2>&1 \
// RUN:   | FileCheck -check-prefix MULTIPLE-OUTPUT-FILES %s
// RUN: %clang -### -S -target x86_64-linux-gnu --cuda-device-only \
// RUN:   --cuda-gpu-arch=sm_20 --cuda-gpu-arch=sm_30 -o foo.s %s 2>&1 \
// RUN:   | FileCheck -check-prefix MULTIPLE-OUTPUT-FILES %s
// RUN: %clang -### -emit-llvm -c -target x86_64-linux-gnu -o foo.s %s 2>&1 \
// RUN:   | FileCheck -check-prefix MULTIPLE-OUTPUT-FILES %s
// MULTIPLE-OUTPUT-FILES: error: cannot specify -o when generating multiple output files
// Make sure we do not get duplicate diagnostics.
// MULTIPLE-OUTPUT-FILES-NOT: error: cannot specify -o when generating multiple output files
