
#include <hip/hip_runtime.h>
// Check that -fsanitize=foo doesn't get passed down to device-side
// compilation.
//
// REQUIRES: clang-driver
//
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 -fsanitize=address %s 2>&1 | \
// RUN:   FileCheck %s

// CHECK-NOT: error: unsupported option '-fsanitize=address'
// CHECK-DAG: "-fcuda-is-device"
// CHECK-NOT: "-fsanitize=address"
// CHECK-DAG: "-triple" "x86_64-unknown-linux-gnu"
// CHECK: "-fsanitize=address"
