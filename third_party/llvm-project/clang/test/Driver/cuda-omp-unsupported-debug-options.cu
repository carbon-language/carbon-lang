
#include <hip/hip_runtime.h>
// REQUIRES: clang-driver
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target
// REQUIRES: zlib

// RUN: %clang -### -target x86_64-linux-gnu -c %s -g -gz 2>&1 \
// RUN: | FileCheck %s --check-prefixes WARN,COMMON
// RUN: %clang -### -target x86_64-linux-gnu -c %s -gdwarf -fdebug-info-for-profiling 2>&1 \
// RUN: | FileCheck %s --check-prefixes WARN,COMMON
// RUN: %clang -### -target x86_64-linux-gnu -c %s -gdwarf-2 -gsplit-dwarf 2>&1 \
// RUN: | FileCheck %s --check-prefixes WARN,COMMON
// RUN: %clang -### -target x86_64-linux-gnu -c %s -gdwarf-3 -glldb 2>&1 \
// RUN: | FileCheck %s --check-prefixes WARN,COMMON
// RUN: %clang -### -target x86_64-linux-gnu -c %s -gdwarf-4 -gcodeview 2>&1 \
// RUN: | FileCheck %s --check-prefixes WARN,COMMON
// RUN: %clang -### -target x86_64-linux-gnu -c %s -gdwarf-5 -gmodules 2>&1 \
// RUN: | FileCheck %s --check-prefixes WARN,COMMON
// RUN: %clang -### -target x86_64-linux-gnu -c %s -ggdb1 -fdebug-macro 2>&1 \
// RUN: | FileCheck %s --check-prefixes WARN,COMMON
// RUN: %clang -### -target x86_64-linux-gnu -c %s -ggdb2 -ggnu-pubnames 2>&1 \
// RUN: | FileCheck %s --check-prefixes WARN,COMMON
// RUN: %clang -### -target x86_64-linux-gnu -c %s -ggdb3 -gdwarf-aranges 2>&1 \
// RUN: | FileCheck %s --check-prefixes WARN,COMMON
// RUN: %clang -### -target x86_64-linux-gnu -c %s -g -gcolumn-info -fdebug-types-section 2>&1 \
// RUN: | FileCheck %s --check-prefixes WARN,COMMON

// Same tests for OpenMP
// RUN: %clang -### -target x86_64-linux-gnu -fopenmp=libomp -fopenmp-targets=nvptx64-nvidia-cuda -c %s \
// RUN:   -g -gz 2>&1 | FileCheck %s --check-prefixes WARN,COMMON
// RUN: %clang -### -target x86_64-linux-gnu -fopenmp=libomp -fopenmp-targets=nvptx64-nvidia-cuda -c %s \
// RUN:   -gdwarf -fdebug-info-for-profiling 2>&1 | FileCheck %s --check-prefixes WARN,COMMON
// RUN: %clang -### -target x86_64-linux-gnu -fopenmp=libomp -fopenmp-targets=nvptx64-nvidia-cuda -c %s \
// RUN:   -gdwarf-2 -gsplit-dwarf 2>&1 | FileCheck %s --check-prefixes WARN,COMMON
// RUN: %clang -### -target x86_64-linux-gnu -fopenmp=libomp -fopenmp-targets=nvptx64-nvidia-cuda -c %s \
// RUN:   -gdwarf-3 -glldb 2>&1 | FileCheck %s --check-prefixes WARN,COMMON
// RUN: %clang -### -target x86_64-linux-gnu -fopenmp=libomp -fopenmp-targets=nvptx64-nvidia-cuda -c %s \
// RUN:   -gdwarf-4 -gcodeview 2>&1 | FileCheck %s --check-prefixes WARN,COMMON
// RUN: %clang -### -target x86_64-linux-gnu -fopenmp=libomp -fopenmp-targets=nvptx64-nvidia-cuda -c %s \
// RUN:   -gdwarf-5 -gmodules 2>&1 | FileCheck %s --check-prefixes WARN,COMMON
// RUN: %clang -### -target x86_64-linux-gnu -fopenmp=libomp -fopenmp-targets=nvptx64-nvidia-cuda -c %s \
// RUN:   -ggdb1 -fdebug-macro 2>&1 | FileCheck %s --check-prefixes WARN,COMMON
// RUN: %clang -### -target x86_64-linux-gnu -fopenmp=libomp -fopenmp-targets=nvptx64-nvidia-cuda -c %s \
// RUN:   -ggdb2 -ggnu-pubnames 2>&1 | FileCheck %s --check-prefixes WARN,COMMON
// RUN: %clang -### -target x86_64-linux-gnu -fopenmp=libomp -fopenmp-targets=nvptx64-nvidia-cuda -c %s \
// RUN:   -ggdb3 -gdwarf-aranges 2>&1 | FileCheck %s --check-prefixes WARN,COMMON
// RUN: %clang -### -target x86_64-linux-gnu -fopenmp=libomp -fopenmp-targets=nvptx64-nvidia-cuda -c %s \
// RUN:   -g -gcolumn-info -fdebug-types-section 2>&1 | FileCheck %s --check-prefixes WARN,COMMON

// RUN: %clang -### -target x86_64-linux-gnu -c %s -gdwarf-5 -gembed-source 2>&1 \
// RUN: | FileCheck %s --check-prefixes WARN-GES,COMMON
// RUN: %clang -### -target x86_64-linux-gnu -c %s -ggdb -gembed-source -gdwarf-5 2>&1 \
// RUN: | FileCheck %s --check-prefixes WARN-GES,COMMON
// RUN: %clang -### -target x86_64-linux-gnu -fopenmp=libomp -fopenmp-targets=nvptx64-nvidia-cuda -c %s \
// RUN:   -gdwarf-5 -gembed-source 2>&1 | FileCheck %s --check-prefixes WARN-GES,COMMON
// RUN: %clang -### -target x86_64-linux-gnu -fopenmp=libomp -fopenmp-targets=nvptx64-nvidia-cuda -c %s \
// RUN:   -ggdb -gembed-source -gdwarf-5 2>&1 | FileCheck %s --check-prefixes WARN-GES,COMMON

// COMMON: warning: debug information option '{{-gz|-fdebug-info-for-profiling|-gsplit-dwarf|-glldb|-gcodeview|-gmodules|-gembed-source|-fdebug-macro|-ggnu-pubnames|-gdwarf-aranges|-fdebug-types-section}}' is not supported
// WARN-SAME: for target 'nvptx64-nvidia-cuda' [-Wunsupported-target-opt]
// WARN-GES-SAME: requires DWARF-5 but target 'nvptx64-nvidia-cuda' only provides DWARF-2 [-Wunsupported-target-opt]
// COMMON-NOT: debug information option '{{.*}}' is not supported for target 'x86
// COMMON: "-triple" "nvptx64-nvidia-cuda"
// COMMON-NOT: {{-compress-debug|-fdebug-info-for-profiling|lldb|codeview|module-format|embed-source|debug-info-macro|gnu-pubnames|generate-arange-section|generate-type-units}}
// COMMON: "-triple" "x86_64
// COMMON-SAME: {{-compress-debug|-fdebug-info-for-profiling|split-dwarf|lldb|codeview|module-format|embed-source|debug-info-macro|gnu-pubnames|generate-arange-section|generate-type-units}}
