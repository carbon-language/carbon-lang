
#include <hip/hip_runtime.h>
// REQUIRES: clang-driver

// RUN: %clang -### -emit-llvm --cuda-device-only \
// RUN:   -nocudalib -nocudainc --offload=spirv32-unknown-unknown -c %s 2>&1 | FileCheck %s

// CHECK: "-cc1" "-triple" "spirv32-unknown-unknown" {{.*}} "-fcuda-is-device" {{.*}}
