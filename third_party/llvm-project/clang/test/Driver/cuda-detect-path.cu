
#include <hip/hip_runtime.h>
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target
// This tests uses the PATH environment variable.
// REQUIRES: !system-windows

// RUN: env PATH=%S/Inputs/CUDA/usr/local/cuda/bin \
// RUN:    %clang -v --target=i386-unknown-linux --sysroot=%S/no-cuda-there \
// RUN:    2>&1 | FileCheck %s
// RUN: env PATH=%S/Inputs/CUDA/usr/local/cuda/bin \
// RUN:    %clang -v --target=i386-apple-macosx --sysroot=%S/no-cuda-there \
// RUN:    2>&1 | FileCheck %s
// RUN: env PATH=%S/Inputs/CUDA/usr/local/cuda/bin \
// RUN:    %clang -v --target=x86_64-unknown-linux --sysroot=%S/no-cuda-there \
// RUN:    2>&1 | FileCheck %s
// RUN: env PATH=%S/Inputs/CUDA/usr/local/cuda/bin \
// RUN:    %clang -v --target=x86_64-apple-macosx --sysroot=%S/no-cuda-there \
// RUN:    2>&1 | FileCheck %s


// Check that we follow ptxas binaries that are symlinks.
// RUN: env PATH=%S/Inputs/CUDA-symlinks/usr/bin \
// RUN:    %clang -v --target=i386-unknown-linux --sysroot=%S/no-cuda-there \
// RUN:    2>&1 | FileCheck %s --check-prefix SYMLINKS
// RUN: env PATH=%S/Inputs/CUDA-symlinks/usr/bin \
// RUN:    %clang -v --target=i386-apple-macosx --sysroot=%S/no-cuda-there \
// RUN:    2>&1 | FileCheck %s --check-prefix SYMLINKS
// RUN: env PATH=%S/Inputs/CUDA-symlinks/usr/bin \
// RUN:    %clang -v --target=x86_64-unknown-linux --sysroot=%S/no-cuda-there \
// RUN:    2>&1 | FileCheck %s --check-prefix SYMLINKS
// RUN: env PATH=%S/Inputs/CUDA-symlinks/usr/bin \
// RUN:    %clang -v --target=x86_64-apple-macosx --sysroot=%S/no-cuda-there \
// RUN:    2>&1 | FileCheck %s --check-prefix SYMLINKS


// We only take a CUDA installation from PATH if it contains libdevice.
// RUN: env PATH=%S/Inputs/CUDA-nolibdevice/usr/local/cuda/bin \
// RUN:    %clang -v --target=i386-unknown-linux --sysroot=%S/no-cuda-there \
// RUN:    2>&1 | FileCheck %s --check-prefix NOCUDA
// RUN: env PATH=%S/Inputs/CUDA-nolibdevice/usr/local/cuda/bin \
// RUN:    %clang -v --target=i386-apple-macosx --sysroot=%S/no-cuda-there \
// RUN:    2>&1 | FileCheck %s --check-prefix NOCUDA
// RUN: env PATH=%S/Inputs/CUDA-nolibdevice/usr/local/cuda/bin \
// RUN:    %clang -v --target=x86_64-unknown-linux --sysroot=%S/no-cuda-there \
// RUN:    2>&1 | FileCheck %s --check-prefix NOCUDA
// RUN: env PATH=%S/Inputs/CUDA-nolibdevice/usr/local/cuda/bin \
// RUN:    %clang -v --target=x86_64-apple-macosx --sysroot=%S/no-cuda-there \
// RUN:    2>&1 | FileCheck %s --check-prefix NOCUDA

// We even require libdevice if -nocudalib is passed to avoid false positives
// if the distribution merges CUDA into /usr and ptxas ends up /usr/bin.
// RUN: env PATH=%S/Inputs/CUDA-nolibdevice/usr/local/cuda/bin \
// RUN:    %clang -v --target=i386-unknown-linux --sysroot=%S/no-cuda-there -nocudalib \
// RUN:    2>&1 | FileCheck %s --check-prefix NOCUDA
// RUN: env PATH=%S/Inputs/CUDA-nolibdevice/usr/local/cuda/bin \
// RUN:    %clang -v --target=i386-apple-macosx --sysroot=%S/no-cuda-there -nocudalib \
// RUN:    2>&1 | FileCheck %s --check-prefix NOCUDA
// RUN: env PATH=%S/Inputs/CUDA-nolibdevice/usr/local/cuda/bin \
// RUN:    %clang -v --target=x86_64-unknown-linux --sysroot=%S/no-cuda-there -nocudalib \
// RUN:    2>&1 | FileCheck %s --check-prefix NOCUDA
// RUN: env PATH=%S/Inputs/CUDA-nolibdevice/usr/local/cuda/bin \
// RUN:    %clang -v --target=x86_64-apple-macosx --sysroot=%S/no-cuda-there -nocudalib \
// RUN:    2>&1 | FileCheck %s --check-prefix NOCUDA


// Check that the CUDA installation in PATH is not taken when passing
// the option --cuda-path-ignore-env.
// RUN: env PATH=%S/Inputs/CUDA/usr/local/cuda/bin \
// RUN:    %clang -v --target=i386-unknown-linux --sysroot=%S/no-cuda-there --cuda-path-ignore-env \
// RUN:    2>&1 | FileCheck %s --check-prefix NOCUDA
// RUN: env PATH=%S/Inputs/CUDA/usr/local/cuda/bin \
// RUN:    %clang -v --target=i386-apple-macosx --sysroot=%S/no-cuda-there --cuda-path-ignore-env \
// RUN:    2>&1 | FileCheck %s --check-prefix NOCUDA
// RUN: env PATH=%S/Inputs/CUDA/usr/local/cuda/bin \
// RUN:    %clang -v --target=x86_64-unknown-linux --sysroot=%S/no-cuda-there --cuda-path-ignore-env \
// RUN:    2>&1 | FileCheck %s --check-prefix NOCUDA
// RUN: env PATH=%S/Inputs/CUDA/usr/local/cuda/bin \
// RUN:    %clang -v --target=x86_64-apple-macosx --sysroot=%S/no-cuda-there --cuda-path-ignore-env \
// RUN:    2>&1 | FileCheck %s --check-prefix NOCUDA

// CHECK: Found CUDA installation: {{.*}}/Inputs/CUDA/usr/local/cuda
// SYMLINKS: Found CUDA installation: {{.*}}/Inputs/CUDA-symlinks/opt/cuda
// NOCUDA-NOT: Found CUDA installation:
