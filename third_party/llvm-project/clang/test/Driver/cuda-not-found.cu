
#include <hip/hip_runtime.h>
// REQUIRES: clang-driver

// Check that we raise an error if we're trying to compile CUDA code but can't
// find a CUDA install, unless -nocudainc was passed.

// RUN: %clang -### --sysroot=%s/no-cuda-there --cuda-path-ignore-env %s 2>&1 | FileCheck %s --check-prefix ERR
// RUN: %clang -### --cuda-path=%s/no-cuda-there %s 2>&1 | FileCheck %s --check-prefix ERR
// ERR: cannot find CUDA installation

// RUN: %clang -### -nocudainc --sysroot=%s/no-cuda-there --cuda-path-ignore-env %s 2>&1 | FileCheck %s --check-prefix OK
// RUN: %clang -### -nocudainc --cuda-path=%s/no-cuda-there %s 2>&1 | FileCheck %s --check-prefix OK
// OK-NOT: cannot find CUDA installation
