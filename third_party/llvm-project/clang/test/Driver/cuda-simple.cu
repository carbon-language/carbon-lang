// Verify that we can parse a simple CUDA file with or without -save-temps
// http://llvm.org/PR22936
// RUN: %clang --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN:        -nocudainc -nocudalib -Werror -fsyntax-only -c %s
//
// Verify that we pass -x cuda-cpp-output to compiler after
// preprocessing a CUDA file
// RUN: %clang --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN:        -Werror -### -save-temps -c %s 2>&1 | FileCheck %s
// CHECK-LABEL: "-cc1"
// CHECK: "-E"
// CHECK: "-x" "cuda"
// CHECK-LABEL: "-cc1"
// CHECK: "-x" "cuda-cpp-output"
//
// Verify that compiler accepts CUDA syntax with "-x cuda-cpp-output".
// RUN: %clang --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN:        -Werror -fsyntax-only -x cuda-cpp-output -c %s

extern "C" int hipConfigureCall(int, int);
extern "C" int __cudaPushCallConfiguration(int, int);

__attribute__((global)) void kernel() {}

void func() {
     kernel<<<1,1>>>();
}
