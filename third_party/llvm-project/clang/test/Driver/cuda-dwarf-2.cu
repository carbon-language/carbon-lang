
#include <hip/hip_runtime.h>
//
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s -g -O1 --no-cuda-noopt-device-debug 2>&1 | \
// RUN:   FileCheck %s -check-prefix DEBUG_DIRECTIVES
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s -g -O3 2>&1 | \
// RUN:   FileCheck %s -check-prefix DEBUG_DIRECTIVES
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s -g -O3 --no-cuda-noopt-device-debug 2>&1 | \
// RUN:   FileCheck %s -check-prefix DEBUG_DIRECTIVES
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s -g0 2>&1 | \
// RUN:   FileCheck %s -check-prefix NO_DEBUG
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s -ggdb0 -O3 --cuda-noopt-device-debug 2>&1 | \
// RUN:   FileCheck %s -check-prefix NO_DEBUG
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s -gline-directives-only -O2 --cuda-noopt-device-debug 2>&1 | \
// RUN:   FileCheck %s -check-prefix DEBUG_DIRECTIVES

// NO_DEBUG-NOT: warning: debug
// DEBUG_DIRECTIVES-NOT: warning: debug
// NO_DEBUG: "-fcuda-is-device"
// NO_DEBUG-NOT: "-debug-info-kind=
// NO_DEBUG: ptxas
// NO_DEBUG-NOT: "-g"
// DEBUG_DIRECTIVES: "-fcuda-is-device"
// DEBUG_DIRECTIVES-SAME: "-debug-info-kind=line-directives-only"
// DEBUG_DIRECTIVES: ptxas
// DEBUG_DIRECTIVES-SAME: "-lineinfo"
// NO_DEBUG: fatbinary
// NO_DEBUG-NOT: "-g"

// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s -g 2>&1 | \
// RUN:   FileCheck %s -check-prefix HAS_DEBUG
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s -g -O0 --cuda-noopt-device-debug 2>&1 | \
// RUN:   FileCheck %s -check-prefix HAS_DEBUG
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s -g -O0 --no-cuda-noopt-device-debug 2>&1 | \
// RUN:   FileCheck %s -check-prefix HAS_DEBUG
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s -g -O3 --cuda-noopt-device-debug 2>&1 | \
// RUN:   FileCheck %s -check-prefix HAS_DEBUG
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s -g2 2>&1 | \
// RUN:   FileCheck %s -check-prefix HAS_DEBUG
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s -ggdb2 -O0 2>&1 | \
// RUN:   FileCheck %s -check-prefix HAS_DEBUG
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s -g3 -O2 --cuda-noopt-device-debug 2>&1 | \
// RUN:   FileCheck %s -check-prefix HAS_DEBUG
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s -ggdb3 -O3 --cuda-noopt-device-debug 2>&1 | \
// RUN:   FileCheck %s -check-prefix HAS_DEBUG
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s -ggdb1 2>&1 | \
// RUN:   FileCheck %s -check-prefix HAS_DEBUG
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s -gline-tables-only -O2 --cuda-noopt-device-debug 2>&1 | \
// RUN:   FileCheck %s -check-prefix HAS_DEBUG

// HAS_DEBUG-NOT: warning: debug
// HAS_DEBUG: "-fcuda-is-device"
// HAS_DEBUG-SAME: "-debug-info-kind={{constructor|line-tables-only}}"
// HAS_DEBUG-SAME: "-dwarf-version=2"
// HAS_DEBUG: ptxas
// HAS_DEBUG-SAME: "-g"
// HAS_DEBUG-SAME: "--dont-merge-basicblocks"
// HAS_DEBUG-SAME: "--return-at-end"
// HAS_DEBUG: fatbinary
// HAS_DEBUG-SAME: "-g"

