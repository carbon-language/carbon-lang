
#include <hip/hip_runtime.h>
// Tests CUDA compilation pipeline construction in Driver.
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// Simple compilation case. Compile device-side to PTX assembly and make sure
// we use it on the host side.
// RUN: %clang -### -target x86_64-linux-gnu -c %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix HOST -check-prefix INCLUDES-DEVICE \
// RUN:    -check-prefix NOLINK %s

// Typical compilation + link case.
// RUN: %clang -### -target x86_64-linux-gnu %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix HOST -check-prefix INCLUDES-DEVICE \
// RUN:    -check-prefix LINK %s

// Verify that --cuda-host-only disables device-side compilation, but doesn't
// disable host-side compilation/linking.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-host-only %s 2>&1 \
// RUN: | FileCheck -check-prefix NODEVICE -check-prefix HOST \
// RUN:    -check-prefix NOINCLUDES-DEVICE -check-prefix LINK %s

// Verify that --cuda-device-only disables host-side compilation and linking.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix NOHOST -check-prefix NOLINK %s

// Check that the last of --cuda-compile-host-device, --cuda-host-only, and
// --cuda-device-only wins.

// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only \
// RUN:    --cuda-host-only %s 2>&1 \
// RUN: | FileCheck -check-prefix NODEVICE -check-prefix HOST \
// RUN:    -check-prefix NOINCLUDES-DEVICE -check-prefix LINK %s

// RUN: %clang -### -target x86_64-linux-gnu --cuda-compile-host-device \
// RUN:    --cuda-host-only %s 2>&1 \
// RUN: | FileCheck -check-prefix NODEVICE -check-prefix HOST \
// RUN:    -check-prefix NOINCLUDES-DEVICE -check-prefix LINK %s

// RUN: %clang -### -target x86_64-linux-gnu --cuda-host-only \
// RUN:    --cuda-device-only %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix NOHOST -check-prefix NOLINK %s

// RUN: %clang -### -target x86_64-linux-gnu --cuda-compile-host-device \
// RUN:    --cuda-device-only %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix NOHOST -check-prefix NOLINK %s

// RUN: %clang -### -target x86_64-linux-gnu --cuda-host-only \
// RUN:   --cuda-compile-host-device %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix HOST -check-prefix INCLUDES-DEVICE \
// RUN:    -check-prefix LINK %s

// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only \
// RUN:   --cuda-compile-host-device %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix HOST -check-prefix INCLUDES-DEVICE \
// RUN:    -check-prefix LINK %s

// Verify that --cuda-gpu-arch option passes the correct GPU architecture to
// device compilation.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-gpu-arch=sm_30 -c %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix DEVICE-SM30 -check-prefix HOST \
// RUN:    -check-prefix INCLUDES-DEVICE -check-prefix NOLINK %s

// Verify that there is one device-side compilation per --cuda-gpu-arch args
// and that all results are included on the host side.
// RUN: %clang -### -target x86_64-linux-gnu \
// RUN:   --cuda-gpu-arch=sm_35 --cuda-gpu-arch=sm_30 -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes DEVICE,DEVICE-NOSAVE,DEVICE2 \
// RUN:             -check-prefixes DEVICE-SM30,DEVICE2-SM35 \
// RUN:             -check-prefixes INCLUDES-DEVICE,INCLUDES-DEVICE2 \
// RUN:             -check-prefixes HOST,HOST-NOSAVE,NOLINK %s

// Verify that device-side results are passed to the correct tool when
// -save-temps is used.
// RUN: %clang -### -target x86_64-linux-gnu -save-temps -c %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-SAVE \
// RUN:    -check-prefix HOST -check-prefix HOST-SAVE -check-prefix NOLINK %s

// Verify that device-side results are passed to the correct tool when
// -fno-integrated-as is used.
// RUN: %clang -### -target x86_64-linux-gnu -fno-integrated-as -c %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix HOST -check-prefix HOST-NOSAVE \
// RUN:    -check-prefix HOST-AS -check-prefix NOLINK %s

// Verify that --[no-]cuda-gpu-arch arguments are handled correctly.
// a) --no-cuda-gpu-arch=X negates preceding --cuda-gpu-arch=X
// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only \
// RUN:   --cuda-gpu-arch=sm_50 --cuda-gpu-arch=sm_30 \
// RUN:   --no-cuda-gpu-arch=sm_50 \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes ARCH-SM30,NOARCH-SM35,NOARCH-SM50 %s

// b) --no-cuda-gpu-arch=X negates more than one preceding --cuda-gpu-arch=X
// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only \
// RUN:   --cuda-gpu-arch=sm_50 --cuda-gpu-arch=sm_50 --cuda-gpu-arch=sm_30 \
// RUN:   --no-cuda-gpu-arch=sm_50 \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes ARCH-SM30,NOARCH-SM35,NOARCH-SM50 %s

// c) if --no-cuda-gpu-arch=X negates all preceding --cuda-gpu-arch=X
//    we default to sm_35 -- same as if no --cuda-gpu-arch were passed.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only \
// RUN:   --cuda-gpu-arch=sm_50 --cuda-gpu-arch=sm_30 \
// RUN:   --no-cuda-gpu-arch=sm_50 --no-cuda-gpu-arch=sm_30 \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes NOARCH-SM30,ARCH-SM35,NOARCH-SM50 %s

// d) --no-cuda-gpu-arch=X is a no-op if there's no preceding --cuda-gpu-arch=X
// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only \
// RUN:   --cuda-gpu-arch=sm_35 --cuda-gpu-arch=sm_30\
// RUN:   --no-cuda-gpu-arch=sm_50 \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes ARCH-SM30,ARCH-SM35,NOARCH-SM50 %s

// e) --no-cuda-gpu-arch=X does not affect following --cuda-gpu-arch=X
// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only \
// RUN:   --no-cuda-gpu-arch=sm_50 --no-cuda-gpu-arch=sm_30 \
// RUN:   --cuda-gpu-arch=sm_50 --cuda-gpu-arch=sm_30 \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes ARCH-SM30,NOARCH-SM35,ARCH-SM50 %s

// f) --no-cuda-gpu-arch=all negates all preceding --cuda-gpu-arch=X
// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only \
// RUN:   --cuda-gpu-arch=sm_35 --cuda-gpu-arch=sm_30 \
// RUN:   --no-cuda-gpu-arch=all \
// RUN:   --cuda-gpu-arch=sm_50 \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes NOARCH-SM30,NOARCH-SM35,ARCH-SM50 %s

// g) There's no --cuda-gpu-arch=all
// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only \
// RUN:   --cuda-gpu-arch=all \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefix ARCHALLERROR %s


// Verify that --[no-]cuda-include-ptx arguments are handled correctly.
// a) by default we're including PTX for all GPUs.
// RUN: %clang -### -target x86_64-linux-gnu \
// RUN:   --cuda-gpu-arch=sm_35 --cuda-gpu-arch=sm_30 \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes FATBIN-COMMON,PTX-SM35,PTX-SM30 %s

// b) --no-cuda-include-ptx=all disables PTX inclusion for all GPUs
// RUN: %clang -### -target x86_64-linux-gnu \
// RUN:   --cuda-gpu-arch=sm_35 --cuda-gpu-arch=sm_30 \
// RUN:   --no-cuda-include-ptx=all \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes FATBIN-COMMON,NOPTX-SM35,NOPTX-SM30 %s

// c) --no-cuda-include-ptx=sm_XX disables PTX inclusion for that GPU only.
// RUN: %clang -### -target x86_64-linux-gnu \
// RUN:   --cuda-gpu-arch=sm_35 --cuda-gpu-arch=sm_30 \
// RUN:   --no-cuda-include-ptx=sm_35 \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes FATBIN-COMMON,NOPTX-SM35,PTX-SM30 %s
// RUN: %clang -### -target x86_64-linux-gnu \
// RUN:   --cuda-gpu-arch=sm_35 --cuda-gpu-arch=sm_30 \
// RUN:   --no-cuda-include-ptx=sm_30 \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes FATBIN-COMMON,PTX-SM35,NOPTX-SM30 %s

// d) --cuda-include-ptx=all overrides preceding --no-cuda-include-ptx=all
// RUN: %clang -### -target x86_64-linux-gnu \
// RUN:   --cuda-gpu-arch=sm_35 --cuda-gpu-arch=sm_30 \
// RUN:   --no-cuda-include-ptx=all --cuda-include-ptx=all \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes FATBIN-COMMON,PTX-SM35,PTX-SM30 %s

// e) --cuda-include-ptx=all overrides preceding --no-cuda-include-ptx=sm_XX
// RUN: %clang -### -target x86_64-linux-gnu \
// RUN:   --cuda-gpu-arch=sm_35 --cuda-gpu-arch=sm_30 \
// RUN:   --no-cuda-include-ptx=sm_30 --cuda-include-ptx=all \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes FATBIN-COMMON,PTX-SM35,PTX-SM30 %s

// Verify -flto=thin -fwhole-program-vtables handling. This should result in
// both options being passed to the host compilation, with neither passed to
// the device compilation.
// RUN: %clang -### -target x86_64-linux-gnu -c -flto=thin -fwhole-program-vtables %s 2>&1 \
// RUN: | FileCheck -check-prefixes DEVICE,DEVICE-NOSAVE,HOST,INCLUDES-DEVICE,NOLINK,THINLTOWPD %s
// THINLTOWPD-NOT: error: invalid argument '-fwhole-program-vtables' only allowed with '-flto'

// ARCH-SM30: "-cc1"{{.*}}"-target-cpu" "sm_30"
// NOARCH-SM30-NOT: "-cc1"{{.*}}"-target-cpu" "sm_30"
// ARCH-SM35: "-cc1"{{.*}}"-target-cpu" "sm_35"
// NOARCH-SM35-NOT: "-cc1"{{.*}}"-target-cpu" "sm_35"
// ARCH-SM50: "-cc1"{{.*}}"-target-cpu" "sm_50"
// NOARCH-SM50-NOT: "-cc1"{{.*}}"-target-cpu" "sm_50"
// ARCHALLERROR: error: unsupported CUDA gpu architecture: all

// Match device-side preprocessor and compiler phases with -save-temps.
// DEVICE-SAVE: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// DEVICE-SAVE-SAME: "-aux-triple" "x86_64-unknown-linux-gnu"
// DEVICE-SAVE-SAME: "-fcuda-is-device"
// DEVICE-SAVE-SAME: "-x" "cuda"

// DEVICE-SAVE: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// DEVICE-SAVE-SAME: "-aux-triple" "x86_64-unknown-linux-gnu"
// DEVICE-SAVE-SAME: "-fcuda-is-device"
// DEVICE-SAVE-SAME: "-x" "cuda-cpp-output"

// Match the job that produces PTX assembly.
// DEVICE: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// DEVICE-NOSAVE-SAME: "-aux-triple" "x86_64-unknown-linux-gnu"
// THINLTOWPD-NOT: "-flto=thin"
// DEVICE-SAME: "-fcuda-is-device"
// DEVICE-SM30-SAME: "-target-cpu" "sm_30"
// THINLTOWPD-NOT: "-fwhole-program-vtables"
// DEVICE-SAME: "-o" "[[PTXFILE:[^"]*]]"
// DEVICE-NOSAVE-SAME: "-x" "cuda"
// DEVICE-SAVE-SAME: "-x" "ir"

// Match the call to ptxas (which assembles PTX to SASS).
// DEVICE:ptxas
// DEVICE-SM30-DAG: "--gpu-name" "sm_30"
// DEVICE-DAG: "--output-file" "[[CUBINFILE:[^"]*]]"
// DEVICE-DAG: "[[PTXFILE]]"

// Match another device-side compilation.
// DEVICE2: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// DEVICE2-SAME: "-aux-triple" "x86_64-unknown-linux-gnu"
// DEVICE2-SAME: "-fcuda-is-device"
// DEVICE2-SM35-SAME: "-target-cpu" "sm_35"
// DEVICE2-SAME: "-o" "[[PTXFILE2:[^"]*]]"
// DEVICE2-SAME: "-x" "cuda"

// Match another call to ptxas.
// DEVICE2: ptxas
// DEVICE2-SM35-DAG: "--gpu-name" "sm_35"
// DEVICE2-DAG: "--output-file" "[[CUBINFILE2:[^"]*]]"
// DEVICE2-DAG: "[[PTXFILE2]]"

// Match no device-side compilation.
// NODEVICE-NOT: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// NODEVICE-NOT: "-fcuda-is-device"

// INCLUDES-DEVICE:fatbinary
// INCLUDES-DEVICE-DAG: "--create" "[[FATBINARY:[^"]*]]"
// INCLUDES-DEVICE-DAG: "--image=profile=sm_{{[0-9]+}},file=[[CUBINFILE]]"
// INCLUDES-DEVICE-DAG: "--image=profile=compute_{{[0-9]+}},file=[[PTXFILE]]"
// INCLUDES-DEVICE2-DAG: "--image=profile=sm_{{[0-9]+}},file=[[CUBINFILE2]]"
// INCLUDES-DEVICE2-DAG: "--image=profile=compute_{{[0-9]+}},file=[[PTXFILE2]]"

// Match host-side preprocessor job with -save-temps.
// HOST-SAVE: "-cc1" "-triple" "x86_64-unknown-linux-gnu"
// HOST-SAVE-SAME: "-aux-triple" "nvptx64-nvidia-cuda"
// HOST-SAVE-NOT: "-fcuda-is-device"
// HOST-SAVE-SAME: "-x" "cuda"

// Match host-side compilation.
// HOST: "-cc1" "-triple" "x86_64-unknown-linux-gnu"
// HOST-SAME: "-aux-triple" "nvptx64-nvidia-cuda"
// THINLTOWPD-SAME: "-flto=thin"
// HOST-NOT: "-fcuda-is-device"
// There is only one GPU binary after combining it with fatbinary!
// INCLUDES-DEVICE2-NOT: "-fcuda-include-gpubinary"
// INCLUDES-DEVICE-SAME: "-fcuda-include-gpubinary" "[[FATBINARY]]"
// There is only one GPU binary after combining it with fatbinary.
// INCLUDES-DEVICE2-NOT: "-fcuda-include-gpubinary"
// THINLTOWPD-SAME: "-fwhole-program-vtables"
// HOST-SAME: "-o" "[[HOSTOUTPUT:[^"]*]]"
// HOST-NOSAVE-SAME: "-x" "cuda"
// HOST-SAVE-SAME: "-x" "cuda-cpp-output"

// Match external assembler that uses compilation output.
// HOST-AS: "-o" "{{.*}}.o" "[[HOSTOUTPUT]]"

// Match no GPU code inclusion.
// NOINCLUDES-DEVICE-NOT: "-fcuda-include-gpubinary"

// Match no host compilation.
// NOHOST-NOT: "-cc1" "-triple"
// NOHOST-NOT: "-x" "cuda"

// Match linker.
// LINK: "{{.*}}{{ld|link}}{{(.exe)?}}"
// LINK-SAME: "[[HOSTOUTPUT]]"

// Match no linker.
// NOLINK-NOT: "{{.*}}{{ld|link}}{{(.exe)?}}"

// FATBIN-COMMON:fatbinary
// FATBIN-COMMON: "--create" "[[FATBINARY:[^"]*]]"
// FATBIN-COMMON: "--image=profile=sm_30,file=
// PTX-SM30: "--image=profile=compute_30,file=
// NOPTX-SM30-NOT: "--image=profile=compute_30,file=
// FATBIN-COMMON: "--image=profile=sm_35,file=
// PTX-SM35: "--image=profile=compute_35,file=
// NOPTX-SM35-NOT: "--image=profile=compute_35,file=
