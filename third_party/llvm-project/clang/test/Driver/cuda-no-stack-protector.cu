
#include <hip/hip_runtime.h>
// Check that -stack-protector doesn't get passed down to device-side
// compilation.
//
// REQUIRES: clang-driver
//
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 \
// RUN:   -fstack-protector-all %s 2>&1 | \
// RUN: FileCheck %s
//
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 \
// RUN:   -fstack-protector-strong %s 2>&1 | \
// RUN: FileCheck %s
//
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 \
// RUN:   -fstack-protector %s 2>&1 | \
// RUN: FileCheck %s
//
// CHECK-NOT: error: unsupported option '-fstack-protector
// CHECK-DAG: "-fcuda-is-device"
// CHECK-NOT: "-stack-protector"
// CHECK-NOT: "-stack-protector-buffer-size"
// CHECK-DAG: "-triple" "x86_64-unknown-linux-gnu"
// CHECK: "-stack-protector"
