
#include <hip/hip_runtime.h>
// Test clang driver bails out after one error during CUDA compilation.

// REQUIRES: powerpc-registered-target
// REQUIRES: nvptx-registered-target

#ifdef FORCE_ERROR
#error compilation failed
#endif

// RUN: not %clang -target powerpc64le-ibm-linux-gnu -fsyntax-only -nocudalib \
// RUN:   -nocudainc -DFORCE_ERROR %s 2>&1 | FileCheck %s
// RUN: not %clang -target powerpc64le-ibm-linux-gnu -fsyntax-only -nocudalib \
// RUN:   -nocudainc -DFORCE_ERROR --cuda-gpu-arch=sm_35 --cuda-gpu-arch=sm_60 \
// RUN:   %s 2>&1 | FileCheck %s
// RUN: not %clang -target powerpc64le-ibm-linux-gnu -fsyntax-only -nocudalib \
// RUN:   -nocudainc -DFORCE_ERROR --cuda-gpu-arch=sm_35 --cuda-gpu-arch=sm_60 \
// RUN:   --cuda-device-only %s 2>&1 | FileCheck %s

#if defined(ERROR_HOST) && !defined(__CUDA_ARCH__)
#error compilation failed
#endif

#if defined(ERROR_SM35) && (__CUDA_ARCH__ == 350)
#error compilation failed
#endif

#if defined(ERROR_SM60) && (__CUDA_ARCH__ == 600)
#error compilation failed
#endif

// RUN: not %clang -target powerpc64le-ibm-linux-gnu -fsyntax-only -nocudalib \
// RUN:   -nocudainc -DERROR_HOST --cuda-gpu-arch=sm_35 --cuda-gpu-arch=sm_60 \
// RUN:   %s 2>&1 | FileCheck %s
// RUN: not %clang -target powerpc64le-ibm-linux-gnu -fsyntax-only -nocudalib \
// RUN:   -nocudainc -DERROR_SM35 --cuda-gpu-arch=sm_35 --cuda-gpu-arch=sm_60 \
// RUN:   --cuda-device-only %s 2>&1 | FileCheck %s
// RUN: not %clang -target powerpc64le-ibm-linux-gnu -fsyntax-only -nocudalib \
// RUN:   -nocudainc -DERROR_SM60 --cuda-gpu-arch=sm_35 --cuda-gpu-arch=sm_60 \
// RUN:   --cuda-device-only %s 2>&1 | FileCheck %s

// RUN: not %clang -target powerpc64le-ibm-linux-gnu -fsyntax-only -nocudalib \
// RUN:   -nocudainc -DERROR_HOST -DERROR_SM35 --cuda-gpu-arch=sm_35 \
// RUN:   --cuda-gpu-arch=sm_60 %s 2>&1 | FileCheck %s
// RUN: not %clang -target powerpc64le-ibm-linux-gnu -fsyntax-only -nocudalib \
// RUN:   -nocudainc -DERROR_HOST -DERROR_SM60 --cuda-gpu-arch=sm_35 \
// RUN:   --cuda-gpu-arch=sm_60 %s 2>&1 | FileCheck %s
// RUN: not %clang -target powerpc64le-ibm-linux-gnu -fsyntax-only -nocudalib \
// RUN:   -nocudainc -DERROR_SM35 -DERROR_SM60 --cuda-gpu-arch=sm_35 \
// RUN:   --cuda-gpu-arch=sm_60 %s 2>&1 | FileCheck %s


// CHECK: error: compilation failed
// CHECK-NOT: error: compilation failed
