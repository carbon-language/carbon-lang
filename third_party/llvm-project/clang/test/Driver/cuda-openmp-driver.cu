
#include <hip/hip_runtime.h>
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// RUN: %clang -### -target x86_64-linux-gnu -nocudalib -ccc-print-bindings -fgpu-rdc \
// RUN:        --offload-new-driver --offload-arch=sm_35 --offload-arch=sm_70 %s 2>&1 \
// RUN: | FileCheck -check-prefix BINDINGS %s

// BINDINGS: "nvptx64-nvidia-cuda" - "clang", inputs: ["[[INPUT:.+]]"], output: "[[PTX_SM_35:.+]]"
// BINDINGS-NEXT: "nvptx64-nvidia-cuda" - "NVPTX::Assembler", inputs: ["[[PTX_SM_35]]"], output: "[[CUBIN_SM_35:.+]]"
// BINDINGS-NEXT: "nvptx64-nvidia-cuda" - "NVPTX::Linker", inputs: ["[[CUBIN_SM_35]]", "[[PTX_SM_35]]"], output: "[[FATBIN_SM_35:.+]]"
// BINDINGS-NEXT: "nvptx64-nvidia-cuda" - "clang", inputs: ["[[INPUT]]"], output: "[[PTX_SM_70:.+]]"
// BINDINGS-NEXT: "nvptx64-nvidia-cuda" - "NVPTX::Assembler", inputs: ["[[PTX_SM_70:.+]]"], output: "[[CUBIN_SM_70:.+]]"
// BINDINGS-NEXT: "nvptx64-nvidia-cuda" - "NVPTX::Linker", inputs: ["[[CUBIN_SM_70]]", "[[PTX_SM_70:.+]]"], output: "[[FATBIN_SM_70:.+]]"
// BINDINGS-NEXT: "x86_64-unknown-linux-gnu" - "Offload::Packager", inputs: ["[[FATBIN_SM_35]]", "[[FATBIN_SM_70]]"], output: "[[BINARY:.+]]"
// BINDINGS-NEXT: "x86_64-unknown-linux-gnu" - "clang", inputs: ["[[INPUT]]", "[[BINARY]]"], output: "[[HOST_OBJ:.+]]"
// BINDINGS-NEXT: "x86_64-unknown-linux-gnu" - "Offload::Linker", inputs: ["[[HOST_OBJ]]"], output: "a.out"

// RUN: %clang -### -nocudalib --offload-new-driver %s 2>&1 | FileCheck -check-prefix RDC %s
// RDC: error: Using '--offload-new-driver' requires '-fgpu-rdc'

// RUN: %clang -### -target x86_64-linux-gnu -nocudalib -ccc-print-bindings -fgpu-rdc \
// RUN:        --offload-new-driver --offload-arch=sm_35 --offload-arch=sm_70 %s 2>&1 \
// RUN: | FileCheck -check-prefix BINDINGS-HOST %s

// BINDINGS-HOST: # "x86_64-unknown-linux-gnu" - "clang", inputs: ["[[INPUT:.+]]"], output: "[[OUTPUT:.+]]"
// BINDINGS-HOST: # "x86_64-unknown-linux-gnu" - "Offload::Linker", inputs: ["[[OUTPUT]]"], output: "a.out"

// RUN: %clang -### -target x86_64-linux-gnu -nocudalib -ccc-print-bindings -fgpu-rdc \
// RUN:        --offload-new-driver --offload-arch=sm_35 --offload-arch=sm_70 %s 2>&1 \
// RUN: | FileCheck -check-prefix BINDINGS-DEVICE %s

// BINDINGS-DEVICE: # "nvptx64-nvidia-cuda" - "clang", inputs: ["[[INPUT:.+]]"], output: "[[PTX:.+]]"
// BINDINGS-DEVICE: # "nvptx64-nvidia-cuda" - "NVPTX::Assembler", inputs: ["[[PTX]]"], output: "[[CUBIN:.+]]"
// BINDINGS-DEVICE: # "nvptx64-nvidia-cuda" - "NVPTX::Linker", inputs: ["[[CUBIN]]", "[[PTX]]"], output: "{{.*}}.fatbin"

// RUN: %clang -### -target x86_64-linux-gnu -nocudalib --cuda-feature=+ptx61 --offload-arch=sm_70 %s 2>&1 | FileCheck -check-prefix MANUAL-FEATURE %s
// MANUAL-FEATURE: -cc1{{.*}}-target-feature{{.*}}+ptx61

// RUN: %clang -### -target x86_64-linux-gnu -nocudalib -ccc-print-bindings --offload-link %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE-LINK %s

// DEVICE-LINK: "x86_64-unknown-linux-gnu" - "Offload::Linker", inputs: ["[[INPUT:.+]]"], output: "a.out"
