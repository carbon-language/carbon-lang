
#include <hip/hip_runtime.h>
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// -flto causes a switch to llvm-bc object files.
// RUN: %clangxx -nocudainc -nocudalib -ccc-print-phases -c %s -flto 2> %t
// RUN: FileCheck -check-prefix=CHECK-COMPILE-ACTIONS < %t %s
//
// CHECK-COMPILE-ACTIONS: 2: compiler, {1}, ir, (host-cuda)
// CHECK-COMPILE-ACTIONS-NOT: lto-bc
// CHECK-COMPILE-ACTIONS: 12: backend, {11}, lto-bc, (host-cuda)

// RUN: %clangxx -nocudainc -nocudalib -ccc-print-phases %s -flto 2> %t
// RUN: FileCheck -check-prefix=CHECK-COMPILELINK-ACTIONS < %t %s
//
// CHECK-COMPILELINK-ACTIONS: 0: input, "{{.*}}lto.cu", cuda, (host-cuda)
// CHECK-COMPILELINK-ACTIONS: 1: preprocessor, {0}, cuda-cpp-output
// CHECK-COMPILELINK-ACTIONS: 2: compiler, {1}, ir, (host-cuda)
// CHECK-COMPILELINK-ACTIONS: 3: input, "{{.*}}lto.cu", cuda, (device-cuda, sm_{{.*}})
// CHECK-COMPILELINK-ACTIONS: 4: preprocessor, {3}, cuda-cpp-output, (device-cuda, sm_{{.*}})
// CHECK-COMPILELINK-ACTIONS: 5: compiler, {4}, ir, (device-cuda, sm_{{.*}})
// CHECK-COMPILELINK-ACTIONS: 6: backend, {5}, assembler, (device-cuda, sm_{{.*}})
// CHECK-COMPILELINK-ACTIONS: 7: assembler, {6}, object, (device-cuda, sm_{{.*}})
// CHECK-COMPILELINK-ACTIONS: 8: offload, "device-cuda (nvptx{{.*}}-nvidia-cuda:sm_{{.*}})" {7}, object
// CHECK-COMPILELINK-ACTIONS: 9: offload, "device-cuda (nvptx{{.*}}-nvidia-cuda:sm_{{.*}})" {6}, assembler
// CHECK-COMPILELINK-ACTIONS: 10: linker, {8, 9}, cuda-fatbin, (device-cuda)
// CHECK-COMPILELINK-ACTIONS: 11: offload, "host-cuda {{.*}}" {2}, "device-cuda{{.*}}" {10}, ir
// CHECK-COMPILELINK-ACTIONS: 12: backend, {11}, lto-bc, (host-cuda)
// CHECK-COMPILELINK-ACTIONS: 13: linker, {12}, image, (host-cuda)

// llvm-bc and llvm-ll outputs need to match regular suffixes
// (unfortunately).
// RUN: %clangxx %s -nocudainc -nocudalib -flto -save-temps -### 2> %t
// RUN: FileCheck -check-prefix=CHECK-COMPILELINK-SUFFIXES < %t %s
//
// CHECK-COMPILELINK-SUFFIXES: "-o" "[[CPP:.*lto-host.*\.cui]]" "-x" "cuda" "{{.*}}lto.cu"
// CHECK-COMPILELINK-SUFFIXES: "-o" "[[BC:.*lto-host.*\.bc]]" {{.*}}[[CPP]]"
// CHECK-COMPILELINK-SUFFIXES: "-o" "[[OBJ:.*lto-host.*\.o]]" {{.*}}[[BC]]"
// CHECK-COMPILELINK-SUFFIXES: "{{.*}}a.{{(out|exe)}}" {{.*}}[[OBJ]]"

// RUN: %clangxx %s -nocudainc -nocudalib -flto -S -### 2> %t
// RUN: FileCheck -check-prefix=CHECK-COMPILE-SUFFIXES < %t %s
//
// CHECK-COMPILE-SUFFIXES: "-o" "{{.*}}lto.s" "-x" "cuda" "{{.*}}lto.cu"

// RUN: not %clangxx -nocudainc -nocudalib %s -emit-llvm 2>&1 \
// RUN:    | FileCheck --check-prefix=LLVM-LINK %s
// LLVM-LINK: -emit-llvm cannot be used when linking

/// With ld.bfd or gold, link against LLVMgold.
// RUN: %clangxx -nocudainc -nocudalib -target x86_64-unknown-linux-gnu --sysroot %S/Inputs/basic_cross_linux_tree %s \
// RUN:   -fuse-ld=bfd -flto=thin -### 2>&1 | FileCheck --check-prefix=LLVMGOLD %s
// RUN: %clangxx -nocudainc -nocudalib -target x86_64-unknown-linux-gnu --sysroot %S/Inputs/basic_cross_linux_tree %s \
// RUN:   -fuse-ld=gold -flto=full -### 2>&1 | FileCheck --check-prefix=LLVMGOLD %s
//
// LLVMGOLD: "-plugin" "{{.*}}{{[/\\]}}LLVMgold.{{dll|dylib|so}}"

/// lld does not need LLVMgold.
// RUN: %clangxx -nocudainc -nocudalib -target x86_64-unknown-linux-gnu --sysroot %S/Inputs/basic_cross_linux_tree %s \
// RUN:   -fuse-ld=lld -flto=full -### 2>&1 | FileCheck --check-prefix=NO-LLVMGOLD %s
// RUN: %clangxx -nocudainc -nocudalib -target x86_64-unknown-linux-gnu --sysroot %S/Inputs/basic_cross_linux_tree %s \
// RUN:   -fuse-ld=gold -flto=full -fno-lto -### 2>&1 | FileCheck --check-prefix=NO-LLVMGOLD %s
//
// NO-LLVMGOLD-NOT: "-plugin" "{{.*}}{{[/\\]}}LLVMgold.{{dll|dylib|so}}"

// -flto passes along an explicit debugger tuning argument.
// RUN: %clangxx -nocudainc -nocudalib \
// RUN:          -target x86_64-unknown-linux -### %s -flto -glldb 2> %t
// RUN: FileCheck -check-prefix=CHECK-TUNING-LLDB < %t %s
// RUN: %clangxx -nocudainc -nocudalib \
// RUN:          -target x86_64-unknown-linux -### %s -flto -g 2> %t
// RUN: FileCheck -check-prefix=CHECK-NO-TUNING < %t %s
//
// CHECK-TUNING-LLDB:   "-plugin-opt=-debugger-tune=lldb"
// CHECK-NO-TUNING-NOT: "-plugin-opt=-debugger-tune
