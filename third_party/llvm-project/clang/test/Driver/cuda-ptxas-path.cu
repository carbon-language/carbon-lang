
#include <hip/hip_runtime.h>
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// RUN: %clang -### --target=i386-unknown-linux \
// RUN:   --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda \
// RUN:   --ptxas-path=/some/path/to/ptxas %s 2>&1 \
// RUN: | FileCheck %s

// CHECK-NOT: "ptxas"
// CHECK: "/some/path/to/ptxas"
// CHECK-SAME: "--gpu-name" "sm_35"
