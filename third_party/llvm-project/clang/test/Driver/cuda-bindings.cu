
#include <hip/hip_runtime.h>
// Tests the bindings generated for a CUDA offloading target for different
// combinations of:
// - Number of gpu architectures;
// - Host/device-only compilation;
// - User-requested final phase - binary or assembly.
// It parallels cuda-phases.cu test, but verifies whether output file is temporary or not.

// It's hard to check whether file name is temporary in a portable
// way. Instead we check whether we've generated a permanent name on
// device side, which appends '-device-cuda-<triple>' suffix.

// REQUIRES: clang-driver
// REQUIRES: powerpc-registered-target
// REQUIRES: nvptx-registered-target

//
// Test single gpu architecture with complete compilation.
// No intermediary device files should have "-device-cuda..." in the name.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings --cuda-gpu-arch=sm_30 %s 2>&1 \
// RUN: | FileCheck -check-prefix=BIN %s
// BIN: # "nvptx64-nvidia-cuda" - "clang",{{.*}} output:
// BIN-NOT: cuda-bindings-device-cuda-nvptx64
// BIN: # "nvptx64-nvidia-cuda" - "NVPTX::Assembler",{{.*}} output:
// BIN-NOT: cuda-bindings-device-cuda-nvptx64
// BIN: # "nvptx64-nvidia-cuda" - "NVPTX::Linker",{{.*}} output:
// BIN-NOT: cuda-bindings-device-cuda-nvptx64
// BIN: # "powerpc64le-ibm-linux-gnu" - "clang",{{.*}}  output:
// BIN-NOT: cuda-bindings-device-cuda-nvptx64
// BIN: # "powerpc64le-ibm-linux-gnu" - "GNU::Linker", inputs:{{.*}}, output: "a.out"

//
// Test single gpu architecture up to the assemble phase.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings --cuda-gpu-arch=sm_30 %s -S 2>&1 \
// RUN: | FileCheck -check-prefix=ASM %s
// ASM-DAG: # "nvptx64-nvidia-cuda" - "clang",{{.*}} output: "cuda-bindings-cuda-nvptx64-nvidia-cuda-sm_30.s"
// ASM-DAG: # "powerpc64le-ibm-linux-gnu" - "clang",{{.*}} output: "cuda-bindings.s"

//
// Test two gpu architectures with complete compilation.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s 2>&1 \
// RUN: | FileCheck -check-prefix=BIN2 %s
// BIN2: # "nvptx64-nvidia-cuda" - "clang",{{.*}} output:
// BIN2-NOT: cuda-bindings-device-cuda-nvptx64
// BIN2: # "nvptx64-nvidia-cuda" - "NVPTX::Assembler",{{.*}} output:
// BIN2-NOT: cuda-bindings-device-cuda-nvptx64
// BIN2: # "nvptx64-nvidia-cuda" - "clang",{{.*}} output:
// BIN2-NOT: cuda-bindings-device-cuda-nvptx64
// BIN2: # "nvptx64-nvidia-cuda" - "NVPTX::Assembler",{{.*}} output:
// BIN2-NOT: cuda-bindings-device-cuda-nvptx64
// BIN2: # "nvptx64-nvidia-cuda" - "NVPTX::Linker",{{.*}} output:
// BIN2-NOT: cuda-bindings-device-cuda-nvptx64
// BIN2: # "powerpc64le-ibm-linux-gnu" - "clang",{{.*}}  output:
// BIN2-NOT: cuda-bindings-device-cuda-nvptx64
// BIN2: # "powerpc64le-ibm-linux-gnu" - "GNU::Linker", inputs:{{.*}}, output: "a.out"

//
// Test two gpu architectures up to the assemble phase.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings \
// RUN:        --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s -S 2>&1 \
// RUN: | FileCheck -check-prefix=ASM2 %s
// ASM2-DAG: # "nvptx64-nvidia-cuda" - "clang",{{.*}} output: "cuda-bindings-cuda-nvptx64-nvidia-cuda-sm_30.s"
// ASM2-DAG: # "nvptx64-nvidia-cuda" - "clang",{{.*}} output: "cuda-bindings-cuda-nvptx64-nvidia-cuda-sm_35.s"
// ASM2-DAG: # "powerpc64le-ibm-linux-gnu" - "clang",{{.*}} output: "cuda-bindings.s"

//
// Test one or more gpu architecture with complete compilation in host-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings \
// RUN:        --cuda-gpu-arch=sm_30 %s --cuda-host-only 2>&1 \
// RUN: | FileCheck -check-prefix=HBIN %s
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings \
// RUN:        --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s --cuda-host-only 2>&1 \
// RUN: | FileCheck -check-prefix=HBIN %s
// HBIN: # "powerpc64le-ibm-linux-gnu" - "clang",{{.*}}  output:
// HBIN-NOT: cuda-bindings-device-cuda-nvptx64
// HBIN: # "powerpc64le-ibm-linux-gnu" - "GNU::Linker", inputs:{{.*}}, output: "a.out"

//
// Test one or more gpu architecture up to the assemble phase in host-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings \
// RUN:        --cuda-gpu-arch=sm_30 %s --cuda-host-only -S 2>&1 \
// RUN: | FileCheck -check-prefix=HASM %s
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings \
// RUN:        --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s --cuda-host-only -S 2>&1 \
// RUN: | FileCheck -check-prefix=HASM %s
// HASM: # "powerpc64le-ibm-linux-gnu" - "clang",{{.*}} output: "cuda-bindings.s"

//
// Test single gpu architecture with complete compilation in device-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings \
// RUN:        --cuda-gpu-arch=sm_30 %s --cuda-device-only 2>&1 \
// RUN: | FileCheck -check-prefix=DBIN %s
// DBIN: # "nvptx64-nvidia-cuda" - "clang",{{.*}} output:
// DBIN-NOT: cuda-bindings-device-cuda-nvptx64
// DBIN: # "nvptx64-nvidia-cuda" - "NVPTX::Assembler",{{.*}} output: "cuda-bindings-cuda-nvptx64-nvidia-cuda-sm_30.o"

//
// Test single gpu architecture up to the assemble phase in device-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings \
// RUN:        --cuda-gpu-arch=sm_30 %s --cuda-device-only -S 2>&1 \
// RUN: | FileCheck -check-prefix=DASM %s
// DASM: # "nvptx64-nvidia-cuda" - "clang",{{.*}} output: "cuda-bindings-cuda-nvptx64-nvidia-cuda-sm_30.s"

//
// Test two gpu architectures with complete compilation in device-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings \
// RUN:        --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s --cuda-device-only 2>&1 \
// RUN: | FileCheck -check-prefix=DBIN2 %s
// DBIN2: # "nvptx64-nvidia-cuda" - "clang",{{.*}} output:
// DBIN2-NOT: cuda-bindings-device-cuda-nvptx64
// DBIN2: # "nvptx64-nvidia-cuda" - "NVPTX::Assembler",{{.*}} output: "cuda-bindings-cuda-nvptx64-nvidia-cuda-sm_30.o"
// DBIN2: # "nvptx64-nvidia-cuda" - "clang",{{.*}} output:
// DBIN2-NOT: cuda-bindings-device-cuda-nvptx64
// DBIN2: # "nvptx64-nvidia-cuda" - "NVPTX::Assembler",{{.*}} output: "cuda-bindings-cuda-nvptx64-nvidia-cuda-sm_35.o"

//
// Test two gpu architectures up to the assemble phase in device-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings \
// RUN:        --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s --cuda-device-only -S 2>&1 \
// RUN: | FileCheck -check-prefix=DASM2 %s
// DASM2: # "nvptx64-nvidia-cuda" - "clang",{{.*}} output: "cuda-bindings-cuda-nvptx64-nvidia-cuda-sm_30.s"
// DASM2: # "nvptx64-nvidia-cuda" - "clang",{{.*}} output: "cuda-bindings-cuda-nvptx64-nvidia-cuda-sm_35.s"
