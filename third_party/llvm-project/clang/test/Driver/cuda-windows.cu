
#include <hip/hip_runtime.h>
// REQUIRES: clang-driver
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target
//
// RUN: %clang -v --target=i386-pc-windows-msvc \
// RUN:   --sysroot=%S/Inputs/CUDA-windows 2>&1 %s -### | FileCheck %s
// RUN: %clang -v --target=i386-pc-windows-mingw32 \
// RUN:   --sysroot=%S/Inputs/CUDA-windows 2>&1 %s -### | FileCheck %s

// CHECK: Found CUDA installation: {{.*}}/Inputs/CUDA-windows/Program Files/NVIDIA GPU Computing Toolkit/CUDA/v8.0
// CHECK: "-cc1" "-triple" "nvptx-nvidia-cuda"
// CHECK-SAME: "-fms-extensions"
// CHECK-SAME: "-fms-compatibility"
// CHECK-SAME: "-fms-compatibility-version=
