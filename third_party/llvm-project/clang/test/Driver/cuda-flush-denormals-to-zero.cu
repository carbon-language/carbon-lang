
#include <hip/hip_runtime.h>
// Checks that cuda compilation does the right thing when passed
// -fgpu-flush-denormals-to-zero. This should be translated to
// -fdenormal-fp-math-f32=preserve-sign

// RUN: %clang -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=sm_20 -fgpu-flush-denormals-to-zero -nocudainc -nocudalib %s 2>&1 | FileCheck -check-prefix=FTZ %s
// RUN: %clang -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=sm_20 -fno-gpu-flush-denormals-to-zero -nocudainc -nocudalib %s 2>&1 | FileCheck -check-prefix=NOFTZ %s
// RUN: %clang -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=sm_70 -fgpu-flush-denormals-to-zero -nocudainc -nocudalib %s 2>&1 | FileCheck -check-prefix=FTZ %s
// RUN: %clang -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=sm_70 -fno-gpu-flush-denormals-to-zero -nocudainc -nocudalib %s 2>&1 | FileCheck -check-prefix=NOFTZ %s

// Test alias options -f[no-]cuda-flush-denormals-to-zero
// RUN: %clang -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=sm_20 -fcuda-flush-denormals-to-zero -nocudainc -nocudalib %s 2>&1 | FileCheck -check-prefix=FTZ %s
// RUN: %clang -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=sm_20 -fno-cuda-flush-denormals-to-zero -nocudainc -nocudalib %s 2>&1 | FileCheck -check-prefix=NOFTZ %s

// Test explicit argument, with CUDA offload kind
// RUN: %clang -x hip -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=gfx803 -fgpu-flush-denormals-to-zero -nocudainc -nogpulib %s 2>&1 | FileCheck -check-prefix=FTZ %s
// RUN: %clang -x hip -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=gfx803 -fno-gpu-flush-denormals-to-zero -nocudainc -nogpulib %s 2>&1 | FileCheck -check-prefix=NOFTZ %s

// Test explicit argument, with HIP offload kind
// RUN: %clang -x hip -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=gfx803 -fgpu-flush-denormals-to-zero -nocudainc -nogpulib %s 2>&1 | FileCheck -check-prefix=FTZ %s
// RUN: %clang -x hip -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=gfx803 -fno-gpu-flush-denormals-to-zero -nocudainc -nogpulib %s 2>&1 | FileCheck -check-prefix=NOFTZ %s

// RUN: %clang -x hip -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=gfx900 -fgpu-flush-denormals-to-zero -nocudainc -nogpulib %s 2>&1 | FileCheck -check-prefix=FTZ %s
// RUN: %clang -x hip -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=gfx900 -fno-gpu-flush-denormals-to-zero -nocudainc -nogpulib %s 2>&1 | FileCheck -check-prefix=NOFTZ %s

// Test the default changing with no argument based on the subtarget in HIP mode
// RUN: %clang -x hip -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=gfx803 -nocudainc -nogpulib %s 2>&1 | FileCheck -check-prefix=FTZ %s
// RUN: %clang -x hip -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=gfx900 -nocudainc -nogpulib %s 2>&1 | FileCheck -check-prefix=NOFTZ %s

// Test no subtarget, which should get the denormal setting of the default gfx803
// RUN: %clang -x hip -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell -nocudainc -nogpulib %s 2>&1 | FileCheck -check-prefix=FTZ %s

// Test multiple offload archs with different defaults.
// RUN: %clang -x hip -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=gfx803 --cuda-gpu-arch=gfx900 -nocudainc -nogpulib %s 2>&1 | FileCheck -check-prefix=MIXED-DEFAULT-MODE %s
// RUN: %clang -x hip -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell -fgpu-flush-denormals-to-zero --cuda-gpu-arch=gfx803 --cuda-gpu-arch=gfx900 -nocudainc -nogpulib %s 2>&1 | FileCheck -check-prefix=FTZX2 %s
// RUN: %clang -x hip -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell -fno-gpu-flush-denormals-to-zero --cuda-gpu-arch=gfx803 --cuda-gpu-arch=gfx900 -nocudainc -nogpulib %s 2>&1 | FileCheck -check-prefix=NOFTZ %s


// CPUFTZ-NOT: -fdenormal-fp-math

// FTZ-NOT: -fdenormal-fp-math-f32=
// FTZ: "-fdenormal-fp-math-f32=preserve-sign,preserve-sign"

// The default of ieee is omitted
// NOFTZ-NOT: "-fdenormal-fp-math"
// NOFTZ-NOT: "-fdenormal-fp-math-f32"

// MIXED-DEFAULT-MODE-NOT: -denormal-fp-math
// MIXED-DEFAULT-MODE: "-fdenormal-fp-math-f32=preserve-sign,preserve-sign"
// MIXED-DEFAULT-MODE-SAME: "-target-cpu" "gfx803"
// MIXED-DEFAULT-MODE-NOT: -denormal-fp-math

// FTZX2: "-fdenormal-fp-math-f32=preserve-sign,preserve-sign"
// FTZX2-SAME: "-target-cpu" "gfx803"
// FTZX2: "-fdenormal-fp-math-f32=preserve-sign,preserve-sign"
// FTZX2-SAME: "-target-cpu" "gfx900"
