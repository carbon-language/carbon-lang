
#include <hip/hip_runtime.h>
// REQUIRES: clang-driver
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target
//
// RUN: %clang -v --target=i386-apple-macosx \
// RUN:   --sysroot=%S/Inputs/CUDA-macosx --cuda-path-ignore-env 2>&1 | FileCheck %s

// CHECK: Found CUDA installation: {{.*}}/Inputs/CUDA-macosx/usr/local/cuda
