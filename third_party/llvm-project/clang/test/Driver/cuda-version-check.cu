
#include <hip/hip_runtime.h>
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// RUN: %clang --target=x86_64-linux -v -### --cuda-gpu-arch=sm_20 --cuda-path=%S/Inputs/CUDA/usr/local/cuda 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=OK
// RUN: %clang --target=x86_64-linux -v -### --cuda-gpu-arch=sm_20 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=OK
// RUN: %clang --target=x86_64-linux -v -### --cuda-gpu-arch=sm_60 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=OK
// Test version guess when cuda.h has not been found
// RUN: %clang --target=x86_64-linux -v -### --cuda-gpu-arch=sm_60 --cuda-path=%S/Inputs/CUDA-unknown/usr/local/cuda 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=UNKNOWN_VERSION
// Unknown version info present in cuda.h
// RUN: %clang --target=x86_64-linux -v -### --cuda-gpu-arch=sm_60 --cuda-path=%S/Inputs/CUDA-new/usr/local/cuda 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=UNKNOWN_VERSION
// Make sure that we don't warn about CUDA version during C++ compilation.
// RUN: %clang --target=x86_64-linux -v -### -x c++ --cuda-gpu-arch=sm_60 \
// RUN:    --cuda-path=%S/Inputs/CUDA-unknown/usr/local/cuda 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=UNKNOWN_VERSION_CXX

// The installation at Inputs/CUDA is CUDA 7.0, which doesn't support sm_60.
// RUN: %clang --target=x86_64-linux -v -### --cuda-gpu-arch=sm_60 --cuda-path=%S/Inputs/CUDA/usr/local/cuda 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=ERR_SM60

// This should only complain about sm_60, not sm_35.
// RUN: %clang --target=x86_64-linux -v -### --cuda-gpu-arch=sm_60 --cuda-gpu-arch=sm_35 \
// RUN:    --cuda-path=%S/Inputs/CUDA/usr/local/cuda 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=ERR_SM60 --check-prefix=OK_SM35

// We should get two errors here, one for sm_60 and one for sm_61.
// RUN: %clang --target=x86_64-linux -v -### --cuda-gpu-arch=sm_60 --cuda-gpu-arch=sm_61 \
// RUN:    --cuda-path=%S/Inputs/CUDA/usr/local/cuda 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=ERR_SM60 --check-prefix=ERR_SM61

// We should still get an error if we pass -nocudainc, because this compilation
// would invoke ptxas, and we do a version check on that, too.
// RUN: %clang --target=x86_64-linux -v -### --cuda-gpu-arch=sm_60 -nocudainc --cuda-path=%S/Inputs/CUDA/usr/local/cuda 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=ERR_SM60

// If with -nocudainc and -E, we don't touch the CUDA install, so we
// shouldn't get an error.
// RUN: %clang --target=x86_64-linux -v -### -E --cuda-device-only --cuda-gpu-arch=sm_60 -nocudainc \
// RUN:    --cuda-path=%S/Inputs/CUDA/usr/local/cuda 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=OK

// --no-cuda-version-check should suppress all of these errors.
// RUN: %clang --target=x86_64-linux -v -### --cuda-gpu-arch=sm_60 --cuda-path=%S/Inputs/CUDA/usr/local/cuda 2>&1 \
// RUN:    --no-cuda-version-check %s | \
// RUN:    FileCheck %s --check-prefix=OK

// We need to make sure the version check is done only for the device toolchain,
// therefore we should not get an error in host-only mode. We use the -S here
// to avoid the error being produced in case by the assembler tool, which does
// the same check.
// RUN: %clang --target=x86_64-linux -v -### --cuda-gpu-arch=sm_60 --cuda-host-only --cuda-path=%S/Inputs/CUDA/usr/local/cuda -S 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=OK
// RUN: %clang --target=x86_64-linux -v -### --cuda-gpu-arch=sm_60 --cuda-device-only --cuda-path=%S/Inputs/CUDA/usr/local/cuda -S 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=ERR_SM60

// OK-NOT: error: GPU arch

// OK_SM35-NOT: error: GPU arch sm_35

// We should only get one error per architecture.
// ERR_SM20: error: GPU arch sm_20 {{.*}}
// ERR_SM20-NOT: error: GPU arch sm_20

// ERR_SM60: error: GPU arch sm_60 {{.*}}
// ERR_SM60-NOT: error: GPU arch sm_60

// ERR_SM61: error: GPU arch sm_61 {{.*}}
// ERR_SM61-NOT: error: GPU arch sm_61

// UNKNOWN_VERSION: CUDA version is newer than the latest{{.*}} supported version
// UNKNOWN_VERSION_CXX-NOT: unknown CUDA version
