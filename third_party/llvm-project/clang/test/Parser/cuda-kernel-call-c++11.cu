
#include <hip/hip_runtime.h>
// RUN: %clang_cc1 -fsyntax-only -std=c++11 -verify %s

template<typename T=int> struct S {};
template<typename> void f();

template<typename T, typename... V> struct S<T(V...)> {};

template<typename ...T> struct V {};
template<typename ...T> struct V<void(T)...> {};

void foo(void) {
  // In C++11 mode, all of these are expected to parse correctly, and the CUDA
  // language should not interfere with that.

  // expected-no-diagnostics

  S<S<S<int>>> s3;
  S<S<S<>>> s30;

  S<S<S<S<int>>>> s4;
  S<S<S<S<>>>> s40;

  S<S<S<S<S<int>>>>> s5;
  S<S<S<S<S<>>>>> s50;

  (void)(&f<S<S<int>>>==0);
  (void)(&f<S<S<>>>==0);

  S<S<S<void()>>> s6;
}

template<typename ...T>
void bar(T... args) {
  S<S<V<void(T)...>>> s7;
}
