
#include <hip/hip_runtime.h>
// RUN: %clang_cc1 -std=c++14 -fsyntax-only -verify -fcuda-is-device %s

// Check how the force_cuda_host_device pragma interacts with template
// instantiations.

template <typename T>
auto foo() {  // expected-note {{declared here}}
  return T();
}

template <typename T>
struct X {
  void foo(); // expected-note {{declared here}}
};

#pragma clang force_cuda_host_device begin
__attribute__((host)) __attribute__((device)) void test() {
  int n = foo<int>();  // expected-error {{reference to __host__ function 'foo<int>'}}
  X<int>().foo();  // expected-error {{reference to __host__ function 'foo'}}
}
#pragma clang force_cuda_host_device end

// Same thing as above, but within a force_cuda_host_device block without a
// corresponding end.

template <typename T>
T bar() {  // expected-note {{declared here}}
  return T();
}

template <typename T>
struct Y {
  void bar(); // expected-note {{declared here}}
};

#pragma clang force_cuda_host_device begin
__attribute__((host)) __attribute__((device)) void test2() {
  int n = bar<int>();  // expected-error {{reference to __host__ function 'bar<int>'}}
  Y<int>().bar();  // expected-error {{reference to __host__ function 'bar'}}
}
