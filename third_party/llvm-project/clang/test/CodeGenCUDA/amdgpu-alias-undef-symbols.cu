// REQUIRES: amdgpu-registered-target

// RUN: %clang -Xclang -no-opaque-pointers -target x86_64-unknown-linux-gnu --offload-arch=gfx906 --cuda-device-only -nogpulib -nogpuinc -x hip -emit-llvm -S -o - %s \
// RUN:   -fgpu-rdc -O3 -mllvm -amdgpu-early-inline-all=true -mllvm -amdgpu-function-calls=false | \
// RUN:   FileCheck %s

#include "Inputs/hip/hip_runtime.h"

// CHECK: %struct.B = type { i8 }
struct B {

  // CHECK: @_ZN1BC1Ei = hidden unnamed_addr alias void (%struct.B*, i32), void (%struct.B*, i32)* @_ZN1BC2Ei
  __device__ B(int x);
};

__device__ B::B(int x) {
}
