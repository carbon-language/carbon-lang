#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -no-opaque-pointers -triple amdgcn-amd-amdhsa -fcuda-is-device -std=c++11 \
// RUN:   -emit-llvm -o - -x hip %s | FileCheck \
// RUN:   -check-prefixes=COMMON,DEV,NORDC-D %s

// RUN: %clang_cc1 -no-opaque-pointers -triple amdgcn-amd-amdhsa -fcuda-is-device -std=c++11 \
// RUN:   -emit-llvm -fgpu-rdc -cuid=abc -o - -x hip %s > %t.dev
// RUN: cat %t.dev | FileCheck -check-prefixes=COMMON,DEV,RDC-D %s

// RUN: %clang_cc1 -no-opaque-pointers -triple x86_64-gnu-linux -std=c++11 \
// RUN:   -emit-llvm -o - -x hip %s | FileCheck \
// RUN:   -check-prefixes=COMMON,HOST,NORDC %s

// RUN: %clang_cc1 -no-opaque-pointers -triple x86_64-gnu-linux -std=c++11 \
// RUN:   -emit-llvm -fgpu-rdc -cuid=abc -o - -x hip %s > %t.host
// RUN: cat %t.host | FileCheck -check-prefixes=COMMON,HOST,RDC %s

// Check device and host compilation use the same postfix for static
// variable name.

// RUN: cat %t.dev %t.host | FileCheck -check-prefix=POSTFIX %s

#include "Inputs/hip/hip_runtime.h"

struct vec {
  float x,y,z;
};

// DEV-DAG: @x.managed = addrspace(1) externally_initialized global i32 1, align 4
// DEV-DAG: @x = addrspace(1) externally_initialized global i32 addrspace(1)* null
// NORDC-DAG: @x.managed = internal global i32 1
// RDC-DAG: @x.managed = global i32 1
// NORDC-DAG: @x = internal externally_initialized global i32* null
// RDC-DAG: @x = externally_initialized global i32* null
// HOST-DAG: @[[DEVNAMEX:[0-9]+]] = {{.*}}c"x\00"
__managed__ int x = 1;

// DEV-DAG: @v.managed = addrspace(1) externally_initialized global [100 x %struct.vec] zeroinitializer, align 4
// DEV-DAG: @v = addrspace(1) externally_initialized global [100 x %struct.vec] addrspace(1)* null
__managed__ vec v[100];

// DEV-DAG: @v2.managed = addrspace(1) externally_initialized global <{ %struct.vec, [99 x %struct.vec] }> <{ %struct.vec { float 1.000000e+00, float 1.000000e+00, float 1.000000e+00 }, [99 x %struct.vec] zeroinitializer }>, align 4
// DEV-DAG: @v2 = addrspace(1) externally_initialized global <{ %struct.vec, [99 x %struct.vec] }> addrspace(1)* null
__managed__ vec v2[100] = {{1, 1, 1}};

// DEV-DAG: @ex.managed = external addrspace(1) global i32, align 4
// DEV-DAG: @ex = external addrspace(1) externally_initialized global i32 addrspace(1)*
// HOST-DAG: @ex.managed = external global i32
// HOST-DAG: @ex = external externally_initialized global i32*
extern __managed__ int ex;

// NORDC-D-DAG: @_ZL2sx.managed = addrspace(1) externally_initialized global i32 1, align 4
// NORDC-D-DAG: @_ZL2sx = addrspace(1) externally_initialized global i32 addrspace(1)* null
// RDC-D-DAG: @_ZL2sx.static.[[HASH:.*]].managed = addrspace(1) externally_initialized global i32 1, align 4
// RDC-D-DAG: @_ZL2sx.static.[[HASH]] = addrspace(1) externally_initialized global i32 addrspace(1)* null
// HOST-DAG: @_ZL2sx.managed = internal global i32 1
// HOST-DAG: @_ZL2sx = internal externally_initialized global i32* null
// NORDC-DAG: @[[DEVNAMESX:[0-9]+]] = {{.*}}c"_ZL2sx\00"
// RDC-DAG: @[[DEVNAMESX:[0-9]+]] = {{.*}}c"_ZL2sx.static.[[HASH:.*]]\00"

// POSTFIX:  @_ZL2sx.static.[[HASH:.*]] = addrspace(1) externally_initialized global i32 addrspace(1)* null
// POSTFIX: @[[DEVNAMESX:[0-9]+]] = {{.*}}c"_ZL2sx.static.[[HASH]]\00"
static __managed__ int sx = 1;

// DEV-DAG: @llvm.compiler.used
// DEV-SAME-DAG: @x.managed
// DEV-SAME-DAG: @x
// DEV-SAME-DAG: @v.managed
// DEV-SAME-DAG: @v
// DEV-SAME-DAG: @_ZL2sx.managed
// DEV-SAME-DAG: @_ZL2sx

// Force ex and sx mitted in device compilation.
__global__ void foo(int *z) {
  *z = x + ex + sx;
  v[1].x = 2;
}

// Force ex and sx emitted in host compilatioin.
int foo2() {
  return ex + sx;
}

// COMMON-LABEL: define {{.*}}@_Z4loadv()
// DEV:  %ld.managed = load i32 addrspace(1)*, i32 addrspace(1)* addrspace(1)* @x, align 4
// DEV:  %0 = addrspacecast i32 addrspace(1)* %ld.managed to i32*
// DEV:  %1 = load i32, i32* %0, align 4
// DEV:  ret i32 %1
// HOST:  %ld.managed = load i32*, i32** @x, align 4
// HOST:  %0 = load i32, i32* %ld.managed, align 4
// HOST:  ret i32 %0
__device__ __host__ int load() {
  return x;
}

// COMMON-LABEL: define {{.*}}@_Z5storev()
// DEV:  %ld.managed = load i32 addrspace(1)*, i32 addrspace(1)* addrspace(1)* @x, align 4
// DEV:  %0 = addrspacecast i32 addrspace(1)* %ld.managed to i32*
// DEV:  store i32 2, i32* %0, align 4
// HOST:  %ld.managed = load i32*, i32** @x, align 4
// HOST:  store i32 2, i32* %ld.managed, align 4
__device__ __host__ void store() {
  x = 2;
}

// COMMON-LABEL: define {{.*}}@_Z10addr_takenv()
// DEV:  %0 = addrspacecast i32 addrspace(1)* %ld.managed to i32*
// DEV:  store i32* %0, i32** %p.ascast, align 8
// DEV:  %1 = load i32*, i32** %p.ascast, align 8
// DEV:  store i32 3, i32* %1, align 4
// HOST:  %ld.managed = load i32*, i32** @x, align 4
// HOST:  store i32* %ld.managed, i32** %p, align 8
// HOST:  %0 = load i32*, i32** %p, align 8
// HOST:  store i32 3, i32* %0, align 4
__device__ __host__ void addr_taken() {
  int *p = &x;
  *p = 3;
}

// HOST-LABEL: define {{.*}}@_Z5load2v()
// HOST: %ld.managed = load [100 x %struct.vec]*, [100 x %struct.vec]** @v, align 16
// HOST:  %0 = getelementptr inbounds [100 x %struct.vec], [100 x %struct.vec]* %ld.managed, i64 0, i64 1, i32 0
// HOST:  %1 = load float, float* %0, align 4
// HOST:  ret float %1
__device__ __host__ float load2() {
  return v[1].x;
}

// HOST-LABEL: define {{.*}}@_Z5load3v()
// HOST:  %ld.managed = load <{ %struct.vec, [99 x %struct.vec] }>*, <{ %struct.vec, [99 x %struct.vec] }>** @v2, align 16
// HOST:  %0 = bitcast <{ %struct.vec, [99 x %struct.vec] }>* %ld.managed to [100 x %struct.vec]*
// HOST:  %1 = getelementptr inbounds [100 x %struct.vec], [100 x %struct.vec]* %0, i64 0, i64 1, i32 1
// HOST:  %2 = load float, float* %1, align 4
// HOST:  ret float %2
float load3() {
  return v2[1].y;
}

// HOST-LABEL: define {{.*}}@_Z11addr_taken2v()
// HOST:  %ld.managed = load [100 x %struct.vec]*, [100 x %struct.vec]** @v, align 16
// HOST:  %0 = getelementptr inbounds [100 x %struct.vec], [100 x %struct.vec]* %ld.managed, i64 0, i64 1, i32 0
// HOST:  %1 = ptrtoint float* %0 to i64
// HOST:  %ld.managed1 = load <{ %struct.vec, [99 x %struct.vec] }>*, <{ %struct.vec, [99 x %struct.vec] }>** @v2, align 16
// HOST:  %2 = bitcast <{ %struct.vec, [99 x %struct.vec] }>* %ld.managed1 to [100 x %struct.vec]*
// HOST:  %3 = getelementptr inbounds [100 x %struct.vec], [100 x %struct.vec]* %2, i64 0, i64 1, i32 1
// HOST:  %4 = ptrtoint float* %3 to i64
// HOST:  %5 = sub i64 %4, %1
// HOST:  %6 = sdiv exact i64 %5, 4
// HOST:  %7 = sitofp i64 %6 to float
// HOST:  ret float %7
float addr_taken2() {
  return (float)reinterpret_cast<long>(&(v2[1].y)-&(v[1].x));
}

// COMMON-LABEL: define {{.*}}@_Z5load4v()
// DEV:  %ld.managed = load i32 addrspace(1)*, i32 addrspace(1)* addrspace(1)* @ex, align 4
// DEV:  %0 = addrspacecast i32 addrspace(1)* %ld.managed to i32*
// DEV:  %1 = load i32, i32* %0, align 4
// DEV:  ret i32 %1
// HOST:  %ld.managed = load i32*, i32** @ex, align 4
// HOST:  %0 = load i32, i32* %ld.managed, align 4
// HOST:  ret i32 %0
__device__ __host__ int load4() {
  return ex;
}

// HOST-DAG: __hipRegisterManagedVar({{.*}}@x {{.*}}@x.managed {{.*}}@[[DEVNAMEX]]{{.*}}, i64 4, i32 4)
// HOST-DAG: __hipRegisterManagedVar({{.*}}@_ZL2sx {{.*}}@_ZL2sx.managed {{.*}}@[[DEVNAMESX]]
// HOST-NOT: __hipRegisterManagedVar({{.*}}@ex {{.*}}@ex.managed
// HOST-DAG: declare void @__hipRegisterManagedVar(i8**, i8*, i8*, i8*, i64, i32)
