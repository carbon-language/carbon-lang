#include "hip/hip_runtime.h"
// RUN: not %clang_cc1 -std=c++11 -triple x86_64-unknown-linux-gnu \
// RUN:   -emit-llvm -o - %s 2>&1 | FileCheck %s
// RUN: not %clang_cc1 -std=c++11 -triple x86_64-unknown-linux-gnu \
// RUN:   -fcuda-is-device -emit-llvm -o - %s 2>&1 \
// RUN:   | FileCheck %s

// Check no crash due to deferred diagnostics.

#include "Inputs/hip/hip_runtime.h"

// CHECK: error: invalid output constraint '=h' in asm
// CHECK-NOT: core dump
inline __host__ __device__ int foo() {
  short h;
  __asm__("dont care" : "=h"(h) : "f"(0.0), "d"(0.0), "h"(0), "r"(0), "l"(0));
  return 0;
}

void host_fun() {
  foo();
}

__global__ void kernel() {
  foo();
}
