#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -no-opaque-pointers -triple x86_64-unknown-gnu-linux -aux-triple amdgcn-amd-amdhsa \
// RUN:   -emit-llvm -o - -x hip %s | FileCheck -check-prefixes=CHECK,LNX %s
// RUN: %clang_cc1 -no-opaque-pointers -triple x86_64-unknown-windows-msvc -aux-triple amdgcn-amd-amdhsa \
// RUN:   -emit-llvm -o - -x hip %s | FileCheck -check-prefixes=CHECK,MSVC %s

#include "Inputs/hip/hip_runtime.h"

namespace X {
  __global__ void kern1(int *x);
  __device__ int var1;
}

// CHECK: @[[STR1:.*]] = {{.*}} c"_ZN1X5kern1EPi\00"
// CHECK: @[[STR2:.*]] = {{.*}} c"_ZN1X4var1E\00"

// LNX-LABEL: define {{.*}}@_Z4fun1v()
// MSVC-LABEL: define {{.*}} @"?fun1@@YAPEBDXZ"()
// CHECK: ret i8* getelementptr inbounds ({{.*}} @[[STR1]], i64 0, i64 0)
const char *fun1() {
  return __builtin_get_device_side_mangled_name(X::kern1);
}

// LNX-LABEL: define {{.*}}@_Z4fun2v()
// MSVC-LABEL: define {{.*}}@"?fun2@@YAPEBDXZ"()
// CHECK: ret i8* getelementptr inbounds ({{.*}} @[[STR2]], i64 0, i64 0)
__host__ __device__ const char *fun2() {
  return __builtin_get_device_side_mangled_name(X::var1);
}
