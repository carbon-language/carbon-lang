#include "hip/hip_runtime.h"
// RUN: %clang_cc1 %s -emit-llvm -o - -triple=amdgcn-amd-amdhsa \
// RUN:   -fcuda-is-device -target-cpu gfx906 -fsanitize=address \
// RUN:   -O3 -x hip | FileCheck -check-prefixes=MFCHECK %s

// MFCHECK: !llvm.module.flags = !{![[FLAG1:[0-9]+]], ![[FLAG2:[0-9]+]]}
// MFCHECK: ![[FLAG1]] = !{i32 4, !"amdgpu_hostcall", i32 1}

// Test to check hostcall module flag metadata is generated correctly
// when a program has printf call and compiled with -fsanitize=address.
#include "Inputs/hip/hip_runtime.h"
__device__ void non_kernel() {
  printf("sanitized device function");
}

__global__ void kernel() {
  non_kernel();
}

