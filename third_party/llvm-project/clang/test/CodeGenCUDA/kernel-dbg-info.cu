#include "hip/hip_runtime.h"
// RUN: echo "GPU binary would be here" > %t

// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s -O0 \
// RUN:   -fcuda-include-gpubinary %t -debug-info-kind=limited \
// RUN:   -o - -x hip | FileCheck -check-prefixes=CHECK,O0 %s
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -emit-llvm %s -O0 \
// RUN:   -fcuda-include-gpubinary %t -debug-info-kind=limited \
// RUN:   -o - -x hip -fcuda-is-device | FileCheck -check-prefix=DEV %s

// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s -O0 \
// RUN:   -fcuda-include-gpubinary %t -debug-info-kind=limited \
// RUN:   -o - -x hip -debugger-tuning=gdb -dwarf-version=4 \
// RUN:   | FileCheck -check-prefixes=CHECK,O0 %s
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -emit-llvm %s -O0 \
// RUN:   -fcuda-include-gpubinary %t -debug-info-kind=limited \
// RUN:   -o - -x hip -debugger-tuning=gdb -dwarf-version=4 \
// RUN:   -fcuda-is-device | FileCheck -check-prefix=DEV %s

// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s -O3 \
// RUN:   -fcuda-include-gpubinary %t -debug-info-kind=limited \
// RUN:   -o - -x hip -debugger-tuning=gdb -dwarf-version=4 | FileCheck %s
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -emit-llvm %s -O3 \
// RUN:   -fcuda-include-gpubinary %t -debug-info-kind=limited \
// RUN:   -o - -x hip -debugger-tuning=gdb -dwarf-version=4 \
// RUN:   -fcuda-is-device | FileCheck -check-prefix=DEV %s

#include "Inputs/hip/hip_runtime.h"

extern "C" __global__ void ckernel(int *a) {
  *a = 1;
}

// Kernel symbol for launching kernel.
// CHECK: @[[SYM:ckernel]] = constant void (i32*)* @__device_stub__ckernel, align 8

// Device side kernel names
// CHECK: @[[CKERN:[0-9]*]] = {{.*}} c"ckernel\00"

// DEV: define {{.*}}@ckernel{{.*}}!dbg
// DEV:  store {{.*}}!dbg
// DEV:  ret {{.*}}!dbg

// Make sure there is no !dbg between function attributes and '{'
// CHECK: define{{.*}} void @[[CSTUB:__device_stub__ckernel]]{{.*}} #{{[0-9]+}} {
// CHECK-NOT: call {{.*}}@hipLaunchByPtr{{.*}}!dbg
// CHECK: call {{.*}}@hipLaunchByPtr{{.*}}@[[SYM]]
// CHECK-NOT: ret {{.*}}!dbg

// CHECK-LABEL: define {{.*}}@_Z8hostfuncPi{{.*}}!dbg
// O0: call void @[[CSTUB]]{{.*}}!dbg
void hostfunc(int *a) {
  ckernel<<<1, 1>>>(a);
}
