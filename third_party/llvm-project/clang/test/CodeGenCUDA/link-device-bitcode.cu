#include "hip/hip_runtime.h"
// Test for linking with CUDA's libdevice as outlined in
// http://llvm.org/docs/NVPTXUsage.html#linking-with-libdevice
//
// REQUIRES: nvptx-registered-target
//
// Prepare bitcode file to link with
// RUN: %clang_cc1 -triple nvptx-unknown-cuda -emit-llvm-bc \
// RUN:    -disable-llvm-passes -o %t.bc %S/Inputs/device-code.ll
// RUN: %clang_cc1 -triple nvptx-unknown-cuda -emit-llvm-bc \
// RUN:    -disable-llvm-passes -o %t-2.bc %S/Inputs/device-code-2.ll
//
// Make sure function in device-code gets linked in and internalized.
// RUN: %clang_cc1 -triple nvptx-unknown-cuda -fcuda-is-device \
// RUN:    -mlink-builtin-bitcode %t.bc  -emit-llvm \
// RUN:    -disable-llvm-passes -o - %s \
// RUN:    | FileCheck %s -check-prefix CHECK-IR

// Make sure legacy flag name works
// RUN: %clang_cc1 -triple nvptx-unknown-cuda -fcuda-is-device \
// RUN:    -mlink-cuda-bitcode %t.bc  -emit-llvm \
// RUN:    -disable-llvm-passes -o - %s \
// RUN:    | FileCheck %s -check-prefix CHECK-IR
//
// Make sure we can link two bitcode files.
// RUN: %clang_cc1 -triple nvptx-unknown-cuda -fcuda-is-device \
// RUN:    -mlink-builtin-bitcode %t.bc -mlink-builtin-bitcode %t-2.bc \
// RUN:    -emit-llvm -disable-llvm-passes -o - %s \
// RUN:    | FileCheck %s -check-prefix CHECK-IR -check-prefix CHECK-IR-2
//
// Make sure function in device-code gets linked but is not internalized
// without -fcuda-uses-libdevice
// RUN: %clang_cc1 -triple nvptx-unknown-cuda -fcuda-is-device \
// RUN:    -mlink-bitcode-file %t.bc -emit-llvm \
// RUN:    -disable-llvm-passes -o - %s \
// RUN:    | FileCheck %s -check-prefix CHECK-IR-NLD
//
// Make sure NVVMReflect pass is enabled in NVPTX back-end.
// RUN: %clang_cc1 -triple nvptx-unknown-cuda -fcuda-is-device \
// RUN:    -mlink-builtin-bitcode %t.bc -S -o /dev/null %s \
// RUN:    -mllvm -debug-pass=Structure 2>&1 \
// RUN:    | FileCheck %s -check-prefix CHECK-REFLECT

#include "Inputs/hip/hip_runtime.h"

__device__ float device_mul_or_add(float a, float b);
extern "C" __device__ double __nv_sin(double x);
extern "C" __device__ double __nv_exp(double x);

// CHECK-IR-LABEL: define{{.*}} void @_Z26should_not_be_internalizedPf(
// CHECK-PTX-LABEL: .visible .func _Z26should_not_be_internalizedPf(
__device__ void should_not_be_internalized(float *data) {}

// Make sure kernel call has not been internalized.
// CHECK-IR-LABEL: define{{.*}} void @_Z6kernelPfS_
// CHECK-PTX-LABEL: .visible .entry _Z6kernelPfS_(
__global__ __attribute__((used)) void kernel(float *out, float *in) {
  *out = device_mul_or_add(in[0], in[1]);
  *out += __nv_exp(__nv_sin(*out));
  should_not_be_internalized(out);
}

// Make sure device_mul_or_add() is present in IR, is internal and
// calls __nvvm_reflect().
// CHECK-IR-LABEL: define internal float @_Z17device_mul_or_addff(
// CHECK-IR-NLD-LABEL: define{{.*}} float @_Z17device_mul_or_addff(
// CHECK-IR: call i32 @__nvvm_reflect
// CHECK-IR: ret float

// Make sure we've linked in and internalized only needed functions
// from the second bitcode file.
// CHECK-IR-2-LABEL: define internal double @__nv_sin
// CHECK-IR-2-LABEL: define internal double @__nv_exp
// CHECK-IR-2-NOT: double @__unused

// Verify that NVVMReflect pass is among the passes run by NVPTX back-end.
// CHECK-REFLECT: Replace occurrences of __nvvm_reflect() calls with 0/1
