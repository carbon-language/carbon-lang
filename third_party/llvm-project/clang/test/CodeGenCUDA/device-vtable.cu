// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// Make sure we don't emit vtables for classes with methods that have
// inappropriate target attributes. Currently it's mostly needed in
// order to avoid emitting vtables for host-only classes on device
// side where we can't codegen them.

// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -emit-llvm -o - %s \
// RUN:     | FileCheck %s -check-prefix=CHECK-HOST -check-prefix=CHECK-BOTH
// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fcuda-is-device -emit-llvm -o - %s \
// RUN:     | FileCheck %s -check-prefix=CHECK-DEVICE -check-prefix=CHECK-BOTH

#include "Inputs/hip/hip_runtime.h"

struct H  {
  virtual void method();
};
//CHECK-HOST: @_ZTV1H =
//CHECK-HOST-SAME: @_ZN1H6methodEv
//CHECK-DEVICE-NOT: @_ZTV1H =
//CHECK-DEVICE-NOT: @_ZTVN10__cxxabiv117__class_type_infoE
//CHECK-DEVICE-NOT: @_ZTS1H
//CHECK-DEVICE-NOT: @_ZTI1H
struct D  {
   __device__ virtual void method();
};

//CHECK-DEVICE: @_ZTV1D
//CHECK-DEVICE-SAME: @_ZN1D6methodEv
//CHECK-HOST-NOT: @_ZTV1D
//CHECK-DEVICE-NOT: @_ZTVN10__cxxabiv117__class_type_infoE
//CHECK-DEVICE-NOT: @_ZTS1D
//CHECK-DEVICE-NOT: @_ZTI1D
// This is the case with mixed host and device virtual methods.  It's
// impossible to emit a valid vtable in that case because only host or
// only device methods would be available during host or device
// compilation. At the moment Clang (and NVCC) emit NULL pointers for
// unavailable methods,
struct HD  {
  virtual void h_method();
  __device__ virtual void d_method();
};
// CHECK-BOTH: @_ZTV2HD
// CHECK-DEVICE-NOT: @_ZN2HD8h_methodEv
// CHECK-DEVICE-SAME: null
// CHECK-DEVICE-SAME: @_ZN2HD8d_methodEv
// CHECK-HOST-SAME: @_ZN2HD8h_methodEv
// CHECK-HOST-NOT: @_ZN2HD8d_methodEv
// CHECK-HOST-SAME: null
// CHECK-BOTH-SAME: ]
// CHECK-DEVICE-NOT: @_ZTVN10__cxxabiv117__class_type_infoE
// CHECK-DEVICE-NOT: @_ZTS2HD
// CHECK-DEVICE-NOT: @_ZTI2HD

void H::method() {}
//CHECK-HOST: define{{.*}} void @_ZN1H6methodEv

void __device__ D::method() {}
//CHECK-DEVICE: define{{.*}} void @_ZN1D6methodEv

void __device__ HD::d_method() {}
// CHECK-DEVICE: define{{.*}} void @_ZN2HD8d_methodEv
// CHECK-HOST-NOT: define{{.*}} void @_ZN2HD8d_methodEv
void HD::h_method() {}
// CHECK-HOST: define{{.*}} void @_ZN2HD8h_methodEv
// CHECK-DEVICE-NOT: define{{.*}} void @_ZN2HD8h_methodEv

