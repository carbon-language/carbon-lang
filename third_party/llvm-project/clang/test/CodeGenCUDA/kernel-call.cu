#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -target-sdk-version=8.0 -emit-llvm %s -o - \
// RUN: | FileCheck %s --check-prefixes=CUDA-OLD,CHECK
// RUN: %clang_cc1 -target-sdk-version=9.2  -emit-llvm %s -o - \
// RUN: | FileCheck %s --check-prefixes=CUDA-NEW,CHECK
// RUN: %clang_cc1 -x hip -emit-llvm %s -o - \
// RUN: | FileCheck %s --check-prefixes=HIP-OLD,CHECK
// RUN: %clang_cc1 -fhip-new-launch-api -x hip -emit-llvm %s -o - \
// RUN: | FileCheck %s --check-prefixes=HIP-NEW,LEGACY,CHECK
// RUN: %clang_cc1 -fhip-new-launch-api -x hip -emit-llvm %s -o - \
// RUN:   -fgpu-default-stream=legacy \
// RUN:   | FileCheck %s --check-prefixes=HIP-NEW,LEGACY,CHECK
// RUN: %clang_cc1 -fhip-new-launch-api -x hip -emit-llvm %s -o - \
// RUN:   -fgpu-default-stream=per-thread -DHIP_API_PER_THREAD_DEFAULT_STREAM \
// RUN:   | FileCheck %s --check-prefixes=HIP-NEW,PTH,CHECK

#include "Inputs/hip/hip_runtime.h"

// CHECK-LABEL: define{{.*}}g1
// HIP-OLD: call{{.*}}hipSetupArgument
// HIP-OLD: call{{.*}}hipLaunchByPtr
// HIP-NEW: call{{.*}}__hipPopCallConfiguration
// LEGACY: call{{.*}}hipLaunchKernel
// PTH: call{{.*}}hipLaunchKernel_spt
// CUDA-OLD: call{{.*}}hipSetupArgument
// CUDA-OLD: call{{.*}}hipLaunchByPtr
// CUDA-NEW: call{{.*}}__cudaPopCallConfiguration
// CUDA-NEW: call{{.*}}hipLaunchKernel
__global__ void g1(int x) {}

// CHECK-LABEL: define{{.*}}main
int main(void) {
  // HIP-OLD: call{{.*}}hipConfigureCall
  // HIP-NEW: call{{.*}}__hipPushCallConfiguration
  // CUDA-OLD: call{{.*}}hipConfigureCall
  // CUDA-NEW: call{{.*}}__cudaPushCallConfiguration
  // CHECK: icmp
  // CHECK: br
  // CHECK: call{{.*}}g1
  g1<<<1, 1>>>(42);
}
