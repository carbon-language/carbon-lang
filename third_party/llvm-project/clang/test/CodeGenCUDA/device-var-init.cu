#include "hip/hip_runtime.h"
// REQUIRES: nvptx-registered-target
// REQUIRES: amdgpu-registered-target

// Make sure we don't allow dynamic initialization for device
// variables, but accept empty constructors allowed by CUDA.

// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fcuda-is-device -std=c++11 \
// RUN:     -fno-threadsafe-statics -emit-llvm -o - %s | FileCheck -check-prefixes=DEVICE,NVPTX %s
// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -std=c++11 \
// RUN:     -fno-threadsafe-statics -emit-llvm -o - %s | FileCheck -check-prefixes=HOST %s

// RUN: %clang_cc1 -triple amdgcn -fcuda-is-device -std=c++11 \
// RUN:     -fno-threadsafe-statics -emit-llvm -o - %s | FileCheck -check-prefixes=DEVICE,AMDGCN %s

#ifdef __clang__
#include "Inputs/hip/hip_runtime.h"
#endif

// Use the types we share with Sema tests.
#include "Inputs/cuda-initializers.h"

__device__ int d_v;
// DEVICE: @d_v ={{.*}} addrspace(1) externally_initialized global i32 0,
// HOST:   @d_v = internal global i32 undef,
__shared__ int s_v;
// DEVICE: @s_v ={{.*}} addrspace(3) global i32 undef,
// HOST:   @s_v = internal global i32 undef,
__constant__ int c_v;
// DEVICE: addrspace(4) externally_initialized global i32 0,
// HOST:   @c_v = internal global i32 undef,

__device__ int d_v_i = 1;
// DEVICE: @d_v_i ={{.*}} addrspace(1) externally_initialized global i32 1,
// HOST:   @d_v_i = internal global i32 undef,

// For `static` device variables, assume they won't be addressed from the host
// side.
static __device__ int d_s_v_i = 1;
// DEVICE: @_ZL7d_s_v_i = internal addrspace(1) global i32 1,

// Dummy function to keep static variables referenced.
__device__ int foo() {
  return d_s_v_i;
}

// trivial constructor -- allowed
__device__ T d_t;
// DEVICE: @d_t ={{.*}} addrspace(1) externally_initialized global %struct.T zeroinitializer
// HOST:   @d_t = internal global %struct.T undef,
__shared__ T s_t;
// DEVICE: @s_t ={{.*}} addrspace(3) global %struct.T undef,
// HOST:   @s_t = internal global %struct.T undef,
__constant__ T c_t;
// DEVICE: @c_t ={{.*}} addrspace(4) externally_initialized global %struct.T zeroinitializer,
// HOST:   @c_t = internal global %struct.T undef,

__device__ T d_t_i = {2};
// DEVICE: @d_t_i ={{.*}} addrspace(1) externally_initialized global %struct.T { i32 2 },
// HOST:   @d_t_i = internal global %struct.T undef,
__constant__ T c_t_i = {2};
// DEVICE: @c_t_i ={{.*}} addrspace(4) externally_initialized global %struct.T { i32 2 },
// HOST:   @c_t_i = internal global %struct.T undef,

// empty constructor
__device__ EC d_ec;
// DEVICE: @d_ec ={{.*}} addrspace(1) externally_initialized global %struct.EC zeroinitializer,
// HOST:   @d_ec = internal global %struct.EC undef,
__shared__ EC s_ec;
// DEVICE: @s_ec ={{.*}} addrspace(3) global %struct.EC undef,
// HOST:   @s_ec = internal global %struct.EC undef,
__constant__ EC c_ec;
// DEVICE: @c_ec ={{.*}} addrspace(4) externally_initialized global %struct.EC zeroinitializer,
// HOST:   @c_ec = internal global %struct.EC undef

// empty destructor
__device__ ED d_ed;
// DEVICE: @d_ed ={{.*}} addrspace(1) externally_initialized global %struct.ED zeroinitializer,
// HOST:   @d_ed = internal global %struct.ED undef,
__shared__ ED s_ed;
// DEVICE: @s_ed ={{.*}} addrspace(3) global %struct.ED undef,
// HOST:   @s_ed = internal global %struct.ED undef,
__constant__ ED c_ed;
// DEVICE: @c_ed ={{.*}} addrspace(4) externally_initialized global %struct.ED zeroinitializer,
// HOST:   @c_ed = internal global %struct.ED undef,

__device__ ECD d_ecd;
// DEVICE: @d_ecd ={{.*}} addrspace(1) externally_initialized global %struct.ECD zeroinitializer,
// HOST:   @d_ecd = internal global %struct.ECD undef,
__shared__ ECD s_ecd;
// DEVICE: @s_ecd ={{.*}} addrspace(3) global %struct.ECD undef,
// HOST:   @s_ecd = internal global %struct.ECD undef,
__constant__ ECD c_ecd;
// DEVICE: @c_ecd ={{.*}} addrspace(4) externally_initialized global %struct.ECD zeroinitializer,
// HOST:   @c_ecd = internal global %struct.ECD undef,

// empty templated constructor -- allowed with no arguments
__device__ ETC d_etc;
// DEVICE: @d_etc ={{.*}} addrspace(1) externally_initialized global %struct.ETC zeroinitializer,
// HOST:   @d_etc = internal global %struct.ETC undef,
__shared__ ETC s_etc;
// DEVICE: @s_etc ={{.*}} addrspace(3) global %struct.ETC undef,
// HOST:   @s_etc = internal global %struct.ETC undef,
__constant__ ETC c_etc;
// DEVICE: @c_etc ={{.*}} addrspace(4) externally_initialized global %struct.ETC zeroinitializer,
// HOST:   @c_etc = internal global %struct.ETC undef,

__device__ NCFS d_ncfs;
// DEVICE: @d_ncfs ={{.*}} addrspace(1) externally_initialized global %struct.NCFS { i32 3 }
// HOST:   @d_ncfs = internal global %struct.NCFS undef,
__constant__ NCFS c_ncfs;
// DEVICE: @c_ncfs ={{.*}} addrspace(4) externally_initialized global %struct.NCFS { i32 3 }
// HOST:   @c_ncfs = internal global %struct.NCFS undef,

// Regular base class -- allowed
__device__ T_B_T d_t_b_t;
// DEVICE: @d_t_b_t ={{.*}} addrspace(1) externally_initialized global %struct.T_B_T zeroinitializer,
// HOST:   @d_t_b_t = internal global %struct.T_B_T undef,
__shared__ T_B_T s_t_b_t;
// DEVICE: @s_t_b_t ={{.*}} addrspace(3) global %struct.T_B_T undef,
// HOST:   @s_t_b_t = internal global %struct.T_B_T undef,
__constant__ T_B_T c_t_b_t;
// DEVICE: @c_t_b_t ={{.*}} addrspace(4) externally_initialized global %struct.T_B_T zeroinitializer,
// HOST:   @c_t_b_t = internal global %struct.T_B_T undef,

// Incapsulated object of allowed class -- allowed
__device__ T_F_T d_t_f_t;
// DEVICE: @d_t_f_t ={{.*}} addrspace(1) externally_initialized global %struct.T_F_T zeroinitializer,
// HOST:   @d_t_f_t = internal global %struct.T_F_T undef,
__shared__ T_F_T s_t_f_t;
// DEVICE: @s_t_f_t ={{.*}} addrspace(3) global %struct.T_F_T undef,
// HOST:   @s_t_f_t = internal global %struct.T_F_T undef,
__constant__ T_F_T c_t_f_t;
// DEVICE: @c_t_f_t ={{.*}} addrspace(4) externally_initialized global %struct.T_F_T zeroinitializer,
// HOST:   @c_t_f_t = internal global %struct.T_F_T undef,

// array of allowed objects -- allowed
__device__ T_FA_T d_t_fa_t;
// DEVICE: @d_t_fa_t ={{.*}} addrspace(1) externally_initialized global %struct.T_FA_T zeroinitializer,
// HOST:   @d_t_fa_t = internal global %struct.T_FA_T undef,
__shared__ T_FA_T s_t_fa_t;
// DEVICE: @s_t_fa_t ={{.*}} addrspace(3) global %struct.T_FA_T undef,
// HOST:   @s_t_fa_t = internal global %struct.T_FA_T undef,
__constant__ T_FA_T c_t_fa_t;
// DEVICE: @c_t_fa_t ={{.*}} addrspace(4) externally_initialized global %struct.T_FA_T zeroinitializer,
// HOST:   @c_t_fa_t = internal global %struct.T_FA_T undef,


// Calling empty base class initializer is OK
__device__ EC_I_EC d_ec_i_ec;
// DEVICE: @d_ec_i_ec ={{.*}} addrspace(1) externally_initialized global %struct.EC_I_EC zeroinitializer,
// HOST:   @d_ec_i_ec = internal global %struct.EC_I_EC undef,
__shared__ EC_I_EC s_ec_i_ec;
// DEVICE: @s_ec_i_ec ={{.*}} addrspace(3) global %struct.EC_I_EC undef,
// HOST:   @s_ec_i_ec = internal global %struct.EC_I_EC undef,
__constant__ EC_I_EC c_ec_i_ec;
// DEVICE: @c_ec_i_ec ={{.*}} addrspace(4) externally_initialized global %struct.EC_I_EC zeroinitializer,
// HOST:   @c_ec_i_ec = internal global %struct.EC_I_EC undef,

// DEVICE: @_ZZ2dfvE4s_ec = internal addrspace(3) global %struct.EC undef
// DEVICE: @_ZZ2dfvE5s_etc = internal addrspace(3) global %struct.ETC undef

// DEVICE: @_ZZ2dfvE11const_array = internal addrspace(4) constant [5 x i32] [i32 1, i32 2, i32 3, i32 4, i32 5]
// DEVICE: @_ZZ2dfvE9const_int = internal addrspace(4) constant i32 123

// We should not emit global initializers for device-side variables.
// DEVICE-NOT: @__cxx_global_var_init

// Make sure that initialization restrictions do not apply to local
// variables.
__device__ void df() {
  // NVPTX:  %[[ec:.*]] = alloca %struct.EC
  // NVPTX:  %[[ed:.*]] = alloca %struct.ED
  // NVPTX:  %[[ecd:.*]] = alloca %struct.ECD
  // NVPTX:  %[[etc:.*]] = alloca %struct.ETC
  // NVPTX:  %[[uc:.*]] = alloca %struct.UC
  // NVPTX:  %[[ud:.*]] = alloca %struct.UD
  // NVPTX:  %[[eci:.*]] = alloca %struct.ECI
  // NVPTX:  %[[nec:.*]] = alloca %struct.NEC
  // NVPTX:  %[[ned:.*]] = alloca %struct.NED
  // NVPTX:  %[[ncv:.*]] = alloca %struct.NCV
  // NVPTX:  %[[vd:.*]] = alloca %struct.VD
  // NVPTX:  %[[ncf:.*]] = alloca %struct.NCF
  // NVPTX:  %[[ncfs:.*]] = alloca %struct.NCFS
  // NVPTX:  %[[utc:.*]] = alloca %struct.UTC
  // NVPTX:  %[[netc:.*]] = alloca %struct.NETC
  // NVPTX:  %[[ec_i_ec:.*]] = alloca %struct.EC_I_EC
  // NVPTX:  %[[ec_i_ec1:.*]] = alloca %struct.EC_I_EC1
  // NVPTX:  %[[t_v_t:.*]] = alloca %struct.T_V_T
  // NVPTX:  %[[t_b_nec:.*]] = alloca %struct.T_B_NEC
  // NVPTX:  %[[t_f_nec:.*]] = alloca %struct.T_F_NEC
  // NVPTX:  %[[t_fa_nec:.*]] = alloca %struct.T_FA_NEC
  // NVPTX:  %[[t_b_ned:.*]] = alloca %struct.T_B_NED
  // NVPTX:  %[[t_f_ned:.*]] = alloca %struct.T_F_NED
  // NVPTX:  %[[t_fa_ned:.*]] = alloca %struct.T_FA_NED
  // AMDGCN:  %[[ec:.*]] ={{.*}} addrspacecast %struct.EC addrspace(5)* %ec to %struct.EC*
  // AMDGCN:  %[[ed:.*]] ={{.*}} addrspacecast %struct.ED addrspace(5)* %ed to %struct.ED*
  // AMDGCN:  %[[ecd:.*]] ={{.*}} addrspacecast %struct.ECD addrspace(5)* %ecd to %struct.ECD*
  // AMDGCN:  %[[etc:.*]] ={{.*}} addrspacecast %struct.ETC addrspace(5)* %etc to %struct.ETC*
  // AMDGCN:  %[[uc:.*]] ={{.*}} addrspacecast %struct.UC addrspace(5)* %uc to %struct.UC*
  // AMDGCN:  %[[ud:.*]] ={{.*}} addrspacecast %struct.UD addrspace(5)* %ud to %struct.UD*
  // AMDGCN:  %[[eci:.*]] ={{.*}} addrspacecast %struct.ECI addrspace(5)* %eci to %struct.ECI*
  // AMDGCN:  %[[nec:.*]] ={{.*}} addrspacecast %struct.NEC addrspace(5)* %nec to %struct.NEC*
  // AMDGCN:  %[[ned:.*]] ={{.*}} addrspacecast %struct.NED addrspace(5)* %ned to %struct.NED*
  // AMDGCN:  %[[ncv:.*]] ={{.*}} addrspacecast %struct.NCV addrspace(5)* %ncv to %struct.NCV*
  // AMDGCN:  %[[vd:.*]] ={{.*}} addrspacecast %struct.VD addrspace(5)* %vd to %struct.VD*
  // AMDGCN:  %[[ncf:.*]] ={{.*}} addrspacecast %struct.NCF addrspace(5)* %ncf to %struct.NCF*
  // AMDGCN:  %[[ncfs:.*]] ={{.*}} addrspacecast %struct.NCFS addrspace(5)* %ncfs to %struct.NCFS*
  // AMDGCN:  %[[utc:.*]] ={{.*}} addrspacecast %struct.UTC addrspace(5)* %utc to %struct.UTC*
  // AMDGCN:  %[[netc:.*]] ={{.*}} addrspacecast %struct.NETC addrspace(5)* %netc to %struct.NETC*
  // AMDGCN:  %[[ec_i_ec:.*]] ={{.*}} addrspacecast %struct.EC_I_EC addrspace(5)* %ec_i_ec to %struct.EC_I_EC*
  // AMDGCN:  %[[ec_i_ec1:.*]] ={{.*}} addrspacecast %struct.EC_I_EC1 addrspace(5)* %ec_i_ec1 to %struct.EC_I_EC1*
  // AMDGCN:  %[[t_v_t:.*]] ={{.*}} addrspacecast %struct.T_V_T addrspace(5)* %t_v_t to %struct.T_V_T*
  // AMDGCN:  %[[t_b_nec:.*]] ={{.*}} addrspacecast %struct.T_B_NEC addrspace(5)* %t_b_nec to %struct.T_B_NEC*
  // AMDGCN:  %[[t_f_nec:.*]] ={{.*}} addrspacecast %struct.T_F_NEC addrspace(5)* %t_f_nec to %struct.T_F_NEC*
  // AMDGCN:  %[[t_fa_nec:.*]] ={{.*}} addrspacecast %struct.T_FA_NEC addrspace(5)* %t_fa_nec to %struct.T_FA_NEC*
  // AMDGCN:  %[[t_b_ned:.*]] ={{.*}} addrspacecast %struct.T_B_NED addrspace(5)* %t_b_ned to %struct.T_B_NED*
  // AMDGCN:  %[[t_f_ned:.*]] ={{.*}} addrspacecast %struct.T_F_NED addrspace(5)* %t_f_ned to %struct.T_F_NED*
  // AMDGCN:  %[[t_fa_ned:.*]] ={{.*}} addrspacecast %struct.T_FA_NED addrspace(5)* %t_fa_ned to %struct.T_FA_NED*

  T t;
  // DEVICE-NOT: call
  EC ec;
  // DEVICE:  call void @_ZN2ECC1Ev(%struct.EC* {{[^,]*}} %[[ec]])
  ED ed;
  // DEVICE-NOT: call
  ECD ecd;
  // DEVICE:  call void @_ZN3ECDC1Ev(%struct.ECD* {{[^,]*}} %[[ecd]])
  ETC etc;
  // DEVICE:  call void @_ZN3ETCC1IJEEEDpT_(%struct.ETC* {{[^,]*}} %[[etc]])
  UC uc;
  // undefined constructor -- not allowed
  // DEVICE:  call void @_ZN2UCC1Ev(%struct.UC* {{[^,]*}} %[[uc]])
  UD ud;
  // undefined destructor -- not allowed
  // DEVICE-NOT: call
  ECI eci;
  // empty constructor w/ initializer list -- not allowed
  // DEVICE:  call void @_ZN3ECIC1Ev(%struct.ECI* {{[^,]*}} %[[eci]])
  NEC nec;
  // non-empty constructor -- not allowed
  // DEVICE:  call void @_ZN3NECC1Ev(%struct.NEC* {{[^,]*}} %[[nec]])
  // non-empty destructor -- not allowed
  NED ned;
  // no-constructor,  virtual method -- not allowed
  // DEVICE:  call void @_ZN3NCVC1Ev(%struct.NCV* {{[^,]*}} %[[ncv]])
  NCV ncv;
  // DEVICE-NOT: call
  VD vd;
  // DEVICE:  call void @_ZN2VDC1Ev(%struct.VD* {{[^,]*}} %[[vd]])
  NCF ncf;
  // DEVICE:   call void @_ZN3NCFC1Ev(%struct.NCF* {{[^,]*}} %[[ncf]])
  NCFS ncfs;
  // DEVICE:  call void @_ZN4NCFSC1Ev(%struct.NCFS* {{[^,]*}} %[[ncfs]])
  UTC utc;
  // DEVICE:  call void @_ZN3UTCC1IJEEEDpT_(%struct.UTC* {{[^,]*}} %[[utc]])
  NETC netc;
  // DEVICE:  call void @_ZN4NETCC1IJEEEDpT_(%struct.NETC* {{[^,]*}} %[[netc]])
  T_B_T t_b_t;
  // DEVICE-NOT: call
  T_F_T t_f_t;
  // DEVICE-NOT: call
  T_FA_T t_fa_t;
  // DEVICE-NOT: call
  EC_I_EC ec_i_ec;
  // DEVICE:  call void @_ZN7EC_I_ECC1Ev(%struct.EC_I_EC* {{[^,]*}} %[[ec_i_ec]])
  EC_I_EC1 ec_i_ec1;
  // DEVICE:  call void @_ZN8EC_I_EC1C1Ev(%struct.EC_I_EC1* {{[^,]*}} %[[ec_i_ec1]])
  T_V_T t_v_t;
  // DEVICE:  call void @_ZN5T_V_TC1Ev(%struct.T_V_T* {{[^,]*}} %[[t_v_t]])
  T_B_NEC t_b_nec;
  // DEVICE:  call void @_ZN7T_B_NECC1Ev(%struct.T_B_NEC* {{[^,]*}} %[[t_b_nec]])
  T_F_NEC t_f_nec;
  // DEVICE:  call void @_ZN7T_F_NECC1Ev(%struct.T_F_NEC* {{[^,]*}} %[[t_f_nec]])
  T_FA_NEC t_fa_nec;
  // DEVICE:  call void @_ZN8T_FA_NECC1Ev(%struct.T_FA_NEC* {{[^,]*}} %[[t_fa_nec]])
  T_B_NED t_b_ned;
  // DEVICE-NOT: call
  T_F_NED t_f_ned;
  // DEVICE-NOT: call
  T_FA_NED t_fa_ned;
  // DEVICE-NOT: call
  static __shared__ EC s_ec;
  // DEVICE-NOT: call void @_ZN2ECC1Ev(%struct.EC* addrspacecast (%struct.EC addrspace(3)* @_ZZ2dfvE4s_ec to %struct.EC*))
  static __shared__ ETC s_etc;
  // DEVICE-NOT: call void @_ZN3ETCC1IJEEEDpT_(%struct.ETC* addrspacecast (%struct.ETC addrspace(3)* @_ZZ2dfvE5s_etc to %struct.ETC*))

  static const int const_array[] = {1, 2, 3, 4, 5};
  static const int const_int = 123;

  // anchor point separating constructors and destructors
  df(); // DEVICE: call void @_Z2dfv()

  // Verify that we only call non-empty destructors
  // DEVICE-NEXT: call void @_ZN8T_FA_NEDD1Ev(%struct.T_FA_NED* {{[^,]*}} %[[t_fa_ned]])
  // DEVICE-NEXT: call void @_ZN7T_F_NEDD1Ev(%struct.T_F_NED* {{[^,]*}} %[[t_f_ned]])
  // DEVICE-NEXT: call void @_ZN7T_B_NEDD1Ev(%struct.T_B_NED* {{[^,]*}} %[[t_b_ned]])
  // DEVICE-NEXT: call void @_ZN2VDD1Ev(%struct.VD* {{[^,]*}} %[[vd]])
  // DEVICE-NEXT: call void @_ZN3NEDD1Ev(%struct.NED* {{[^,]*}} %[[ned]])
  // DEVICE-NEXT: call void @_ZN2UDD1Ev(%struct.UD* {{[^,]*}} %[[ud]])
  // DEVICE-NEXT: call void @_ZN3ECDD1Ev(%struct.ECD* {{[^,]*}} %[[ecd]])
  // DEVICE-NEXT: call void @_ZN2EDD1Ev(%struct.ED* {{[^,]*}} %[[ed]])

  // DEVICE-NEXT: ret void
}

// We should not emit global init function.
// DEVICE-NOT: @_GLOBAL__sub_I
