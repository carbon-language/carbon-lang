
#include <hip/hip_runtime.h>
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// RUN: %clang_cc1 -no-opaque-pointers -std=c++11 -fcuda-is-device -triple nvptx64-nvidia-cuda -emit-llvm -o - %s | FileCheck --check-prefix=DEVICE %s
// RUN: echo "GPU binary would be here" > %t
// RUN: %clang_cc1 -no-opaque-pointers -std=c++11 -triple x86_64-unknown-linux-gnu -target-sdk-version=8.0 -fcuda-include-gpubinary %t -emit-llvm -o - %s | FileCheck --check-prefix=HOST %s

struct surfaceReference {
  int desc;
};

template <typename T, int dim = 1>
struct __attribute__((device_builtin_surface_type)) surface : public surfaceReference {
};

// Partial specialization over `void`.
template<int dim>
struct __attribute__((device_builtin_surface_type)) surface<void, dim> : public surfaceReference {
};

// On the device side, surface references are represented as `i64` handles.
// DEVICE: @surf ={{.*}} addrspace(1) externally_initialized global i64 undef, align 4
// On the host side, they remain in the original type.
// HOST: @surf = internal global %struct.surface
// HOST: @0 = private unnamed_addr constant [5 x i8] c"surf\00"
surface<void, 2> surf;

__attribute__((device)) int suld_2d_zero(surface<void, 2>, int, int) asm("llvm.nvvm.suld.2d.i32.zero");

// DEVICE-LABEL: i32 @_Z3fooii(i32 noundef %x, i32 noundef %y)
// DEVICE: call i64 @llvm.nvvm.texsurf.handle.internal.p1i64(i64 addrspace(1)* @surf)
// DEVICE: call noundef i32 @llvm.nvvm.suld.2d.i32.zero(i64 %{{.*}}, i32 noundef %{{.*}}, i32 noundef %{{.*}})
__attribute__((device)) int foo(int x, int y) {
  return suld_2d_zero(surf, x, y);
}

// HOST: define internal void @[[PREFIX:__cuda]]_register_globals
// Texture references need registering with correct arguments.
// HOST: call void @[[PREFIX]]RegisterSurface(i8** %0, i8*{{.*}}({{.*}}@surf{{.*}}), i8*{{.*}}({{.*}}@0{{.*}}), i8*{{.*}}({{.*}}@0{{.*}}), i32 2, i32 0)

// They also need annotating in metadata.
// DEVICE: !0 = !{i64 addrspace(1)* @surf, !"surface", i32 1}
