#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -x hip -std=c++11 -triple amdgcn -fcuda-is-device -emit-llvm %s -o - | FileCheck %s
#include "Inputs/hip/hip_runtime.h"

// CHECK-LABEL: @_Z24atomic32_op_singlethreadPiii
// CHECK: cmpxchg i32* {{%[0-9]+}}, i32 {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: cmpxchg weak i32* {{%[0-9]+}}, i32 {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("singlethread-one-as") monotonic monotonic, align 4
// CHECK: atomicrmw xchg i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: atomicrmw add i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: atomicrmw and i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: atomicrmw or i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: atomicrmw xor i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: atomicrmw min i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: atomicrmw max i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: load atomic i32, i32* {{%[0-9]+}} syncscope("singlethread-one-as") monotonic, align 4
// CHECK: store atomic i32 %{{.*}}, i32* %{{.*}} syncscope("singlethread-one-as") monotonic, align 4
__device__ int atomic32_op_singlethread(int *ptr, int val, int desired) {
  bool flag = __hip_atomic_compare_exchange_strong(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_exchange(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_fetch_add(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_fetch_and(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_fetch_or(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_fetch_xor(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_load(ptr, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  __hip_atomic_store(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  return flag ? val : desired;
}

// CHECK-LABEL: @_Z25atomicu32_op_singlethreadPjjj
// CHECK: atomicrmw umin i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: atomicrmw umax i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("singlethread-one-as")
__device__ unsigned int atomicu32_op_singlethread(unsigned int *ptr, unsigned int val, unsigned int desired) {
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  return val;
}

// CHECK-LABEL: @_Z21atomic32_op_wavefrontPiii
// CHECK: cmpxchg i32* {{%[0-9]+}}, i32 {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: cmpxchg weak i32* {{%[0-9]+}}, i32 {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("wavefront-one-as") monotonic monotonic, align 4
// CHECK: atomicrmw xchg i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: atomicrmw add i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: atomicrmw and i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: atomicrmw or i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: atomicrmw xor i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: atomicrmw min i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: atomicrmw max i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: load atomic i32, i32* {{%[0-9]+}} syncscope("wavefront-one-as") monotonic, align 4
// CHECK: store atomic i32 %{{.*}}, i32* %{{.*}} syncscope("wavefront-one-as") monotonic, align 4
__device__ int atomic32_op_wavefront(int *ptr, int val, int desired) {
  bool flag = __hip_atomic_compare_exchange_strong(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_exchange(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_fetch_add(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_fetch_and(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_fetch_or(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_fetch_xor(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_load(ptr, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  __hip_atomic_store(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  return flag ? val : desired;
}

// CHECK-LABEL: @_Z22atomicu32_op_wavefrontPjjj
// CHECK: atomicrmw umin i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: atomicrmw umax i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("wavefront-one-as")
__device__ unsigned int atomicu32_op_wavefront(unsigned int *ptr, unsigned int val, unsigned int desired) {
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  return val;
}

// CHECK-LABEL: @_Z21atomic32_op_workgroupPiii
// CHECK: cmpxchg i32* {{%[0-9]+}}, i32 {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: cmpxchg weak i32* {{%[0-9]+}}, i32 {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("workgroup-one-as") monotonic monotonic, align 4
// CHECK: atomicrmw xchg i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: atomicrmw add i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: atomicrmw and i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: atomicrmw or i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: atomicrmw xor i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: atomicrmw min i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: atomicrmw max i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: store atomic i32 %{{.*}}, i32* %{{.*}} syncscope("workgroup-one-as") monotonic, align 4
__device__ int atomic32_op_workgroup(int *ptr, int val, int desired) {
  bool flag = __hip_atomic_compare_exchange_strong(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_exchange(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_fetch_add(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_fetch_and(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_fetch_or(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_fetch_xor(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  __hip_atomic_store(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  return flag ? val : desired;
}

// CHECK-LABEL: @_Z22atomicu32_op_workgroupPjjj
// CHECK: atomicrmw umin i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: atomicrmw umax i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("workgroup-one-as")
__device__ unsigned int atomicu32_op_workgroup(unsigned int *ptr, unsigned int val, unsigned int desired) {
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  return val;
}

// CHECK-LABEL: @_Z17atomic32_op_agentPiii
// CHECK: cmpxchg i32* {{%[0-9]+}}, i32 {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: cmpxchg weak i32* {{%[0-9]+}}, i32 {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("agent-one-as") monotonic monotonic, align 4
// CHECK: atomicrmw xchg i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: atomicrmw add i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: atomicrmw and i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: atomicrmw or i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: atomicrmw xor i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: atomicrmw min i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: atomicrmw max i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: store atomic i32 %{{.*}}, i32* %{{.*}} syncscope("agent-one-as") monotonic, align 4
__device__ int atomic32_op_agent(int *ptr, int val, int desired) {
  bool flag = __hip_atomic_compare_exchange_strong(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_exchange(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_fetch_add(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_fetch_and(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_fetch_or(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_fetch_xor(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  __hip_atomic_store(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  return flag ? val : desired;
}

// CHECK-LABEL: @_Z18atomicu32_op_agentPjjj
// CHECK: atomicrmw umin i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: atomicrmw umax i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("agent-one-as")
__device__ unsigned int atomicu32_op_agent(unsigned int *ptr, unsigned int val, unsigned int desired) {
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  return val;
}

// CHECK-LABEL: @_Z18atomic32_op_systemPiii
// CHECK: cmpxchg i32* {{%[0-9]+}}, i32 {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("one-as")
// CHECK: cmpxchg weak i32* {{%[0-9]+}}, i32 {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("one-as") monotonic monotonic, align 4
// CHECK: atomicrmw xchg i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("one-as")
// CHECK: atomicrmw add i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("one-as")
// CHECK: atomicrmw and i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("one-as")
// CHECK: atomicrmw or i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("one-as")
// CHECK: atomicrmw xor i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("one-as")
// CHECK: atomicrmw min i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("one-as")
// CHECK: atomicrmw max i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("one-as")
// CHECK: load i32, i32* %{{.*}}, align 4
// CHECK: store atomic i32 %{{.*}}, i32* %{{.*}} syncscope("one-as") monotonic, align 4
__device__ int atomic32_op_system(int *ptr, int val, int desired) {
  bool flag = __hip_atomic_compare_exchange_strong(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_exchange(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_fetch_add(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_fetch_and(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_fetch_or(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_fetch_xor(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_load(ptr, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  __hip_atomic_store(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  return flag ? val : desired;
}

// CHECK-LABEL: @_Z19atomicu32_op_systemPjjj
// CHECK: atomicrmw umin i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("one-as")
// CHECK: atomicrmw umax i32* {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("one-as")
__device__ unsigned int atomicu32_op_system(unsigned int *ptr, unsigned int val, unsigned int desired) {
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  return val;
}

// CHECK-LABEL: @_Z24atomic64_op_singlethreadPxS_xx
// CHECK: cmpxchg i64* {{%[0-9]+}}, i64 {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: cmpxchg weak i64* {{%[0-9]+}}, i64 {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("singlethread-one-as") monotonic monotonic, align 8
// CHECK: atomicrmw xchg i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: atomicrmw add i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: atomicrmw and i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: atomicrmw or i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: atomicrmw xor i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: atomicrmw min i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: atomicrmw max i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: store atomic i64 %{{.*}}, i64* %{{.*}} syncscope("singlethread-one-as") monotonic, align 8
__device__ long long atomic64_op_singlethread(long long *ptr, long long *ptr2, long long val, long long desired) {
  bool flag = __hip_atomic_compare_exchange_strong(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_exchange(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_fetch_add(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_fetch_and(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_fetch_or(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_fetch_xor(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  __hip_atomic_store(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  return flag ? val : desired;
}

// CHECK-LABEL: @_Z25atomicu64_op_singlethreadPyS_yy
// CHECK: atomicrmw umin i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: atomicrmw umax i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: load atomic i64, i64* %{{.*}} syncscope("singlethread-one-as") monotonic, align 8
// CHECK: store atomic i64 %{{.*}}, i64* %{{.*}} syncscope("singlethread-one-as") monotonic, align 8
__device__ unsigned long long atomicu64_op_singlethread(unsigned long long *ptr, unsigned long long *ptr2, unsigned long long val, unsigned long long desired) {
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_load(ptr, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  __hip_atomic_store(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  return val;
}

// CHECK-LABEL: @_Z21atomic64_op_wavefrontPxS_xx
// CHECK: cmpxchg i64* {{%[0-9]+}}, i64 {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: cmpxchg weak i64* {{%[0-9]+}}, i64 {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("wavefront-one-as") monotonic monotonic, align 8
// CHECK: atomicrmw xchg i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: atomicrmw add i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: atomicrmw and i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: atomicrmw or i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: atomicrmw xor i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: atomicrmw min i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: atomicrmw max i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: load atomic i64, i64* {{%[0-9]+}} syncscope("wavefront-one-as") monotonic, align 8
// CHECK: store atomic i64 %{{.*}}, i64* %{{.*}} syncscope("wavefront-one-as") monotonic, align 8
__device__ long long atomic64_op_wavefront(long long *ptr, long long *ptr2, long long val, long long desired) {
  bool flag = __hip_atomic_compare_exchange_strong(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_exchange(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_fetch_add(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_fetch_and(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_fetch_or(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_fetch_xor(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_load(ptr, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  __hip_atomic_store(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  return flag ? val : desired;
}

// CHECK-LABEL: @_Z22atomicu64_op_wavefrontPyS_yy
// CHECK: atomicrmw umin i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: atomicrmw umax i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: load atomic i64, i64* {{%[0-9]+}} syncscope("wavefront-one-as") monotonic, align 8
// CHECK: store atomic i64 %{{.*}}, i64* %{{.*}} syncscope("wavefront-one-as") monotonic, align 8
__device__ unsigned long long atomicu64_op_wavefront(unsigned long long *ptr, unsigned long long *ptr2, unsigned long long val, unsigned long long desired) {
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_load(ptr, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  __hip_atomic_store(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  return val;
}

// CHECK-LABEL: @_Z21atomic64_op_workgroupPxS_xx
// CHECK: cmpxchg i64* {{%[0-9]+}}, i64 {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: cmpxchg weak i64* {{%[0-9]+}}, i64 {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("workgroup-one-as") monotonic monotonic, align 8
// CHECK: atomicrmw xchg i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: atomicrmw add i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: atomicrmw and i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: atomicrmw or i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: atomicrmw xor i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: atomicrmw min i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: atomicrmw max i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: store atomic i64 %{{.*}}, i64* %{{.*}} syncscope("workgroup-one-as") monotonic, align 8
__device__ long long atomic64_op_workgroup(long long *ptr, long long *ptr2, long long val, long long desired) {
  bool flag = __hip_atomic_compare_exchange_strong(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_exchange(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_fetch_add(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_fetch_and(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_fetch_or(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_fetch_xor(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  __hip_atomic_store(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  return flag ? val : desired;
}

// CHECK-LABEL: @_Z22atomicu64_op_workgroupPyS_yy
// CHECK: atomicrmw umin i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: atomicrmw umax i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: store atomic i64 %{{.*}}, i64* %{{.*}} syncscope("workgroup-one-as") monotonic, align 8
__device__ unsigned long long atomicu64_op_workgroup(unsigned long long *ptr, unsigned long long *ptr2, unsigned long long val, unsigned long long desired) {
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  __hip_atomic_store(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  return val;
}

// CHECK-LABEL: @_Z17atomic64_op_agentPxS_xx
// CHECK: cmpxchg i64* {{%[0-9]+}}, i64 {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: cmpxchg weak i64* {{%[0-9]+}}, i64 {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("agent-one-as") monotonic monotonic, align 8
// CHECK: atomicrmw xchg i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: atomicrmw add i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: atomicrmw and i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: atomicrmw or i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: atomicrmw xor i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: atomicrmw min i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: atomicrmw max i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: store atomic i64 %{{.*}}, i64* %{{.*}} syncscope("agent-one-as") monotonic, align 8
__device__ long long atomic64_op_agent(long long *ptr, long long *ptr2, long long val, long long desired) {
  bool flag = __hip_atomic_compare_exchange_strong(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_exchange(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_fetch_add(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_fetch_and(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_fetch_or(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_fetch_xor(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  __hip_atomic_store(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  return flag ? val : desired;
}

// CHECK-LABEL: @_Z18atomicu64_op_agentPyS_yy
// CHECK: atomicrmw umin i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: atomicrmw umax i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: store atomic i64 %{{.*}}, i64* %{{.*}} syncscope("agent-one-as") monotonic, align 8
__device__ unsigned long long atomicu64_op_agent(unsigned long long *ptr, unsigned long long *ptr2, unsigned long long val, unsigned long long desired) {
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  __hip_atomic_store(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  return val;
}

// CHECK-LABEL: @_Z18atomic64_op_systemPxS_xx
// CHECK: cmpxchg i64* {{%[0-9]+}}, i64 {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("one-as")
// CHECK: cmpxchg weak i64* {{%[0-9]+}}, i64 {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("one-as") monotonic monotonic, align 8
// CHECK: atomicrmw xchg i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("one-as")
// CHECK: atomicrmw add i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("one-as")
// CHECK: atomicrmw and i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("one-as")
// CHECK: atomicrmw or i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("one-as")
// CHECK: atomicrmw xor i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("one-as")
// CHECK: atomicrmw min i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("one-as")
// CHECK: atomicrmw max i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("one-as")
// CHECK: load i64, i64* %{{.*}}, align 8
// CHECK: store atomic i64 %{{.*}}, i64* %{{.*}} syncscope("one-as") monotonic, align 8
__device__ long long atomic64_op_system(long long *ptr, long long *ptr2, long long val, long long desired) {
  bool flag = __hip_atomic_compare_exchange_strong(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_exchange(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_fetch_add(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_fetch_and(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_fetch_or(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_fetch_xor(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_load(ptr, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  __hip_atomic_store(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  return flag ? val : desired;
}

// CHECK-LABEL: @_Z19atomicu64_op_systemPyS_yy
// CHECK: atomicrmw umin i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("one-as")
// CHECK: atomicrmw umax i64* {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("one-as")
// CHECK: load i64, i64* %{{.*}}, align 8
// CHECK: store atomic i64 %{{.*}}, i64* %{{.*}} syncscope("one-as") monotonic, align 8
__device__ unsigned long long atomicu64_op_system(unsigned long long *ptr, unsigned long long *ptr2, unsigned long long val, unsigned long long desired) {
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_load(ptr, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  __hip_atomic_store(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  return val;
}
