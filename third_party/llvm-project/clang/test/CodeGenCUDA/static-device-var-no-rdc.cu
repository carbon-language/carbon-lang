#include "hip/hip_runtime.h"
// REQUIRES: x86-registered-target
// REQUIRES: amdgpu-registered-target

// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -std=c++11 \
// RUN:   -emit-llvm -o - -x hip %s | FileCheck -check-prefix=DEV %s

// RUN: %clang_cc1 -triple x86_64-gnu-linux -std=c++11 \
// RUN:   -emit-llvm -o - -x hip %s | FileCheck -check-prefix=HOST %s

// Negative tests.

// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -std=c++11 \
// RUN:   -emit-llvm -o - -x hip %s | FileCheck -check-prefix=DEV-NEG %s

// RUN: %clang_cc1 -triple x86_64-gnu-linux -std=c++11 \
// RUN:   -emit-llvm -o - -x hip %s | FileCheck -check-prefix=HOST-NEG %s

#include "Inputs/hip/hip_runtime.h"

// Test function scope static device variable, which should not be externalized.
// DEV-DAG: @_ZZ6kernelPiPPKiE1w = internal addrspace(4) constant i32 1
// DEV-DAG: @_ZZ6kernelPiPPKiE21local_static_constant = internal addrspace(4) constant i32 42
// DEV-DAG: @_ZZ6kernelPiPPKiE19local_static_device = internal addrspace(1) constant i32 43

// Check a static device variable referenced by host function is externalized.
// DEV-DAG: @_ZL1x ={{.*}} addrspace(1) externally_initialized global i32 0
// HOST-DAG: @_ZL1x = internal global i32 undef
// HOST-DAG: @[[DEVNAMEX:[0-9]+]] = {{.*}}c"_ZL1x\00"

static __device__ int x;

// Check a static device variables referenced only by device functions and kernels
// is not externalized.
// DEV-DAG: @_ZL2x2 = internal addrspace(1) global i32 0
static __device__ int x2;

// Check a static device variable referenced by host device function is externalized.
// DEV-DAG: @_ZL2x3 ={{.*}} addrspace(1) externally_initialized global i32 0
static __device__ int x3;

// Check a static device variable referenced in file scope is externalized.
// DEV-DAG: @_ZL2x4 ={{.*}} addrspace(1) externally_initialized global i32 0
static __device__ int x4;
int& x4_ref = x4;

// Check a static device variable in anonymous namespace.
// DEV-DAG: @_ZN12_GLOBAL__N_12x5E ={{.*}} addrspace(1) externally_initialized global i32 0
namespace {
static __device__ int x5;
}

// Check a static constant variable referenced by host is externalized.
// DEV-DAG: @_ZL1y ={{.*}} addrspace(4) externally_initialized global i32 0
// HOST-DAG: @_ZL1y = internal global i32 undef
// HOST-DAG: @[[DEVNAMEY:[0-9]+]] = {{.*}}c"_ZL1y\00"

static __constant__ int y;

// Test static host variable, which should not be externalized nor registered.
// HOST-DAG: @_ZL1z = internal global i32 0
// DEV-NEG-NOT: @_ZL1z
static int z;

// Test implicit static constant variable, which should not be externalized.
// HOST-DAG: @_ZL2z2 = internal constant i32 456
// DEV-DAG: @_ZL2z2 = internal addrspace(4) constant i32 456

static constexpr int z2 = 456;

// Test static device variable in inline function, which should not be
// externalized nor registered.
// DEV-DAG: @_ZZ6devfunPPKiE1p = linkonce_odr addrspace(4) constant i32 2, comdat

// Check a static device variable referenced by host function only is externalized.
// DEV-DAG: @_ZL1w ={{.*}} addrspace(1) externally_initialized global i32 0
// HOST-DAG: @_ZL1w = internal global i32 undef
// HOST-DAG: @[[DEVNAMEW:[0-9]+]] = {{.*}}c"_ZL1w\00"

static __device__ int w;

// Test non-ODR-use of static device var should not be emitted or registered.
// DEV-NEG-NOT: @_ZL1u
// HOST-NEG-NOT: @_ZL1u

static __device__ int u;

inline __device__ void devfun(const int ** b) {
  const static int p = 2;
  b[0] = &p;
  b[1] = &x2;
}

__global__ void kernel(int *a, const int **b) {
  const static int w = 1;
  const static __constant__ int local_static_constant = 42;
  const static __device__ int local_static_device = 43;
  a[0] = x;
  a[1] = y;
  a[2] = x2;
  a[3] = x3;
  a[4] = x4;
  a[5] = x5;
  a[6] = sizeof(u);
  b[0] = &w;
  b[1] = &z2;
  b[2] = &local_static_constant;
  b[3] = &local_static_device;
  devfun(b);
}

__host__ __device__ void hdf(int *a) {
  a[0] = x3;
}

int* getDeviceSymbol(int *x);

void foo(const int **a) {
  getDeviceSymbol(&x);
  getDeviceSymbol(&x5);
  getDeviceSymbol(&y);
  getDeviceSymbol(&w);
  z = 123;
  a[0] = &z2;
  decltype(u) tmp;
}

// HOST-DAG: __hipRegisterVar({{.*}}@_ZL1x {{.*}}@[[DEVNAMEX]]
// HOST-DAG: __hipRegisterVar({{.*}}@_ZL1y {{.*}}@[[DEVNAMEY]]
// HOST-DAG: __hipRegisterVar({{.*}}@_ZL1w {{.*}}@[[DEVNAMEW]]
// HOST-NEG-NOT: __hipRegisterVar({{.*}}@_ZL1u
// HOST-NEG-NOT: __hipRegisterVar({{.*}}@_ZZ6kernelPiPPKiE1w
// HOST-NEG-NOT: __hipRegisterVar({{.*}}@_ZZ6devfunPPKiE1p
