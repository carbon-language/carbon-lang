#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -fapply-global-visibility-to-externs -fvisibility default -emit-llvm -o - %s | FileCheck --check-prefix=CHECK-DEFAULT %s
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -fapply-global-visibility-to-externs -fvisibility protected -emit-llvm -o - %s | FileCheck --check-prefix=CHECK-PROTECTED %s
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -fapply-global-visibility-to-externs -fvisibility hidden -emit-llvm -o - %s | FileCheck --check-prefix=CHECK-HIDDEN %s

#include "Inputs/hip/hip_runtime.h"

// CHECK-DEFAULT: @c ={{.*}} addrspace(4) externally_initialized global
// CHECK-DEFAULT: @g ={{.*}} addrspace(1) externally_initialized global
// CHECK-PROTECTED: @c = protected addrspace(4) externally_initialized global
// CHECK-PROTECTED: @g = protected addrspace(1) externally_initialized global
// CHECK-HIDDEN: @c = protected addrspace(4) externally_initialized global
// CHECK-HIDDEN: @g = protected addrspace(1) externally_initialized global
__constant__ int c;
__device__ int g;

// CHECK-DEFAULT: @e = external addrspace(1) global
// CHECK-PROTECTED: @e = external protected addrspace(1) global
// CHECK-HIDDEN: @e = external protected addrspace(1) global
extern __device__ int e;

// dummy one to hold reference to `e`.
__device__ int f() {
  return e;
}

// CHECK-DEFAULT: define{{.*}} amdgpu_kernel void @_Z3foov()
// CHECK-PROTECTED: define protected amdgpu_kernel void @_Z3foov()
// CHECK-HIDDEN: define protected amdgpu_kernel void @_Z3foov()
__global__ void foo() {
  g = c;
}
