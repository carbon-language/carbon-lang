#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -std=c++11 -x hip -triple x86_64-linux-gnu -aux-triple amdgcn-amd-amdhsa -emit-llvm %s -o - | FileCheck %s --check-prefix=HOST
// RUN: %clang_cc1 -std=c++11 -x hip -triple x86_64-pc-windows-msvc -aux-triple amdgcn-amd-amdhsa -emit-llvm %s -o - | FileCheck %s --check-prefix=MSVC
// RUN: %clang_cc1 -std=c++11 -x hip -triple amdgcn-amd-amdhsa -fcuda-is-device -emit-llvm %s -o - | FileCheck %s --check-prefix=DEVICE

#include "Inputs/hip/hip_runtime.h"

// HOST: @0 = private unnamed_addr constant [43 x i8] c"_Z2k0IZZ2f1PfENKUlS0_E_clES0_EUlfE_EvS0_T_\00", align 1
// HOST: @1 = private unnamed_addr constant [60 x i8] c"_Z2k1IZ2f1PfEUlfE_Z2f1S0_EUlffE_Z2f1S0_EUlfE0_EvS0_T_T0_T1_\00", align 1
// Check that, on MSVC, the same device kernel mangling name is generated.
// MSVC: @0 = private unnamed_addr constant [43 x i8] c"_Z2k0IZZ2f1PfENKUlS0_E_clES0_EUlfE_EvS0_T_\00", align 1
// MSVC: @1 = private unnamed_addr constant [60 x i8] c"_Z2k1IZ2f1PfEUlfE_Z2f1S0_EUlffE_Z2f1S0_EUlfE0_EvS0_T_T0_T1_\00", align 1

__device__ float d0(float x) {
  return [](float x) { return x + 1.f; }(x);
}

__device__ float d1(float x) {
  return [](float x) { return x * 2.f; }(x);
}

// DEVICE: amdgpu_kernel void @_Z2k0IZZ2f1PfENKUlS0_E_clES0_EUlfE_EvS0_T_(
// DEVICE: define internal noundef float @_ZZZ2f1PfENKUlS_E_clES_ENKUlfE_clEf(
template <typename F>
__global__ void k0(float *p, F f) {
  p[0] = f(p[0]) + d0(p[1]) + d1(p[2]);
}

// DEVICE: amdgpu_kernel void @_Z2k1IZ2f1PfEUlfE_Z2f1S0_EUlffE_Z2f1S0_EUlfE0_EvS0_T_T0_T1_(
// DEVICE: define internal noundef float @_ZZ2f1PfENKUlfE_clEf(
// DEVICE: define internal noundef float @_ZZ2f1PfENKUlffE_clEff(
// DEVICE: define internal noundef float @_ZZ2f1PfENKUlfE0_clEf(
template <typename F0, typename F1, typename F2>
__global__ void k1(float *p, F0 f0, F1 f1, F2 f2) {
  p[0] = f0(p[0]) + f1(p[1], p[2]) + f2(p[3]);
}

void f0(float *p) {
  [](float *p) {
    *p = 1.f;
  }(p);
}

// The inner/outer lambdas are required to be mangled following ODR but their
// linkages are still required to keep the original `internal` linkage.

// HOST: define internal void @_ZZ2f1PfENKUlS_E_clES_(
void f1(float *p) {
  [](float *p) {
    k0<<<1,1>>>(p, [] __device__ (float x) { return x + 3.f; });
  }(p);
  k1<<<1,1>>>(p,
              [] __device__ (float x) { return x + 4.f; },
              [] __device__ (float x, float y) { return x * y; },
              [] __device__ (float x) { return x + 5.f; });
}
// HOST: @__hip_register_globals
// HOST: __hipRegisterFunction{{.*}}@_Z2k0IZZ2f1PfENKUlS0_E_clES0_EUlfE_EvS0_T_{{.*}}@0
// HOST: __hipRegisterFunction{{.*}}@_Z2k1IZ2f1PfEUlfE_Z2f1S0_EUlffE_Z2f1S0_EUlfE0_EvS0_T_T0_T1_{{.*}}@1
// MSVC: __hipRegisterFunction{{.*}}@"??$k0@V<lambda_1>@?0???R1?0??f1@@YAXPEAM@Z@QEBA@0@Z@@@YAXPEAMV<lambda_1>@?0???R0?0??f1@@YAX0@Z@QEBA@0@Z@@Z{{.*}}@0
// MSVC: __hipRegisterFunction{{.*}}@"??$k1@V<lambda_2>@?0??f1@@YAXPEAM@Z@V<lambda_3>@?0??2@YAX0@Z@V<lambda_4>@?0??2@YAX0@Z@@@YAXPEAMV<lambda_2>@?0??f1@@YAX0@Z@V<lambda_3>@?0??1@YAX0@Z@V<lambda_4>@?0??1@YAX0@Z@@Z{{.*}}@1
