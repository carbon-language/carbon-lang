#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa \
// RUN:     -fcuda-is-device -emit-llvm -o - -x hip %s \
// RUN:     | FileCheck -check-prefixes=CHECK,DEFAULT %s
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa --gpu-max-threads-per-block=1024 \
// RUN:     -fcuda-is-device -emit-llvm -o - -x hip %s \
// RUN:     | FileCheck -check-prefixes=CHECK,MAX1024 %s
// RUN: %clang_cc1 -triple nvptx \
// RUN:     -fcuda-is-device -emit-llvm -o - %s | FileCheck %s \
// RUN:     -check-prefix=NAMD
// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -emit-llvm \
// RUN:     -verify -o - -x hip %s | FileCheck -check-prefix=NAMD %s

#include "Inputs/hip/hip_runtime.h"

__global__ void flat_work_group_size_default() {
// CHECK: define{{.*}} amdgpu_kernel void @_Z28flat_work_group_size_defaultv() [[FLAT_WORK_GROUP_SIZE_DEFAULT:#[0-9]+]]
}

__attribute__((amdgpu_flat_work_group_size(32, 64))) // expected-no-diagnostics
__global__ void flat_work_group_size_32_64() {
// CHECK: define{{.*}} amdgpu_kernel void @_Z26flat_work_group_size_32_64v() [[FLAT_WORK_GROUP_SIZE_32_64:#[0-9]+]]
}
__attribute__((amdgpu_waves_per_eu(2))) // expected-no-diagnostics
__global__ void waves_per_eu_2() {
// CHECK: define{{.*}} amdgpu_kernel void @_Z14waves_per_eu_2v() [[WAVES_PER_EU_2:#[0-9]+]]
}
__attribute__((amdgpu_num_sgpr(32))) // expected-no-diagnostics
__global__ void num_sgpr_32() {
// CHECK: define{{.*}} amdgpu_kernel void @_Z11num_sgpr_32v() [[NUM_SGPR_32:#[0-9]+]]
}
__attribute__((amdgpu_num_vgpr(64))) // expected-no-diagnostics
__global__ void num_vgpr_64() {
// CHECK: define{{.*}} amdgpu_kernel void @_Z11num_vgpr_64v() [[NUM_VGPR_64:#[0-9]+]]
}

// Make sure this is silently accepted on other targets.
// NAMD-NOT: "amdgpu-flat-work-group-size"
// NAMD-NOT: "amdgpu-waves-per-eu"
// NAMD-NOT: "amdgpu-num-vgpr"
// NAMD-NOT: "amdgpu-num-sgpr"

// DEFAULT-DAG: attributes [[FLAT_WORK_GROUP_SIZE_DEFAULT]] = {{.*}}"amdgpu-flat-work-group-size"="1,1024"{{.*}}"uniform-work-group-size"="true"
// MAX1024-DAG: attributes [[FLAT_WORK_GROUP_SIZE_DEFAULT]] = {{.*}}"amdgpu-flat-work-group-size"="1,1024"
// CHECK-DAG: attributes [[FLAT_WORK_GROUP_SIZE_32_64]] = {{.*}}"amdgpu-flat-work-group-size"="32,64"
// CHECK-DAG: attributes [[WAVES_PER_EU_2]] = {{.*}}"amdgpu-waves-per-eu"="2"
// CHECK-DAG: attributes [[NUM_SGPR_32]] = {{.*}}"amdgpu-num-sgpr"="32"
// CHECK-DAG: attributes [[NUM_VGPR_64]] = {{.*}}"amdgpu-num-vgpr"="64"
