// RUN: %clang_cc1 "-triple" "nvptx-nvidia-cuda" -emit-llvm -fcuda-is-device -o - %s | FileCheck %s


#include <hip/hip_runtime.h>
#include "__clang_cuda_builtin_vars.h"

// CHECK: define{{.*}} void @_Z6kernelPi(i32* noundef %out)
__attribute__((global))
void kernel(int *out) {
  int i = 0;
  out[i++] = threadIdx.x; // CHECK: call i32 @llvm.nvvm.read.ptx.sreg.tid.x()
  out[i++] = threadIdx.y; // CHECK: call i32 @llvm.nvvm.read.ptx.sreg.tid.y()
  out[i++] = threadIdx.z; // CHECK: call i32 @llvm.nvvm.read.ptx.sreg.tid.z()

  out[i++] = blockIdx.x; // CHECK: call i32 @llvm.nvvm.read.ptx.sreg.ctaid.x()
  out[i++] = blockIdx.y; // CHECK: call i32 @llvm.nvvm.read.ptx.sreg.ctaid.y()
  out[i++] = blockIdx.z; // CHECK: call i32 @llvm.nvvm.read.ptx.sreg.ctaid.z()

  out[i++] = blockDim.x; // CHECK: call i32 @llvm.nvvm.read.ptx.sreg.ntid.x()
  out[i++] = blockDim.y; // CHECK: call i32 @llvm.nvvm.read.ptx.sreg.ntid.y()
  out[i++] = blockDim.z; // CHECK: call i32 @llvm.nvvm.read.ptx.sreg.ntid.z()

  out[i++] = gridDim.x; // CHECK: call i32 @llvm.nvvm.read.ptx.sreg.nctaid.x()
  out[i++] = gridDim.y; // CHECK: call i32 @llvm.nvvm.read.ptx.sreg.nctaid.y()
  out[i++] = gridDim.z; // CHECK: call i32 @llvm.nvvm.read.ptx.sreg.nctaid.z()

  out[i++] = warpSize; // CHECK: store i32 32,

  // CHECK: ret void
}
