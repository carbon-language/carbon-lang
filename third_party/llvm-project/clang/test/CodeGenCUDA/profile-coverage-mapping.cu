#include "hip/hip_runtime.h"
// RUN: echo "GPU binary would be here" > %t
// RUN: %clang_cc1 -fprofile-instrument=clang -triple x86_64-linux-gnu -target-sdk-version=8.0 -fcuda-include-gpubinary %t -emit-llvm -o - %s | FileCheck --check-prefix=PGOGEN %s
// RUN: %clang_cc1 -fprofile-instrument=clang -fcoverage-mapping -triple x86_64-linux-gnu -target-sdk-version=8.0 -fcuda-include-gpubinary %t -emit-llvm -o - %s | FileCheck --check-prefix=COVMAP %s
// RUN: %clang_cc1 -fprofile-instrument=clang -fcoverage-mapping -dump-coverage-mapping -triple x86_64-linux-gnu -target-sdk-version=8.0 -fcuda-include-gpubinary %t -emit-llvm-only -o - %s | FileCheck --check-prefix=MAPPING %s

#include "Inputs/hip/hip_runtime.h"

// PGOGEN-NOT: @__profn_{{.*kernel.*}} =
// COVMAP-COUNT-2: section "__llvm_covfun", comdat
// COVMAP-NOT: section "__llvm_covfun", comdat
// MAPPING-NOT: {{.*dfn.*}}:
// MAPPING-NOT: {{.*kernel.*}}:

__device__ void dfn(int i) {}

__global__ void kernel(int i) { dfn(i); }

void host(void) {
  kernel<<<1, 1>>>(1);
}
