#include "hip/hip_runtime.h"
// REQUIRES: amdgpu-registered-target
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -x hip %s \
// RUN:   -std=c++17 -O3 -mllvm -amdgpu-internalize-symbols -emit-llvm -o - \
// RUN:   | FileCheck -check-prefix=DEV %s
// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -x hip %s \
// RUN:   -std=c++17 -O3 -emit-llvm -o - | FileCheck -check-prefix=HOST %s

// Negative tests.

// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -x hip %s \
// RUN:   -std=c++17 -O3 -mllvm -amdgpu-internalize-symbols -emit-llvm -o - \
// RUN:   | FileCheck -check-prefix=DEV-NEG %s
// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -x hip %s \
// RUN:   -std=c++17 -O3 -emit-llvm -o - | FileCheck -check-prefix=HOST-NEG %s

#include "Inputs/hip/hip_runtime.h"

// DEV-DAG: @v1
__device__ int v1;

// DEV-DAG: @v2
__constant__ int v2;

// Check device variables used by neither host nor device functioins are not kept.

// DEV-NEG-NOT: @_ZL2v3
static __device__ int v3;

// Check device variables used by host functions are kept.

// DEV-DAG: @u1
__device__ int u1;

// DEV-DAG: @u2
__constant__ int u2;

// Check host-used static device var is in llvm.compiler.used.
// DEV-DAG: @_ZL2u3
static __device__ int u3;

// Check device-used static device var is emitted but is not in llvm.compiler.used.
// DEV-DAG: @_ZL2u4
static __device__ int u4;

// Check device variables with used attribute are always kept.
// DEV-DAG: @u5
__device__ __attribute__((used)) int u5;

// Test external device variable ODR-used by host code is not emitted or registered.
// DEV-NEG-NOT: @ext_var
extern __device__ int ext_var;

// DEV-DAG: @inline_var = linkonce_odr addrspace(1) externally_initialized global i32 0
__device__ inline int inline_var;

template<typename T>
using func_t = T (*) (T, T);

template <typename T>
__device__ T add_func (T x, T y)
{
  return x + y;
}

// DEV-DAG: @_Z10p_add_funcIiE = linkonce_odr addrspace(1) externally_initialized global i32 (i32, i32)* @_Z8add_funcIiET_S0_S0_
template <typename T>
__device__ func_t<T> p_add_func = add_func<T>;

// Check non-constant constexpr variables ODR-used by host code only is not emitted.
// DEV-NEG-NOT: constexpr_var1a
// DEV-NEG-NOT: constexpr_var1b
constexpr int constexpr_var1a = 1;
inline constexpr int constexpr_var1b = 1;

// Check constant constexpr variables ODR-used by host code only.
// Non-inline constexpr variable has internal linkage, therefore it is not accessible by host and not kept.
// Inline constexpr variable has linkonce_ord linkage, therefore it can be accessed by host and kept.
// DEV-NEG-NOT: constexpr_var2a
// DEV-DAG: @constexpr_var2b = linkonce_odr addrspace(4) externally_initialized constant i32 2
__constant__ constexpr int constexpr_var2a = 2;
inline __constant__ constexpr int constexpr_var2b = 2;

void use(func_t<int> p);
__host__ __device__ void use(const int *p);

// Check static device variable in host function.
// DEV-DAG:  @_ZZ4fun1vE11static_var1 = addrspace(1) externally_initialized global i32 3
void fun1() {
  static __device__ int static_var1 = 3;
  use(&u1);
  use(&u2);
  use(&u3);
  use(&ext_var);
  use(&inline_var);
  use(p_add_func<int>);
  use(&constexpr_var1a);
  use(&constexpr_var1b);
  use(&constexpr_var2a);
  use(&constexpr_var2b);
  use(&static_var1);
}

// Check static variable in host device function.
// DEV-DAG:  @_ZZ4fun2vE11static_var2 = internal addrspace(1) global i32 4
// DEV-DAG:  @_ZZ4fun2vE11static_var3 = addrspace(1) global i32 4
__host__ __device__ void fun2() {
  static int static_var2 = 4;
  static __device__ int static_var3 = 4;
  use(&static_var2);
  use(&static_var3);
}

__global__ void kern1(int **x) {
  *x = &u4;
  fun2();
}

// Check static variables of lambda functions.

// Lambda functions are implicit host device functions.
// Default static variables in lambda functions should be treated
// as host variables on host side, therefore should not be forced
// to be emitted on device.

// DEV-DAG: @_ZZZN21TestStaticVarInLambda3funEvENKUlPcE_clES0_E4var2 = addrspace(1) externally_initialized global i32 5
// DEV-NEG-NOT: @_ZZZN21TestStaticVarInLambda3funEvENKUlPcE_clES0_E4var1
namespace TestStaticVarInLambda {
class A {
public:
  A(char *);
};
void fun() {
  (void) [](char *c) {
    static A var1(c);
    static __device__ int var2 = 5;
    (void) var1;
    (void) var2;
  };
}
}

// Check implicit constant variable ODR-used by host code is not emitted.

// AST contains instantiation of al<ar>, which triggers AST instantiation
// of x::al<ar>::am, which triggers AST instatiation of x::ap<ar>,
// which triggers AST instantiation of aw<ar>::c, which has type
// ar. ar has base class x which has member ah. x::ah is initialized
// with function pointer pointing to ar:as, which returns an object
// of type ou. The constexpr aw<ar>::c is an implicit constant variable
// which is ODR-used by host function x::ap<ar>. An incorrect implementation
// will force aw<ar>::c to be emitted on device side, which will trigger
// emit of x::as and further more ctor of ou and variable o.
// The ODR-use of aw<ar>::c in x::ap<ar> should be treated as a host variable
// instead of device variable.

// DEV-NEG-NOT: _ZN16TestConstexprVar1oE
namespace TestConstexprVar {
char o;
class ou {
public:
  ou(char) { __builtin_strlen(&o); }
};
template < typename ao > struct aw { static constexpr ao c; };
class x {
protected:
  typedef ou (*y)(const x *);
  constexpr x(y ag) : ah(ag) {}
  template < bool * > struct ak;
  template < typename > struct al {
    static bool am;
    static ak< &am > an;
  };
  template < typename ao > static x ap() { (void)aw< ao >::c; return x(nullptr); }
  y ah;
};
template < typename ao > bool x::al< ao >::am(&ap< ao >);
class ar : x {
public:
  constexpr ar() : x(as) {}
  static ou as(const x *) { return 0; }
  al< ar > av;
};
}

// Check the exact list of variables to ensure @_ZL2u4 is not among them.
// DEV: @llvm.compiler.used = {{[^@]*}} @_Z10p_add_funcIiE
// DEV-SAME: {{^[^@]*}} @_ZL2u3
// DEV-SAME: {{^[^@]*}} @_ZZ4fun1vE11static_var1
// DEV-SAME: {{^[^@]*}} @_ZZZN21TestStaticVarInLambda3funEvENKUlPcE_clES0_E4var2
// DEV-SAME: {{^[^@]*}} @constexpr_var2b
// DEV-SAME: {{^[^@]*}} @inline_var
// DEV-SAME: {{^[^@]*}} @u1
// DEV-SAME: {{^[^@]*}} @u2
// DEV-SAME: {{^[^@]*}} @u5
// DEV-SAME: {{^[^@]*$}}

// HOST-DAG: hipRegisterVar{{.*}}@u1
// HOST-DAG: hipRegisterVar{{.*}}@u2
// HOST-DAG: hipRegisterVar{{.*}}@_ZL2u3
// HOST-DAG: hipRegisterVar{{.*}}@constexpr_var2b
// HOST-DAG: hipRegisterVar{{.*}}@u5
// HOST-DAG: hipRegisterVar{{.*}}@inline_var
// HOST-DAG: hipRegisterVar{{.*}}@_Z10p_add_funcIiE
// HOST-NEG-NOT: hipRegisterVar{{.*}}@_ZZ4fun1vE11static_var1
// HOST-NEG-NOT: hipRegisterVar{{.*}}@_ZZ4fun2vE11static_var2
// HOST-NEG-NOT: hipRegisterVar{{.*}}@_ZZ4fun2vE11static_var3
// HOST-NEG-NOT: hipRegisterVar{{.*}}@_ZZZN21TestStaticVarInLambda3funEvENKUlPcE_clES0_E4var2
// HOST-NEG-NOT: hipRegisterVar{{.*}}@_ZZZN21TestStaticVarInLambda3funEvENKUlPcE_clES0_E4var1
// HOST-NEG-NOT: hipRegisterVar{{.*}}@ext_var
// HOST-NEG-NOT: hipRegisterVar{{.*}}@_ZL2u4
// HOST-NEG-NOT: hipRegisterVar{{.*}}@constexpr_var1a
// HOST-NEG-NOT: hipRegisterVar{{.*}}@constexpr_var1b
// HOST-NEG-NOT: hipRegisterVar{{.*}}@constexpr_var2a
