#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -triple amdgcn -fcuda-is-device -emit-llvm -x hip %s -o - | FileCheck %s
#include "Inputs/hip/hip_runtime.h"

// CHECK: define{{.*}} amdgpu_kernel void @_ZN1A6kernelEv
class A {
public:
  static __global__ void kernel(){}
};

// CHECK: define{{.*}} void @_Z10non_kernelv
__device__ void non_kernel(){}

// CHECK: define{{.*}} amdgpu_kernel void @_Z6kerneli
__global__ void kernel(int x) {
  non_kernel();
}

// CHECK: define{{.*}} amdgpu_kernel void @_Z11EmptyKernelIvEvv
template <typename T>
__global__ void EmptyKernel(void) {}

struct Dummy {
  /// Type definition of the EmptyKernel kernel entry point
  typedef void (*EmptyKernelPtr)();
  EmptyKernelPtr Empty() { return EmptyKernel<void>; } 
};

// CHECK: define{{.*}} amdgpu_kernel void @_Z15template_kernelI1AEvT_{{.*}} #[[ATTR:[0-9][0-9]*]]
template<class T>
__global__ void template_kernel(T x) {}

void launch(void *f);

int main() {
  Dummy D;
  launch((void*)A::kernel);
  launch((void*)kernel);
  launch((void*)template_kernel<A>);
  launch((void*)D.Empty());
  return 0;
}
// CHECK: attributes #[[ATTR]] = {{.*}}"amdgpu-flat-work-group-size"="1,1024"
