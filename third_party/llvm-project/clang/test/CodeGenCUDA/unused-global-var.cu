#include "hip/hip_runtime.h"
// REQUIRES: amdgpu-registered-target
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -x hip %s \
// RUN:   -std=c++11 -O3 -mllvm -amdgpu-internalize-symbols -emit-llvm -o - \
// RUN:   -target-cpu gfx906 | FileCheck %s
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -x hip %s \
// RUN:   -std=c++11 -O3 -mllvm -amdgpu-internalize-symbols -emit-llvm -o - \
// RUN:   -target-cpu gfx906 | FileCheck -check-prefix=NEGCHK %s

#include "Inputs/hip/hip_runtime.h"

// AMDGPU internalize unused global variables for whole-program compilation
// (-fno-gpu-rdc for each TU, or -fgpu-rdc for LTO), which are then
// eliminated by global DCE. If there are invisible unused address space casts
// for global variables, these dead users need to be eliminated by global
// DCE before internalization. This test makes sure unused global variables
// are eliminated.

// CHECK-DAG: @v1
__device__ int v1;

// CHECK-DAG: @v2
__constant__ int v2;

// Check unused device/constant variables are eliminated.

// NEGCHK-NOT: @_ZL2v3
constexpr int v3 = 1;

// Check managed variables are always kept.

// CHECK-DAG: @v4
__managed__ int v4;

// Check used device/constant variables are not eliminated.
// CHECK-DAG: @u1
__device__ int u1;

// CHECK-DAG: @u2
__constant__ int u2;

// Check u3 is kept because its address is taken.
// CHECK-DAG: @_ZL2u3
constexpr int u3 = 2;

// Check u4 is not kept because it is not ODR-use.
// NEGCHK-NOT: @_ZL2u4
constexpr int u4 = 3;

__device__ int fun1(const int& x);

__global__ void kern1(int *x) {
  *x = u1 + u2 + fun1(u3) + u4;
}
