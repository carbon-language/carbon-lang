#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -triple nvptx -fcuda-is-device -emit-llvm -o - %s \
// RUN:   | FileCheck -check-prefix=NORDC %s
// RUN: %clang_cc1 -triple nvptx -fcuda-is-device -emit-llvm -o - %s \
// RUN:   | FileCheck -check-prefix=NORDC-NEG %s
// RUN: %clang_cc1 -triple nvptx -fcuda-is-device -fgpu-rdc -emit-llvm -o - %s \
// RUN:   | FileCheck -check-prefix=RDC %s
// RUN: %clang_cc1 -triple nvptx -fcuda-is-device -fgpu-rdc -emit-llvm -o - %s \
// RUN:   | FileCheck -check-prefix=RDC-NEG %s

#include "Inputs/hip/hip_runtime.h"

template <typename T> __device__ void func() {}
template <typename T> __global__ void kernel() {}

template __device__ void func<int>();
// NORDC:     define internal void @_Z4funcIiEvv()
// RDC:       define weak_odr void @_Z4funcIiEvv()

template __global__ void kernel<int>();
// NORDC:     define void @_Z6kernelIiEvv()
// RDC:       define weak_odr void @_Z6kernelIiEvv()

// Ensure that unused static device function is eliminated
static __device__ void static_func() {}
// NORDC-NEG-NOT: define{{.*}} void @_ZL13static_funcv()
// RDC-NEG-NOT:   define{{.*}} void @_ZL13static_funcv[[FILEID:.*]]()

// Ensure that kernel function has external or weak_odr
// linkage regardless static specifier
static __global__ void static_kernel() {}
// NORDC:     define void @_ZL13static_kernelv()
// RDC:       define weak_odr void @_ZL13static_kernelv[[FILEID:.*]]()
