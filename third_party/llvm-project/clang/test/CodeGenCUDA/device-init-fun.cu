// REQUIRES: amdgpu-registered-target

// RUN: %clang_cc1 -triple amdgcn -fcuda-is-device -std=c++11 \
// RUN:     -fgpu-allow-device-init -x hip \
// RUN:     -fno-threadsafe-statics -emit-llvm -o - %s \
// RUN:     | FileCheck %s

#include "Inputs/hip/hip_runtime.h"

// CHECK: define internal amdgpu_kernel void @_GLOBAL__sub_I_device_init_fun.cu() #[[ATTR:[0-9]*]]
// CHECK: attributes #[[ATTR]] = {{.*}}"device-init"

__device__ void f();

struct A {
  __device__ A() { f(); }
};

__device__ A a;
