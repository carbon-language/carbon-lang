#include "hip/hip_runtime.h"
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// RUN: %clang_cc1 -std=c++11 -fcuda-is-device -triple nvptx64-nvidia-cuda -emit-llvm -o - %s | FileCheck --check-prefix=DEVICE %s
// RUN: echo "GPU binary would be here" > %t
// RUN: %clang_cc1 -std=c++11 -triple x86_64-unknown-linux-gnu -target-sdk-version=8.0 -fcuda-include-gpubinary %t -emit-llvm -o - %s | FileCheck --check-prefix=HOST %s

struct textureReference {
  int desc;
};

enum ReadMode {
  ElementType = 0,
  NormalizedFloat = 1
};

template <typename T, int dim = 1, enum ReadMode mode = ElementType>
struct __attribute__((device_builtin_texture_type)) texture : public textureReference {
};

// On the device side, texture references are represented as `i64` handles.
// DEVICE: @tex ={{.*}} addrspace(1) externally_initialized global i64 undef, align 4
// DEVICE: @norm ={{.*}} addrspace(1) externally_initialized global i64 undef, align 4
// On the host side, they remain in the original type.
// HOST: @tex = internal global %struct.texture
// HOST: @norm = internal global %struct.texture
// HOST: @0 = private unnamed_addr constant [4 x i8] c"tex\00"
// HOST: @1 = private unnamed_addr constant [5 x i8] c"norm\00"
texture<float, 2, ElementType> tex;
texture<float, 2, NormalizedFloat> norm;

struct v4f {
  float x, y, z, w;
};

__attribute__((device)) v4f tex2d_ld(texture<float, 2, ElementType>, float, float) asm("llvm.nvvm.tex.unified.2d.v4f32.f32");
__attribute__((device)) v4f tex2d_ld(texture<float, 2, NormalizedFloat>, int, int) asm("llvm.nvvm.tex.unified.2d.v4f32.s32");

// DEVICE-LABEL: float @_Z3fooff(float %x, float %y)
// DEVICE: call i64 @llvm.nvvm.texsurf.handle.internal.p1i64(i64 addrspace(1)* @tex)
// DEVICE: call %struct.v4f @llvm.nvvm.tex.unified.2d.v4f32.f32(i64 %{{.*}}, float %{{.*}}, float %{{.*}})
// DEVICE: call i64 @llvm.nvvm.texsurf.handle.internal.p1i64(i64 addrspace(1)* @norm)
// DEVICE: call %struct.v4f @llvm.nvvm.tex.unified.2d.v4f32.s32(i64 %{{.*}}, i32 %{{.*}}, i32 %{{.*}})
__attribute__((device)) float foo(float x, float y) {
  return tex2d_ld(tex, x, y).x + tex2d_ld(norm, int(x), int(y)).x;
}

// HOST: define internal void @[[PREFIX:__cuda]]_register_globals
// Texture references need registering with correct arguments.
// HOST: call void @[[PREFIX]]RegisterTexture(i8** %0, i8*{{.*}}({{.*}}@tex{{.*}}), i8*{{.*}}({{.*}}@0{{.*}}), i8*{{.*}}({{.*}}@0{{.*}}), i32 2, i32 0, i32 0)
// HOST: call void @[[PREFIX]]RegisterTexture(i8** %0, i8*{{.*}}({{.*}}@norm{{.*}}), i8*{{.*}}({{.*}}@1{{.*}}), i8*{{.*}}({{.*}}@1{{.*}}), i32 2, i32 1, i32 0)

// They also need annotating in metadata.
// DEVICE: !0 = !{i64 addrspace(1)* @tex, !"texture", i32 1}
// DEVICE: !1 = !{i64 addrspace(1)* @norm, !"texture", i32 1}
