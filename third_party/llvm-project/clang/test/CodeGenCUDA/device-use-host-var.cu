// RUN: %clang_cc1 -no-opaque-pointers -std=c++14 -triple amdgcn-amd-amdhsa \
// RUN:   -fcuda-is-device -emit-llvm -o - -x hip %s | FileCheck %s
// RUN: %clang_cc1 -no-opaque-pointers -std=c++14 -triple amdgcn-amd-amdhsa \
// RUN:   -fcuda-is-device -emit-llvm -o - -x hip %s | FileCheck -check-prefix=NEG %s

#include "Inputs/hip/hip_runtime.h"

struct A {
  int x;
};

// Check the situation of B<T> has empty ctor but B<int> has non-empty ctor.
// Make sure const B<int> variables are not promoted to constant variables.
template<typename T>
struct B {
  T x;
  B() {}
  B(T _x) { x = _x; }
  static const B<T> y;
};

template<>
struct B<int> {
  int x;
  B() { x = 1; }
  static const B<int> y;
};

template<typename T>
const B<T> B<T>::y;

const B<int> B<int>::y;

template<typename T>
T temp_fun(T x) {
  return B<T>::y.x;
}

// Check template variable with empty default ctor but non-empty initializer
// ctor is not promoted.
template<typename T>
const B<T> b = B<T>(-1);

constexpr int constexpr_var = 1;
constexpr A constexpr_struct{2};
constexpr A constexpr_array[4] = {0, 0, 0, 3};
constexpr char constexpr_str[] = "abcd";
const int const_var = 4;
const A const_struct{5};
const A const_array[] = {0, 0, 0, 6};
const char const_str[] = "xyz";

// Check const variables used by host only are not emitted.
const int var_host_only = 7;

// CHECK-DAG: @_ZL13constexpr_str.const = private unnamed_addr addrspace(4) constant [5 x i8] c"abcd\00"
// CHECK-DAG: @_ZL13constexpr_var = internal addrspace(4) constant i32 1
// CHECK-DAG: @_ZL16constexpr_struct = internal addrspace(4) constant %struct.A { i32 2 }
// CHECK-DAG: @_ZL15constexpr_array = internal addrspace(4) constant [4 x %struct.A] [%struct.A zeroinitializer, %struct.A zeroinitializer, %struct.A zeroinitializer, %struct.A { i32 3 }]
// CHECK-DAG: @_ZL9const_var = internal addrspace(4) constant i32 4
// CHECK-DAG: @_ZL12const_struct = internal addrspace(4) constant %struct.A { i32 5 }
// CHECK-DAG: @_ZL11const_array = internal addrspace(4) constant [4 x %struct.A] [%struct.A zeroinitializer, %struct.A zeroinitializer, %struct.A zeroinitializer, %struct.A { i32 6 }]
// CHECK-DAG: @_ZL9const_str = internal addrspace(4) constant [4 x i8] c"xyz\00"

// NEG-NOT: @_ZN1BIiE1yE
// NEG-NOT: @_Z1bIdE
// NEG-NOT: @_ZL13var_host_only
// NEG-NOT: {{^}}@{{.*}} = external

// CHECK-LABEL: define{{.*}}@_Z7dev_funPiPPKi
// CHECK: store i32 1
// CHECK: store i32 2
// CHECK: store i32 3
// CHECK: load i8, i8* getelementptr {{.*}} @_ZL13constexpr_str.const
// CHECK: store i32 4
// CHECK: store i32 5
// CHECK: store i32 6
// CHECK: load i8, i8* getelementptr {{.*}} @_ZL9const_str
// CHECK: store i32* {{.*}}@_ZL13constexpr_var
// CHECK: store i32* getelementptr {{.*}} @_ZL16constexpr_struct
// CHECK: store i32* getelementptr {{.*}} @_ZL15constexpr_array
// CHECK: store i32* {{.*}}@_ZL9const_var
// CHECK: store i32* getelementptr {{.*}} @_ZL12const_struct
// CHECK: store i32* getelementptr {{.*}} @_ZL11const_array
__device__ void dev_fun(int *out, const int **out2) {
  *out = constexpr_var;
  *out = constexpr_struct.x;
  *out = constexpr_array[3].x;
  *out = constexpr_str[3];
  *out = const_var;
  *out = const_struct.x;
  *out = const_array[3].x;
  *out = const_str[3];
  *out2 = &constexpr_var;
  *out2 = &constexpr_struct.x;
  *out2 = &constexpr_array[3].x;
  *out2 = &const_var;
  *out2 = &const_struct.x;
  *out2 = &const_array[3].x;
}

void fun() {
  temp_fun(1);
  (void) b<double>;
  (void) var_host_only;
}
