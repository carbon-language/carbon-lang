#include "hip/hip_runtime.h"
// REQUIRES: x86-registered-target, nvptx-registered-target, amdgpu-registered-target

// By default CUDA uses -ffp-contract=fast, HIP uses -ffp-contract=fast-honor-pragmas.
// we should fuse multiply/add into fma instruction.
// In IR, fmul/fadd instructions with contract flag are emitted.
// In backend
//    nvptx -  assumes fast fp fuse option, which fuses
//             mult/add insts disregarding contract flag and
//             llvm.fmuladd intrinsics.
//    amdgcn - assumes standard fp fuse option, which only
//             fuses mult/add insts with contract flag and
//             llvm.fmuladd intrinsics.

// RUN: %clang_cc1 -fcuda-is-device -triple nvptx-nvidia-cuda -S \
// RUN:   -disable-llvm-passes -o - %s \
// RUN:   | FileCheck -check-prefixes=COMMON,NV-ON %s
// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -S \
// RUN:   -target-cpu gfx906 -disable-llvm-passes -o - -x hip %s \
// RUN:   | FileCheck -check-prefixes=COMMON,AMD-ON %s
// RUN: %clang_cc1 -fcuda-is-device -triple nvptx-nvidia-cuda -S \
// RUN:   -O3 -o - %s \
// RUN:   | FileCheck -check-prefixes=COMMON,NV-OPT-FAST %s
// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -S \
// RUN:   -O3 -target-cpu gfx906 -o - -x hip %s \
// RUN:   | FileCheck -check-prefixes=COMMON,AMD-OPT-FASTSTD %s

// Check separate compile/backend steps corresponding to -save-temps.

// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm \
// RUN:   -O3 -disable-llvm-passes -target-cpu gfx906 -o %t.ll -x hip %s
// RUN: cat %t.ll  | FileCheck -check-prefixes=COMMON,AMD-OPT-FAST-IR %s
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -S \
// RUN:   -O3 -target-cpu gfx906 -o - -x ir %t.ll \
// RUN:   | FileCheck -check-prefixes=COMMON,AMD-OPT-FASTSTD %s

// Explicit -ffp-contract=fast
// In IR, fmul/fadd instructions with contract flag are emitted.
// In backend
//    nvptx/amdgcn - assumes fast fp fuse option, which fuses
//                   mult/add insts disregarding contract flag and
//                   llvm.fmuladd intrinsics.

// RUN: %clang_cc1 -fcuda-is-device -triple nvptx-nvidia-cuda -S \
// RUN:   -ffp-contract=fast -disable-llvm-passes -o - %s \
// RUN:   | FileCheck -check-prefixes=COMMON,NV-ON %s
// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -S \
// RUN:   -target-cpu gfx906 -disable-llvm-passes -o - -x hip %s \
// RUN:   -ffp-contract=fast \
// RUN:   | FileCheck -check-prefixes=COMMON,AMD-ON %s
// RUN: %clang_cc1 -fcuda-is-device -triple nvptx-nvidia-cuda -S \
// RUN:   -O3 -o - %s \
// RUN:   -ffp-contract=fast \
// RUN:   | FileCheck -check-prefixes=COMMON,NV-OPT-FAST %s
// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -S \
// RUN:   -O3 -target-cpu gfx906 -o - -x hip %s \
// RUN:   -ffp-contract=fast \
// RUN:   | FileCheck -check-prefixes=COMMON,AMD-OPT-FAST %s

// Check separate compile/backend steps corresponding to -save-temps.
// When input is IR, -ffp-contract has no effect. Backend uses default
// default FP fuse option.

// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm \
// RUN:   -ffp-contract=fast \
// RUN:   -O3 -disable-llvm-passes -target-cpu gfx906 -o %t.ll -x hip %s
// RUN: cat %t.ll  | FileCheck -check-prefixes=COMMON,AMD-OPT-FAST-IR %s
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -S \
// RUN:   -O3 -target-cpu gfx906 -o - -x ir %t.ll \
// RUN:   | FileCheck -check-prefixes=COMMON,AMD-OPT-FASTSTD %s

// Explicit -ffp-contract=fast-honor-pragmas
// In IR, fmul/fadd instructions with contract flag are emitted.
// In backend
//    nvptx/amdgcn - assumes standard fp fuse option, which only
//                   fuses mult/add insts with contract flag or
//                   llvm.fmuladd intrinsics.

// RUN: %clang_cc1 -fcuda-is-device -triple nvptx-nvidia-cuda -S \
// RUN:   -ffp-contract=fast-honor-pragmas -disable-llvm-passes -o - %s \
// RUN:   | FileCheck -check-prefixes=COMMON,NV-ON %s
// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -S \
// RUN:   -target-cpu gfx906 -disable-llvm-passes -o - -x hip %s \
// RUN:   -ffp-contract=fast-honor-pragmas \
// RUN:   | FileCheck -check-prefixes=COMMON,AMD-ON %s
// RUN: %clang_cc1 -fcuda-is-device -triple nvptx-nvidia-cuda -S \
// RUN:   -O3 -o - %s \
// RUN:   -ffp-contract=fast-honor-pragmas \
// RUN:   | FileCheck -check-prefixes=COMMON,NV-OPT-FASTSTD %s
// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -S \
// RUN:   -O3 -target-cpu gfx906 -o - -x hip %s \
// RUN:   -ffp-contract=fast-honor-pragmas \
// RUN:   | FileCheck -check-prefixes=COMMON,AMD-OPT-FASTSTD %s

// Check separate compile/backend steps corresponding to -save-temps.
// When input is IR, -ffp-contract has no effect. Backend uses default
// default FP fuse option.

// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm \
// RUN:   -ffp-contract=fast-honor-pragmas \
// RUN:   -O3 -disable-llvm-passes -target-cpu gfx906 -o %t.ll -x hip %s
// RUN: cat %t.ll  | FileCheck -check-prefixes=COMMON,AMD-OPT-FAST-IR %s
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -S \
// RUN:   -O3 -target-cpu gfx906 -o - -x ir %t.ll \
// RUN:   | FileCheck -check-prefixes=COMMON,AMD-OPT-FASTSTD %s

// Explicit -ffp-contract=on -- fusing by front-end.
// In IR,
//    mult/add in the same statement - llvm.fmuladd intrinsic emitted
//    mult/add in different statement -  fmul/fadd instructions without
//                                       contract flag are emitted.
// In backend
//    nvptx/amdgcn - assumes standard fp fuse option, which only
//                   fuses mult/add insts with contract flag or
//                   llvm.fmuladd intrinsics.

// RUN: %clang_cc1 -fcuda-is-device -triple nvptx-nvidia-cuda -S \
// RUN:   -ffp-contract=on -disable-llvm-passes -o - %s \
// RUN:   | FileCheck -check-prefixes=COMMON,NV-ON %s
// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -S \
// RUN:   -target-cpu gfx906 -disable-llvm-passes -o - -x hip %s \
// RUN:   -ffp-contract=on \
// RUN:   | FileCheck -check-prefixes=COMMON,AMD-ON %s
// RUN: %clang_cc1 -fcuda-is-device -triple nvptx-nvidia-cuda -S \
// RUN:   -O3 -o - %s \
// RUN:   -ffp-contract=on \
// RUN:   | FileCheck -check-prefixes=COMMON,NV-OPT-ON %s
// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -S \
// RUN:   -O3 -target-cpu gfx906 -o - -x hip %s \
// RUN:   -ffp-contract=on \
// RUN:   | FileCheck -check-prefixes=COMMON,AMD-OPT-ON %s

// Check separate compile/backend steps corresponding to -save-temps.

// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm \
// RUN:   -ffp-contract=on \
// RUN:   -O3 -disable-llvm-passes -target-cpu gfx906 -o %t.ll -x hip %s
// RUN: cat %t.ll  | FileCheck -check-prefixes=COMMON,AMD-OPT-ON-IR %s
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -S \
// RUN:   -O3 -target-cpu gfx906 -o - -x ir %t.ll \
// RUN:   | FileCheck -check-prefixes=COMMON,AMD-OPT-ON %s

// Explicit -ffp-contract=off should disable instruction fusing.
// In IR, fmul/fadd instructions without contract flag are emitted.
// In backend
//    nvptx/amdgcn - assumes standard fp fuse option, which only
//                   fuses mult/add insts with contract flag or
//                   llvm.fmuladd intrinsics.

// RUN: %clang_cc1 -fcuda-is-device -triple nvptx-nvidia-cuda -S \
// RUN:   -ffp-contract=off -disable-llvm-passes -o - %s \
// RUN:   | FileCheck -check-prefixes=COMMON,NV-OFF %s
// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -S \
// RUN:   -target-cpu gfx906 -disable-llvm-passes -o - -x hip %s \
// RUN:   -ffp-contract=off \
// RUN:   | FileCheck -check-prefixes=COMMON,AMD-OFF %s
// RUN: %clang_cc1 -fcuda-is-device -triple nvptx-nvidia-cuda -S \
// RUN:   -O3 -o - %s \
// RUN:   -ffp-contract=off \
// RUN:   | FileCheck -check-prefixes=COMMON,NV-OPT-OFF %s
// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -S \
// RUN:   -O3 -target-cpu gfx906 -o - -x hip %s \
// RUN:   -ffp-contract=off \
// RUN:   | FileCheck -check-prefixes=COMMON,AMD-OPT-OFF %s

// Check separate compile/backend steps corresponding to -save-temps.

// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm \
// RUN:   -ffp-contract=off \
// RUN:   -O3 -disable-llvm-passes -target-cpu gfx906 -o %t.ll -x hip %s
// RUN: cat %t.ll  | FileCheck -check-prefixes=COMMON,AMD-OPT-OFF-IR %s
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -S \
// RUN:   -O3 -target-cpu gfx906 -o - -x ir %t.ll \
// RUN:   | FileCheck -check-prefixes=COMMON,AMD-OPT-OFF %s

#include "Inputs/hip/hip_runtime.h"

// Test multiply/add in the same statement, which can be emitted as FMA when
// fp-contract is on or fast.
__host__ __device__ float func(float a, float b, float c) { return a + b * c; }
// COMMON-LABEL: _Z4funcfff
// NV-ON:       fma.rn.f32
// NV-ON-NEXT:  st.param.f32
// AMD-ON:       v_fmac_f32_e64
// AMD-ON-NEXT:  s_setpc_b64

// NV-OFF:      mul.rn.f32
// NV-OFF-NEXT: add.rn.f32
// NV-OFF-NEXT: st.param.f32
// AMD-OFF:      v_mul_f32_e64
// AMD-OFF-NEXT: v_add_f32_e64
// AMD-OFF-NEXT: s_setpc_b64

// NV-OPT-FAST: fma.rn.f32
// NV-OPT-FAST-NEXT: st.param.f32
// NV-OPT-FASTSTD: fma.rn.f32
// NV-OPT-FASTSTD-NEXT: st.param.f32
// NV-OPT-ON: fma.rn.f32
// NV-OPT-ON-NEXT: st.param.f32
// NV-OPT-OFF: mul.rn.f32
// NV-OPT-OFF-NEXT: add.rn.f32
// NV-OPT-OFF-NEXT: st.param.f32

// AMD-OPT-FAST-IR: fmul contract float
// AMD-OPT-FAST-IR: fadd contract float
// AMD-OPT-ON-IR: @llvm.fmuladd.f32
// AMD-OPT-OFF-IR: fmul float
// AMD-OPT-OFF-IR: fadd float

// AMD-OPT-FAST: v_fmac_f32_e32
// AMD-OPT-FAST-NEXT: s_setpc_b64
// AMD-OPT-FASTSTD: v_fmac_f32_e32
// AMD-OPT-FASTSTD-NEXT: s_setpc_b64
// AMD-OPT-ON: v_fmac_f32_e32
// AMD-OPT-ON-NEXT: s_setpc_b64
// AMD-OPT-OFF: v_mul_f32_e32
// AMD-OPT-OFF-NEXT: v_add_f32_e32
// AMD-OPT-OFF-NEXT: s_setpc_b64

// Test multiply/add in the different statements, which can be emitted as
// FMA when fp-contract is fast but not on.
__host__ __device__ float func2(float a, float b, float c) {
  float t = b * c;
  return t + a;
}
// COMMON-LABEL: _Z5func2fff
// NV-OPT-FAST: fma.rn.f32
// NV-OPT-FAST-NEXT: st.param.f32
// NV-OPT-FASTSTD: fma.rn.f32
// NV-OPT-FASTSTD-NEXT: st.param.f32
// NV-OPT-ON: mul.rn.f32
// NV-OPT-ON: add.rn.f32
// NV-OPT-ON-NEXT: st.param.f32
// NV-OPT-OFF: mul.rn.f32
// NV-OPT-OFF: add.rn.f32
// NV-OPT-OFF-NEXT: st.param.f32

// AMD-OPT-FAST-IR: fmul contract float
// AMD-OPT-FAST-IR: fadd contract float
// AMD-OPT-ON-IR: fmul float
// AMD-OPT-ON-IR: fadd float
// AMD-OPT-OFF-IR: fmul float
// AMD-OPT-OFF-IR: fadd float

// AMD-OPT-FAST: v_fmac_f32_e32
// AMD-OPT-FAST-NEXT: s_setpc_b64
// AMD-OPT-FASTSTD: v_fmac_f32_e32
// AMD-OPT-FASTSTD-NEXT: s_setpc_b64
// AMD-OPT-ON: v_mul_f32_e32
// AMD-OPT-ON-NEXT: v_add_f32_e32
// AMD-OPT-ON-NEXT: s_setpc_b64
// AMD-OPT-OFF: v_mul_f32_e32
// AMD-OPT-OFF-NEXT: v_add_f32_e32
// AMD-OPT-OFF-NEXT: s_setpc_b64

// Test multiply/add in the different statements, which is forced
// to be compiled with fp contract on. fmul/fadd without contract
// flags are emitted in IR. In nvptx, they are emitted as FMA in
// fp-contract is fast but not on, as nvptx backend uses the same
// fp fuse option as front end, whereas fast fp fuse option in
// backend fuses fadd/fmul disregarding contract flag. In amdgcn
// they are not fused as amdgcn always use standard fp fusion
// option which respects contract flag.
  __host__ __device__ float func3(float a, float b, float c) {
#pragma clang fp contract(on)
  float t = b * c;
  return t + a;
}
// COMMON-LABEL: _Z5func3fff
// NV-OPT-FAST: fma.rn.f32
// NV-OPT-FAST-NEXT: st.param.f32
// NV-OPT-FASTSTD: mul.rn.f32
// NV-OPT-FASTSTD: add.rn.f32
// NV-OPT-FASTSTD-NEXT: st.param.f32
// NV-OPT-ON: mul.rn.f32
// NV-OPT-ON: add.rn.f32
// NV-OPT-ON-NEXT: st.param.f32
// NV-OPT-OFF: mul.rn.f32
// NV-OPT-OFF: add.rn.f32
// NV-OPT-OFF-NEXT: st.param.f32

// AMD-OPT-FAST-IR: fmul float
// AMD-OPT-FAST-IR: fadd float
// AMD-OPT-ON-IR: fmul float
// AMD-OPT-ON-IR: fadd float
// AMD-OPT-OFF-IR: fmul float
// AMD-OPT-OFF-IR: fadd float

// AMD-OPT-FAST: v_fmac_f32_e32
// AMD-OPT-FAST-NEXT: s_setpc_b64
// AMD-OPT-FASTSTD: v_mul_f32_e32
// AMD-OPT-FASTSTD-NEXT: v_add_f32_e32
// AMD-OPT-FASTSTD-NEXT: s_setpc_b64
// AMD-OPT-ON: v_mul_f32_e32
// AMD-OPT-ON-NEXT: v_add_f32_e32
// AMD-OPT-ON-NEXT: s_setpc_b64
// AMD-OPT-OFF: v_mul_f32_e32
// AMD-OPT-OFF-NEXT: v_add_f32_e32
// AMD-OPT-OFF-NEXT: s_setpc_b64
