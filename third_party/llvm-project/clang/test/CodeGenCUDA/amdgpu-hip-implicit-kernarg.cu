#include "hip/hip_runtime.h"
// REQUIRES: amdgpu-registered-target
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -emit-llvm -x hip -o - %s | FileCheck %s
#include "Inputs/hip/hip_runtime.h"

__global__ void hip_kernel_temp() {
}

// CHECK: attributes {{.*}} = {{.*}} "amdgpu-implicitarg-num-bytes"="56"
