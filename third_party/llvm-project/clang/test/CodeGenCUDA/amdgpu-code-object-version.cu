
#include <hip/hip_runtime.h>
// Create module flag for code object version.

// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm \
// RUN:   -o - %s | FileCheck %s -check-prefix=NONE

// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm \
// RUN:   -mcode-object-version=2 -o - %s | FileCheck -check-prefix=NONE %s

// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm \
// RUN:   -mcode-object-version=3 -o - %s | FileCheck -check-prefix=NONE %s

// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm \
// RUN:   -mcode-object-version=4 -o - %s | FileCheck -check-prefix=NONE %s

// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm \
// RUN:   -mcode-object-version=5 -o - %s | FileCheck -check-prefix=V5 %s

// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm \
// RUN:   -mcode-object-version=none -o - %s | FileCheck %s -check-prefix=NONE

// RUN: not %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm \
// RUN:   -mcode-object-version=4.1 -o - %s 2>&1| FileCheck %s -check-prefix=INV

// V5: !{{.*}} = !{i32 1, !"amdgpu_code_object_version", i32 500}
// NONE-NOT: !{{.*}} = !{i32 1, !"amdgpu_code_object_version",
// INV: error: invalid value '4.1' in '-mcode-object-version=4.1'
