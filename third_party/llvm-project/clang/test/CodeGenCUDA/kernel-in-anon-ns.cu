#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -cuid=abc \
// RUN:   -aux-triple x86_64-unknown-linux-gnu -std=c++11 -fgpu-rdc \
// RUN:   -emit-llvm -o - -x hip %s > %t.dev

// RUN: %clang_cc1 -triple x86_64-gnu-linux -cuid=abc \
// RUN:   -aux-triple amdgcn-amd-amdhsa -std=c++11 -fgpu-rdc \
// RUN:   -emit-llvm -o - -x hip %s > %t.host

// RUN: cat %t.dev %t.host | FileCheck -check-prefixes=HIP,COMMON %s

// RUN: echo "GPU binary" > %t.fatbin

// RUN: %clang_cc1 -triple nvptx -fcuda-is-device -cuid=abc \
// RUN:   -aux-triple x86_64-unknown-linux-gnu -std=c++11 -fgpu-rdc \
// RUN:   -emit-llvm -o - %s > %t.dev

// RUN: %clang_cc1 -triple x86_64-gnu-linux -cuid=abc \
// RUN:   -aux-triple nvptx -std=c++11 -fgpu-rdc -fcuda-include-gpubinary %t.fatbin \
// RUN:   -emit-llvm -o - %s > %t.host

// RUN: cat %t.dev %t.host | FileCheck -check-prefixes=CUDA,COMMON %s

#include "Inputs/hip/hip_runtime.h"

// HIP-DAG: define weak_odr {{.*}}void @[[KERN1:_ZN12_GLOBAL__N_16kernelEv\.intern\.b04fd23c98500190]](
// HIP-DAG: define weak_odr {{.*}}void @[[KERN2:_Z8tempKernIN12_GLOBAL__N_11XEEvT_\.intern\.b04fd23c98500190]](
// HIP-DAG: define weak_odr {{.*}}void @[[KERN3:_Z8tempKernIN12_GLOBAL__N_1UlvE_EEvT_\.intern\.b04fd23c98500190]](

// CUDA-DAG: define weak_odr {{.*}}void @[[KERN1:_ZN12_GLOBAL__N_16kernelEv__intern__b04fd23c98500190]](
// CUDA-DAG: define weak_odr {{.*}}void @[[KERN2:_Z8tempKernIN12_GLOBAL__N_11XEEvT___intern__b04fd23c98500190]](
// CUDA-DAG: define weak_odr {{.*}}void @[[KERN3:_Z8tempKernIN12_GLOBAL__N_1UlvE_EEvT___intern__b04fd23c98500190]](

// COMMON-DAG: @[[STR1:.*]] = {{.*}} c"[[KERN1]]\00"
// COMMON-DAG: @[[STR2:.*]] = {{.*}} c"[[KERN2]]\00"
// COMMON-DAG: @[[STR3:.*]] = {{.*}} c"[[KERN3]]\00"

// COMMON-DAG: call i32 @__{{.*}}RegisterFunction({{.*}}@[[STR1]]
// COMMON-DAG: call i32 @__{{.*}}RegisterFunction({{.*}}@[[STR2]]
// COMMON-DAG: call i32 @__{{.*}}RegisterFunction({{.*}}@[[STR3]]


template <typename T>
__global__ void tempKern(T x) {}

namespace {
  __global__ void kernel() {}
  struct X {};
  X x;
  auto lambda = [](){};
}

void test() {
  kernel<<<1, 1>>>();

  tempKern<<<1, 1>>>(x);

  tempKern<<<1, 1>>>(lambda);
}
