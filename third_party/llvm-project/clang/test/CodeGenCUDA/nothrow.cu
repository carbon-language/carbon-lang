#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -std=c++11 -fcxx-exceptions -fexceptions -fcuda-is-device \
// RUN:   -triple nvptx-nvidia-cuda -emit-llvm -disable-llvm-passes -o - %s | \
// RUN: FileCheck -check-prefix DEVICE %s

// RUN: %clang_cc1 -std=c++11 -fcxx-exceptions -fexceptions \
// RUN:   -triple x86_64-unknown-linux-gnu -emit-llvm -disable-llvm-passes -o - %s | \
// RUN:  FileCheck -check-prefix HOST %s

#include "Inputs/hip/hip_runtime.h"

__host__ __device__ void f();

// HOST: define{{.*}} void @_Z7host_fnv() [[HOST_ATTR:#[0-9]+]]
void host_fn() { f(); }

// DEVICE: define{{.*}} void @_Z3foov() [[DEVICE_ATTR:#[0-9]+]]
__device__ void foo() {
  // DEVICE: call void @_Z1fv
  f();
}

// DEVICE: define{{.*}} void @_Z12foo_noexceptv() [[DEVICE_ATTR:#[0-9]+]]
__device__ void foo_noexcept() noexcept {
  // DEVICE: call void @_Z1fv
  f();
}

// This is nounwind only on the device side.
// CHECK: define{{.*}} void @_Z3foov() [[DEVICE_ATTR:#[0-9]+]]
__host__ __device__ void bar() { f(); }

// DEVICE: define{{.*}} void @_Z3bazv() [[DEVICE_ATTR:#[0-9]+]]
__global__ void baz() { f(); }

// DEVICE: attributes [[DEVICE_ATTR]] = {
// DEVICE-SAME: nounwind
// HOST: attributes [[HOST_ATTR]] = {
// HOST-NOT: nounwind
// HOST-SAME: }
