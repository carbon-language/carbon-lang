// RUN: %clang_cc1 -std=c++14 %s -emit-llvm -o - -triple nvptx \
// RUN:   -fcuda-is-device | FileCheck --check-prefixes=CXX14 %s
// RUN: %clang_cc1 -std=c++17 %s -emit-llvm -o - -triple nvptx \
// RUN:   -fcuda-is-device | FileCheck --check-prefixes=CXX17 %s

#include "Inputs/hip/hip_runtime.h"

// COM: @_ZL1a = internal {{.*}}constant i32 7
constexpr int a = 7;
__constant__ const int &use_a = a;

namespace B {
 // COM: @_ZN1BL1bE = internal {{.*}}constant i32 9
  constexpr int b = 9;
}
__constant__ const int &use_B_b = B::b;

struct Q {
  // CXX14: @_ZN1Q2k2E = {{.*}}externally_initialized constant i32 6
  // CXX17: @_ZN1Q2k2E = internal {{.*}}constant i32 6
  // CXX14: @_ZN1Q2k1E = available_externally {{.*}}constant i32 5
  // CXX17: @_ZN1Q2k1E = {{.*}} externally_initialized constant i32 5
  static constexpr int k1 = 5;
  static constexpr int k2 = 6;
};
constexpr int Q::k2;

__constant__ const int &use_Q_k1 = Q::k1;
__constant__ const int &use_Q_k2 = Q::k2;

template<typename T> struct X {
  // CXX14: @_ZN1XIiE1aE = available_externally {{.*}}constant i32 123
  // CXX17: @_ZN1XIiE1aE = {{.*}}externally_initialized constant i32 123
  static constexpr int a = 123;
};
__constant__ const int &use_X_a = X<int>::a;

template <typename T, T a, T b> struct A {
  // CXX14: @_ZN1AIiLi1ELi2EE1xE = available_externally {{.*}}constant i32 2
  // CXX17: @_ZN1AIiLi1ELi2EE1xE = {{.*}}externally_initialized constant i32 2
  constexpr static T x = a * b;
};
__constant__ const int &y = A<int, 1, 2>::x;
