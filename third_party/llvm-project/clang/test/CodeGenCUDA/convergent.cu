// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// RUN: %clang_cc1 -fcuda-is-device -triple nvptx-nvidia-cuda -emit-llvm \
// RUN:   -disable-llvm-passes -o - %s | FileCheck -allow-deprecated-dag-overlap -check-prefix DEVICE %s

// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -emit-llvm \
// RUN:   -disable-llvm-passes -o - %s | \
// RUN:  FileCheck -allow-deprecated-dag-overlap -check-prefix HOST %s

#include "Inputs/hip/hip_runtime.h"

// DEVICE: Function Attrs:
// DEVICE-SAME: convergent
// DEVICE-NEXT: define{{.*}} void @_Z3foov
__device__ void foo() {}

// HOST: Function Attrs:
// HOST-NOT: convergent
// HOST-NEXT: define{{.*}} void @_Z3barv
// DEVICE: Function Attrs:
// DEVICE-SAME: convergent
// DEVICE-NEXT: define{{.*}} void @_Z3barv
__host__ __device__ void baz();
__host__ __device__ void bar() {
  // DEVICE: call void @_Z3bazv() [[CALL_ATTR:#[0-9]+]]
  baz();
  // DEVICE: call i32 asm "trap;", "=l"() [[ASM_ATTR:#[0-9]+]]
  int x;
  asm ("trap;" : "=l"(x));
  // DEVICE: call void asm sideeffect "trap;", ""() [[ASM_ATTR:#[0-9]+]]
  asm volatile ("trap;");
}

// DEVICE: declare void @_Z3bazv() [[BAZ_ATTR:#[0-9]+]]
// DEVICE: attributes [[BAZ_ATTR]] = {
// DEVICE-SAME: convergent
// DEVICE-SAME: }
// DEVICE-DAG: attributes [[CALL_ATTR]] = { convergent
// DEVICE-DAG: attributes [[ASM_ATTR]] = { convergent

// HOST: declare void @_Z3bazv() [[BAZ_ATTR:#[0-9]+]]
// HOST: attributes [[BAZ_ATTR]] = {
// HOST-NOT: convergent
// HOST-SAME: }
