#include "hip/hip_runtime.h"
// REQUIRES: x86-registered-target
// REQUIRES: amdgpu-registered-target

// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -emit-llvm -x hip %s -o - | FileCheck --check-prefixes=COMMON,CHECK %s
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -emit-llvm -x hip %s -disable-O0-optnone -o - | opt -S -O2 | FileCheck %s --check-prefixes=COMMON,OPT
// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -emit-llvm -x hip %s -o - | FileCheck -check-prefix=HOST %s

#include "Inputs/hip/hip_runtime.h"

// Coerced struct from `struct S` without all generic pointers lowered into
// global ones.

// On the host-side compilation, generic pointer won't be coerced.
// HOST-NOT: %struct.S.coerce
// HOST-NOT: %struct.T.coerce

// HOST: define{{.*}} void @_Z22__device_stub__kernel1Pi(i32* %x)
// COMMON-LABEL: define{{.*}} amdgpu_kernel void @_Z7kernel1Pi(i32 addrspace(1)*{{.*}} %x.coerce)
// CHECK:     ={{.*}} addrspacecast [[TYPE:.*]] addrspace(1)* %{{.*}} to [[TYPE]]*
// CHECK-NOT: ={{.*}} addrspacecast [[TYPE:.*]] addrspace(1)* %{{.*}} to [[TYPE]]*
// OPT: [[VAL:%.*]] = load i32, i32 addrspace(1)* %x.coerce, align 4
// OPT: [[INC:%.*]] = add nsw i32 [[VAL]], 1
// OPT: store i32 [[INC]], i32 addrspace(1)* %x.coerce, align 4
// OPT: ret void
__global__ void kernel1(int *x) {
  x[0]++;
}

// HOST: define{{.*}} void @_Z22__device_stub__kernel2Ri(i32* nonnull align 4 dereferenceable(4) %x)
// COMMON-LABEL: define{{.*}} amdgpu_kernel void @_Z7kernel2Ri(i32 addrspace(1)*{{.*}} nonnull align 4 dereferenceable(4) %x.coerce)
// CHECK:     ={{.*}} addrspacecast [[TYPE:.*]] addrspace(1)* %{{.*}} to [[TYPE]]*
// CHECK-NOT: ={{.*}} addrspacecast [[TYPE:.*]] addrspace(1)* %{{.*}} to [[TYPE]]*
// OPT: [[VAL:%.*]] = load i32, i32 addrspace(1)* %x.coerce, align 4
// OPT: [[INC:%.*]] = add nsw i32 [[VAL]], 1
// OPT: store i32 [[INC]], i32 addrspace(1)* %x.coerce, align 4
// OPT: ret void
__global__ void kernel2(int &x) {
  x++;
}

// HOST: define{{.*}} void @_Z22__device_stub__kernel3PU3AS2iPU3AS1i(i32 addrspace(2)* %x, i32 addrspace(1)* %y)
// CHECK-LABEL: define{{.*}} amdgpu_kernel void  @_Z7kernel3PU3AS2iPU3AS1i(i32 addrspace(2)*{{.*}} %x, i32 addrspace(1)*{{.*}} %y)
// CHECK-NOT: ={{.*}} addrspacecast [[TYPE:.*]] addrspace(1)* %{{.*}} to [[TYPE]]*
__global__ void kernel3(__attribute__((address_space(2))) int *x,
                        __attribute__((address_space(1))) int *y) {
  y[0] = x[0];
}

// COMMON-LABEL: define{{.*}} void @_Z4funcPi(i32*{{.*}} %x)
// CHECK-NOT: ={{.*}} addrspacecast [[TYPE:.*]] addrspace(1)* %{{.*}} to [[TYPE]]*
__device__ void func(int *x) {
  x[0]++;
}

struct S {
  int *x;
  float *y;
};
// `by-val` struct is passed by-indirect-alias (a mix of by-ref and indirect
// by-val). However, the enhanced address inferring pass should be able to
// assume they are global pointers.
//
// HOST: define{{.*}} void @_Z22__device_stub__kernel41S(i32* %s.coerce0, float* %s.coerce1)
// COMMON-LABEL: define{{.*}} amdgpu_kernel void @_Z7kernel41S(%struct.S addrspace(4)*{{.*}} byref(%struct.S) align 8 %0)
// OPT: [[R0:%.*]] = getelementptr inbounds %struct.S, %struct.S addrspace(4)* %0, i64 0, i32 0
// OPT: [[P0:%.*]] = load i32*, i32* addrspace(4)* [[R0]], align 8
// OPT: [[G0:%.*]] ={{.*}} addrspacecast i32* [[P0]] to i32 addrspace(1)*
// OPT: [[R1:%.*]] = getelementptr inbounds %struct.S, %struct.S addrspace(4)* %0, i64 0, i32 1
// OPT: [[P1:%.*]] = load float*, float* addrspace(4)* [[R1]], align 8
// OPT: [[G1:%.*]] ={{.*}} addrspacecast float* [[P1]] to float addrspace(1)*
// OPT: [[V0:%.*]] = load i32, i32 addrspace(1)* [[G0]], align 4
// OPT: [[INC:%.*]] = add nsw i32 [[V0]], 1
// OPT: store i32 [[INC]], i32 addrspace(1)* [[G0]], align 4
// OPT: [[V1:%.*]] = load float, float addrspace(1)* [[G1]], align 4
// OPT: [[ADD:%.*]] = fadd contract float [[V1]], 1.000000e+00
// OPT: store float [[ADD]], float addrspace(1)* [[G1]], align 4
// OPT: ret void
__global__ void kernel4(struct S s) {
  s.x[0]++;
  s.y[0] += 1.f;
}

// If a pointer to struct is passed, only the pointer itself is coerced into the global one.
// HOST: define{{.*}} void @_Z22__device_stub__kernel5P1S(%struct.S* %s)
// COMMON-LABEL: define{{.*}} amdgpu_kernel void @_Z7kernel5P1S(%struct.S addrspace(1)*{{.*}} %s.coerce)
__global__ void kernel5(struct S *s) {
  s->x[0]++;
  s->y[0] += 1.f;
}

struct T {
  float *x[2];
};
// `by-val` array is passed by-indirect-alias (a mix of by-ref and indirect
// by-val). However, the enhanced address inferring pass should be able to
// assume they are global pointers.
//
// HOST: define{{.*}} void @_Z22__device_stub__kernel61T(float* %t.coerce0, float* %t.coerce1)
// COMMON-LABEL: define{{.*}} amdgpu_kernel void @_Z7kernel61T(%struct.T addrspace(4)*{{.*}} byref(%struct.T) align 8 %0)
// OPT: [[R0:%.*]] = getelementptr inbounds %struct.T, %struct.T addrspace(4)* %0, i64 0, i32 0, i64 0
// OPT: [[P0:%.*]] = load float*, float* addrspace(4)* [[R0]], align 8
// OPT: [[G0:%.*]] ={{.*}} addrspacecast float* [[P0]] to float addrspace(1)*
// OPT: [[R1:%.*]] = getelementptr inbounds %struct.T, %struct.T addrspace(4)* %0, i64 0, i32 0, i64 1
// OPT: [[P1:%.*]] = load float*, float* addrspace(4)* [[R1]], align 8
// OPT: [[G1:%.*]] ={{.*}} addrspacecast float* [[P1]] to float addrspace(1)*
// OPT: [[V0:%.*]] = load float, float addrspace(1)* [[G0]], align 4
// OPT: [[ADD0:%.*]] = fadd contract float [[V0]], 1.000000e+00
// OPT: store float [[ADD0]], float addrspace(1)* [[G0]], align 4
// OPT: [[V1:%.*]] = load float, float addrspace(1)* [[G1]], align 4
// OPT: [[ADD1:%.*]] = fadd contract float [[V1]], 2.000000e+00
// OPT: store float [[ADD1]], float addrspace(1)* [[G1]], align 4
// OPT: ret void
__global__ void kernel6(struct T t) {
  t.x[0][0] += 1.f;
  t.x[1][0] += 2.f;
}

// Check that coerced pointers retain the noalias attribute when qualified with __restrict.
// HOST: define{{.*}} void @_Z22__device_stub__kernel7Pi(i32* noalias %x)
// COMMON-LABEL: define{{.*}} amdgpu_kernel void @_Z7kernel7Pi(i32 addrspace(1)* noalias{{.*}} %x.coerce)
__global__ void kernel7(int *__restrict x) {
  x[0]++;
}

// Single element struct.
struct SS {
  float *x;
};
// HOST: define{{.*}} void @_Z22__device_stub__kernel82SS(float* %a.coerce)
// COMMON-LABEL: define{{.*}} amdgpu_kernel void @_Z7kernel82SS(float addrspace(1)*{{.*}} %a.coerce)
// CHECK:     ={{.*}} addrspacecast [[TYPE:.*]] addrspace(1)* %{{.*}} to [[TYPE]]*
// CHECK-NOT: ={{.*}} addrspacecast [[TYPE:.*]] addrspace(1)* %{{.*}} to [[TYPE]]*
// OPT: [[VAL:%.*]] = load float, float addrspace(1)* %a.coerce, align 4
// OPT: [[INC:%.*]] = fadd contract float [[VAL]], 3.000000e+00
// OPT: store float [[INC]], float addrspace(1)* %a.coerce, align 4
// OPT: ret void
__global__ void kernel8(struct SS a) {
  *a.x += 3.f;
}
