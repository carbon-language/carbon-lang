#include "hip/hip_runtime.h"
// Check that when we link a bitcode module into a file using
// -mlink-builtin-bitcode, we apply the same attributes to the functions in that
// bitcode module as we apply to functions we generate.
//
// In particular, we check that ftz and unsafe-math are propagated into the
// bitcode library as appropriate.

// Build the bitcode library.  This is not built in CUDA mode, otherwise it
// might have incompatible attributes.  This mirrors how libdevice is built.
// RUN: %clang_cc1 -x c++ -fconvergent-functions -emit-llvm-bc -DLIB \
// RUN:   %s -o %t.bc -triple nvptx-unknown-unknown

// RUN: %clang_cc1 -x cuda %s -emit-llvm -mlink-builtin-bitcode %t.bc -o - \
// RUN:   -fcuda-is-device -triple nvptx-unknown-unknown \
// RUN: | FileCheck %s --check-prefix=CHECK --check-prefix=NOFTZ --check-prefix=NOFAST

// RUN: %clang_cc1 -x cuda %s -emit-llvm -mlink-builtin-bitcode %t.bc \
// RUN:   -fdenormal-fp-math-f32=preserve-sign -o - \
// RUN:   -fcuda-is-device -triple nvptx-unknown-unknown \
// RUN: | FileCheck %s --check-prefix=CHECK --check-prefix=FTZ \
// RUN:   --check-prefix=NOFAST

// RUN: %clang_cc1 -x cuda %s -emit-llvm -mlink-builtin-bitcode %t.bc \
// RUN:   -fdenormal-fp-math-f32=preserve-sign -o - \
// RUN:   -fcuda-is-device -menable-unsafe-fp-math -triple nvptx-unknown-unknown \
// RUN: | FileCheck %s --check-prefix=CHECK --check-prefix=FAST

// Wrap everything in extern "C" so we don't have to worry about name mangling
// in the IR.
extern "C" {
#ifdef LIB

// This function is defined in the library and only declared in the main
// compilation.
void lib_fn() {}

#else

#include "Inputs/hip/hip_runtime.h"
__device__ void lib_fn();
__global__ void kernel() { lib_fn(); }

#endif
}

// The kernel and lib function should have the same attributes.
// CHECK: define{{.*}} void @kernel() [[kattr:#[0-9]+]]
// CHECK: define internal void @lib_fn() [[fattr:#[0-9]+]]

// FIXME: These -NOT checks do not work as intended and do not check on the same
// line.

// Check the attribute list for kernel.
// CHECK: attributes [[kattr]] = {

// CHECK-SAME: convergent
// CHECK-SAME: norecurse

// FTZ-NOT: "denormal-fp-math"
// FTZ-SAME: "denormal-fp-math-f32"="preserve-sign,preserve-sign"
// NOFTZ-NOT: "denormal-fp-math-f32"

// CHECK-SAME: "no-trapping-math"="true"

// FAST-SAME: "unsafe-fp-math"="true"
// NOFAST-NOT: "unsafe-fp-math"="true"

// Check the attribute list for lib_fn.
// CHECK: attributes [[fattr]] = {

// CHECK-SAME: convergent
// CHECK-NOT: norecurse

// FTZ-NOT: "denormal-fp-math"
// NOFTZ-NOT: "denormal-fp-math"

// FTZ-SAME: "denormal-fp-math-f32"="preserve-sign,preserve-sign"
// NOFTZ-NOT: "denormal-fp-math-f32"

// CHECK-SAME: "no-trapping-math"="true"

// FAST-SAME: "unsafe-fp-math"="true"
// NOFAST-NOT: "unsafe-fp-math"="true"
