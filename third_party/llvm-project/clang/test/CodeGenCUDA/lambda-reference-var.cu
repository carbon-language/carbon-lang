#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -no-opaque-pointers -x hip -emit-llvm -std=c++11 %s -o - \
// RUN:   -triple x86_64-linux-gnu \
// RUN:   | FileCheck -check-prefix=HOST %s
// RUN: %clang_cc1 -no-opaque-pointers -x hip -emit-llvm -std=c++11 %s -o - \
// RUN:   -triple amdgcn-amd-amdhsa -fcuda-is-device \
// RUN:   | FileCheck -check-prefix=DEV %s

#include "Inputs/hip/hip_runtime.h"

// HOST: %[[T1:.*]] = type <{ i32*, i32, [4 x i8] }>
// HOST: %[[T2:.*]] = type { i32*, i32** }
// HOST: %[[T3:.*]] = type <{ i32*, i32, [4 x i8] }>
// DEV: %[[T1:.*]] = type { i32* }
// DEV: %[[T2:.*]] = type { i32** }
// DEV: %[[T3:.*]] = type <{ i32*, i32, [4 x i8] }>
int global_host_var;
__device__ int global_device_var;

template<class F>
__global__ void kern(F f) { f(); }

// DEV-LABEL: @_ZZ27dev_capture_dev_ref_by_copyPiENKUlvE_clEv(
// DEV: %[[VAL:.*]] = load i32, i32* addrspacecast (i32 addrspace(1)* @global_device_var to i32*)
// DEV: store i32 %[[VAL]]
__device__ void dev_capture_dev_ref_by_copy(int *out) {
  int &ref = global_device_var;
  [=](){ *out = ref;}();
}

// DEV-LABEL: @_ZZ28dev_capture_dev_rval_by_copyPiENKUlvE_clEv(
// DEV: store i32 3
__device__ void dev_capture_dev_rval_by_copy(int *out) {
  constexpr int a = 1;
  constexpr int b = 2;
  constexpr int c = a + b;
  [=](){ *out = c;}();
}

// DEV-LABEL: @_ZZ26dev_capture_dev_ref_by_refPiENKUlvE_clEv(
// DEV: %[[VAL:.*]] = load i32, i32* addrspacecast (i32 addrspace(1)* @global_device_var to i32*)
// DEV: %[[VAL2:.*]] = add nsw i32 %[[VAL]], 1
// DEV: store i32 %[[VAL2]], i32* addrspacecast (i32 addrspace(1)* @global_device_var to i32*)
// DEV: %[[VAL:.*]] = load i32, i32* addrspacecast (i32 addrspace(1)* @global_device_var to i32*)
// DEV: store i32 %[[VAL]]
__device__ void dev_capture_dev_ref_by_ref(int *out) {
  int &ref = global_device_var;
  [&](){ ref++; *out = ref;}();
}

// DEV-LABEL: define{{.*}} void @_Z7dev_refPi(
// DEV: %[[VAL:.*]] = load i32, i32* addrspacecast (i32 addrspace(1)* @global_device_var to i32*)
// DEV: %[[VAL2:.*]] = add nsw i32 %[[VAL]], 1
// DEV: store i32 %[[VAL2]], i32* addrspacecast (i32 addrspace(1)* @global_device_var to i32*)
// DEV: %[[VAL:.*]] = load i32, i32* addrspacecast (i32 addrspace(1)* @global_device_var to i32*)
// DEV: store i32 %[[VAL]]
__device__ void dev_ref(int *out) {
  int &ref = global_device_var;
  ref++;
  *out = ref;
}

// DEV-LABEL: @_ZZ14dev_lambda_refPiENKUlvE_clEv(
// DEV: %[[VAL:.*]] = load i32, i32* addrspacecast (i32 addrspace(1)* @global_device_var to i32*)
// DEV: %[[VAL2:.*]] = add nsw i32 %[[VAL]], 1
// DEV: store i32 %[[VAL2]], i32* addrspacecast (i32 addrspace(1)* @global_device_var to i32*)
// DEV: %[[VAL:.*]] = load i32, i32* addrspacecast (i32 addrspace(1)* @global_device_var to i32*)
// DEV: store i32 %[[VAL]]
__device__ void dev_lambda_ref(int *out) {
  [=](){
    int &ref = global_device_var;
    ref++;
    *out = ref;
  }();
}

// HOST-LABEL: @_ZZ29host_capture_host_ref_by_copyPiENKUlvE_clEv(
// HOST: %[[VAL:.*]] = load i32, i32* @global_host_var
// HOST: store i32 %[[VAL]]
void host_capture_host_ref_by_copy(int *out) {
  int &ref = global_host_var;
  [=](){ *out = ref;}();
}

// HOST-LABEL: @_ZZ28host_capture_host_ref_by_refPiENKUlvE_clEv(
// HOST: %[[CAP:.*]] = getelementptr inbounds %[[T2]], %[[T2]]* %this1, i32 0, i32 0
// HOST: %[[REF:.*]] = load i32*, i32** %[[CAP]]
// HOST: %[[VAL:.*]] = load i32, i32* %[[REF]]
// HOST: %[[VAL2:.*]] = add nsw i32 %[[VAL]], 1
// HOST: store i32 %[[VAL2]], i32* %[[REF]]
// HOST: %[[VAL:.*]] = load i32, i32* @global_host_var
// HOST: store i32 %[[VAL]]
void host_capture_host_ref_by_ref(int *out) {
  int &ref = global_host_var;
  [&](){ ref++; *out = ref;}();
}

// HOST-LABEL: define{{.*}} void @_Z8host_refPi(
// HOST: %[[VAL:.*]] = load i32, i32* @global_host_var
// HOST: %[[VAL2:.*]] = add nsw i32 %[[VAL]], 1
// HOST: store i32 %[[VAL2]], i32* @global_host_var
// HOST: %[[VAL:.*]] = load i32, i32* @global_host_var
// HOST: store i32 %[[VAL]]
void host_ref(int *out) {
  int &ref = global_host_var;
  ref++;
  *out = ref;
}

// HOST-LABEL: @_ZZ15host_lambda_refPiENKUlvE_clEv(
// HOST: %[[VAL:.*]] = load i32, i32* @global_host_var
// HOST: %[[VAL2:.*]] = add nsw i32 %[[VAL]], 1
// HOST: store i32 %[[VAL2]], i32* @global_host_var
// HOST: %[[VAL:.*]] = load i32, i32* @global_host_var
// HOST: store i32 %[[VAL]]
void host_lambda_ref(int *out) {
  [=](){
    int &ref = global_host_var;
    ref++;
    *out = ref;
  }();
}

// HOST-LABEL: define{{.*}} void @_Z28dev_capture_host_ref_by_copyPi(
// HOST: %[[CAP:.*]] = getelementptr inbounds %[[T3]], %[[T3]]* %{{.*}}, i32 0, i32 1
// HOST: %[[VAL:.*]] = load i32, i32* @global_host_var
// HOST: store i32 %[[VAL]], i32* %[[CAP]]
// DEV-LABEL: define internal void @_ZZ28dev_capture_host_ref_by_copyPiENKUlvE_clEv(
// DEV: %[[CAP:.*]] = getelementptr inbounds %[[T3]], %[[T3]]* %this1, i32 0, i32 1
// DEV: %[[VAL:.*]] = load i32, i32* %[[CAP]]
// DEV: store i32 %[[VAL]]
void dev_capture_host_ref_by_copy(int *out) {
  int &ref = global_host_var;
  kern<<<1, 1>>>([=]__device__() { *out = ref;});
}

