#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -no-opaque-pointers -x hip -emit-llvm -std=c++11 %s -o - \
// RUN:   -triple x86_64-linux-gnu \
// RUN:   | FileCheck -check-prefix=HOST %s
// RUN: %clang_cc1 -no-opaque-pointers -x hip -emit-llvm -std=c++11 %s -o - \
// RUN:   -triple amdgcn-amd-amdhsa -fcuda-is-device \
// RUN:   | FileCheck -check-prefix=DEV %s

#include "Inputs/hip/hip_runtime.h"

// Device side kernel name.
// HOST: @[[KERN_CAPTURE:[0-9]+]] = {{.*}} c"_Z1gIZ12test_capturevEUlvE_EvT_\00"
// HOST: @[[KERN_RESOLVE:[0-9]+]] = {{.*}} c"_Z1gIZ12test_resolvevEUlvE_EvT_\00"

// Check functions emitted for test_capture in host compilation.
// Check lambda is not emitted in host compilation.
// HOST-LABEL: define{{.*}} void @_Z12test_capturev
// HOST:  call void @_Z19test_capture_helperIZ12test_capturevEUlvE_EvT_
// HOST-LABEL: define internal void @_Z19test_capture_helperIZ12test_capturevEUlvE_EvT_
// HOST:  call void @_Z16__device_stub__gIZ12test_capturevEUlvE_EvT_
// HOST-NOT: define{{.*}}@_ZZ4mainENKUlvE_clEv

// Check functions emitted for test_resolve in host compilation.
// Check host version of template function 'overloaded' is emitted and called
// by the lambda function.
// HOST-LABEL: define{{.*}} void @_Z12test_resolvev
// HOST:  call void @_Z19test_resolve_helperIZ12test_resolvevEUlvE_EvT_()
// HOST-LABEL: define internal void @_Z19test_resolve_helperIZ12test_resolvevEUlvE_EvT_
// HOST:  call void @_Z16__device_stub__gIZ12test_resolvevEUlvE_EvT_
// HOST:  call void @_ZZ12test_resolvevENKUlvE_clEv
// HOST-LABEL: define internal void @_ZZ12test_resolvevENKUlvE_clEv
// HOST:  call noundef i32 @_Z10overloadedIiET_v
// HOST-LABEL: define linkonce_odr noundef i32 @_Z10overloadedIiET_v
// HOST:  ret i32 2

// Check kernel is registered with correct device side kernel name.
// HOST: @__hipRegisterFunction({{.*}}@[[KERN_CAPTURE]]
// HOST: @__hipRegisterFunction({{.*}}@[[KERN_RESOLVE]]

// DEV: @a ={{.*}} addrspace(1) externally_initialized global i32 0

// Check functions emitted for test_capture in device compilation.
// Check lambda is emitted in device compilation and accessing device variable.
// DEV-LABEL: define{{.*}} amdgpu_kernel void @_Z1gIZ12test_capturevEUlvE_EvT_
// DEV:  call void @_ZZ12test_capturevENKUlvE_clEv
// DEV-LABEL: define internal void @_ZZ12test_capturevENKUlvE_clEv
// DEV:  store i32 1, i32* addrspacecast (i32 addrspace(1)* @a to i32*)

// Check functions emitted for test_resolve in device compilation.
// Check device version of template function 'overloaded' is emitted and called
// by the lambda function.
// DEV-LABEL: define{{.*}} amdgpu_kernel void @_Z1gIZ12test_resolvevEUlvE_EvT_
// DEV:  call void @_ZZ12test_resolvevENKUlvE_clEv
// DEV-LABEL: define internal void @_ZZ12test_resolvevENKUlvE_clEv
// DEV:  call noundef i32 @_Z10overloadedIiET_v
// DEV-LABEL: define linkonce_odr noundef i32 @_Z10overloadedIiET_v
// DEV:  ret i32 1

__device__ int a;

template<class T>
__device__ T overloaded() { return 1; }

template<class T>
__host__ T overloaded() { return 2; }

template<class F>
__global__ void g(F f) { f(); }

template<class F>
void test_capture_helper(F f) { g<<<1,1>>>(f); }

template<class F>
void test_resolve_helper(F f) { g<<<1,1>>>(f); f(); }

// Test capture of device variable in lambda function.
void test_capture(void) {
  test_capture_helper([](){ a = 1;});
}

// Test resolving host/device function in lambda function.
// Callee should resolve to correct host/device function based on where
// the lambda function is called, not where it is defined.
void test_resolve(void) {
  test_resolve_helper([](){ overloaded<int>();});
}
