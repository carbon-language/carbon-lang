#include "hip/hip_runtime.h"
// Make sure that __global__ functions are emitted along with correct
// annotations and are added to @llvm.used to prevent their elimination.
// REQUIRES: nvptx-registered-target
//
// RUN: %clang_cc1 -no-opaque-pointers %s -triple nvptx-unknown-unknown -fcuda-is-device -emit-llvm -o - | FileCheck %s

#include "Inputs/hip/hip_runtime.h"

// CHECK-LABEL: define{{.*}} void @device_function
extern "C"
__device__ void device_function() {}

// CHECK-LABEL: define{{.*}} void @global_function
extern "C"
__global__ void global_function() {
  // CHECK: call void @device_function
  device_function();
}

// Make sure host-instantiated kernels are preserved on device side.
template <typename T> __global__ void templated_kernel(T param) {}
// CHECK-DAG: define{{.*}} void @_Z16templated_kernelIiEvT_(

namespace {
__global__ void anonymous_ns_kernel() {}
// CHECK-DAG: define{{.*}} void @_ZN12_GLOBAL__N_119anonymous_ns_kernelEv(
}

void host_function() {
  templated_kernel<<<0, 0>>>(0);
  anonymous_ns_kernel<<<0,0>>>();
}

// CHECK: !{{[0-9]+}} = !{void ()* @global_function, !"kernel", i32 1}
// CHECK: !{{[0-9]+}} = !{void (i32)* @_Z16templated_kernelIiEvT_, !"kernel", i32 1}
