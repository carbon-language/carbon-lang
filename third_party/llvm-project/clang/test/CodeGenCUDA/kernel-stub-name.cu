#include "hip/hip_runtime.h"
// RUN: echo "GPU binary would be here" > %t

// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s \
// RUN:     -fcuda-include-gpubinary %t -o - -x hip\
// RUN:   | FileCheck -check-prefixes=CHECK,GNU %s

// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s \
// RUN:     -fcuda-include-gpubinary %t -o - -x hip\
// RUN:   | FileCheck -check-prefix=NEG %s

// RUN: %clang_cc1 -triple x86_64-pc-windows-msvc -emit-llvm %s \
// RUN:     -aux-triple amdgcn-amd-amdhsa -fcuda-include-gpubinary \
// RUN:     %t -o - -x hip\
// RUN:   | FileCheck -check-prefixes=CHECK,MSVC %s

// RUN: %clang_cc1 -triple x86_64-pc-windows-msvc -emit-llvm %s \
// RUN:     -aux-triple amdgcn-amd-amdhsa -fcuda-include-gpubinary \
// RUN:     %t -o - -x hip\
// RUN:   | FileCheck -check-prefix=NEG %s

#include "Inputs/hip/hip_runtime.h"

// Check kernel handles are emitted for non-MSVC target but not for MSVC target.

// GNU: @[[HCKERN:ckernel]] = constant void ()* @[[CSTUB:__device_stub__ckernel]], align 8
// GNU: @[[HNSKERN:_ZN2ns8nskernelEv]] = constant void ()* @[[NSSTUB:_ZN2ns23__device_stub__nskernelEv]], align 8
// GNU: @[[HTKERN:_Z10kernelfuncIiEvv]] = linkonce_odr constant void ()* @[[TSTUB:_Z25__device_stub__kernelfuncIiEvv]], comdat, align 8
// GNU: @[[HDKERN:_Z11kernel_declv]] = external constant void ()*, align 8

// MSVC: @[[HCKERN:ckernel]] = dso_local constant void ()* @[[CSTUB:__device_stub__ckernel]], align 8
// MSVC: @[[HNSKERN:"\?nskernel@ns@@YAXXZ.*"]] = dso_local constant void ()* @[[NSSTUB:"\?__device_stub__nskernel@ns@@YAXXZ"]], align 8
// MSVC: @[[HTKERN:"\?\?\$kernelfunc@H@@YAXXZ.*"]] = linkonce_odr dso_local constant void ()* @[[TSTUB:"\?\?\$__device_stub__kernelfunc@H@@YAXXZ.*"]], comdat, align 8
// MSVC: @[[HDKERN:"\?kernel_decl@@YAXXZ.*"]] = external dso_local constant void ()*, align 8

extern "C" __global__ void ckernel() {}

namespace ns {
__global__ void nskernel() {}
} // namespace ns

template<class T>
__global__ void kernelfunc() {}

__global__ void kernel_decl();

extern "C" void (*kernel_ptr)();
extern "C" void *void_ptr;

extern "C" void launch(void *kern);

// Device side kernel names

// CHECK: @[[CKERN:[0-9]*]] = {{.*}} c"ckernel\00"
// CHECK: @[[NSKERN:[0-9]*]] = {{.*}} c"_ZN2ns8nskernelEv\00"
// CHECK: @[[TKERN:[0-9]*]] = {{.*}} c"_Z10kernelfuncIiEvv\00"

// Non-template kernel stub functions

// CHECK: define{{.*}}@[[CSTUB]]
// CHECK: call{{.*}}@hipLaunchByPtr{{.*}}@[[HCKERN]]

// CHECK: define{{.*}}@[[NSSTUB]]
// CHECK: call{{.*}}@hipLaunchByPtr{{.*}}@[[HNSKERN]]

// Check kernel stub is called for triple chevron.

// CHECK-LABEL: define{{.*}}@fun1()
// CHECK: call void @[[CSTUB]]()
// CHECK: call void @[[NSSTUB]]()
// CHECK: call void @[[TSTUB]]()
// GNU: call void @[[DSTUB:_Z26__device_stub__kernel_declv]]()
// MSVC: call void @[[DSTUB:"\?__device_stub__kernel_decl@@YAXXZ"]]()

extern "C" void fun1(void) {
  ckernel<<<1, 1>>>();
  ns::nskernel<<<1, 1>>>();
  kernelfunc<int><<<1, 1>>>();
  kernel_decl<<<1, 1>>>();
}

// Template kernel stub functions

// CHECK: define{{.*}}@[[TSTUB]]
// CHECK: call{{.*}}@hipLaunchByPtr{{.*}}@[[HTKERN]]

// Check declaration of stub function for external kernel.

// CHECK: declare{{.*}}@[[DSTUB]]

// Check kernel handle is used for passing the kernel as a function pointer.

// CHECK-LABEL: define{{.*}}@fun2()
// CHECK: call void @launch({{.*}}[[HCKERN]]
// CHECK: call void @launch({{.*}}[[HNSKERN]]
// CHECK: call void @launch({{.*}}[[HTKERN]]
// CHECK: call void @launch({{.*}}[[HDKERN]]
extern "C" void fun2() {
  launch((void *)ckernel);
  launch((void *)ns::nskernel);
  launch((void *)kernelfunc<int>);
  launch((void *)kernel_decl);
}

// Check kernel handle is used for assigning a kernel to a function pointer.

// CHECK-LABEL: define{{.*}}@fun3()
// CHECK:  store void ()* bitcast (void ()** @[[HCKERN]] to void ()*), void ()** @kernel_ptr, align 8
// CHECK:  store void ()* bitcast (void ()** @[[HCKERN]] to void ()*), void ()** @kernel_ptr, align 8
// CHECK:  store i8* bitcast (void ()** @[[HCKERN]] to i8*), i8** @void_ptr, align 8
// CHECK:  store i8* bitcast (void ()** @[[HCKERN]] to i8*), i8** @void_ptr, align 8
extern "C" void fun3() {
  kernel_ptr = ckernel;
  kernel_ptr = &ckernel;
  void_ptr = (void *)ckernel;
  void_ptr = (void *)&ckernel;
}

// Check kernel stub is loaded from kernel handle when function pointer is
// used with triple chevron.

// CHECK-LABEL: define{{.*}}@fun4()
// CHECK:  store void ()* bitcast (void ()** @[[HCKERN]] to void ()*), void ()** @kernel_ptr
// CHECK:  call i32 @{{.*hipConfigureCall}}
// CHECK:  %[[HANDLE:.*]] = load void ()*, void ()** @kernel_ptr, align 8
// CHECK:  %[[CAST:.*]] = bitcast void ()* %[[HANDLE]] to void ()**
// CHECK:  %[[STUB:.*]] = load void ()*, void ()** %[[CAST]], align 8
// CHECK:  call void %[[STUB]]()
extern "C" void fun4() {
  kernel_ptr = ckernel;
  kernel_ptr<<<1,1>>>();
}

// Check kernel handle is passed to a function.

// CHECK-LABEL: define{{.*}}@fun5()
// CHECK:  store void ()* bitcast (void ()** @[[HCKERN]] to void ()*), void ()** @kernel_ptr
// CHECK:  %[[HANDLE:.*]] = load void ()*, void ()** @kernel_ptr, align 8
// CHECK:  %[[CAST:.*]] = bitcast void ()* %[[HANDLE]] to i8*
// CHECK:  call void @launch(i8* %[[CAST]])
extern "C" void fun5() {
  kernel_ptr = ckernel;
  launch((void *)kernel_ptr);
}

// Check kernel handle is registered.

// CHECK-LABEL: define{{.*}}@__hip_register_globals
// CHECK: call{{.*}}@__hipRegisterFunction{{.*}}@[[HCKERN]]{{.*}}@[[CKERN]]
// CHECK: call{{.*}}@__hipRegisterFunction{{.*}}@[[HNSKERN]]{{.*}}@[[NSKERN]]
// CHECK: call{{.*}}@__hipRegisterFunction{{.*}}@[[HTKERN]]{{.*}}@[[TKERN]]
// NEG-NOT: call{{.*}}@__hipRegisterFunction{{.*}}__device_stub
// NEG-NOT: call{{.*}}@__hipRegisterFunction{{.*}}kernel_decl
