
#include <hip/hip_runtime.h>
// REQUIRES: x86-registered-target
// REQUIRES: amdgpu-registered-target
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -emit-llvm -o - -fcuda-is-device -x hip %s | FileCheck --check-prefix=DEV %s
// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm -o - -x hip %s | FileCheck --check-prefix=HOST %s

// DEV-NOT: llvm.dependent-libraries
// HOST: llvm.dependent-libraries
#pragma comment(lib, "libabc")
