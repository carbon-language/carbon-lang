// REQUIRES: amdgpu-registered-target
// RUN: %clang_cc1 -no-opaque-pointers -triple amdgcn-amd-amdhsa -fcuda-is-device -x hip %s \
// RUN:   -emit-llvm -o - | FileCheck -check-prefix=DEV %s
// RUN: %clang_cc1 -no-opaque-pointers -triple x86_64-unknown-linux-gnu -x hip %s \
// RUN:   -emit-llvm -o - | FileCheck -check-prefix=HOST %s

// Negative tests.

// RUN: %clang_cc1 -no-opaque-pointers -triple amdgcn-amd-amdhsa -fcuda-is-device -x hip %s \
// RUN:   -emit-llvm -o - | FileCheck -check-prefix=DEV-NEG %s

#include "Inputs/hip/hip_runtime.h"

// Test const var initialized with address of a const var.
// Both are promoted to device side.

// DEV-DAG: @_ZN5Test1L1aE = internal addrspace(4) constant i32 1
// DEV-DAG: @_ZN5Test11B2p1E = addrspace(4) externally_initialized constant i32* addrspacecast (i32 addrspace(4)* @_ZN5Test1L1aE to i32*)
// DEV-DAG: @_ZN5Test11B2p2E = addrspace(4) externally_initialized constant i32* addrspacecast (i32 addrspace(4)* @_ZN5Test1L1aE to i32*)
// DEV-DAG: @_ZN5Test12b2E = addrspace(1) externally_initialized global i32 1
// HOST-DAG: @_ZN5Test1L1aE = internal constant i32 1
// HOST-DAG: @_ZN5Test11B2p1E = constant i32* @_ZN5Test1L1aE
// HOST-DAG: @_ZN5Test11B2p2E = internal constant i32* undef
// HOST-DAG: @_ZN5Test12b1E = global i32 1
// HOST-DAG: @_ZN5Test12b2E = internal global i32 undef
namespace Test1 {
const int a = 1;

struct B {
    static const int *const p1;
    static __device__ const int *const p2;
};
const int *const B::p1 = &a;
__device__ const int *const B::p2 = &a;
int b1 = B::p1 == B::p2;
__device__ int b2 = B::p1 == B::p2;
}

// Test const var initialized with address of a non-cost var.
// Neither is promoted to device side.

// DEV-NEG-NOT: @_ZN5Test2L1aE
// DEV-NEG-NOT: @_ZN5Test21B1pE
// HOST-DAG: @_ZN5Test21aE = global i32 1
// HOST-DAG: @_ZN5Test21B1pE = constant i32* @_ZN5Test21aE

namespace Test2 {
int a = 1;

struct B {
    static int *const p;
};
int *const B::p = &a;
}
