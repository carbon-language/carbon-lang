#include "hip/hip_runtime.h"
// RUN: %clang_cc1 %s -emit-llvm -o - -triple -amdgcn-amd-amdhsa \
// RUN:  -target-cpu gfx906 -fcuda-is-device -x hip \
// RUN:  | FileCheck --check-prefixes=COMMON,CRDIV %s
// RUN: %clang_cc1 %s -emit-llvm -o - -triple -amdgcn-amd-amdhsa \
// RUN:  -target-cpu gfx906 -fcuda-is-device -x hip \
// RUN:  -fno-hip-fp32-correctly-rounded-divide-sqrt \
// RUN:  | FileCheck --check-prefixes=COMMON,NCRDIV %s

#include "Inputs/hip/hip_runtime.h"

typedef __attribute__(( ext_vector_type(4) )) float float4;

// COMMON-LABEL: @_Z11spscalardiv
// COMMON: fdiv{{.*}},
// NCRDIV: !fpmath ![[MD:[0-9]+]]
// CRDIV-NOT: !fpmath
__device__ float spscalardiv(float a, float b) {
  return a / b;
}

// COMMON-LABEL: @_Z11spvectordiv
// COMMON: fdiv{{.*}},
// NCRDIV: !fpmath ![[MD]]
// CRDIV-NOT: !fpmath
__device__ float4 spvectordiv(float4 a, float4 b) {
  return a / b;
}

// COMMON-LABEL: @_Z11dpscalardiv
// COMMON-NOT: !fpmath
__device__ double dpscalardiv(double a, double b) {
  return a / b;
}

// NCRDIV: ![[MD]] = !{float 2.500000e+00}
