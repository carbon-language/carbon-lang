#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -target-cpu gfx906 \
// RUN:   -emit-llvm -o - %s | FileCheck %s

#include "Inputs/hip/hip_runtime.h"

// CHECK-LABEL: define {{.*}}@_ZN1AC2Ev(%struct.A* nonnull align 8 dereferenceable(8) %this)
// CHECK: store %struct.A* %this, %struct.A** %this.addr.ascast
// CHECK: %this1 = load %struct.A*, %struct.A** %this.addr.ascast
// CHECK: %[[VTFIELD:.*]] = bitcast %struct.A* %this1 to i32 (...)* addrspace(1)**
// CHECK: store i32 (...)* addrspace(1)* bitcast{{.*}} @_ZTV1A{{.*}}, i32 (...)* addrspace(1)** %[[VTFIELD]]
struct A {
  __device__ virtual void vf() {}
};

__global__ void kern() {
  A a;
}
