#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -no-opaque-pointers %s -triple nvptx-unknown-unknown -fcuda-is-device -emit-llvm -o - | FileCheck %s

#include "Inputs/hip/hip_runtime.h"

#define MAX_THREADS_PER_BLOCK 256
#define MIN_BLOCKS_PER_MP     2

// Test both max threads per block and Min cta per sm.
extern "C" {
__global__ void
__launch_bounds__( MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP )
Kernel1()
{
}
}

// CHECK: !{{[0-9]+}} = !{void ()* @Kernel1, !"maxntidx", i32 256}
// CHECK: !{{[0-9]+}} = !{void ()* @Kernel1, !"minctasm", i32 2}

// Test only max threads per block. Min cta per sm defaults to 0, and
// CodeGen doesn't output a zero value for minctasm.
extern "C" {
__global__ void
__launch_bounds__( MAX_THREADS_PER_BLOCK )
Kernel2()
{
}
}

// CHECK: !{{[0-9]+}} = !{void ()* @Kernel2, !"maxntidx", i32 256}

template <int max_threads_per_block>
__global__ void
__launch_bounds__(max_threads_per_block)
Kernel3()
{
}

template __global__ void Kernel3<MAX_THREADS_PER_BLOCK>();
// CHECK: !{{[0-9]+}} = !{void ()* @{{.*}}Kernel3{{.*}}, !"maxntidx", i32 256}

template <int max_threads_per_block, int min_blocks_per_mp>
__global__ void
__launch_bounds__(max_threads_per_block, min_blocks_per_mp)
Kernel4()
{
}
template __global__ void Kernel4<MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP>();

// CHECK: !{{[0-9]+}} = !{void ()* @{{.*}}Kernel4{{.*}}, !"maxntidx", i32 256}
// CHECK: !{{[0-9]+}} = !{void ()* @{{.*}}Kernel4{{.*}}, !"minctasm", i32 2}

const int constint = 100;
template <int max_threads_per_block, int min_blocks_per_mp>
__global__ void
__launch_bounds__(max_threads_per_block + constint,
                  min_blocks_per_mp + max_threads_per_block)
Kernel5()
{
}
template __global__ void Kernel5<MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP>();

// CHECK: !{{[0-9]+}} = !{void ()* @{{.*}}Kernel5{{.*}}, !"maxntidx", i32 356}
// CHECK: !{{[0-9]+}} = !{void ()* @{{.*}}Kernel5{{.*}}, !"minctasm", i32 258}

// Make sure we don't emit negative launch bounds values.
__global__ void
__launch_bounds__( -MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP )
Kernel6()
{
}
// CHECK-NOT: !{{[0-9]+}} = !{void ()* @{{.*}}Kernel6{{.*}}, !"maxntidx",
// CHECK:     !{{[0-9]+}} = !{void ()* @{{.*}}Kernel6{{.*}}, !"minctasm",

__global__ void
__launch_bounds__( MAX_THREADS_PER_BLOCK, -MIN_BLOCKS_PER_MP )
Kernel7()
{
}
// CHECK:     !{{[0-9]+}} = !{void ()* @{{.*}}Kernel7{{.*}}, !"maxntidx",
// CHECK-NOT: !{{[0-9]+}} = !{void ()* @{{.*}}Kernel7{{.*}}, !"minctasm",

const char constchar = 12;
__global__ void __launch_bounds__(constint, constchar) Kernel8() {}
// CHECK:     !{{[0-9]+}} = !{void ()* @{{.*}}Kernel8{{.*}}, !"maxntidx", i32 100
// CHECK:     !{{[0-9]+}} = !{void ()* @{{.*}}Kernel8{{.*}}, !"minctasm", i32 12
