#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -no-opaque-pointers -emit-llvm %s -o - -fcuda-is-device -triple nvptx-unknown-unknown -debug-info-kind=limited -dwarf-version=2 -debugger-tuning=gdb | FileCheck %s

#include "Inputs/hip/hip_runtime.h"

// CHECK-DAG: ![[FILEVAR0:[0-9]+]] = distinct !DIGlobalVariable(name: "FileVar0", scope: !{{[0-9]+}}, file: !{{[0-9]+}}, line: {{[0-9]+}}, type: !{{[0-9]+}}, isLocal: false, isDefinition: true)
// CHECK-DAG: !DIGlobalVariableExpression(var: ![[FILEVAR0]], expr: !DIExpression())
__device__ int FileVar0;
// CHECK-DAG: ![[FILEVAR1:[0-9]+]] = distinct !DIGlobalVariable(name: "FileVar1", scope: !{{[0-9]+}}, file: !{{[0-9]+}}, line: {{[0-9]+}}, type: !{{[0-9]+}}, isLocal: false, isDefinition: true)
// CHECK-DAG: !DIGlobalVariableExpression(var: ![[FILEVAR1]], expr: !DIExpression(DW_OP_constu, 8, DW_OP_swap, DW_OP_xderef))
__device__ __shared__ int FileVar1;
// CHECK-DAG: ![[FILEVAR2:[0-9]+]] = distinct !DIGlobalVariable(name: "FileVar2", scope: !{{[0-9]+}}, file: !{{[0-9]+}}, line: {{[0-9]+}}, type: !{{[0-9]+}}, isLocal: false, isDefinition: true)
// CHECK-DAG: !DIGlobalVariableExpression(var: ![[FILEVAR2]], expr: !DIExpression(DW_OP_constu, 4, DW_OP_swap, DW_OP_xderef))
__device__ __constant__ int FileVar2;

__device__ void kernel1(
    // CHECK-DAG: ![[ARG:[0-9]+]] = !DILocalVariable(name: "Arg", arg: {{[0-9]+}}, scope: !{{[0-9]+}}, file: !{{[0-9]+}}, line: {{[0-9]+}}, type: !{{[0-9]+}})
    // CHECK-DAG: call void @llvm.dbg.declare(metadata i32* {{.*}}, metadata ![[ARG]], metadata !DIExpression()), !dbg !{{[0-9]+}}
    int Arg) {
    // CHECK-DAG: ![[FUNCVAR0:[0-9]+]] = distinct !DIGlobalVariable(name: "FuncVar0", scope: !{{[0-9]+}}, file: !{{[0-9]+}}, line: {{[0-9]+}}, type: !{{[0-9]+}}, isLocal: true, isDefinition: true)
    // CHECK-DAG: !DIGlobalVariableExpression(var: ![[FUNCVAR0]], expr: !DIExpression(DW_OP_constu, 8, DW_OP_swap, DW_OP_xderef))
  __shared__ int FuncVar0;
  // CHECK-DAG: ![[FUNCVAR1:[0-9]+]] = !DILocalVariable(name: "FuncVar1", scope: !{{[0-9]+}}, file: !{{[0-9]+}}, line: {{[0-9]+}}, type: !{{[0-9]+}})
  // CHECK-DAG: call void @llvm.dbg.declare(metadata i32* {{.*}}, metadata ![[FUNCVAR1]], metadata !DIExpression()), !dbg !{{[0-9]+}}
  int FuncVar1;
}
