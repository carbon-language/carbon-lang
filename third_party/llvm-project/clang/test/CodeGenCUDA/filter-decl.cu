#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -triple %itanium_abi_triple -emit-llvm %s -o - | FileCheck -check-prefix=CHECK-HOST %s
// RUN: %clang_cc1 -triple %itanium_abi_triple -emit-llvm %s -o - -fcuda-is-device | FileCheck -check-prefix=CHECK-DEVICE %s

#include "Inputs/hip/hip_runtime.h"

// This has to be at the top of the file as that's where file-scope
// asm ends up.
// CHECK-HOST: module asm "file scope asm is host only"
// CHECK-DEVICE-NOT: module asm "file scope asm is host only"
__asm__("file scope asm is host only");

// CHECK-HOST: constantdata = internal global
// CHECK-DEVICE: constantdata = {{(dso_local )?}}externally_initialized global
__constant__ char constantdata[256];

// CHECK-HOST: devicedata = internal global
// CHECK-DEVICE: devicedata = {{(dso_local )?}}externally_initialized global
__device__ char devicedata[256];

// CHECK-HOST: shareddata = internal global
// CHECK-DEVICE: shareddata = {{(dso_local )?}}global
__shared__ char shareddata[256];

// CHECK-HOST: hostdata = {{(dso_local )?}}global
// CHECK-DEVICE-NOT: hostdata = global
char hostdata[256];

// CHECK-HOST: define{{.*}}implicithostonlyfunc
// CHECK-DEVICE-NOT: define{{.*}}implicithostonlyfunc
void implicithostonlyfunc(void) {}

// CHECK-HOST: define{{.*}}explicithostonlyfunc
// CHECK-DEVICE-NOT: define{{.*}}explicithostonlyfunc
__host__ void explicithostonlyfunc(void) {}

// CHECK-HOST-NOT: define{{.*}}deviceonlyfunc
// CHECK-DEVICE: define{{.*}}deviceonlyfunc
__device__ void deviceonlyfunc(void) {}

// CHECK-HOST: define{{.*}}hostdevicefunc
// CHECK-DEVICE: define{{.*}}hostdevicefunc
__host__  __device__ void hostdevicefunc(void) {}

// CHECK-HOST: define{{.*}}globalfunc
// CHECK-DEVICE: define{{.*}}globalfunc
__global__ void globalfunc(void) {}
