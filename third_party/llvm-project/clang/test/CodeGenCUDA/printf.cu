// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// RUN: %clang_cc1 -no-opaque-pointers -triple nvptx64-nvidia-cuda -fcuda-is-device -emit-llvm \
// RUN:   -o - %s | FileCheck %s

#include "Inputs/hip/hip_runtime.h"

extern "C" __device__ int vprintf(const char*, const char*);

// Check a simple call to printf end-to-end.
// CHECK: [[SIMPLE_PRINTF_TY:%[a-zA-Z0-9_]+]] = type { i32, i64, double }
__device__ int CheckSimple() {
  // CHECK: [[BUF:%[a-zA-Z0-9_]+]] = alloca [[SIMPLE_PRINTF_TY]]
  // CHECK: [[FMT:%[0-9]+]] = load{{.*}}%fmt
  const char* fmt = "%d %lld %f";
  // CHECK: [[PTR0:%[0-9]+]] = getelementptr inbounds [[SIMPLE_PRINTF_TY]], [[SIMPLE_PRINTF_TY]]* [[BUF]], i32 0, i32 0
  // CHECK: store i32 1, i32* [[PTR0]], align 4
  // CHECK: [[PTR1:%[0-9]+]] = getelementptr inbounds [[SIMPLE_PRINTF_TY]], [[SIMPLE_PRINTF_TY]]* [[BUF]], i32 0, i32 1
  // CHECK: store i64 2, i64* [[PTR1]], align 8
  // CHECK: [[PTR2:%[0-9]+]] = getelementptr inbounds [[SIMPLE_PRINTF_TY]], [[SIMPLE_PRINTF_TY]]* [[BUF]], i32 0, i32 2
  // CHECK: store double 3.0{{[^,]*}}, double* [[PTR2]], align 8
  // CHECK: [[BUF_CAST:%[0-9]+]] = bitcast [[SIMPLE_PRINTF_TY]]* [[BUF]] to i8*
  // CHECK: [[RET:%[0-9]+]] = call i32 @vprintf(i8* [[FMT]], i8* [[BUF_CAST]])
  // CHECK: ret i32 [[RET]]
  return printf(fmt, 1, 2ll, 3.0);
}

__device__ void CheckNoArgs() {
  // CHECK: call i32 @vprintf({{.*}}, i8* null){{$}}
  printf("hello, world!");
}

// Check that printf's alloca happens in the entry block, not inside the if
// statement.
__device__ bool foo();
__device__ void CheckAllocaIsInEntryBlock() {
  // CHECK: alloca %printf_args
  // CHECK: call {{.*}} @_Z3foov()
  if (foo()) {
    printf("%d", 42);
  }
}
