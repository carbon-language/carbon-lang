// RUN: %clang_cc1 -no-opaque-pointers -triple amdgcn -fcuda-is-device \
// RUN:   -emit-llvm -o - -x hip %s \
// RUN:   | FileCheck -check-prefixes=DEV,NORDC %s
// RUN: %clang_cc1 -no-opaque-pointers -triple amdgcn -fcuda-is-device \
// RUN:   -fgpu-rdc -cuid=abc -emit-llvm -o - -x hip %s \
// RUN:   | FileCheck -check-prefixes=DEV,RDC %s
// RUN: %clang_cc1 -no-opaque-pointers -triple x86_64-unknown-gnu-linux \
// RUN:   -emit-llvm -o - -x hip %s \
// RUN:   | FileCheck -check-prefixes=HOST,NORDC-H %s
// RUN: %clang_cc1 -no-opaque-pointers -triple x86_64-unknown-gnu-linux \
// RUN:   -fgpu-rdc -cuid=abc -emit-llvm -o - -x hip %s \
// RUN:   | FileCheck -check-prefixes=HOST,RDC-H %s
// RUN: %clang_cc1 -no-opaque-pointers -triple nvptx -fcuda-is-device \
// RUN:   -fgpu-rdc -cuid=abc -emit-llvm -o - %s \
// RUN:   | FileCheck -check-prefixes=CUDA %s

#include "Inputs/hip/hip_runtime.h"

// DEV-DAG: @v1 = addrspace(1) externally_initialized global i32 0
// NORDC-H-DAG: @v1 = internal global i32 undef
// RDC-H-DAG: @v1 = global i32 undef
__device__ int v1;
// DEV-DAG: @v2 = addrspace(4) externally_initialized global i32 0
// NORDC-H-DAG: @v2 = internal global i32 undef
// RDC-H-DAG: @v2 = global i32 undef
__constant__ int v2;
// DEV-DAG: @v3 = addrspace(1) externally_initialized global i32 addrspace(1)* null
// NORDC-H-DAG: @v3 = internal externally_initialized global i32* null
// RDC-H-DAG: @v3 = externally_initialized global i32* null
#if __HIP__
__managed__ int v3;
#endif

// DEV-DAG: @ev1 = external addrspace(1) global i32
// HOST-DAG: @ev1 = external global i32
extern __device__ int ev1;
// DEV-DAG: @ev2 = external addrspace(4) global i32
// HOST-DAG: @ev2 = external global i32
extern __constant__ int ev2;
// DEV-DAG: @ev3 = external addrspace(1) externally_initialized global i32 addrspace(1)*
// HOST-DAG: @ev3 = external externally_initialized global i32*
#if __HIP__
extern __managed__ int ev3;
#endif

// NORDC-DAG: @_ZL3sv1 = addrspace(1) externally_initialized global i32 0
// RDC-DAG: @_ZL3sv1.static.[[HASH:.*]] = addrspace(1) externally_initialized global i32 0
// HOST-DAG: @_ZL3sv1 = internal global i32 undef
// CUDA-DAG: @_ZL3sv1__static__[[HASH:.*]] = addrspace(1) externally_initialized global i32 0
static __device__ int sv1;
// NORDC-DAG: @_ZL3sv2 = addrspace(4) externally_initialized global i32 0
// RDC-DAG: @_ZL3sv2.static.[[HASH]] = addrspace(4) externally_initialized global i32 0
// HOST-DAG: @_ZL3sv2 = internal global i32 undef
// CUDA-DAG: @_ZL3sv2__static__[[HASH]] = addrspace(4) externally_initialized global i32 0
static __constant__ int sv2;
// NORDC-DAG: @_ZL3sv3 = addrspace(1) externally_initialized global i32 addrspace(1)* null
// RDC-DAG: @_ZL3sv3.static.[[HASH]] = addrspace(1) externally_initialized global i32 addrspace(1)* null
// HOST-DAG: @_ZL3sv3 = internal externally_initialized global i32* null
#if __HIP__
static __managed__ int sv3;
#endif

__device__ __host__ int work(int *x);

__device__ __host__ int fun1() {
  return work(&ev1) + work(&ev2) + work(&sv1) + work(&sv2)
#if __HIP__
    + work(&ev3) + work(&sv3)
#endif
    ;
}

// HOST: hipRegisterVar({{.*}}@v1
// HOST: hipRegisterVar({{.*}}@v2
// HOST: hipRegisterManagedVar({{.*}}@v3
// HOST-NOT: hipRegisterVar({{.*}}@ev1
// HOST-NOT: hipRegisterVar({{.*}}@ev2
// HOST-NOT: hipRegisterManagedVar({{.*}}@ev3
// HOST: hipRegisterVar({{.*}}@_ZL3sv1
// HOST: hipRegisterVar({{.*}}@_ZL3sv2
// HOST: hipRegisterManagedVar({{.*}}@_ZL3sv3
