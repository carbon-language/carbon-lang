#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -x hip -triple amdgcn-amd-amdhsa -fcuda-is-device \
// RUN:     -emit-llvm %s -o - | FileCheck -check-prefix=AMDGCN %s
// RUN: %clang_cc1 -x cuda -triple nvptx64-nvidia-cuda- -fcuda-is-device \
// RUN:     -emit-llvm %s -o - | FileCheck -check-prefix=NVPTX %s
#include "Inputs/hip/hip_runtime.h"

struct A {
  int a[32];
  float *p;
};

// AMDGCN: define{{.*}} amdgpu_kernel void @_Z6kernel1A(%struct.A addrspace(4)* byref(%struct.A) align 8 %{{.+}})
// NVPTX: define{{.*}} void @_Z6kernel1A(%struct.A* noundef byval(%struct.A) align 8 %x)
__global__ void kernel(A x) {
}

class Kernel {
public:
  // AMDGCN: define{{.*}} amdgpu_kernel void @_ZN6Kernel12memberKernelE1A(%struct.A addrspace(4)* byref(%struct.A) align 8 %{{.+}})
  // NVPTX: define{{.*}} void @_ZN6Kernel12memberKernelE1A(%struct.A* noundef byval(%struct.A) align 8 %x)
  static __global__ void memberKernel(A x){}
  template<typename T> static __global__ void templateMemberKernel(T x) {}
};


template <typename T>
__global__ void templateKernel(T x) {}

void launch(void*);

void test() {
  Kernel K;
  // AMDGCN: define{{.*}} amdgpu_kernel void @_Z14templateKernelI1AEvT_(%struct.A addrspace(4)* byref(%struct.A) align 8 %{{.+}}
  // NVPTX: define{{.*}} void @_Z14templateKernelI1AEvT_(%struct.A* noundef byval(%struct.A) align 8 %x)
  launch((void*)templateKernel<A>);

  // AMDGCN: define{{.*}} amdgpu_kernel void @_ZN6Kernel20templateMemberKernelI1AEEvT_(%struct.A addrspace(4)* byref(%struct.A) align 8 %{{.+}}
  // NVPTX: define{{.*}} void @_ZN6Kernel20templateMemberKernelI1AEEvT_(%struct.A* noundef byval(%struct.A) align 8 %x)
  launch((void*)Kernel::templateMemberKernel<A>);
}
