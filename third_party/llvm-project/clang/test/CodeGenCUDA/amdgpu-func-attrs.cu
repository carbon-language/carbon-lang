// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa \
// RUN:     -fcuda-is-device -emit-llvm -o - -x hip %s \
// RUN:     | FileCheck -check-prefixes=NO-UNSAFE-FP-ATOMICS %s
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa \
// RUN:     -fcuda-is-device -emit-llvm -o - -x hip %s \
// RUN:     -munsafe-fp-atomics \
// RUN:     | FileCheck -check-prefixes=UNSAFE-FP-ATOMICS %s
// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -emit-llvm \
// RUN:     -o - -x hip %s -munsafe-fp-atomics \
// RUN:     | FileCheck -check-prefix=NO-UNSAFE-FP-ATOMICS %s

#include "Inputs/hip/hip_runtime.h"

__device__ void test() {
// UNSAFE-FP-ATOMICS: define{{.*}} void @_Z4testv() [[ATTR:#[0-9]+]]
}


// Make sure this is silently accepted on other targets.
// NO-UNSAFE-FP-ATOMICS-NOT: "amdgpu-unsafe-fp-atomics"

// UNSAFE-FP-ATOMICS-DAG: attributes [[ATTR]] = {{.*}}"amdgpu-unsafe-fp-atomics"="true"
