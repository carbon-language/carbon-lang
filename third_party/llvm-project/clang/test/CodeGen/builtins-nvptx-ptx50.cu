
#include <hip/hip_runtime.h>
// RUN: %clang_cc1 -no-opaque-pointers -triple nvptx64-unknown-unknown -target-cpu sm_60 \
// RUN:            -fcuda-is-device -S -emit-llvm -o - -x cuda %s \
// RUN:   | FileCheck -check-prefix=CHECK %s
//
// RUN: %clang_cc1 -no-opaque-pointers -triple nvptx-unknown-unknown -target-cpu sm_50 \
// RUN:   -fcuda-is-device -S -o /dev/null -x cuda -verify %s

#define __device__ __attribute__((device))
#define __global__ __attribute__((global))
#define __shared__ __attribute__((shared))
#define __constant__ __attribute__((constant))

// We have to keep all builtins that depend on particular target feature in the
// same function, because the codegen will stop after the very first function
// that encounters an error, so -verify will not be able to find errors in
// subsequent functions.

// CHECK-LABEL: test_fn
__device__ void test_fn(double d, double* double_ptr) {
  // CHECK: atomicrmw fadd double* {{.*}} seq_cst, align 8
  // expected-error@+1 {{'__nvvm_atom_add_gen_d' needs target feature sm_60}}
  __nvvm_atom_add_gen_d(double_ptr, d);
}
