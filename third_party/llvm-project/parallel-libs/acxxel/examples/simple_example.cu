#include "hip/hip_runtime.h"
//===--- simple_example.cu - Simple example of using Acxxel ---------------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//===----------------------------------------------------------------------===//
///
/// This file is a simple example of using Acxxel.
///
//===----------------------------------------------------------------------===//

/// [Example simple saxpy]
#include "acxxel.h"

#include <array>
#include <cstdio>
#include <cstdlib>

// A standard CUDA kernel.
__global__ void saxpyKernel(float A, float *X, float *Y, int N) {
  int I = (blockDim.x * blockIdx.x) + threadIdx.x;
  if (I < N)
    X[I] = A * X[I] + Y[I];
}

// A host library wrapping the CUDA kernel. All Acxxel calls are in here.
template <size_t N>
void saxpy(float A, std::array<float, N> &X, const std::array<float, N> &Y) {
  // Get the CUDA platform and make a CUDA stream.
  acxxel::Platform *CUDA = acxxel::getCUDAPlatform().getValue();
  acxxel::Stream Stream = CUDA->createStream().takeValue();

  // Allocate space for device arrays.
  auto DeviceX = CUDA->mallocD<float>(N).takeValue();
  auto DeviceY = CUDA->mallocD<float>(N).takeValue();

  // Copy X and Y out to the device.
  Stream.syncCopyHToD(X, DeviceX).syncCopyHToD(Y, DeviceY);

  // Launch the kernel using triple-chevron notation.
  saxpyKernel<<<1, N, 0, Stream>>>(A, DeviceX, DeviceY, N);

  // Copy the results back to the host.
  acxxel::Status Status = Stream.syncCopyDToH(DeviceX, X).takeStatus();

  // Check for any errors.
  if (Status.isError()) {
    std::fprintf(stderr, "Error performing acxxel saxpy: %s\n",
                 Status.getMessage().c_str());
    std::exit(EXIT_FAILURE);
  }
}
/// [Example simple saxpy]

/// [Example CUDA simple saxpy]
template <size_t N>
void cudaSaxpy(float A, std::array<float, N> &X, std::array<float, N> &Y) {
  // This size is needed all over the place, so give it a name.
  constexpr size_t Size = N * sizeof(float);

  // Allocate space for device arrays.
  float *DeviceX;
  float *DeviceY;
  hipMalloc(&DeviceX, Size);
  hipMalloc(&DeviceY, Size);

  // Copy X and Y out to the device.
  hipMemcpy(DeviceX, X.data(), Size, hipMemcpyHostToDevice);
  hipMemcpy(DeviceY, Y.data(), Size, hipMemcpyHostToDevice);

  // Launch the kernel using triple-chevron notation.
  saxpyKernel<<<1, N>>>(A, DeviceX, DeviceY, N);

  // Copy the results back to the host.
  hipMemcpy(X.data(), DeviceX, Size, hipMemcpyDeviceToHost);

  // Free resources.
  hipFree(DeviceX);
  hipFree(DeviceY);

  // Check for any errors.
  hipError_t Error = hipGetLastError();
  if (Error) {
    std::fprintf(stderr, "Error performing cudart saxpy: %s\n",
                 hipGetErrorString(Error));
    std::exit(EXIT_FAILURE);
  }
}
/// [Example CUDA simple saxpy]

template <typename F> void testSaxpy(F &&SaxpyFunction) {
  float A = 2.f;
  std::array<float, 3> X = {{0.f, 1.f, 2.f}};
  std::array<float, 3> Y = {{3.f, 4.f, 5.f}};
  std::array<float, 3> Expected = {{3.f, 6.f, 9.f}};
  SaxpyFunction(A, X, Y);
  for (int I = 0; I < 3; ++I)
    if (X[I] != Expected[I]) {
      std::fprintf(stderr, "Result mismatch at index %d, %f != %f\n", I, X[I],
                   Expected[I]);
      std::exit(EXIT_FAILURE);
    }
}

int main() {
  testSaxpy(saxpy<3>);
  testSaxpy(cudaSaxpy<3>);
}
