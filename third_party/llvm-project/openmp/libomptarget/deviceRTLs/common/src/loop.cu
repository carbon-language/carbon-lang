#include "hip/hip_runtime.h"
//===------------ loop.cu - NVPTX OpenMP loop constructs --------- CUDA -*-===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//===----------------------------------------------------------------------===//
//
// This file contains the implementation of the KMPC interface
// for the loop construct plus other worksharing constructs that use the same
// interface as loops.
//
//===----------------------------------------------------------------------===//
#pragma omp declare target

#include "common/omptarget.h"
#include "target/shuffle.h"
#include "target_impl.h"

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
// template class that encapsulate all the helper functions
//
// T is loop iteration type (32 | 64)  (unsigned | signed)
// ST is the signed version of T
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////

template <typename T, typename ST> class omptarget_nvptx_LoopSupport {
public:
  ////////////////////////////////////////////////////////////////////////////////
  // Loop with static scheduling with chunk

  // Generic implementation of OMP loop scheduling with static policy
  /*! \brief Calculate initial bounds for static loop and stride
   *  @param[in] loc location in code of the call (not used here)
   *  @param[in] global_tid global thread id
   *  @param[in] schetype type of scheduling (see omptarget-nvptx.h)
   *  @param[in] plastiter pointer to last iteration
   *  @param[in,out] pointer to loop lower bound. it will contain value of
   *  lower bound of first chunk
   *  @param[in,out] pointer to loop upper bound. It will contain value of
   *  upper bound of first chunk
   *  @param[in,out] pointer to loop stride. It will contain value of stride
   *  between two successive chunks executed by the same thread
   *  @param[in] loop increment bump
   *  @param[in] chunk size
   */

  // helper function for static chunk
  INLINE static void ForStaticChunk(int &last, T &lb, T &ub, ST &stride,
                                    ST chunk, T entityId, T numberOfEntities) {
    // each thread executes multiple chunks all of the same size, except
    // the last one

    // distance between two successive chunks
    stride = numberOfEntities * chunk;
    lb = lb + entityId * chunk;
    T inputUb = ub;
    ub = lb + chunk - 1; // Clang uses i <= ub
    // Say ub' is the begining of the last chunk. Then who ever has a
    // lower bound plus a multiple of the increment equal to ub' is
    // the last one.
    T beginingLastChunk = inputUb - (inputUb % chunk);
    last = ((beginingLastChunk - lb) % stride) == 0;
  }

  ////////////////////////////////////////////////////////////////////////////////
  // Loop with static scheduling without chunk

  // helper function for static no chunk
  INLINE static void ForStaticNoChunk(int &last, T &lb, T &ub, ST &stride,
                                      ST &chunk, T entityId,
                                      T numberOfEntities) {
    // No chunk size specified.  Each thread or warp gets at most one
    // chunk; chunks are all almost of equal size
    T loopSize = ub - lb + 1;

    chunk = loopSize / numberOfEntities;
    T leftOver = loopSize - chunk * numberOfEntities;

    if (entityId < leftOver) {
      chunk++;
      lb = lb + entityId * chunk;
    } else {
      lb = lb + entityId * chunk + leftOver;
    }

    T inputUb = ub;
    ub = lb + chunk - 1; // Clang uses i <= ub
    last = lb <= inputUb && inputUb <= ub;
    stride = loopSize; // make sure we only do 1 chunk per warp
  }

  ////////////////////////////////////////////////////////////////////////////////
  // Support for Static Init

  INLINE static void for_static_init(int32_t gtid, int32_t schedtype,
                                     int32_t *plastiter, T *plower, T *pupper,
                                     ST *pstride, ST chunk,
                                     bool IsSPMDExecutionMode) {
    // When IsRuntimeUninitialized is true, we assume that the caller is
    // in an L0 parallel region and that all worker threads participate.

    // Assume we are in teams region or that we use a single block
    // per target region
    ST numberOfActiveOMPThreads = GetNumberOfOmpThreads(IsSPMDExecutionMode);

    // All warps that are in excess of the maximum requested, do
    // not execute the loop
    PRINT(LD_LOOP,
          "OMP Thread %d: schedule type %d, chunk size = %lld, mytid "
          "%d, num tids %d\n",
          (int)gtid, (int)schedtype, (long long)chunk, (int)gtid,
          (int)numberOfActiveOMPThreads);
    ASSERT0(LT_FUSSY, gtid < numberOfActiveOMPThreads,
            "current thread is not needed here; error");

    // copy
    int lastiter = 0;
    T lb = *plower;
    T ub = *pupper;
    ST stride = *pstride;
    // init
    switch (SCHEDULE_WITHOUT_MODIFIERS(schedtype)) {
    case kmp_sched_static_chunk: {
      if (chunk > 0) {
        ForStaticChunk(lastiter, lb, ub, stride, chunk, gtid,
                       numberOfActiveOMPThreads);
        break;
      }
    } // note: if chunk <=0, use nochunk
    case kmp_sched_static_balanced_chunk: {
      if (chunk > 0) {
        // round up to make sure the chunk is enough to cover all iterations
        T tripCount = ub - lb + 1; // +1 because ub is inclusive
        T span = (tripCount + numberOfActiveOMPThreads - 1) /
                 numberOfActiveOMPThreads;
        // perform chunk adjustment
        chunk = (span + chunk - 1) & ~(chunk - 1);

        ASSERT0(LT_FUSSY, ub >= lb, "ub must be >= lb.");
        T oldUb = ub;
        ForStaticChunk(lastiter, lb, ub, stride, chunk, gtid,
                       numberOfActiveOMPThreads);
        if (ub > oldUb)
          ub = oldUb;
        break;
      }
    } // note: if chunk <=0, use nochunk
    case kmp_sched_static_nochunk: {
      ForStaticNoChunk(lastiter, lb, ub, stride, chunk, gtid,
                       numberOfActiveOMPThreads);
      break;
    }
    case kmp_sched_distr_static_chunk: {
      if (chunk > 0) {
        ForStaticChunk(lastiter, lb, ub, stride, chunk, GetOmpTeamId(),
                       GetNumberOfOmpTeams());
        break;
      } // note: if chunk <=0, use nochunk
    }
    case kmp_sched_distr_static_nochunk: {
      ForStaticNoChunk(lastiter, lb, ub, stride, chunk, GetOmpTeamId(),
                       GetNumberOfOmpTeams());
      break;
    }
    case kmp_sched_distr_static_chunk_sched_static_chunkone: {
      ForStaticChunk(lastiter, lb, ub, stride, chunk,
                     numberOfActiveOMPThreads * GetOmpTeamId() + gtid,
                     GetNumberOfOmpTeams() * numberOfActiveOMPThreads);
      break;
    }
    default: {
      ASSERT(LT_FUSSY, 0, "unknown schedtype %d", (int)schedtype);
      PRINT(LD_LOOP, "unknown schedtype %d, revert back to static chunk\n",
            (int)schedtype);
      ForStaticChunk(lastiter, lb, ub, stride, chunk, gtid,
                     numberOfActiveOMPThreads);
      break;
    }
    }
    // copy back
    *plastiter = lastiter;
    *plower = lb;
    *pupper = ub;
    *pstride = stride;
    PRINT(LD_LOOP,
          "Got sched: Active %d, total %d: lb %lld, ub %lld, stride %lld, last "
          "%d\n",
          (int)numberOfActiveOMPThreads, (int)GetNumberOfWorkersInTeam(),
          (long long)(*plower), (long long)(*pupper), (long long)(*pstride),
          (int)lastiter);
  }

  ////////////////////////////////////////////////////////////////////////////////
  // Support for dispatch Init

  INLINE static int OrderedSchedule(kmp_sched_t schedule) {
    return schedule >= kmp_sched_ordered_first &&
           schedule <= kmp_sched_ordered_last;
  }

  INLINE static void dispatch_init(kmp_Ident *loc, int32_t threadId,
                                   kmp_sched_t schedule, T lb, T ub, ST st,
                                   ST chunk) {
    if (isRuntimeUninitialized()) {
      // In SPMD mode no need to check parallelism level - dynamic scheduling
      // may appear only in L2 parallel regions with lightweight runtime.
      ASSERT0(LT_FUSSY, __kmpc_is_spmd_exec_mode(), "Expected non-SPMD mode.");
      return;
    }
    int tid = GetLogicalThreadIdInBlock();
    omptarget_nvptx_TaskDescr *currTaskDescr = getMyTopTaskDescriptor(tid);
    T tnum = GetNumberOfOmpThreads(__kmpc_is_spmd_exec_mode());
    T tripCount = ub - lb + 1; // +1 because ub is inclusive
    ASSERT0(LT_FUSSY, threadId < tnum,
            "current thread is not needed here; error");

    /* Currently just ignore the monotonic and non-monotonic modifiers
     * (the compiler isn't producing them * yet anyway).
     * When it is we'll want to look at them somewhere here and use that
     * information to add to our schedule choice. We shouldn't need to pass
     * them on, they merely affect which schedule we can legally choose for
     * various dynamic cases. (In particular, whether or not a stealing scheme
     * is legal).
     */
    schedule = SCHEDULE_WITHOUT_MODIFIERS(schedule);

    // Process schedule.
    if (tnum == 1 || tripCount <= 1 || OrderedSchedule(schedule)) {
      if (OrderedSchedule(schedule))
        __kmpc_barrier(loc, threadId);
      PRINT(LD_LOOP,
            "go sequential as tnum=%ld, trip count %lld, ordered sched=%d\n",
            (long)tnum, (long long)tripCount, (int)schedule);
      schedule = kmp_sched_static_chunk;
      chunk = tripCount; // one thread gets the whole loop
    } else if (schedule == kmp_sched_runtime) {
      // process runtime
      omp_sched_t rtSched = currTaskDescr->GetRuntimeSched();
      chunk = currTaskDescr->RuntimeChunkSize();
      switch (rtSched) {
      case omp_sched_static: {
        if (chunk > 0)
          schedule = kmp_sched_static_chunk;
        else
          schedule = kmp_sched_static_nochunk;
        break;
      }
      case omp_sched_auto: {
        schedule = kmp_sched_static_chunk;
        chunk = 1;
        break;
      }
      case omp_sched_dynamic:
      case omp_sched_guided: {
        schedule = kmp_sched_dynamic;
        break;
      }
      }
      PRINT(LD_LOOP, "Runtime sched is %d with chunk %lld\n", (int)schedule,
            (long long)chunk);
    } else if (schedule == kmp_sched_auto) {
      schedule = kmp_sched_static_chunk;
      chunk = 1;
      PRINT(LD_LOOP, "Auto sched is %d with chunk %lld\n", (int)schedule,
            (long long)chunk);
    } else {
      PRINT(LD_LOOP, "Dyn sched is %d with chunk %lld\n", (int)schedule,
            (long long)chunk);
      ASSERT(LT_FUSSY,
             schedule == kmp_sched_dynamic || schedule == kmp_sched_guided,
             "unknown schedule %d & chunk %lld\n", (int)schedule,
             (long long)chunk);
    }

    // init schedules
    if (schedule == kmp_sched_static_chunk) {
      ASSERT0(LT_FUSSY, chunk > 0, "bad chunk value");
      // save sched state
      omptarget_nvptx_threadPrivateContext->ScheduleType(tid) = schedule;
      // save ub
      omptarget_nvptx_threadPrivateContext->LoopUpperBound(tid) = ub;
      // compute static chunk
      ST stride;
      int lastiter = 0;
      ForStaticChunk(lastiter, lb, ub, stride, chunk, threadId, tnum);
      // save computed params
      omptarget_nvptx_threadPrivateContext->Chunk(tid) = chunk;
      omptarget_nvptx_threadPrivateContext->NextLowerBound(tid) = lb;
      omptarget_nvptx_threadPrivateContext->Stride(tid) = stride;
      PRINT(LD_LOOP,
            "dispatch init (static chunk) : num threads = %d, ub =  %" PRId64
            ", next lower bound = %llu, stride = %llu\n",
            (int)tnum,
            omptarget_nvptx_threadPrivateContext->LoopUpperBound(tid),
            (unsigned long long)
                omptarget_nvptx_threadPrivateContext->NextLowerBound(tid),
            (unsigned long long)omptarget_nvptx_threadPrivateContext->Stride(
                tid));
    } else if (schedule == kmp_sched_static_balanced_chunk) {
      ASSERT0(LT_FUSSY, chunk > 0, "bad chunk value");
      // save sched state
      omptarget_nvptx_threadPrivateContext->ScheduleType(tid) = schedule;
      // save ub
      omptarget_nvptx_threadPrivateContext->LoopUpperBound(tid) = ub;
      // compute static chunk
      ST stride;
      int lastiter = 0;
      // round up to make sure the chunk is enough to cover all iterations
      T span = (tripCount + tnum - 1) / tnum;
      // perform chunk adjustment
      chunk = (span + chunk - 1) & ~(chunk - 1);

      T oldUb = ub;
      ForStaticChunk(lastiter, lb, ub, stride, chunk, threadId, tnum);
      ASSERT0(LT_FUSSY, ub >= lb, "ub must be >= lb.");
      if (ub > oldUb)
        ub = oldUb;
      // save computed params
      omptarget_nvptx_threadPrivateContext->Chunk(tid) = chunk;
      omptarget_nvptx_threadPrivateContext->NextLowerBound(tid) = lb;
      omptarget_nvptx_threadPrivateContext->Stride(tid) = stride;
      PRINT(LD_LOOP,
            "dispatch init (static chunk) : num threads = %d, ub =  %" PRId64
            ", next lower bound = %llu, stride = %llu\n",
            (int)tnum,
            omptarget_nvptx_threadPrivateContext->LoopUpperBound(tid),
            (unsigned long long)
                omptarget_nvptx_threadPrivateContext->NextLowerBound(tid),
            (unsigned long long)omptarget_nvptx_threadPrivateContext->Stride(
                tid));
    } else if (schedule == kmp_sched_static_nochunk) {
      ASSERT0(LT_FUSSY, chunk == 0, "bad chunk value");
      // save sched state
      omptarget_nvptx_threadPrivateContext->ScheduleType(tid) = schedule;
      // save ub
      omptarget_nvptx_threadPrivateContext->LoopUpperBound(tid) = ub;
      // compute static chunk
      ST stride;
      int lastiter = 0;
      ForStaticNoChunk(lastiter, lb, ub, stride, chunk, threadId, tnum);
      // save computed params
      omptarget_nvptx_threadPrivateContext->Chunk(tid) = chunk;
      omptarget_nvptx_threadPrivateContext->NextLowerBound(tid) = lb;
      omptarget_nvptx_threadPrivateContext->Stride(tid) = stride;
      PRINT(LD_LOOP,
            "dispatch init (static nochunk) : num threads = %d, ub = %" PRId64
            ", next lower bound = %llu, stride = %llu\n",
            (int)tnum,
            omptarget_nvptx_threadPrivateContext->LoopUpperBound(tid),
            (unsigned long long)
                omptarget_nvptx_threadPrivateContext->NextLowerBound(tid),
            (unsigned long long)omptarget_nvptx_threadPrivateContext->Stride(
                tid));
    } else if (schedule == kmp_sched_dynamic || schedule == kmp_sched_guided) {
      // save data
      omptarget_nvptx_threadPrivateContext->ScheduleType(tid) = schedule;
      if (chunk < 1)
        chunk = 1;
      omptarget_nvptx_threadPrivateContext->Chunk(tid) = chunk;
      omptarget_nvptx_threadPrivateContext->LoopUpperBound(tid) = ub;
      omptarget_nvptx_threadPrivateContext->NextLowerBound(tid) = lb;
      __kmpc_barrier(loc, threadId);
      if (tid == 0) {
        omptarget_nvptx_threadPrivateContext->Cnt() = 0;
        __kmpc_impl_threadfence_block();
      }
      __kmpc_barrier(loc, threadId);
      PRINT(LD_LOOP,
            "dispatch init (dyn) : num threads = %d, lb = %llu, ub = %" PRId64
            ", chunk %" PRIu64 "\n",
            (int)tnum,
            (unsigned long long)
                omptarget_nvptx_threadPrivateContext->NextLowerBound(tid),
            omptarget_nvptx_threadPrivateContext->LoopUpperBound(tid),
            omptarget_nvptx_threadPrivateContext->Chunk(tid));
    }
  }

  ////////////////////////////////////////////////////////////////////////////////
  // Support for dispatch next

  INLINE static uint64_t Shuffle(__kmpc_impl_lanemask_t active, int64_t val,
                                 int leader) {
    uint32_t lo, hi;
    __kmpc_impl_unpack(val, lo, hi);
    hi = __kmpc_impl_shfl_sync(active, hi, leader);
    lo = __kmpc_impl_shfl_sync(active, lo, leader);
    return __kmpc_impl_pack(lo, hi);
  }

  INLINE static uint64_t NextIter() {
    __kmpc_impl_lanemask_t active = __kmpc_impl_activemask();
    uint32_t leader = __kmpc_impl_ffs(active) - 1;
    uint32_t change = __kmpc_impl_popc(active);
    __kmpc_impl_lanemask_t lane_mask_lt = __kmpc_impl_lanemask_lt();
    unsigned int rank = __kmpc_impl_popc(active & lane_mask_lt);
    uint64_t warp_res;
    if (rank == 0) {
      warp_res = __kmpc_atomic_add(
          (unsigned long long *)&omptarget_nvptx_threadPrivateContext->Cnt(),
          (unsigned long long)change);
    }
    warp_res = Shuffle(active, warp_res, leader);
    return warp_res + rank;
  }

  INLINE static int DynamicNextChunk(T &lb, T &ub, T chunkSize,
                                     T loopLowerBound, T loopUpperBound) {
    T N = NextIter();
    lb = loopLowerBound + N * chunkSize;
    ub = lb + chunkSize - 1; // Clang uses i <= ub

    // 3 result cases:
    //  a. lb and ub < loopUpperBound --> NOT_FINISHED
    //  b. lb < loopUpperBound and ub >= loopUpperBound: last chunk -->
    //  NOT_FINISHED
    //  c. lb and ub >= loopUpperBound: empty chunk --> FINISHED
    // a.
    if (lb <= loopUpperBound && ub < loopUpperBound) {
      PRINT(LD_LOOPD, "lb %lld, ub %lld, loop ub %lld; not finished\n",
            (long long)lb, (long long)ub, (long long)loopUpperBound);
      return NOT_FINISHED;
    }
    // b.
    if (lb <= loopUpperBound) {
      PRINT(LD_LOOPD, "lb %lld, ub %lld, loop ub %lld; clip to loop ub\n",
            (long long)lb, (long long)ub, (long long)loopUpperBound);
      ub = loopUpperBound;
      return LAST_CHUNK;
    }
    // c. if we are here, we are in case 'c'
    lb = loopUpperBound + 2;
    ub = loopUpperBound + 1;
    PRINT(LD_LOOPD, "lb %lld, ub %lld, loop ub %lld; finished\n", (long long)lb,
          (long long)ub, (long long)loopUpperBound);
    return FINISHED;
  }

  INLINE static int dispatch_next(kmp_Ident *loc, int32_t gtid, int32_t *plast,
                                  T *plower, T *pupper, ST *pstride) {
    if (isRuntimeUninitialized()) {
      // In SPMD mode no need to check parallelism level - dynamic scheduling
      // may appear only in L2 parallel regions with lightweight runtime.
      ASSERT0(LT_FUSSY, __kmpc_is_spmd_exec_mode(), "Expected non-SPMD mode.");
      if (*plast)
        return DISPATCH_FINISHED;
      *plast = 1;
      return DISPATCH_NOTFINISHED;
    }
    // ID of a thread in its own warp

    // automatically selects thread or warp ID based on selected implementation
    int tid = GetLogicalThreadIdInBlock();
    ASSERT0(LT_FUSSY, gtid < GetNumberOfOmpThreads(__kmpc_is_spmd_exec_mode()),
            "current thread is not needed here; error");
    // retrieve schedule
    kmp_sched_t schedule =
        omptarget_nvptx_threadPrivateContext->ScheduleType(tid);

    // xxx reduce to one
    if (schedule == kmp_sched_static_chunk ||
        schedule == kmp_sched_static_nochunk) {
      T myLb = omptarget_nvptx_threadPrivateContext->NextLowerBound(tid);
      T ub = omptarget_nvptx_threadPrivateContext->LoopUpperBound(tid);
      // finished?
      if (myLb > ub) {
        PRINT(LD_LOOP, "static loop finished with myLb %lld, ub %lld\n",
              (long long)myLb, (long long)ub);
        return DISPATCH_FINISHED;
      }
      // not finished, save current bounds
      ST chunk = omptarget_nvptx_threadPrivateContext->Chunk(tid);
      *plower = myLb;
      T myUb = myLb + chunk - 1; // Clang uses i <= ub
      if (myUb > ub)
        myUb = ub;
      *pupper = myUb;
      *plast = (int32_t)(myUb == ub);

      // increment next lower bound by the stride
      ST stride = omptarget_nvptx_threadPrivateContext->Stride(tid);
      omptarget_nvptx_threadPrivateContext->NextLowerBound(tid) = myLb + stride;
      PRINT(LD_LOOP, "static loop continues with myLb %lld, myUb %lld\n",
            (long long)*plower, (long long)*pupper);
      return DISPATCH_NOTFINISHED;
    }
    ASSERT0(LT_FUSSY,
            schedule == kmp_sched_dynamic || schedule == kmp_sched_guided,
            "bad sched");
    T myLb, myUb;
    int finished = DynamicNextChunk(
        myLb, myUb, omptarget_nvptx_threadPrivateContext->Chunk(tid),
        omptarget_nvptx_threadPrivateContext->NextLowerBound(tid),
        omptarget_nvptx_threadPrivateContext->LoopUpperBound(tid));

    if (finished == FINISHED)
      return DISPATCH_FINISHED;

    // not finished (either not finished or last chunk)
    *plast = (int32_t)(finished == LAST_CHUNK);
    *plower = myLb;
    *pupper = myUb;
    *pstride = 1;

    PRINT(LD_LOOP,
          "Got sched: active %d, total %d: lb %lld, ub %lld, stride = %lld, "
          "last %d\n",
          (int)GetNumberOfOmpThreads(__kmpc_is_spmd_exec_mode()),
          (int)GetNumberOfWorkersInTeam(), (long long)*plower,
          (long long)*pupper, (long long)*pstride, (int)*plast);
    return DISPATCH_NOTFINISHED;
  }

  INLINE static void dispatch_fini() {
    // nothing
  }

  ////////////////////////////////////////////////////////////////////////////////
  // end of template class that encapsulate all the helper functions
  ////////////////////////////////////////////////////////////////////////////////
};

////////////////////////////////////////////////////////////////////////////////
// KMP interface implementation (dyn loops)
////////////////////////////////////////////////////////////////////////////////

// init
EXTERN void __kmpc_dispatch_init_4(kmp_Ident *loc, int32_t tid,
                                   int32_t schedule, int32_t lb, int32_t ub,
                                   int32_t st, int32_t chunk) {
  PRINT0(LD_IO, "call kmpc_dispatch_init_4\n");
  omptarget_nvptx_LoopSupport<int32_t, int32_t>::dispatch_init(
      loc, tid, (kmp_sched_t)schedule, lb, ub, st, chunk);
}

EXTERN void __kmpc_dispatch_init_4u(kmp_Ident *loc, int32_t tid,
                                    int32_t schedule, uint32_t lb, uint32_t ub,
                                    int32_t st, int32_t chunk) {
  PRINT0(LD_IO, "call kmpc_dispatch_init_4u\n");
  omptarget_nvptx_LoopSupport<uint32_t, int32_t>::dispatch_init(
      loc, tid, (kmp_sched_t)schedule, lb, ub, st, chunk);
}

EXTERN void __kmpc_dispatch_init_8(kmp_Ident *loc, int32_t tid,
                                   int32_t schedule, int64_t lb, int64_t ub,
                                   int64_t st, int64_t chunk) {
  PRINT0(LD_IO, "call kmpc_dispatch_init_8\n");
  omptarget_nvptx_LoopSupport<int64_t, int64_t>::dispatch_init(
      loc, tid, (kmp_sched_t)schedule, lb, ub, st, chunk);
}

EXTERN void __kmpc_dispatch_init_8u(kmp_Ident *loc, int32_t tid,
                                    int32_t schedule, uint64_t lb, uint64_t ub,
                                    int64_t st, int64_t chunk) {
  PRINT0(LD_IO, "call kmpc_dispatch_init_8u\n");
  omptarget_nvptx_LoopSupport<uint64_t, int64_t>::dispatch_init(
      loc, tid, (kmp_sched_t)schedule, lb, ub, st, chunk);
}

// next
EXTERN int __kmpc_dispatch_next_4(kmp_Ident *loc, int32_t tid, int32_t *p_last,
                                  int32_t *p_lb, int32_t *p_ub, int32_t *p_st) {
  PRINT0(LD_IO, "call kmpc_dispatch_next_4\n");
  return omptarget_nvptx_LoopSupport<int32_t, int32_t>::dispatch_next(
      loc, tid, p_last, p_lb, p_ub, p_st);
}

EXTERN int __kmpc_dispatch_next_4u(kmp_Ident *loc, int32_t tid, int32_t *p_last,
                                   uint32_t *p_lb, uint32_t *p_ub,
                                   int32_t *p_st) {
  PRINT0(LD_IO, "call kmpc_dispatch_next_4u\n");
  return omptarget_nvptx_LoopSupport<uint32_t, int32_t>::dispatch_next(
      loc, tid, p_last, p_lb, p_ub, p_st);
}

EXTERN int __kmpc_dispatch_next_8(kmp_Ident *loc, int32_t tid, int32_t *p_last,
                                  int64_t *p_lb, int64_t *p_ub, int64_t *p_st) {
  PRINT0(LD_IO, "call kmpc_dispatch_next_8\n");
  return omptarget_nvptx_LoopSupport<int64_t, int64_t>::dispatch_next(
      loc, tid, p_last, p_lb, p_ub, p_st);
}

EXTERN int __kmpc_dispatch_next_8u(kmp_Ident *loc, int32_t tid, int32_t *p_last,
                                   uint64_t *p_lb, uint64_t *p_ub,
                                   int64_t *p_st) {
  PRINT0(LD_IO, "call kmpc_dispatch_next_8u\n");
  return omptarget_nvptx_LoopSupport<uint64_t, int64_t>::dispatch_next(
      loc, tid, p_last, p_lb, p_ub, p_st);
}

// fini
EXTERN void __kmpc_dispatch_fini_4(kmp_Ident *loc, int32_t tid) {
  PRINT0(LD_IO, "call kmpc_dispatch_fini_4\n");
  omptarget_nvptx_LoopSupport<int32_t, int32_t>::dispatch_fini();
}

EXTERN void __kmpc_dispatch_fini_4u(kmp_Ident *loc, int32_t tid) {
  PRINT0(LD_IO, "call kmpc_dispatch_fini_4u\n");
  omptarget_nvptx_LoopSupport<uint32_t, int32_t>::dispatch_fini();
}

EXTERN void __kmpc_dispatch_fini_8(kmp_Ident *loc, int32_t tid) {
  PRINT0(LD_IO, "call kmpc_dispatch_fini_8\n");
  omptarget_nvptx_LoopSupport<int64_t, int64_t>::dispatch_fini();
}

EXTERN void __kmpc_dispatch_fini_8u(kmp_Ident *loc, int32_t tid) {
  PRINT0(LD_IO, "call kmpc_dispatch_fini_8u\n");
  omptarget_nvptx_LoopSupport<uint64_t, int64_t>::dispatch_fini();
}

////////////////////////////////////////////////////////////////////////////////
// KMP interface implementation (static loops)
////////////////////////////////////////////////////////////////////////////////

EXTERN void __kmpc_for_static_init_4(kmp_Ident *loc, int32_t global_tid,
                                     int32_t schedtype, int32_t *plastiter,
                                     int32_t *plower, int32_t *pupper,
                                     int32_t *pstride, int32_t incr,
                                     int32_t chunk) {
  PRINT0(LD_IO, "call kmpc_for_static_init_4\n");
  omptarget_nvptx_LoopSupport<int32_t, int32_t>::for_static_init(
      global_tid, schedtype, plastiter, plower, pupper, pstride, chunk,
      __kmpc_is_spmd_exec_mode());
}

EXTERN void __kmpc_for_static_init_4u(kmp_Ident *loc, int32_t global_tid,
                                      int32_t schedtype, int32_t *plastiter,
                                      uint32_t *plower, uint32_t *pupper,
                                      int32_t *pstride, int32_t incr,
                                      int32_t chunk) {
  PRINT0(LD_IO, "call kmpc_for_static_init_4u\n");
  omptarget_nvptx_LoopSupport<uint32_t, int32_t>::for_static_init(
      global_tid, schedtype, plastiter, plower, pupper, pstride, chunk,
      __kmpc_is_spmd_exec_mode());
}

EXTERN void __kmpc_for_static_init_8(kmp_Ident *loc, int32_t global_tid,
                                     int32_t schedtype, int32_t *plastiter,
                                     int64_t *plower, int64_t *pupper,
                                     int64_t *pstride, int64_t incr,
                                     int64_t chunk) {
  PRINT0(LD_IO, "call kmpc_for_static_init_8\n");
  omptarget_nvptx_LoopSupport<int64_t, int64_t>::for_static_init(
      global_tid, schedtype, plastiter, plower, pupper, pstride, chunk,
      __kmpc_is_spmd_exec_mode());
}

EXTERN void __kmpc_for_static_init_8u(kmp_Ident *loc, int32_t global_tid,
                                      int32_t schedtype, int32_t *plastiter,
                                      uint64_t *plower, uint64_t *pupper,
                                      int64_t *pstride, int64_t incr,
                                      int64_t chunk) {
  PRINT0(LD_IO, "call kmpc_for_static_init_8u\n");
  omptarget_nvptx_LoopSupport<uint64_t, int64_t>::for_static_init(
      global_tid, schedtype, plastiter, plower, pupper, pstride, chunk,
      __kmpc_is_spmd_exec_mode());
}

EXTERN void __kmpc_distribute_static_init_4(kmp_Ident *loc, int32_t global_tid,
                                            int32_t schedtype,
                                            int32_t *plastiter, int32_t *plower,
                                            int32_t *pupper, int32_t *pstride,
                                            int32_t incr, int32_t chunk) {
  PRINT0(LD_IO, "call kmpc_distribute_static_init_4\n");
  omptarget_nvptx_LoopSupport<int32_t, int32_t>::for_static_init(
      global_tid, schedtype, plastiter, plower, pupper, pstride, chunk,
      __kmpc_is_spmd_exec_mode());
}

EXTERN void __kmpc_distribute_static_init_4u(kmp_Ident *loc, int32_t global_tid,
                                             int32_t schedtype,
                                             int32_t *plastiter,
                                             uint32_t *plower, uint32_t *pupper,
                                             int32_t *pstride, int32_t incr,
                                             int32_t chunk) {
  PRINT0(LD_IO, "call kmpc_distribute_static_init_4u\n");
  omptarget_nvptx_LoopSupport<uint32_t, int32_t>::for_static_init(
      global_tid, schedtype, plastiter, plower, pupper, pstride, chunk,
      __kmpc_is_spmd_exec_mode());
}

EXTERN void __kmpc_distribute_static_init_8(kmp_Ident *loc, int32_t global_tid,
                                            int32_t schedtype,
                                            int32_t *plastiter, int64_t *plower,
                                            int64_t *pupper, int64_t *pstride,
                                            int64_t incr, int64_t chunk) {
  PRINT0(LD_IO, "call kmpc_distribute_static_init_8\n");
  omptarget_nvptx_LoopSupport<int64_t, int64_t>::for_static_init(
      global_tid, schedtype, plastiter, plower, pupper, pstride, chunk,
      __kmpc_is_spmd_exec_mode());
}

EXTERN void __kmpc_distribute_static_init_8u(kmp_Ident *loc, int32_t global_tid,
                                             int32_t schedtype,
                                             int32_t *plastiter,
                                             uint64_t *plower, uint64_t *pupper,
                                             int64_t *pstride, int64_t incr,
                                             int64_t chunk) {
  PRINT0(LD_IO, "call kmpc_distribute_static_init_8u\n");
  omptarget_nvptx_LoopSupport<uint64_t, int64_t>::for_static_init(
      global_tid, schedtype, plastiter, plower, pupper, pstride, chunk,
      __kmpc_is_spmd_exec_mode());
}

EXTERN
void __kmpc_for_static_init_4_simple_spmd(kmp_Ident *loc, int32_t global_tid,
                                          int32_t schedtype, int32_t *plastiter,
                                          int32_t *plower, int32_t *pupper,
                                          int32_t *pstride, int32_t incr,
                                          int32_t chunk) {
  PRINT0(LD_IO, "call kmpc_for_static_init_4_simple_spmd\n");
  omptarget_nvptx_LoopSupport<int32_t, int32_t>::for_static_init(
      global_tid, schedtype, plastiter, plower, pupper, pstride, chunk,
      /*IsSPMDExecutionMode=*/true);
}

EXTERN
void __kmpc_for_static_init_4u_simple_spmd(kmp_Ident *loc, int32_t global_tid,
                                           int32_t schedtype,
                                           int32_t *plastiter, uint32_t *plower,
                                           uint32_t *pupper, int32_t *pstride,
                                           int32_t incr, int32_t chunk) {
  PRINT0(LD_IO, "call kmpc_for_static_init_4u_simple_spmd\n");
  omptarget_nvptx_LoopSupport<uint32_t, int32_t>::for_static_init(
      global_tid, schedtype, plastiter, plower, pupper, pstride, chunk,
      /*IsSPMDExecutionMode=*/true);
}

EXTERN
void __kmpc_for_static_init_8_simple_spmd(kmp_Ident *loc, int32_t global_tid,
                                          int32_t schedtype, int32_t *plastiter,
                                          int64_t *plower, int64_t *pupper,
                                          int64_t *pstride, int64_t incr,
                                          int64_t chunk) {
  PRINT0(LD_IO, "call kmpc_for_static_init_8_simple_spmd\n");
  omptarget_nvptx_LoopSupport<int64_t, int64_t>::for_static_init(
      global_tid, schedtype, plastiter, plower, pupper, pstride, chunk,
      /*IsSPMDExecutionMode=*/true);
}

EXTERN
void __kmpc_for_static_init_8u_simple_spmd(kmp_Ident *loc, int32_t global_tid,
                                           int32_t schedtype,
                                           int32_t *plastiter, uint64_t *plower,
                                           uint64_t *pupper, int64_t *pstride,
                                           int64_t incr, int64_t chunk) {
  PRINT0(LD_IO, "call kmpc_for_static_init_8u_simple_spmd\n");
  omptarget_nvptx_LoopSupport<uint64_t, int64_t>::for_static_init(
      global_tid, schedtype, plastiter, plower, pupper, pstride, chunk,
      /*IsSPMDExecutionMode=*/true);
}

EXTERN
void __kmpc_for_static_init_4_simple_generic(kmp_Ident *loc, int32_t global_tid,
                                             int32_t schedtype,
                                             int32_t *plastiter,
                                             int32_t *plower, int32_t *pupper,
                                             int32_t *pstride, int32_t incr,
                                             int32_t chunk) {
  PRINT0(LD_IO, "call kmpc_for_static_init_4_simple_generic\n");
  omptarget_nvptx_LoopSupport<int32_t, int32_t>::for_static_init(
      global_tid, schedtype, plastiter, plower, pupper, pstride, chunk,
      /*IsSPMDExecutionMode=*/false);
}

EXTERN
void __kmpc_for_static_init_4u_simple_generic(
    kmp_Ident *loc, int32_t global_tid, int32_t schedtype, int32_t *plastiter,
    uint32_t *plower, uint32_t *pupper, int32_t *pstride, int32_t incr,
    int32_t chunk) {
  PRINT0(LD_IO, "call kmpc_for_static_init_4u_simple_generic\n");
  omptarget_nvptx_LoopSupport<uint32_t, int32_t>::for_static_init(
      global_tid, schedtype, plastiter, plower, pupper, pstride, chunk,
      /*IsSPMDExecutionMode=*/false);
}

EXTERN
void __kmpc_for_static_init_8_simple_generic(kmp_Ident *loc, int32_t global_tid,
                                             int32_t schedtype,
                                             int32_t *plastiter,
                                             int64_t *plower, int64_t *pupper,
                                             int64_t *pstride, int64_t incr,
                                             int64_t chunk) {
  PRINT0(LD_IO, "call kmpc_for_static_init_8_simple_generic\n");
  omptarget_nvptx_LoopSupport<int64_t, int64_t>::for_static_init(
      global_tid, schedtype, plastiter, plower, pupper, pstride, chunk,
      /*IsSPMDExecutionMode=*/false);
}

EXTERN
void __kmpc_for_static_init_8u_simple_generic(
    kmp_Ident *loc, int32_t global_tid, int32_t schedtype, int32_t *plastiter,
    uint64_t *plower, uint64_t *pupper, int64_t *pstride, int64_t incr,
    int64_t chunk) {
  PRINT0(LD_IO, "call kmpc_for_static_init_8u_simple_generic\n");
  omptarget_nvptx_LoopSupport<uint64_t, int64_t>::for_static_init(
      global_tid, schedtype, plastiter, plower, pupper, pstride, chunk,
      /*IsSPMDExecutionMode=*/false);
}

EXTERN void __kmpc_distribute_static_fini(kmp_Ident *loc, int32_t global_tid) {
  PRINT0(LD_IO, "call kmpc_distribute_static_fini\n");
}

EXTERN void __kmpc_for_static_fini(kmp_Ident *loc, int32_t global_tid) {
  PRINT0(LD_IO, "call kmpc_for_static_fini\n");
}

#pragma omp end declare target
