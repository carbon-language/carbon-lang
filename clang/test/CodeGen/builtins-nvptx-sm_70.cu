
#include <hip/hip_runtime.h>
// RUN: %clang_cc1 -triple nvptx64-unknown-unknown -target-cpu sm_70 \
// RUN:            -fcuda-is-device -target-feature +ptx60 \
// RUN:            -S -emit-llvm -o - -x cuda %s \
// RUN:   | FileCheck -check-prefix=CHECK_M16 %s
// RUN: %clang_cc1 -triple nvptx64-unknown-unknown -target-cpu sm_70 \
// RUN:            -fcuda-is-device -target-feature +ptx61 -DPTX61 \
// RUN:            -S -emit-llvm -o - -x cuda %s \
// RUN:   | FileCheck -check-prefixes=CHECK_M16,CHECK_M32_M8 %s
// Make sure builtins still work with the latest combination of GPU & PTX.
// RUN: %clang_cc1 -triple nvptx64-unknown-unknown -target-cpu sm_86 \
// RUN:            -fcuda-is-device -target-feature +ptx72 -DPTX61 \
// RUN:            -S -emit-llvm -o - -x cuda %s \
// RUN:   | FileCheck -check-prefixes=CHECK_M16,CHECK_M32_M8 %s
// RUN: %clang_cc1 -triple nvptx-unknown-unknown -target-cpu sm_60 \
// RUN:   -DPTX61 -fcuda-is-device -S -o /dev/null -x cuda -verify=pre-sm_70 %s
// RUN: %clang_cc1 -triple nvptx-unknown-unknown \
// RUN:   -target-cpu sm_70 -target-feature +ptx60 \
// RUN:   -DPTX61 -fcuda-is-device -S -o /dev/null -x cuda -verify=pre-ptx61 %s

#if !defined(CUDA_VERSION)
#define __device__ __attribute__((device))
#define __global__ __attribute__((global))
#define __shared__ __attribute__((shared))
#define __constant__ __attribute__((constant))

typedef unsigned long long uint64_t;
#endif
// We have to keep all builtins that depend on particular target feature in the
// same function, because the codegen will stop after the very first function
// that encounters an error, so -verify will not be able to find errors in
// subsequent functions.

// CHECK-LABEL: nvvm_wmma_m16n16k16
__device__ void nvvm_wmma_m16n16k16(int *src, int *dst,
                                    float *fsrc, float *fdst,
                                    int ldm) {
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.load.a.row.stride.f16
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_ld_a' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_ld_a(dst, src, ldm, 0);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.load.a.col.stride.f16
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_ld_a' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_ld_a(dst, src+1, ldm, 1);

  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.load.b.row.stride.f16
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_ld_b' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_ld_b(dst, src, ldm, 0);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.load.b.col.stride.f16
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_ld_b' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_ld_b(dst, src+2, ldm, 1);

  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.load.c.row.stride.f16
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_ld_c_f16' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_ld_c_f16(dst, src, ldm, 0);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.load.c.col.stride.f16
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_ld_c_f16' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_ld_c_f16(dst, src, ldm, 1);

  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.load.c.row.stride.f32
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_ld_c_f32' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_ld_c_f32(fdst, fsrc, ldm, 0);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.load.c.col.stride.f32
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_ld_c_f32' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_ld_c_f32(fdst, fsrc, ldm, 1);

  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.store.d.row.stride.f16
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_st_c_f16' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_st_c_f16(dst, src, ldm, 0);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.store.d.col.stride.f16
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_st_c_f16' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_st_c_f16(dst, src, ldm, 1);

  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.store.d.row.stride.f32
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_st_c_f32' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_st_c_f32(fdst, fsrc, ldm, 0);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.store.d.col.stride.f32
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_st_c_f32' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_st_c_f32(fdst, fsrc, ldm, 1);

  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.row.row.f16.f16
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f16f16' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f16f16(dst, src, src, src, 0, 0);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.row.row.f16.f16.satfinite
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f16f16' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f16f16(dst, src, src, src, 0, 1);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.row.col.f16.f16
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f16f16' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f16f16(dst, src, src, src, 1, 0);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.row.col.f16.f16.satfinite
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f16f16' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f16f16(dst, src, src, src, 1, 1);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.col.row.f16.f16
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f16f16' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f16f16(dst, src, src, src, 2, 0);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.col.row.f16.f16.satfinite
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f16f16' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f16f16(dst, src, src, src, 2, 1);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.col.col.f16.f16
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f16f16' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f16f16(dst, src, src, src, 3, 0);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.col.col.f16.f16.satfinite
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f16f16' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f16f16(dst, src, src, src, 3, 1);

  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.row.row.f16.f32
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f16f32' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f16f32(dst, src, src, fsrc, 0, 0);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.row.row.f16.f32.satfinite
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f16f32' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f16f32(dst, src, src, fsrc, 0, 1);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.row.col.f16.f32
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f16f32' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f16f32(dst, src, src, fsrc, 1, 0);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.row.col.f16.f32.satfinite
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f16f32' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f16f32(dst, src, src, fsrc, 1, 1);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.col.row.f16.f32
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f16f32' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f16f32(dst, src, src, fsrc, 2, 0);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.col.row.f16.f32.satfinite
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f16f32' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f16f32(dst, src, src, fsrc, 2, 1);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.col.col.f16.f32
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f16f32' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f16f32(dst, src, src, fsrc, 3, 0);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.col.col.f16.f32.satfinite
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f16f32' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f16f32(dst, src, src, fsrc, 3, 1);

  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.row.row.f32.f16
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f32f16' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f32f16(fdst, src, src, src, 0, 0);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.row.row.f32.f16.satfinite
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f32f16' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f32f16(fdst, src, src, src, 0, 1);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.row.col.f32.f16
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f32f16' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f32f16(fdst, src, src, src, 1, 0);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.row.col.f32.f16.satfinite
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f32f16' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f32f16(fdst, src, src, src, 1, 1);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.col.row.f32.f16
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f32f16' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f32f16(fdst, src, src, src, 2, 0);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.col.row.f32.f16.satfinite
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f32f16' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f32f16(fdst, src, src, src, 2, 1);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.col.col.f32.f16
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f32f16' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f32f16(fdst, src, src, src, 3, 0);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.col.col.f32.f16.satfinite
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f32f16' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f32f16(fdst, src, src, src, 3, 1);

  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.row.row.f32.f32
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f32f32' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f32f32(fdst, src, src, fsrc, 0, 0);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.row.row.f32.f32.satfinite
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f32f32' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f32f32(fdst, src, src, fsrc, 0, 1);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.row.col.f32.f32
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f32f32' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f32f32(fdst, src, src, fsrc, 1, 0);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.row.col.f32.f32.satfinite
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f32f32' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f32f32(fdst, src, src, fsrc, 1, 1);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.col.row.f32.f32
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f32f32' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f32f32(fdst, src, src, fsrc, 2, 0);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.col.row.f32.f32.satfinite
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f32f32' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f32f32(fdst, src, src, fsrc, 2, 1);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.col.col.f32.f32
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f32f32' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f32f32(fdst, src, src, fsrc, 3, 0);
  // CHECK_M16: call {{.*}} @llvm.nvvm.wmma.m16n16k16.mma.col.col.f32.f32.satfinite
  // pre-sm_70-error-re@+1 {{'__hmma_m16n16k16_mma_f32f32' needs target feature (sm_70{{.*}},(ptx60{{.*}}}}
  __hmma_m16n16k16_mma_f32f32(fdst, src, src, fsrc, 3, 1);
}

#ifdef PTX61
// CHECK-LABEL: nvvm_wmma_m32n8k16
__device__ void nvvm_wmma_m32n8k16(int *src, int *dst,
                                    float *fsrc, float *fdst,
                                    int ldm) {
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.load.a.row.stride.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_ld_a' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_ld_a(dst, src, ldm, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.load.a.col.stride.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_ld_a' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_ld_a(dst, src+1, ldm, 1);

  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.load.b.row.stride.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_ld_b' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_ld_b(dst, src, ldm, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.load.b.col.stride.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_ld_b' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_ld_b(dst, src+2, ldm, 1);

  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.load.c.row.stride.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_ld_c_f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_ld_c_f16(dst, src, ldm, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.load.c.col.stride.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_ld_c_f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_ld_c_f16(dst, src, ldm, 1);

  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.load.c.row.stride.f32
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_ld_c_f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_ld_c_f32(fdst, fsrc, ldm, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.load.c.col.stride.f32
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_ld_c_f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_ld_c_f32(fdst, fsrc, ldm, 1);

  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.store.d.row.stride.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_st_c_f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_st_c_f16(dst, src, ldm, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.store.d.col.stride.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_st_c_f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_st_c_f16(dst, src, ldm, 1);

  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.store.d.row.stride.f32
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_st_c_f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_st_c_f32(fdst, fsrc, ldm, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.store.d.col.stride.f32
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_st_c_f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_st_c_f32(fdst, fsrc, ldm, 1);

  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.row.row.f16.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f16f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f16f16(dst, src, src, src, 0, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.row.row.f16.f16.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f16f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f16f16(dst, src, src, src, 0, 1);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.row.col.f16.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f16f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f16f16(dst, src, src, src, 1, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.row.col.f16.f16.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f16f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f16f16(dst, src, src, src, 1, 1);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.col.row.f16.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f16f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f16f16(dst, src, src, src, 2, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.col.row.f16.f16.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f16f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f16f16(dst, src, src, src, 2, 1);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.col.col.f16.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f16f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f16f16(dst, src, src, src, 3, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.col.col.f16.f16.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f16f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f16f16(dst, src, src, src, 3, 1);

  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.row.row.f16.f32
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f16f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f16f32(dst, src, src, fsrc, 0, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.row.row.f16.f32.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f16f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f16f32(dst, src, src, fsrc, 0, 1);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.row.col.f16.f32
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f16f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f16f32(dst, src, src, fsrc, 1, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.row.col.f16.f32.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f16f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f16f32(dst, src, src, fsrc, 1, 1);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.col.row.f16.f32
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f16f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f16f32(dst, src, src, fsrc, 2, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.col.row.f16.f32.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f16f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f16f32(dst, src, src, fsrc, 2, 1);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.col.col.f16.f32
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f16f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f16f32(dst, src, src, fsrc, 3, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.col.col.f16.f32.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f16f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f16f32(dst, src, src, fsrc, 3, 1);

  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.row.row.f32.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f32f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f32f16(fdst, src, src, src, 0, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.row.row.f32.f16.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f32f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f32f16(fdst, src, src, src, 0, 1);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.row.col.f32.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f32f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f32f16(fdst, src, src, src, 1, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.row.col.f32.f16.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f32f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f32f16(fdst, src, src, src, 1, 1);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.col.row.f32.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f32f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f32f16(fdst, src, src, src, 2, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.col.row.f32.f16.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f32f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f32f16(fdst, src, src, src, 2, 1);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.col.col.f32.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f32f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f32f16(fdst, src, src, src, 3, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.col.col.f32.f16.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f32f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f32f16(fdst, src, src, src, 3, 1);

  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.row.row.f32.f32
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f32f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f32f32(fdst, src, src, fsrc, 0, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.row.row.f32.f32.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f32f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f32f32(fdst, src, src, fsrc, 0, 1);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.row.col.f32.f32
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f32f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f32f32(fdst, src, src, fsrc, 1, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.row.col.f32.f32.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f32f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f32f32(fdst, src, src, fsrc, 1, 1);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.col.row.f32.f32
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f32f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f32f32(fdst, src, src, fsrc, 2, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.col.row.f32.f32.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f32f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f32f32(fdst, src, src, fsrc, 2, 1);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.col.col.f32.f32
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f32f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f32f32(fdst, src, src, fsrc, 3, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m32n8k16.mma.col.col.f32.f32.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m32n8k16_mma_f32f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m32n8k16_mma_f32f32(fdst, src, src, fsrc, 3, 1);


  // m8n32k16 variants.

  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.load.a.row.stride.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_ld_a' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_ld_a(dst, src, ldm, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.load.a.col.stride.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_ld_a' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_ld_a(dst, src+1, ldm, 1);

  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.load.b.row.stride.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_ld_b' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_ld_b(dst, src, ldm, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.load.b.col.stride.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_ld_b' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_ld_b(dst, src+2, ldm, 1);

  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.load.c.row.stride.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_ld_c_f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_ld_c_f16(dst, src, ldm, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.load.c.col.stride.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_ld_c_f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_ld_c_f16(dst, src, ldm, 1);

  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.load.c.row.stride.f32
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_ld_c_f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_ld_c_f32(fdst, fsrc, ldm, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.load.c.col.stride.f32
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_ld_c_f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_ld_c_f32(fdst, fsrc, ldm, 1);

  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.store.d.row.stride.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_st_c_f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_st_c_f16(dst, src, ldm, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.store.d.col.stride.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_st_c_f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_st_c_f16(dst, src, ldm, 1);

  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.store.d.row.stride.f32
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_st_c_f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_st_c_f32(fdst, fsrc, ldm, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.store.d.col.stride.f32
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_st_c_f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_st_c_f32(fdst, fsrc, ldm, 1);

  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.row.row.f16.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f16f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f16f16(dst, src, src, src, 0, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.row.row.f16.f16.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f16f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f16f16(dst, src, src, src, 0, 1);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.row.col.f16.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f16f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f16f16(dst, src, src, src, 1, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.row.col.f16.f16.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f16f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f16f16(dst, src, src, src, 1, 1);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.col.row.f16.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f16f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f16f16(dst, src, src, src, 2, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.col.row.f16.f16.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f16f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f16f16(dst, src, src, src, 2, 1);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.col.col.f16.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f16f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f16f16(dst, src, src, src, 3, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.col.col.f16.f16.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f16f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f16f16(dst, src, src, src, 3, 1);

  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.row.row.f16.f32
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f16f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f16f32(dst, src, src, fsrc, 0, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.row.row.f16.f32.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f16f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f16f32(dst, src, src, fsrc, 0, 1);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.row.col.f16.f32
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f16f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f16f32(dst, src, src, fsrc, 1, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.row.col.f16.f32.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f16f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f16f32(dst, src, src, fsrc, 1, 1);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.col.row.f16.f32
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f16f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f16f32(dst, src, src, fsrc, 2, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.col.row.f16.f32.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f16f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f16f32(dst, src, src, fsrc, 2, 1);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.col.col.f16.f32
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f16f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f16f32(dst, src, src, fsrc, 3, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.col.col.f16.f32.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f16f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f16f32(dst, src, src, fsrc, 3, 1);

  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.row.row.f32.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f32f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f32f16(fdst, src, src, src, 0, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.row.row.f32.f16.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f32f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f32f16(fdst, src, src, src, 0, 1);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.row.col.f32.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f32f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f32f16(fdst, src, src, src, 1, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.row.col.f32.f16.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f32f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f32f16(fdst, src, src, src, 1, 1);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.col.row.f32.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f32f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f32f16(fdst, src, src, src, 2, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.col.row.f32.f16.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f32f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f32f16(fdst, src, src, src, 2, 1);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.col.col.f32.f16
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f32f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f32f16(fdst, src, src, src, 3, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.col.col.f32.f16.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f32f16' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f32f16(fdst, src, src, src, 3, 1);

  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.row.row.f32.f32
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f32f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f32f32(fdst, src, src, fsrc, 0, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.row.row.f32.f32.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f32f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f32f32(fdst, src, src, fsrc, 0, 1);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.row.col.f32.f32
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f32f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f32f32(fdst, src, src, fsrc, 1, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.row.col.f32.f32.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f32f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f32f32(fdst, src, src, fsrc, 1, 1);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.col.row.f32.f32
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f32f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f32f32(fdst, src, src, fsrc, 2, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.col.row.f32.f32.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f32f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f32f32(fdst, src, src, fsrc, 2, 1);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.col.col.f32.f32
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f32f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f32f32(fdst, src, src, fsrc, 3, 0);
  // CHECK_M32_M8: call {{.*}} @llvm.nvvm.wmma.m8n32k16.mma.col.col.f32.f32.satfinite
  // pre-ptx61-error-re@+1 {{'__hmma_m8n32k16_mma_f32f32' needs target feature (sm_70{{.*}},(ptx61{{.*}}}}
  __hmma_m8n32k16_mma_f32f32(fdst, src, src, fsrc, 3, 1);
}
#endif
