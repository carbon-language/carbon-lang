
#include <hip/hip_runtime.h>
// Tests that "sm_XX" gets correctly converted to "compute_YY" when we invoke
// fatbinary.
//
// REQUIRES: clang-driver
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM20 %s
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_21 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM21 %s
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_30 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM30 %s
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_32 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM32 %s
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_35 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM35 %s
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_37 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM37 %s
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_50 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM50 %s
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_52 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM52 %s
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_53 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM53 %s
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_60 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM60 %s
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_61 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM61 %s
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_62 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM62 %s
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_70 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM70 %s
// RUN: %clang -x hip -### -target x86_64-linux-gnu -c --cuda-gpu-arch=gfx600 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX600 %s
// RUN: %clang -x hip -### -target x86_64-linux-gnu -c --cuda-gpu-arch=gfx601 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX601 %s
// RUN: %clang -x hip -### -target x86_64-linux-gnu -c --cuda-gpu-arch=gfx602 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX602 %s
// RUN: %clang -x hip -### -target x86_64-linux-gnu -c --cuda-gpu-arch=gfx700 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX700 %s
// RUN: %clang -x hip -### -target x86_64-linux-gnu -c --cuda-gpu-arch=gfx701 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX701 %s
// RUN: %clang -x hip -### -target x86_64-linux-gnu -c --cuda-gpu-arch=gfx702 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX702 %s
// RUN: %clang -x hip -### -target x86_64-linux-gnu -c --cuda-gpu-arch=gfx703 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX703 %s
// RUN: %clang -x hip -### -target x86_64-linux-gnu -c --cuda-gpu-arch=gfx704 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX704 %s
// RUN: %clang -x hip -### -target x86_64-linux-gnu -c --cuda-gpu-arch=gfx705 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX705 %s
// RUN: %clang -x hip -### -target x86_64-linux-gnu -c --cuda-gpu-arch=gfx801 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX801 %s
// RUN: %clang -x hip -### -target x86_64-linux-gnu -c --cuda-gpu-arch=gfx802 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX802 %s
// RUN: %clang -x hip -### -target x86_64-linux-gnu -c --cuda-gpu-arch=gfx803 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX803 %s
// RUN: %clang -x hip -### -target x86_64-linux-gnu -c --cuda-gpu-arch=gfx805 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX805 %s
// RUN: %clang -x hip -### -target x86_64-linux-gnu -c --cuda-gpu-arch=gfx810 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX810 %s
// RUN: %clang -x hip -### -target x86_64-linux-gnu -c --cuda-gpu-arch=gfx900 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX900 %s
// RUN: %clang -x hip -### -target x86_64-linux-gnu -c --cuda-gpu-arch=gfx902 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX902 %s

// CUDA: ptxas
// CUDA-SAME: -m64
// CUDA: fatbinary

// HIP: clang-offload-bundler

// SM20:--image=profile=sm_20{{.*}}--image=profile=compute_20
// SM21:--image=profile=sm_21{{.*}}--image=profile=compute_20
// SM30:--image=profile=sm_30{{.*}}--image=profile=compute_30
// SM32:--image=profile=sm_32{{.*}}--image=profile=compute_32
// SM35:--image=profile=sm_35{{.*}}--image=profile=compute_35
// SM37:--image=profile=sm_37{{.*}}--image=profile=compute_37
// SM50:--image=profile=sm_50{{.*}}--image=profile=compute_50
// SM52:--image=profile=sm_52{{.*}}--image=profile=compute_52
// SM53:--image=profile=sm_53{{.*}}--image=profile=compute_53
// SM60:--image=profile=sm_60{{.*}}--image=profile=compute_60
// SM61:--image=profile=sm_61{{.*}}--image=profile=compute_61
// SM62:--image=profile=sm_62{{.*}}--image=profile=compute_62
// SM70:--image=profile=sm_70{{.*}}--image=profile=compute_70
// GFX600:-targets=host-x86_64-unknown-linux,hipv4-amdgcn-amd-amdhsa--gfx600
// GFX601:-targets=host-x86_64-unknown-linux,hipv4-amdgcn-amd-amdhsa--gfx601
// GFX602:-targets=host-x86_64-unknown-linux,hipv4-amdgcn-amd-amdhsa--gfx602
// GFX700:-targets=host-x86_64-unknown-linux,hipv4-amdgcn-amd-amdhsa--gfx700
// GFX701:-targets=host-x86_64-unknown-linux,hipv4-amdgcn-amd-amdhsa--gfx701
// GFX702:-targets=host-x86_64-unknown-linux,hipv4-amdgcn-amd-amdhsa--gfx702
// GFX703:-targets=host-x86_64-unknown-linux,hipv4-amdgcn-amd-amdhsa--gfx703
// GFX704:-targets=host-x86_64-unknown-linux,hipv4-amdgcn-amd-amdhsa--gfx704
// GFX705:-targets=host-x86_64-unknown-linux,hipv4-amdgcn-amd-amdhsa--gfx705
// GFX801:-targets=host-x86_64-unknown-linux,hipv4-amdgcn-amd-amdhsa--gfx801
// GFX802:-targets=host-x86_64-unknown-linux,hipv4-amdgcn-amd-amdhsa--gfx802
// GFX803:-targets=host-x86_64-unknown-linux,hipv4-amdgcn-amd-amdhsa--gfx803
// GFX805:-targets=host-x86_64-unknown-linux,hipv4-amdgcn-amd-amdhsa--gfx805
// GFX810:-targets=host-x86_64-unknown-linux,hipv4-amdgcn-amd-amdhsa--gfx810
// GFX900:-targets=host-x86_64-unknown-linux,hipv4-amdgcn-amd-amdhsa--gfx900
// GFX902:-targets=host-x86_64-unknown-linux,hipv4-amdgcn-amd-amdhsa--gfx902
