
#include <hip/hip_runtime.h>
// Check that profiling/coverage arguments doen't get passed down to device-side
// compilation.
//
// REQUIRES: clang-driver
//
// XRUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 \
// XRUN:   -fprofile-generate %s 2>&1 | \
// XRUN:   FileCheck --check-prefixes=CHECK,PROF %s
//
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 \
// RUN:   -fprofile-instr-generate %s 2>&1 | \
// RUN:   FileCheck --check-prefixes=CHECK,PROF %s
//
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 \
// RUN:   -coverage %s 2>&1 | \
// RUN:   FileCheck --check-prefixes=CHECK,GCOV %s
//
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 \
// RUN:   -ftest-coverage %s 2>&1 | \
// RUN:   FileCheck --check-prefixes=CHECK,GCOV %s
//
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20   \
// RUN:   -fprofile-instr-generate -fcoverage-mapping %s 2>&1 | \
// RUN:   FileCheck --check-prefixes=CHECK,PROF %s
//
//
// CHECK-NOT: error: unsupported option '-fprofile
// CHECK-NOT: error: invalid argument
// CHECK-DAG: "-fcuda-is-device"
// CHECK-NOT: "-f{{[^"]*coverage.*}}"
// CHECK-NOT: "-fprofile{{[^"]*}}"
// CHECK: "-triple" "x86_64-unknown-linux-gnu"
// PROF:      "-fprofile{{.*}}"
// GCOV:      "-ftest-coverage"
