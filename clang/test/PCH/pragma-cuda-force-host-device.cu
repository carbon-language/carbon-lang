
#include <hip/hip_runtime.h>
// RUN: %clang_cc1 -emit-pch %s -o %t
// RUN: %clang_cc1 -verify -verify-ignore-unexpected=note -include-pch %t -fsyntax-only %s

#ifndef HEADER
#define HEADER

#pragma clang force_cuda_host_device begin
#pragma clang force_cuda_host_device begin
#pragma clang force_cuda_host_device end

void hd1() {}

#else

void hd2() {}

#pragma clang force_cuda_host_device end

void host_only() {}

__attribute__((device)) void device() {
  hd1();
  hd2();
  host_only(); // expected-error {{no matching function for call}}
}

#endif
