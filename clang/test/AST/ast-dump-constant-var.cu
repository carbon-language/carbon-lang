// RUN: %clang_cc1 -std=c++14 -ast-dump -x hip %s | FileCheck -check-prefixes=CHECK,HOST %s
// RUN: %clang_cc1 -std=c++14 -ast-dump -fcuda-is-device -x hip %s | FileCheck -check-prefixes=CHECK,DEV %s

#include "Inputs/hip/hip_runtime.h"

// CHECK-LABEL: VarDecl {{.*}} m1 'int'
// CHECK-NEXT: CUDAConstantAttr {{.*}}cuda.h
__constant__ int m1;

// CHECK-LABEL: VarDecl {{.*}} m2 'int'
// CHECK-NEXT: CUDAConstantAttr {{.*}}cuda.h
// CHECK-NOT: CUDAConstantAttr
__constant__ __constant__ int m2;

// CHECK-LABEL: VarDecl {{.*}} m3 'const int'
// HOST-NOT: CUDAConstantAttr
// DEV-NOT: CUDAConstantAttr {{.*}}cuda.h
// DEV: CUDAConstantAttr {{.*}}Implicit
// DEV-NOT: CUDAConstantAttr {{.*}}cuda.h
constexpr int m3 = 1;

// CHECK-LABEL: VarDecl {{.*}} m3a 'const int'
// CHECK-NOT: CUDAConstantAttr {{.*}}Implicit
// CHECK: CUDAConstantAttr {{.*}}cuda.h
// CHECK-NOT: CUDAConstantAttr {{.*}}Implicit
constexpr __constant__ int m3a = 2;

// CHECK-LABEL: VarDecl {{.*}} m3b 'const int'
// CHECK-NOT: CUDAConstantAttr {{.*}}Implicit
// CHECK: CUDAConstantAttr {{.*}}cuda.h
// CHECK-NOT: CUDAConstantAttr {{.*}}Implicit
__constant__ constexpr int m3b = 3;
