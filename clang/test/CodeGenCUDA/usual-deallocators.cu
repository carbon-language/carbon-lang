#include "hip/hip_runtime.h"
// RUN: %clang_cc1 %s --std=c++11 -triple nvptx-unknown-unknown -fcuda-is-device \
// RUN:   -emit-llvm -o - | FileCheck %s --check-prefixes=COMMON,DEVICE
// RUN: %clang_cc1 %s --std=c++11 -triple nvptx-unknown-unknown \
// RUN:   -emit-llvm -o - | FileCheck %s --check-prefixes=COMMON,HOST
// RUN: %clang_cc1 %s --std=c++17 -triple nvptx-unknown-unknown -fcuda-is-device \
// RUN:   -emit-llvm -o - | FileCheck %s --check-prefixes=COMMON,DEVICE
// RUN: %clang_cc1 %s --std=c++17 -triple nvptx-unknown-unknown \
// RUN:   -emit-llvm -o - | FileCheck %s --check-prefixes=COMMON,HOST

#include "Inputs/hip/hip_runtime.h"
extern "C" __host__ void host_fn();
extern "C" __device__ void dev_fn();
extern "C" __host__ __device__ void hd_fn();

// Destructors are handled a bit differently, compared to regular functions.
// Make sure we do trigger kernel generation on the GPU side even if it's only
// referenced by the destructor.
template<typename T> __global__ void f(T) {}
template<typename T> struct A {
  ~A() { f<<<1, 1>>>(T()); }
};

// HOST-LABEL: @a
A<int> a;
// HOST-LABEL: define linkonce_odr void @_ZN1AIiED1Ev
// search further down for the deice-side checks for @_Z1fIiEvT_

struct H1D1 {
  __host__ void operator delete(void *) { host_fn(); };
  __device__ void operator delete(void *) { dev_fn(); };
};

struct H1D2 {
  __host__ void operator delete(void *) { host_fn(); };
  __device__ void operator delete(void *, __SIZE_TYPE__) { dev_fn(); };
};

struct H2D1 {
  __host__ void operator delete(void *, __SIZE_TYPE__) { host_fn(); };
  __device__ void operator delete(void *) { dev_fn(); };
};

struct H2D2 {
  __host__ void operator delete(void *, __SIZE_TYPE__) { host_fn(); };
  __device__ void operator delete(void *, __SIZE_TYPE__) { dev_fn(); };
};

struct H1D1D2 {
  __host__ void operator delete(void *) { host_fn(); };
  __device__ void operator delete(void *) { dev_fn(); };
  __device__ void operator delete(void *, __SIZE_TYPE__) { dev_fn(); };
};

struct H1H2D1 {
  __host__ void operator delete(void *) { host_fn(); };
  __host__ void operator delete(void *, __SIZE_TYPE__) { host_fn(); };
  __device__ void operator delete(void *) { dev_fn(); };
};

struct H1H2D2 {
  __host__ void operator delete(void *) { host_fn(); };
  __host__ void operator delete(void *, __SIZE_TYPE__) { host_fn(); };
  __device__ void operator delete(void *, __SIZE_TYPE__) { dev_fn(); };
};

struct H1H2D1D2 {
  __host__ void operator delete(void *) { host_fn(); };
  __host__ void operator delete(void *, __SIZE_TYPE__) { host_fn(); };
  __device__ void operator delete(void *) { dev_fn(); };
  __device__ void operator delete(void *, __SIZE_TYPE__) { dev_fn(); };
};


template <typename T>
__host__ __device__ void test_hd(void *p) {
  T *t = (T *)p;
  delete t;
}

// Make sure we call the right variant of usual deallocator.
__host__ __device__ void tests_hd(void *t) {
  // COMMON-LABEL: define linkonce_odr void @_Z7test_hdI4H1D1EvPv
  // COMMON: call void @_ZN4H1D1dlEPv
  test_hd<H1D1>(t);
  // COMMON-LABEL: define linkonce_odr void @_Z7test_hdI4H1D2EvPv
  // DEVICE: call void @_ZN4H1D2dlEPvj(i8* {{.*}}, i32 1)
  // HOST:   call void @_ZN4H1D2dlEPv(i8* {{.*}})
  test_hd<H1D2>(t);
  // COMMON-LABEL: define linkonce_odr void @_Z7test_hdI4H2D1EvPv
  // DEVICE: call void @_ZN4H2D1dlEPv(i8* {{.*}})
  // HOST:   call void @_ZN4H2D1dlEPvj(i8* %3, i32 1)
  test_hd<H2D1>(t);
  // COMMON-LABEL: define linkonce_odr void @_Z7test_hdI4H2D2EvPv
  // COMMON: call void @_ZN4H2D2dlEPvj(i8* {{.*}}, i32 1)
  test_hd<H2D2>(t);
  // COMMON-LABEL: define linkonce_odr void @_Z7test_hdI6H1D1D2EvPv
  // COMMON: call void @_ZN6H1D1D2dlEPv(i8* %3)
  test_hd<H1D1D2>(t);
  // COMMON-LABEL: define linkonce_odr void @_Z7test_hdI6H1H2D1EvPv
  // COMMON: call void @_ZN6H1H2D1dlEPv(i8* {{.*}})
  test_hd<H1H2D1>(t);
  // COMMON-LABEL: define linkonce_odr void @_Z7test_hdI6H1H2D2EvPv
  // DEVICE: call void @_ZN6H1H2D2dlEPvj(i8* {{.*}}, i32 1)
  // HOST:   call void @_ZN6H1H2D2dlEPv(i8* {{.*}})
  test_hd<H1H2D2>(t);
  // COMMON-LABEL: define linkonce_odr void @_Z7test_hdI8H1H2D1D2EvPv
  // COMMON: call void @_ZN8H1H2D1D2dlEPv(i8* {{.*}})
  test_hd<H1H2D1D2>(t);
}

// Make sure that we've generated the kernel used by A::~A.
// DEVICE-LABEL: define dso_local void @_Z1fIiEvT_

// Make sure we've picked deallocator for the correct side of compilation.

// COMMON-LABEL: define  linkonce_odr void @_ZN4H1D1dlEPv(i8* %0)
// DEVICE: call void @dev_fn()
// HOST:   call void @host_fn()

// DEVICE-LABEL: define  linkonce_odr void @_ZN4H1D2dlEPvj(i8* %0, i32 %1)
// DEVICE: call void @dev_fn()
// HOST-LABEL: define linkonce_odr void @_ZN4H1D2dlEPv(i8* %0)
// HOST: call void @host_fn()

// DEVICE-LABEL: define  linkonce_odr void @_ZN4H2D1dlEPv(i8* %0)
// DEVICE: call void @dev_fn()
// HOST-LABEL:  define linkonce_odr void @_ZN4H2D1dlEPvj(i8* %0, i32 %1)
// HOST: call void @host_fn()

// COMMON-LABEL: define  linkonce_odr void @_ZN4H2D2dlEPvj(i8* %0, i32 %1)
// DEVICE: call void @dev_fn()
// HOST: call void @host_fn()

// COMMON-LABEL: define  linkonce_odr void @_ZN6H1D1D2dlEPv(i8* %0)
// DEVICE: call void @dev_fn()
// HOST: call void @host_fn()

// COMMON-LABEL: define  linkonce_odr void @_ZN6H1H2D1dlEPv(i8* %0)
// DEVICE: call void @dev_fn()
// HOST: call void @host_fn()

// DEVICE-LABEL: define  linkonce_odr void @_ZN6H1H2D2dlEPvj(i8* %0, i32 %1)
// DEVICE: call void @dev_fn()
// HOST-LABEL: define linkonce_odr void @_ZN6H1H2D2dlEPv(i8* %0)
// HOST: call void @host_fn()

// COMMON-LABEL: define  linkonce_odr void @_ZN8H1H2D1D2dlEPv(i8* %0)
// DEVICE: call void @dev_fn()
// HOST: call void @host_fn()

// DEVICE: !0 = !{void (i32)* @_Z1fIiEvT_, !"kernel", i32 1}
