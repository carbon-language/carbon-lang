
#include <hip/hip_runtime.h>
// Create a sample address sanitizer bitcode library.

// RUN: %clang_cc1 -x ir -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm-bc \
// RUN:   -disable-llvm-passes -o %t.asanrtl.bc %S/Inputs/amdgpu-asanrtl.ll

// Check sanitizer runtime library functions survive
// optimizations without being removed or parameters altered.

// RUN: %clang_cc1 %s -emit-llvm -o - -triple=amdgcn-amd-amdhsa \
// RUN:   -fcuda-is-device -target-cpu gfx906 -fsanitize=address \
// RUN:   -mlink-bitcode-file %t.asanrtl.bc -x hip \
// RUN:   | FileCheck -check-prefixes=ASAN %s

// RUN: %clang_cc1 %s -emit-llvm -o - -triple=amdgcn-amd-amdhsa \
// RUN:   -fcuda-is-device -target-cpu gfx906 -fsanitize=address \
// RUN:   -O3 -mlink-bitcode-file %t.asanrtl.bc -x hip \
// RUN:   | FileCheck -check-prefixes=ASAN %s

// RUN: %clang_cc1 %s -emit-llvm -o - -triple=amdgcn-amd-amdhsa \
// RUN:   -fcuda-is-device -target-cpu gfx906 -x hip \
// RUN:   | FileCheck %s

// REQUIRES: amdgpu-registered-target

// ASAN-DAG: define weak void @__amdgpu_device_library_preserve_asan_functions()
// ASAN-DAG: @__amdgpu_device_library_preserve_asan_functions_ptr = weak addrspace(1) constant void ()* @__amdgpu_device_library_preserve_asan_functions
// ASAN-DAG: @llvm.compiler.used = {{.*}}@__amdgpu_device_library_preserve_asan_functions_ptr
// ASAN-DAG: define weak void @__asan_report_load1(i64 %{{.*}})

// CHECK-NOT: @__amdgpu_device_library_preserve_asan_functions
// CHECK-NOT: @__asan_report_load1
