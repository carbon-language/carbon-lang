
#include <hip/hip_runtime.h>
// RUN: %clang_cc1 %s -emit-llvm -o - -triple=amdgcn-amd-amdhsa \
// RUN:   -fcuda-is-device -target-cpu gfx906 -fsanitize=address \
// RUN:   -x hip | FileCheck -check-prefix=ASAN %s

// RUN: %clang_cc1 %s -emit-llvm -o - -triple=amdgcn-amd-amdhsa \
// RUN:   -fcuda-is-device -target-cpu gfx906 -x hip \
// RUN:   | FileCheck %s

// REQUIRES: amdgpu-registered-target

// ASAN-DAG: declare void @__amdgpu_device_library_preserve_asan_functions()
// ASAN-DAG: @__amdgpu_device_library_preserve_asan_functions_ptr = weak addrspace(1) constant void ()* @__amdgpu_device_library_preserve_asan_functions
// ASAN-DAG: @llvm.compiler.used = {{.*}}@__amdgpu_device_library_preserve_asan_functions_ptr

// CHECK-NOT: @__amdgpu_device_library_preserve_asan_functions_ptr
