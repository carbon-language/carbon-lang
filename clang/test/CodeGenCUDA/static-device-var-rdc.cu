#include "hip/hip_runtime.h"
// REQUIRES: x86-registered-target
// REQUIRES: amdgpu-registered-target

// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device \
// RUN:   -std=c++11 -fgpu-rdc -emit-llvm -o - -x hip %s | FileCheck \
// RUN:   -check-prefixes=DEV,INT-DEV %s

// RUN: %clang_cc1 -triple x86_64-gnu-linux \
// RUN:   -std=c++11 -fgpu-rdc -emit-llvm -o - -x hip %s | FileCheck \
// RUN:   -check-prefixes=HOST,INT-HOST %s

// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -cuid=abc \
// RUN:   -std=c++11 -fgpu-rdc -emit-llvm -o - -x hip %s > %t.dev
// RUN: cat %t.dev | FileCheck -check-prefixes=DEV,EXT-DEV %s

// RUN: %clang_cc1 -triple x86_64-gnu-linux -cuid=abc \
// RUN:   -std=c++11 -fgpu-rdc -emit-llvm -o - -x hip %s > %t.host
// RUN: cat %t.host | FileCheck -check-prefixes=HOST,EXT-HOST %s

// Check host and device compilations use the same postfixes for static
// variable names.

// RUN: cat %t.dev %t.host | FileCheck -check-prefix=POSTFIX %s

// Negative tests.

// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device \
// RUN:   -std=c++11 -fgpu-rdc -emit-llvm -o - -x hip %s | FileCheck \
// RUN:   -check-prefix=DEV-NEG %s

// RUN: %clang_cc1 -triple x86_64-gnu-linux \
// RUN:   -std=c++11 -fgpu-rdc -emit-llvm -o - -x hip %s | FileCheck \
// RUN:   -check-prefix=HOST-NEG %s

// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -cuid=abc \
// RUN:   -std=c++11 -fgpu-rdc -emit-llvm -o - -x hip %s > %t.dev
// RUN: cat %t.dev | FileCheck -check-prefix=DEV-NEG %s

// RUN: %clang_cc1 -triple x86_64-gnu-linux -cuid=abc \
// RUN:   -std=c++11 -fgpu-rdc -emit-llvm -o - -x hip %s > %t.host
// RUN: cat %t.host | FileCheck -check-prefix=HOST-NEG %s


#include "Inputs/hip/hip_runtime.h"

// Test function scope static device variable, which should not be externalized.
// DEV-DAG: @_ZZ6kernelPiPPKiE1w = internal addrspace(4) constant i32 1


// HOST-DAG: @_ZL1x = internal global i32 undef
// HOST-DAG: @_ZL1y = internal global i32 undef

// Test normal static device variables
// INT-DEV-DAG: @_ZL1x = addrspace(1) externally_initialized global i32 0
// INT-HOST-DAG: @[[DEVNAMEX:[0-9]+]] = {{.*}}c"_ZL1x\00"

// Test externalized static device variables
// EXT-DEV-DAG: @_ZL1x__static__[[HASH:.*]] = addrspace(1) externally_initialized global i32 0
// EXT-HOST-DAG: @[[DEVNAMEX:[0-9]+]] = {{.*}}c"_ZL1x__static__[[HASH:.*]]\00"

// POSTFIX: @_ZL1x__static__[[HASH:.*]] = addrspace(1) externally_initialized global i32 0
// POSTFIX: @[[DEVNAMEX:[0-9]+]] = {{.*}}c"_ZL1x__static__[[HASH]]\00"

static __device__ int x;

// Test static device variables not used by host code should not be externalized
// DEV-DAG: @_ZL2x2 = internal addrspace(1) global i32 0

static __device__ int x2;

// Test normal static device variables
// INT-DEV-DAG: @_ZL1y = addrspace(4) externally_initialized global i32 0
// INT-HOST-DAG: @[[DEVNAMEY:[0-9]+]] = {{.*}}c"_ZL1y\00"

// Test externalized static device variables
// EXT-DEV-DAG: @_ZL1y__static__[[HASH]] = addrspace(4) externally_initialized global i32 0
// EXT-HOST-DAG: @[[DEVNAMEY:[0-9]+]] = {{.*}}c"_ZL1y__static__[[HASH]]\00"

static __constant__ int y;

// Test static host variable, which should not be externalized nor registered.
// HOST-DAG: @_ZL1z = internal global i32 0
// DEV-NEG-NOT: @_ZL1z
static int z;

// Test non-ODR-use of static device variable is not emitted or registered.
// DEV-NEG-NOT: @_ZL1u
// HOST-NEG-NOT: @_ZL1u
static __device__ int u;

// Test static device variable in inline function, which should not be
// externalized nor registered.
// DEV-DAG: @_ZZ6devfunPPKiE1p = linkonce_odr addrspace(4) constant i32 2, comdat

inline __device__ void devfun(const int ** b) {
  const static int p = 2;
  b[0] = &p;
}

__global__ void kernel(int *a, const int **b) {
  const static int w = 1;
  a[0] = x;
  a[1] = y;
  a[2] = sizeof(u);
  b[0] = &w;
  b[1] = &x2;
  devfun(b);
}

int* getDeviceSymbol(int *x);

void foo() {
  getDeviceSymbol(&x);
  getDeviceSymbol(&y);
  z = 123;
  decltype(u) tmp;
}

// HOST-DAG: __hipRegisterVar({{.*}}@_ZL1x {{.*}}@[[DEVNAMEX]]
// HOST-DAG: __hipRegisterVar({{.*}}@_ZL1y {{.*}}@[[DEVNAMEY]]
// HOST-NEG-NOT: __hipRegisterVar({{.*}}@_ZL2x2
// HOST-NEG-NOT: __hipRegisterVar({{.*}}@_ZZ6kernelPiPPKiE1w
// HOST-NEG-NOT: __hipRegisterVar({{.*}}@_ZZ6devfunPPKiE1p
// HOST-NEG-NOT: __hipRegisterVar({{.*}}@_ZL1u
