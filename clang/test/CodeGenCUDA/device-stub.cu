#include "hip/hip_runtime.h"
// RUN: echo "GPU binary would be here" > %t
// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s \
// RUN:     -target-sdk-version=8.0 -fcuda-include-gpubinary %t -o - \
// RUN:   | FileCheck -allow-deprecated-dag-overlap %s \
// RUN:       --check-prefixes=ALL,LNX,NORDC,CUDA,CUDANORDC,CUDA-OLD
// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s \
// RUN:     -target-sdk-version=8.0  -fcuda-include-gpubinary %t \
// RUN:     -o - -DNOGLOBALS \
// RUN:   | FileCheck -allow-deprecated-dag-overlap %s \
// RUN:     -check-prefixes=NOGLOBALS,CUDANOGLOBALS
// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s \
// RUN:     -target-sdk-version=8.0 -fgpu-rdc -fcuda-include-gpubinary %t \
// RUN:     -o - \
// RUN:   | FileCheck -allow-deprecated-dag-overlap %s \
// RUN:       --check-prefixes=ALL,LNX,RDC,CUDA,CUDARDC,CUDA-OLD
// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s \
// RUN:     -target-sdk-version=8.0 -o - \
// RUN:   | FileCheck -allow-deprecated-dag-overlap %s -check-prefix=NOGPUBIN

// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s       \
// RUN:     -target-sdk-version=9.2 -fcuda-include-gpubinary %t -o - \
// RUN:   | FileCheck %s -allow-deprecated-dag-overlap \
// RUN:       --check-prefixes=ALL,LNX,NORDC,CUDA,CUDANORDC,CUDA-NEW
// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s \
// RUN:     -target-sdk-version=9.2 -fcuda-include-gpubinary %t -o -  -DNOGLOBALS \
// RUN:   | FileCheck -allow-deprecated-dag-overlap %s \
// RUN:       --check-prefixes=NOGLOBALS,CUDANOGLOBALS
// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s \
// RUN:     -target-sdk-version=9.2 -fgpu-rdc -fcuda-include-gpubinary %t -o - \
// RUN:   | FileCheck %s -allow-deprecated-dag-overlap \
// RUN:       --check-prefixes=ALL,LNX,RDC,CUDA,CUDARDC,CUDA-NEW
// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s -std=c++17 \
// RUN:     -target-sdk-version=9.2 -fcuda-include-gpubinary %t -o - \
// RUN:   | FileCheck %s -allow-deprecated-dag-overlap \
// RUN:       --check-prefixes=ALL,LNX,NORDC,CUDA,CUDANORDC,CUDA-NEW,LNX_17,NORDC17
// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s -std=c++17 \
// RUN:     -target-sdk-version=9.2 -fgpu-rdc -fcuda-include-gpubinary %t -o - \
// RUN:   | FileCheck %s -allow-deprecated-dag-overlap \
// RUN:       --check-prefixes=ALL,LNX,RDC,CUDA,CUDARDC,CUDA-NEW,LNX_17,RDC17
// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s \
// RUN:     -target-sdk-version=9.2 -o - \
// RUN:   | FileCheck -allow-deprecated-dag-overlap %s -check-prefix=NOGPUBIN

// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s \
// RUN:     -fcuda-include-gpubinary %t -o - -x hip\
// RUN:   | FileCheck -allow-deprecated-dag-overlap %s --check-prefixes=ALL,LNX,NORDC,HIP,HIPEF
// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s \
// RUN:     -fcuda-include-gpubinary %t -o -  -DNOGLOBALS -x hip \
// RUN:   | FileCheck -allow-deprecated-dag-overlap %s -check-prefixes=NOGLOBALS,HIPNOGLOBALS
// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s \
// RUN:     -fgpu-rdc -fcuda-include-gpubinary %t -o - -x hip \
// RUN:   | FileCheck -allow-deprecated-dag-overlap %s --check-prefixes=ALL,LNX,RDC,HIP,HIPEF
// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s -o - -x hip\
// RUN:   | FileCheck -allow-deprecated-dag-overlap %s -check-prefixes=ALL,LNX,NORDC,HIP,HIPNEF

// RUN: %clang_cc1 -triple x86_64-pc-windows-msvc -aux-triple amdgcn -emit-llvm %s \
// RUN:     -fcuda-include-gpubinary %t -o - -x hip\
// RUN:   | FileCheck -allow-deprecated-dag-overlap %s --check-prefixes=ALL,WIN

#include "Inputs/hip/hip_runtime.h"

#ifndef NOGLOBALS
// NORDC-DAG: @device_var = internal global i32
// RDC-DAG: @device_var = global i32
// WIN-DAG: @"?device_var@@3HA" = internal global i32
__device__ int device_var;

// NORDC-DAG: @constant_var = internal global i32
// RDC-DAG: @constant_var = global i32
// WIN-DAG: @"?constant_var@@3HA" = internal global i32
__constant__ int constant_var;

// NORDC-DAG: @shared_var = internal global i32
// RDC-DAG: @shared_var = global i32
// WIN-DAG: @"?shared_var@@3HA" = internal global i32
__shared__ int shared_var;

// Make sure host globals don't get internalized...
// LNX-DAG: @host_var ={{.*}} global i32
// WIN-DAG: @"?host_var@@3HA" = dso_local global i32
int host_var;
// ... and that extern vars remain external.
// LNX-DAG: @ext_host_var = external global i32
// WIN-DAG: @"?ext_host_var@@3HA" = external dso_local global i32
extern int ext_host_var;

// external device-side variables -> extern references to their shadows.
// LNX-DAG: @ext_device_var = external global i32
// WIN-DAG: @"?ext_device_var@@3HA" = external dso_local global i32
extern __device__ int ext_device_var;
// LNX-DAG: @ext_device_var = external global i32
// WIN-DAG: @"?ext_constant_var@@3HA" = external dso_local global i32
extern __constant__ int ext_constant_var;

// external device-side variables with definitions should generate
// definitions for the shadows.
// NORDC-DAG: @ext_device_var_def = internal global i32 undef,
// RDC-DAG: @ext_device_var_def = global i32 undef,
// WIN-DAG: @"?ext_device_var_def@@3HA" = internal global i32 undef
extern __device__ int ext_device_var_def;
__device__ int ext_device_var_def = 1;
// NORDC-DAG: @ext_device_var_def = internal global i32 undef,
// RDC-DAG: @ext_device_var_def = global i32 undef,
// WIN-DAG: @"?ext_constant_var_def@@3HA" = internal global i32 undef
__constant__ int ext_constant_var_def = 2;

#if __cplusplus > 201402L
// NORDC17: @inline_var = internal global i32 undef, comdat, align 4{{$}}
// RDC17: @inline_var = linkonce_odr global i32 undef, comdat, align 4{{$}}
// NORDC17-NOT: @inline_var2 =
// RDC17-NOT: @inline_var2 =
// NORDC17: @_ZN1C17member_inline_varE = internal constant i32 undef, comdat, align 4{{$}}
// RDC17: @_ZN1C17member_inline_varE = linkonce_odr constant i32 undef, comdat, align 4{{$}}
// Check inline variable ODR-used by host is emitted on host and registered.
__device__ inline int inline_var = 3;
// Check inline variable not ODR-used by host is not emitted on host or registered.
__device__ inline int inline_var2 = 5;
struct C {
  __device__ static constexpr int member_inline_var = 4;
};
#endif

void use_pointers() {
  const int *p;
  p = &device_var;
  p = &constant_var;
  p = &shared_var;
  p = &host_var;
  p = &ext_device_var;
  p = &ext_constant_var;
  p = &ext_host_var;
#if __cplusplus > 201402L
  p = &inline_var;
  decltype(inline_var2) tmp;
  p = &C::member_inline_var;
#endif
}

__device__ void device_use() {
#if __cplusplus > 201402L
  const int *p = &inline_var2;
#endif
}

// Make sure that all parts of GPU code init/cleanup are there:
// * constant unnamed string with the device-side kernel name to be passed to
//   __hipRegisterFunction/__cudaRegisterFunction.
// ALL: @0 = private unnamed_addr constant [18 x i8] c"_Z10kernelfunciii\00"
// * constant unnamed string with the device-side kernel name to be passed to
//   __hipRegisterVar/__cudaRegisterVar.
// ALL: @1 = private unnamed_addr constant [11 x i8] c"device_var\00"
// ALL: @2 = private unnamed_addr constant [13 x i8] c"constant_var\00"
// ALL: @3 = private unnamed_addr constant [19 x i8] c"ext_device_var_def\00"
// ALL: @4 = private unnamed_addr constant [21 x i8] c"ext_constant_var_def\00"
// * constant unnamed string with GPU binary
// CUDA: @[[FATBIN:.*]] = private constant{{.*GPU binary would be here.*}}\00",
// HIPEF: @[[FATBIN:.*]] = private constant{{.*GPU binary would be here.*}}\00",{{.*}}align 4096
// HIPNEF: @[[FATBIN:__hip_fatbin]] = external constant i8, section ".hip_fatbin"
// CUDANORDC-SAME: section ".nv_fatbin", align 8
// CUDARDC-SAME: section "__nv_relfatbin", align 8
// * constant struct that wraps GPU binary
// ALL: @__[[PREFIX:cuda|hip]]_fatbin_wrapper = internal constant
// LNX-SAME: { i32, i32, i8*, i8* }
// CUDA-SAME: { i32 1180844977, i32 1,
// HIP-SAME: { i32 1212764230, i32 1,
// CUDA-SAME: i8* getelementptr inbounds ({{.*}}@[[FATBIN]], i64 0, i64 0),
// HIPEF-SAME: i8* getelementptr inbounds ({{.*}}@[[FATBIN]], i64 0, i64 0),
// HIPNEF-SAME:  i8* @[[FATBIN]],
// LNX-SAME: i8* null }
// CUDA-SAME: section ".nvFatBinSegment"
// HIP-SAME: section ".hipFatBinSegment"
// * variable to save GPU binary handle after initialization
// CUDANORDC: @__[[PREFIX]]_gpubin_handle = internal global i8** null
// HIPNEF: @__[[PREFIX]]_gpubin_handle = linkonce hidden global i8** null
// * constant unnamed string with NVModuleID
// CUDARDC: [[MODULE_ID_GLOBAL:@.*]] = private constant
// CUDARDC-SAME: c"[[MODULE_ID:.+]]\00", section "__nv_module_id", align 32
// * Make sure our constructor was added to global ctor list.
// LNX: @llvm.global_ctors = appending global {{.*}}@__[[PREFIX]]_module_ctor
// * Alias to global symbol containing the NVModuleID.
// CUDARDC: @__fatbinwrap[[MODULE_ID]] ={{.*}} alias { i32, i32, i8*, i8* }
// CUDARDC-SAME: { i32, i32, i8*, i8* }* @__[[PREFIX]]_fatbin_wrapper

// Test that we build the correct number of calls to hipSetupArgument followed
// by a call to hipLaunchByPtr.

// LNX: define{{.*}}kernelfunc

// New launch sequence stores arguments into local buffer and passes array of
// pointers to them directly to hipLaunchKernel
// CUDA-NEW: alloca
// CUDA-NEW: store
// CUDA-NEW: store
// CUDA-NEW: store
// CUDA-NEW: call{{.*}}__cudaPopCallConfiguration
// CUDA-NEW: call{{.*}}hipLaunchKernel

// Legacy style launch sequence sets up arguments by passing them to
// [cuda|hip]SetupArgument.
// CUDA-OLD: call{{.*}}[[PREFIX]]SetupArgument
// CUDA-OLD: call{{.*}}[[PREFIX]]SetupArgument
// CUDA-OLD: call{{.*}}[[PREFIX]]SetupArgument
// CUDA-OLD: call{{.*}}[[PREFIX]]Launch

// HIP: call{{.*}}[[PREFIX]]SetupArgument
// HIP: call{{.*}}[[PREFIX]]SetupArgument
// HIP: call{{.*}}[[PREFIX]]SetupArgument
// HIP: call{{.*}}[[PREFIX]]Launch
__global__ void kernelfunc(int i, int j, int k) {}

// Test that we've built correct kernel launch sequence.
// LNX: define{{.*}}hostfunc
// CUDA-OLD: call{{.*}}[[PREFIX]]ConfigureCall
// CUDA-NEW: call{{.*}}__cudaPushCallConfiguration
// HIP: call{{.*}}[[PREFIX]]ConfigureCall
// LNX: call{{.*}}kernelfunc
void hostfunc(void) { kernelfunc<<<1, 1>>>(1, 1, 1); }
#endif

// Test that we've built a function to register kernels and global vars.
// ALL: define internal void @__[[PREFIX]]_register_globals
// ALL: call{{.*}}[[PREFIX]]RegisterFunction(i8** %0, {{.*}}kernelfunc{{[^,]*}}, {{[^@]*}}@0
// ALL-DAG: call void {{.*}}[[PREFIX]]RegisterVar(i8** %0, {{.*}}device_var{{[^,]*}}, {{[^@]*}}@1, {{.*}}i32 0, {{i32|i64}} 4, i32 0, i32 0
// ALL-DAG: call void {{.*}}[[PREFIX]]RegisterVar(i8** %0, {{.*}}constant_var{{[^,]*}}, {{[^@]*}}@2, {{.*}}i32 0, {{i32|i64}} 4, i32 1, i32 0
// ALL-DAG: call void {{.*}}[[PREFIX]]RegisterVar(i8** %0, {{.*}}ext_device_var_def{{[^,]*}}, {{[^@]*}}@3, {{.*}}i32 0, {{i32|i64}} 4, i32 0, i32 0
// ALL-DAG: call void {{.*}}[[PREFIX]]RegisterVar(i8** %0, {{.*}}ext_constant_var_def{{[^,]*}}, {{[^@]*}}@4, {{.*}}i32 0, {{i32|i64}} 4, i32 1, i32 0
// LNX_17-DAG: [[PREFIX]]RegisterVar(i8** %0, {{.*}}inline_var
// LNX_17-NOT: [[PREFIX]]RegisterVar(i8** %0, {{.*}}inline_var2
// ALL: ret void

// Test that we've built a constructor.
// LNX: define internal void @__[[PREFIX]]_module_ctor

// In separate mode it calls __[[PREFIX]]RegisterFatBinary(&__[[PREFIX]]_fatbin_wrapper)
// HIP only register fat binary once.
// HIP: load i8**, i8*** @__hip_gpubin_handle
// HIP-NEXT: icmp eq i8** {{.*}}, null
// HIP-NEXT: br i1 {{.*}}, label %if, label %exit
// HIP: if:
// CUDANORDC: call{{.*}}[[PREFIX]]RegisterFatBinary{{.*}}__[[PREFIX]]_fatbin_wrapper
//   .. stores return value in __[[PREFIX]]_gpubin_handle
// CUDANORDC-NEXT: store{{.*}}__[[PREFIX]]_gpubin_handle
//   .. and then calls __[[PREFIX]]_register_globals
// HIP: call{{.*}}[[PREFIX]]RegisterFatBinary{{.*}}__[[PREFIX]]_fatbin_wrapper
//   .. stores return value in __[[PREFIX]]_gpubin_handle
// HIP-NEXT: store{{.*}}__[[PREFIX]]_gpubin_handle
//   .. and then calls __[[PREFIX]]_register_globals
// HIP-NEXT: br label %exit
// HIP: exit:
// HIP-NEXT: load i8**, i8*** @__hip_gpubin_handle
// CUDANORDC-NEXT: call void @__[[PREFIX]]_register_globals
// HIP-NEXT: call void @__[[PREFIX]]_register_globals
// * In separate mode we also register a destructor.
// CUDANORDC-NEXT: call i32 @atexit(void (i8*)* @__[[PREFIX]]_module_dtor)
// HIP-NEXT: call i32 @atexit(void (i8*)* @__[[PREFIX]]_module_dtor)

// With relocatable device code we call __[[PREFIX]]RegisterLinkedBinary%NVModuleID%
// CUDARDC: call{{.*}}__[[PREFIX]]RegisterLinkedBinary[[MODULE_ID]](
// CUDARDC-SAME: __[[PREFIX]]_register_globals, {{.*}}__[[PREFIX]]_fatbin_wrapper
// CUDARDC-SAME: [[MODULE_ID_GLOBAL]]

// Test that we've created destructor.
// CUDANORDC: define internal void @__[[PREFIX]]_module_dtor
// HIP: define internal void @__[[PREFIX]]_module_dtor
// CUDANORDC: load{{.*}}__[[PREFIX]]_gpubin_handle
// HIP: load{{.*}}__[[PREFIX]]_gpubin_handle
// CUDANORDC-NEXT: call void @__[[PREFIX]]UnregisterFatBinary
// HIP-NEXT: icmp ne i8** {{.*}}, null
// HIP-NEXT: br i1 {{.*}}, label %if, label %exit
// HIP: if:
// HIP-NEXT: call void @__[[PREFIX]]UnregisterFatBinary
// HIP-NEXT: store i8** null, i8*** @__hip_gpubin_handle
// HIP-NEXT: br label %exit
// HIP: exit:

// There should be no __[[PREFIX]]_register_globals if we have no
// device-side globals, but we still need to register GPU binary.
// Skip GPU binary string first.
// CUDANOGLOBALS-NOT: @{{.*}} = private constant{{.*}}
// HIPNOGLOBALS-NOT: @{{.*}} = internal constant{{.*}}
// NOGLOBALS-NOT: define internal void @__{{.*}}_register_globals
// NOGLOBALS-NOT: define internal void @__{{cuda|hip}}_module_ctor
// NOGLOBALS-NOT: call{{.*}}{{cuda|hip}}RegisterFatBinary{{.*}}__{{cuda|hip}}_fatbin_wrapper
// NOGLOBALS-NOT: call void @__{{cuda|hip}}_register_globals
// NOGLOBALS-NOT: define internal void @__{{cuda|hip}}_module_dtor
// NOGLOBALS-NOT: call void @__{{cuda|hip}}UnregisterFatBinary

// There should be no constructors/destructors if we have no GPU binary.
// NOGPUBIN-NOT: define internal void @__{{cuda|hip}}_register_globals
// NOGPUBIN-NOT: define internal void @__{{cuda|hip}}_module_ctor
// NOGPUBIN-NOT: define internal void @__{{cuda|hip}}_module_dtor
