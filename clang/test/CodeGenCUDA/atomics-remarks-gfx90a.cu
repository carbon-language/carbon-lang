// RUN: %clang_cc1 %s -triple=amdgcn-amd-amdhsa -fcuda-is-device \
// RUN:   -target-cpu gfx90a -Rpass=atomic-expand -S -o - 2>&1 | \
// RUN:   FileCheck %s --check-prefix=GFX90A-CAS

// REQUIRES: amdgpu-registered-target

#include "Inputs/hip/hip_runtime.h"
#include <stdatomic.h>

// GFX90A-CAS: A compare and swap loop was generated for an atomic fadd operation at system memory scope
// GFX90A-CAS-LABEL: _Z14atomic_add_casPf
// GFX90A-CAS:  flat_atomic_cmpswap v0, v[2:3], v[4:5] glc
// GFX90A-CAS:  s_cbranch_execnz
__device__ float atomic_add_cas(float *p) {
  return __atomic_fetch_add(p, 1.0f, memory_order_relaxed);
}
