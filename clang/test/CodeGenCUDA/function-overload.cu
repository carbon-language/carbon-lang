// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// Make sure we handle target overloads correctly.  Most of this is checked in
// sema, but special functions like constructors and destructors are here.
//
// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -emit-llvm -o - %s \
// RUN:     | FileCheck -check-prefix=CHECK-BOTH -check-prefix=CHECK-HOST %s
// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fcuda-is-device -emit-llvm -o - %s \
// RUN:     | FileCheck -check-prefix=CHECK-BOTH -check-prefix=CHECK-DEVICE %s

#include "Inputs/hip/hip_runtime.h"

// Check constructors/destructors for D/H functions
#ifdef __CUDA_ARCH__
__device__
#endif
int x;
struct s_cd_dh {
  __host__ s_cd_dh() { x = 11; }
  __device__ s_cd_dh() { x = 12; }
};

struct s_cd_hd {
  __host__ __device__ s_cd_hd() { x = 31; }
  __host__ __device__ ~s_cd_hd() { x = 32; }
};

// CHECK-BOTH: define{{.*}} void @_Z7wrapperv
#if defined(__CUDA_ARCH__)
__device__
#else
__host__
#endif
void wrapper() {
  s_cd_dh scddh;
  // CHECK-BOTH: call void @_ZN7s_cd_dhC1Ev(
  s_cd_hd scdhd;
  // CHECK-BOTH: call void @_ZN7s_cd_hdC1Ev

  // CHECK-BOTH: call void @_ZN7s_cd_hdD1Ev(
}
// CHECK-BOTH: ret void

// Now it's time to check what's been generated for the methods we used.

// CHECK-BOTH: define linkonce_odr void @_ZN7s_cd_dhC2Ev(
// CHECK-HOST:   store i32 11,
// CHECK-DEVICE: store i32 12,
// CHECK-BOTH: ret void

// CHECK-BOTH: define linkonce_odr void @_ZN7s_cd_hdC2Ev(
// CHECK-BOTH:   store i32 31,
// CHECK-BOTH: ret void

// CHECK-BOTH: define linkonce_odr void @_ZN7s_cd_hdD2Ev(
// CHECK-BOTH: store i32 32,
// CHECK-BOTH: ret void
