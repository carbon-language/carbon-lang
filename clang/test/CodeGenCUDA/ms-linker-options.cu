
#include <hip/hip_runtime.h>
// RUN: %clang_cc1 -emit-llvm -o - -fcuda-is-device -fms-extensions -x hip %s \
// RUN:   -fno-autolink -triple amdgcn-amd-amdhsa \
// RUN:   | FileCheck -check-prefix=DEV %s
// RUN: %clang_cc1 -emit-llvm -o - -fms-extensions -x hip %s -triple \
// RUN:    x86_64-pc-windows-msvc | FileCheck -check-prefix=HOST %s
// RUN: %clang_cc1 -emit-llvm -o - -fcuda-is-device -fms-extensions %s \
// RUN:   -fno-autolink -triple amdgcn-amd-amdhsa \
// RUN:   | FileCheck -check-prefix=DEV %s
// RUN: %clang_cc1 -emit-llvm -o - -fms-extensions %s -triple \
// RUN:    x86_64-pc-windows-msvc | FileCheck -check-prefix=HOST %s

// DEV-NOT: llvm.linker.options
// DEV-NOT: llvm.dependent-libraries
// HOST: lvm.linker.options
// HOST: "/DEFAULTLIB:libcpmt.lib"
// HOST: "/FAILIFMISMATCH:\22myLib_version=9\22"

#pragma comment(lib, "libcpmt")
#pragma detect_mismatch("myLib_version", "9")
