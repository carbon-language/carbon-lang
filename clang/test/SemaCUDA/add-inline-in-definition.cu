// RUN: %clang_cc1 -std=c++11 -fcuda-is-device -fsyntax-only -verify %s
// RUN: %clang_cc1 -std=c++11 -fsyntax-only -verify %s

#include "Inputs/hip/hip_runtime.h"

#ifndef __CUDA_ARCH__
// expected-no-diagnostics
#endif

// When compiling for device, foo()'s call to host_fn() is an error, because
// foo() is known-emitted.
//
// The trickiness here comes from the fact that the FunctionDecl bar() sees
// foo() does not have the "inline" keyword, so we might incorrectly think that
// foo() is a priori known-emitted.  This would prevent us from marking foo()
// as known-emitted when we see the call from bar() to foo(), which would
// prevent us from emitting an error for foo()'s call to host_fn() when we
// eventually see it.

void host_fn() {}
#ifdef __CUDA_ARCH__
  // expected-note@-2 {{declared here}}
#endif

__host__ __device__ void foo();
__device__ void bar() {
  foo();
#ifdef __CUDA_ARCH__
  // expected-note@-2 {{called by 'bar'}}
#endif
}
inline __host__ __device__ void foo() {
  host_fn();
#ifdef __CUDA_ARCH__
  // expected-error@-2 {{reference to __host__ function}}
#endif
}

// This is similar to the above, except there's no error here.  This code used
// to trip an assertion due to us noticing, when emitting the definition of
// boom(), that T::operator S() was (incorrectly) considered a priori
// known-emitted.
struct S {};
struct T {
  __device__ operator S() const;
};
__device__ inline T::operator S() const { return S(); }

__device__ T t;
__device__ void boom() {
  S s = t;
}
