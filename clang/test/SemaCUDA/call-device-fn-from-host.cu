#include "hip/hip_runtime.h"
// RUN: %clang_cc1 %s --std=c++11 -triple x86_64-unknown-linux -emit-llvm -o - \
// RUN:   -verify -verify-ignore-unexpected=note
// RUN: %clang_cc1 %s --std=c++11 -triple x86_64-unknown-linux -emit-llvm -o - \
// RUN:   -verify=expected,omp -verify-ignore-unexpected=note -fopenmp

// Note: This test won't work with -fsyntax-only, because some of these errors
// are emitted during codegen.

#include "Inputs/hip/hip_runtime.h"

__device__ void device_fn() {}
// expected-note@-1 5 {{'device_fn' declared here}}

struct S {
  __device__ S() {}
  // expected-note@-1 2 {{'S' declared here}}
  __device__ ~S() { device_fn(); }
  // expected-note@-1 {{'~S' declared here}}
  int x;
};

struct T {
  __host__ __device__ void hd() { device_fn(); }
  // expected-error@-1 {{reference to __device__ function 'device_fn' in __host__ __device__ function}}

  // No error; this is (implicitly) inline and is never called, so isn't
  // codegen'ed.
  __host__ __device__ void hd2() { device_fn(); }

  __host__ __device__ void hd3();

  __device__ void d() {}
  // expected-note@-1 {{'d' declared here}}
};

__host__ __device__ void T::hd3() {
  device_fn();
  // expected-error@-1 {{reference to __device__ function 'device_fn' in __host__ __device__ function}}
}

template <typename T> __host__ __device__ void hd2() { device_fn(); }
// expected-error@-1 {{reference to __device__ function 'device_fn' in __host__ __device__ function}}
void host_fn() { hd2<int>(); }

__host__ __device__ void hd() { device_fn(); }
// expected-error@-1 {{reference to __device__ function 'device_fn' in __host__ __device__ function}}

// No error because this is never instantiated.
template <typename T> __host__ __device__ void hd3() { device_fn(); }

__host__ __device__ void local_var() {
  S s;
  // expected-error@-1 {{reference to __device__ function 'S' in __host__ __device__ function}}
}

__host__ __device__ void placement_new(char *ptr) {
  ::new(ptr) S();
  // expected-error@-1 {{reference to __device__ function 'S' in __host__ __device__ function}}
}

__host__ __device__ void explicit_destructor(S *s) {
  s->~S();
  // expected-error@-1 {{reference to __device__ function '~S' in __host__ __device__ function}}
}

__host__ __device__ void hd_member_fn() {
  T t;
  // Necessary to trigger an error on T::hd.  It's (implicitly) inline, so
  // isn't codegen'ed until we call it.
  t.hd();
}

__host__ __device__ void h_member_fn() {
  T t;
  t.d();
  // expected-error@-1 {{reference to __device__ function 'd' in __host__ __device__ function}}
}

__host__ __device__ void fn_ptr() {
  auto* ptr = &device_fn;
  // expected-error@-1 {{reference to __device__ function 'device_fn' in __host__ __device__ function}}
}

template <typename T>
__host__ __device__ void fn_ptr_template() {
  auto* ptr = &device_fn;  // Not an error because the template isn't instantiated.
}

// Launching a kernel from a host function does not result in code generation
// for it, so calling HD function which calls a D function should not trigger
// errors.
static __host__ __device__ void hd_func() { device_fn(); }
__global__ void kernel() { hd_func(); }
void host_func(void) { kernel<<<1, 1>>>(); }

// Should allow host function call kernel template with device function argument.
__device__ void f();
template<void(*F)()> __global__ void t() { F(); }
__host__ void g() { t<f><<<1,1>>>(); }
