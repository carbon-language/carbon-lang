// RUN: %clang_cc1 -x hip -std=c++11 -triple amdgcn -fcuda-is-device -verify -fsyntax-only %s

#include "Inputs/hip/hip_runtime.h"

__device__ int test_hip_atomic_load(int *pi32, unsigned int *pu32, long long *pll, unsigned long long *pull, float *fp, double *dbl) {
  int val = __hip_atomic_load(0);      // expected-error {{too few arguments to function call, expected 3, have 1}}
  val = __hip_atomic_load(0, 0, 0, 0); // expected-error {{too many arguments to function call, expected 3, have 4}}
  val = __hip_atomic_load(0, 0, 0);    // expected-error {{address argument to atomic builtin must be a pointer ('int' invalid)}}
  val = __hip_atomic_load(pi32, 0, 0); // expected-error {{synchronization scope argument to atomic operation is invalid}}
  val = __hip_atomic_load(pi32, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_load(pi32, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_load(pi32, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_load(pi32, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_load(pi32, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_load(pi32, __ATOMIC_RELAXED, 6); // expected-error {{synchronization scope argument to atomic operation is invalid}}
  val = __hip_atomic_load(pi32, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_load(pi32, __ATOMIC_SEQ_CST, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_load(pi32, __ATOMIC_CONSUME, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_load(pi32, __ATOMIC_ACQUIRE, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_load(pi32, __ATOMIC_ACQ_REL, __HIP_MEMORY_SCOPE_SINGLETHREAD); // expected-warning{{memory order argument to atomic operation is invalid}}
  val = __hip_atomic_load(pu32, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_load(pll, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_load(pull, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_load(fp, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_load(dbl, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  return val;
}

__device__ int test_hip_atomic_store(int *pi32, unsigned int *pu32, long long *pll, unsigned long long *pull, float *fp, double *dbl,
                                     int i32, unsigned int u32, long long i64, unsigned long long u64, float f32, double f64) {
  __hip_atomic_store(0);             // expected-error {{too few arguments to function call, expected 4, have 1}}
  __hip_atomic_store(0, 0, 0, 0, 0); // expected-error {{too many arguments to function call, expected 4, have 5}}
  __hip_atomic_store(0, 0, 0, 0);    // expected-error {{address argument to atomic builtin must be a pointer ('int' invalid)}}
  __hip_atomic_store(pi32, 0, 0, 0); // expected-error {{synchronization scope argument to atomic operation is invalid}}
  __hip_atomic_store(pi32, 0, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  __hip_atomic_store(pi32, 0, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  __hip_atomic_store(pi32, 0, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  __hip_atomic_store(pi32, 0, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  __hip_atomic_store(pi32, 0, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  __hip_atomic_store(pi32, 0, __ATOMIC_RELAXED, 6); // expected-error {{synchronization scope argument to atomic operation is invalid}}
  __hip_atomic_store(pi32, 0, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  __hip_atomic_store(pi32, 0, __ATOMIC_SEQ_CST, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  __hip_atomic_store(pi32, 0, __ATOMIC_CONSUME, __HIP_MEMORY_SCOPE_SINGLETHREAD); // expected-warning{{memory order argument to atomic operation is invalid}}
  __hip_atomic_store(pi32, 0, __ATOMIC_ACQUIRE, __HIP_MEMORY_SCOPE_SINGLETHREAD); // expected-warning{{memory order argument to atomic operation is invalid}}
  __hip_atomic_store(pi32, 0, __ATOMIC_ACQ_REL, __HIP_MEMORY_SCOPE_SINGLETHREAD); // expected-warning{{memory order argument to atomic operation is invalid}}
  __hip_atomic_store(pi32, i32, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  __hip_atomic_store(pi32, i32, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  __hip_atomic_store(pu32, u32, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  __hip_atomic_store(pll, i64, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  __hip_atomic_store(pull, u64, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  __hip_atomic_store(fp, f32, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  __hip_atomic_store(dbl, f64, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  __hip_atomic_store(pi32, u32, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  __hip_atomic_store(pi32, i64, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  __hip_atomic_store(pi32, u64, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  __hip_atomic_store(pll, i32, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  __hip_atomic_store(fp, i32, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  __hip_atomic_store(fp, i64, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  __hip_atomic_store(dbl, i64, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  __hip_atomic_store(dbl, i32, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  return 0;
}

__device__ bool test_hip_atomic_cmpxchg_weak(int *ptr, int val, int desired) {
  bool flag = __hip_atomic_compare_exchange_weak(0);                                     // expected-error {{too few arguments to function call, expected 6, have 1}}
  flag = __hip_atomic_compare_exchange_weak(0, 0, 0, 0, 0, 0, 0);                        // expected-error {{too many arguments to function call, expected 6, have 7}}
  flag = __hip_atomic_compare_exchange_weak(0, 0, 0, 0, 0, 0);                           // expected-error {{address argument to atomic builtin must be a pointer ('int' invalid)}}
  flag = __hip_atomic_compare_exchange_weak(ptr, 0, 0, 0, 0, 0);                         // expected-error {{synchronization scope argument to atomic operation is invalid}}, expected-warning {{null passed to a callee that requires a non-null argument}}
  flag = __hip_atomic_compare_exchange_weak(ptr, 0, 0, 0, 0, __HIP_MEMORY_SCOPE_SYSTEM); // expected-warning {{null passed to a callee that requires a non-null argument}}
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_CONSUME, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_SEQ_CST, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_CONSUME, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_ACQUIRE, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_ACQ_REL, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_SEQ_CST, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_CONSUME, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_ACQUIRE, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_ACQ_REL, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  return flag;
}
