#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -fsyntax-only -verify %s

#include "Inputs/hip/hip_runtime.h"

__global__ void g1(int x) {}

template <typename T> void t1(T arg) {
  g1<<<arg, arg>>>(1);
}

void h1(int x) {}
int h2(int x) { return 1; }

int main(void) {
  g1<<<1, 1>>>(42);
  g1(42); // expected-error {{call to global function 'g1' not configured}}
  g1<<<1>>>(42); // expected-error {{too few execution configuration arguments to kernel function call}}
  g1<<<1, 1, 0, 0, 0>>>(42); // expected-error {{too many execution configuration arguments to kernel function call}}

  t1(1);

  h1<<<1, 1>>>(42); // expected-error {{kernel call to non-global function 'h1'}}

  int (*fp)(int) = h2;
  fp<<<1, 1>>>(42); // expected-error {{must have void return type}}

  g1<<<undeclared, 1>>>(42); // expected-error {{use of undeclared identifier 'undeclared'}}
}
