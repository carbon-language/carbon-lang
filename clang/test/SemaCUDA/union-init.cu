#include "hip/hip_runtime.h"
// RUN: %clang_cc1 %s --std=c++11 -triple x86_64-linux-unknown -fsyntax-only -o - -verify

#include "Inputs/hip/hip_runtime.h"

struct A {
  int a;
  __device__ A() { a = 1; }
  __device__ ~A() { a = 2; }
};

// This can be a global var since ctor/dtors of data members are not called.
union B {
  A a;
  __device__ B() {}
  __device__ ~B() {}
};

// This cannot be a global var since it has a dynamic ctor.
union C {
  A a;
  __device__ C() { a.a = 3; }
  __device__ ~C() {}
};

// This cannot be a global var since it has a dynamic dtor.
union D {
  A a;
  __device__ D() { }
  __device__ ~D() { a.a = 4; }
};

__device__ B b;
__device__ C c;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, __shared__, and __managed__ variables.}}
__device__ D d;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, __shared__, and __managed__ variables.}}

__device__ void foo() {
  __shared__ B b;
  __shared__ C c;
  // expected-error@-1 {{initialization is not supported for __shared__ variables.}}
  __shared__ D d;
  // expected-error@-1 {{initialization is not supported for __shared__ variables.}}
}
