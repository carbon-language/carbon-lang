// RUN: %clang_cc1 -std=gnu++11 -fsyntax-only -verify %s

#include "Inputs/hip/hip_runtime.h"

//------------------------------------------------------------------------------
// Test 1: collision between two bases

struct A1_with_host_ctor {
  A1_with_host_ctor() {}
};

struct B1_with_device_ctor {
  __device__ B1_with_device_ctor() {}
};

struct C1_with_collision : A1_with_host_ctor, B1_with_device_ctor {
};

// expected-note@-3 {{candidate constructor (the implicit default constructor) not viable}}
// expected-note@-4 {{implicit default constructor inferred target collision: call to both __host__ and __device__ members}}
// expected-note@-5 {{candidate constructor (the implicit copy constructor) not viable}}
// expected-note@-6 {{candidate constructor (the implicit move constructor) not viable}}

void hostfoo1() {
  C1_with_collision c; // expected-error {{no matching constructor}}
}

//------------------------------------------------------------------------------
// Test 2: collision between two fields

struct C2_with_collision {
  A1_with_host_ctor aa;
  B1_with_device_ctor bb;
};

// expected-note@-5 {{candidate constructor (the implicit default constructor}} not viable
// expected-note@-6 {{implicit default constructor inferred target collision: call to both __host__ and __device__ members}}
// expected-note@-7 {{candidate constructor (the implicit copy constructor}} not viable
// expected-note@-8 {{candidate constructor (the implicit move constructor}} not viable

void hostfoo2() {
  C2_with_collision c; // expected-error {{no matching constructor}}
}

//------------------------------------------------------------------------------
// Test 3: collision between a field and a base

struct C3_with_collision : A1_with_host_ctor {
  B1_with_device_ctor bb;
};

// expected-note@-4 {{candidate constructor (the implicit default constructor}} not viable
// expected-note@-5 {{implicit default constructor inferred target collision: call to both __host__ and __device__ members}}
// expected-note@-6 {{candidate constructor (the implicit copy constructor}} not viable
// expected-note@-7 {{candidate constructor (the implicit move constructor}} not viable

void hostfoo3() {
  C3_with_collision c; // expected-error {{no matching constructor}}
}

//------------------------------------------------------------------------------
// Test 4: collision on resolving a copy ctor

struct A4_with_host_copy_ctor {
  A4_with_host_copy_ctor() {}
  A4_with_host_copy_ctor(const A4_with_host_copy_ctor&) {}
};

struct B4_with_device_copy_ctor {
  B4_with_device_copy_ctor() {}
  __device__ B4_with_device_copy_ctor(const B4_with_device_copy_ctor&) {}
};

struct C4_with_collision : A4_with_host_copy_ctor, B4_with_device_copy_ctor {
};

// expected-note@-3 {{candidate constructor (the implicit copy constructor) not viable: call to invalid function from __host__ function}}
// expected-note@-4 {{implicit copy constructor inferred target collision: call to both __host__ and __device__ members}}
// expected-note@-5 {{candidate constructor (the implicit default constructor) not viable: requires 0 arguments, but 1 was provided}}

void hostfoo4() {
  C4_with_collision c;
  C4_with_collision c2 = c; // expected-error {{no matching constructor for initialization of 'C4_with_collision'}}
}

//------------------------------------------------------------------------------
// Test 5: collision on resolving a move ctor

struct A5_with_host_move_ctor {
  A5_with_host_move_ctor() {}
  A5_with_host_move_ctor(A5_with_host_move_ctor&&) {}
// expected-note@-1 {{copy constructor is implicitly deleted because 'A5_with_host_move_ctor' has a user-declared move constructor}}
};

struct B5_with_device_move_ctor {
  B5_with_device_move_ctor() {}
  __device__ B5_with_device_move_ctor(B5_with_device_move_ctor&&) {}
};

struct C5_with_collision : A5_with_host_move_ctor, B5_with_device_move_ctor {
};
// expected-note@-2 {{deleted}}

void hostfoo5() {
  C5_with_collision c;
  // What happens here:
  // This tries to find the move ctor. Since the move ctor is deleted due to
  // collision, it then looks for a copy ctor. But copy ctors are implicitly
  // deleted when move ctors are declared explicitly.
  C5_with_collision c2(static_cast<C5_with_collision&&>(c)); // expected-error {{call to implicitly-deleted}}
}
