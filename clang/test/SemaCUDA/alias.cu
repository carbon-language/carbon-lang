
#include <hip/hip_runtime.h>
// RUN: %clang_cc1 -triple nvptx-unknown-cuda -fsyntax-only -fcuda-is-device -verify -DEXPECT_ERR %s
// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -fsyntax-only -verify %s

// The alias attribute is not allowed in CUDA device code.
void bar();
__attribute__((alias("bar"))) void foo();
#ifdef EXPECT_ERR
// expected-error@-2 {{CUDA does not support aliases}}
#else
// expected-no-diagnostics
#endif
