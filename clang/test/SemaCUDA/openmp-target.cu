// RUN: %clang_cc1 -triple x86_64 -verify=expected,dev \
// RUN:            -verify-ignore-unexpected=note \
// RUN:            -fopenmp -fopenmp-version=50 -fopenmp-targets=amdgcn-amd-amdhsa -o - %s
// RUN: %clang_cc1 -triple x86_64 -verify -verify-ignore-unexpected=note\
// RUN:            -fopenmp -fopenmp-version=50 -fopenmp-targets=amdgcn-amd-amdhsa -o - -x c++ %s
// RUN: %clang_cc1 -triple x86_64 -verify=dev -verify-ignore-unexpected=note\
// RUN:            -fcuda-is-device -o - %s

#if __CUDA__
#include "Inputs/hip/hip_runtime.h"
__device__ void cu_devf();
#endif

void bazz() {}
#pragma omp declare target to(bazz) device_type(nohost)
void bazzz() {bazz();}
#pragma omp declare target to(bazzz) device_type(nohost)
void any() {bazz();} // expected-error {{function with 'device_type(nohost)' is not available on host}}
void host1() {bazz();} // expected-error {{function with 'device_type(nohost)' is not available on host}}
#pragma omp declare target to(host1) device_type(host)
void host2() {bazz();} // expected-error {{function with 'device_type(nohost)' is not available on host}}
#pragma omp declare target to(host2)
void device() {host1();}
#pragma omp declare target to(device) device_type(nohost)
void host3() {host1();}
#pragma omp declare target to(host3)

#pragma omp declare target
void any1() {any();}
void any2() {host1();}
void any3() {device();} // expected-error {{function with 'device_type(nohost)' is not available on host}}
void any4() {any2();}
#pragma omp end declare target

void any5() {any();}
void any6() {host1();}
void any7() {device();} // expected-error {{function with 'device_type(nohost)' is not available on host}}
void any8() {any2();}

#if __CUDA__
void cu_hostf() { cu_devf(); } // dev-error {{no matching function for call to 'cu_devf'}}
__device__ void cu_devf2() { cu_hostf(); } // dev-error{{no matching function for call to 'cu_hostf'}}
#endif
